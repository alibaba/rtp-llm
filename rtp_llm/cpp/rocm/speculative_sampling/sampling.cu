#include "hip/hip_runtime.h"
#include <sampling.cuh>
#include <ATen/Utils.h> 
#include <ATen/core/Generator.h>
#include <ATen/hip/HIPGeneratorImpl.h>

#include <ATen/hip/detail/UnpackRaw.cuh>
#include <mutex>

#include "hip/hip_runtime.h"

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/amd_detail/amd_warp_sync_functions.h>

#include <hipcub/block/block_adjacent_difference.hpp>
#include <hipcub/hipcub.hpp>
#include <limits>
#include <numeric>
#include <tuple>

#include "utils.cuh"
#include "vec_dtypes.cuh"

using namespace hipcub;

#define DISPATCH_DETERMINISTIC(deterministic, DETERMINISTIC, ...) \
  if (deterministic) {                                            \
    constexpr bool DETERMINISTIC = true;                          \
    __VA_ARGS__                                                   \
  } else {                                                        \
    constexpr bool DETERMINISTIC = false;                         \
    __VA_ARGS__                                                   \
  }

#define DISPATCH_COMPUTE_CAP_NUM_THREADS(compute_capacity, BLOCK_THREADS, ...) \
  if (compute_capacity.first >= 8) {                                           \
    constexpr uint32_t BLOCK_THREADS = 1024;                                   \
    __VA_ARGS__                                                                \
  } else {                                                                     \
    constexpr uint32_t BLOCK_THREADS = 512;                                    \
    __VA_ARGS__                                                                \
  }

#define DISPATCH_SOFTMAX_CACHE_INPUT(cache_input, CACHE_INPUT, ...) \
  if (cache_input) {                                                \
    constexpr bool CACHE_INPUT = true;                              \
    __VA_ARGS__                                                     \
  } else {                                                          \
    constexpr bool CACHE_INPUT = false;                             \
    __VA_ARGS__                                                     \
  }


constexpr BlockScanAlgorithm SCAN_ALGO = BLOCK_SCAN_WARP_SCANS;
constexpr BlockReduceAlgorithm REDUCE_ALGO = BLOCK_REDUCE_WARP_REDUCTIONS;

template <typename T>
struct ValueCount {
  T value;
  int count;

  __device__ ValueCount operator+(const ValueCount& other) const {
    return {value + other.value, count + other.count};
  }
  __device__ ValueCount& operator+=(const ValueCount& other) {
    value += other.value;
    count += other.count;
    return *this;
  }
};

struct BoolDiffOp {
  __device__ __forceinline__ bool operator()(const bool& lhs, const bool& rhs) const {
    return lhs != rhs;
  }
};

struct Float2SoftmaxReduceOp {
  __device__ __forceinline__ float2 operator()(const float2& a, const float2& b) const {
    if (isinf(a.x)) return b;
    if (isinf(b.x)) return a;

    float new_max = max(a.x, b.x);
    float new_denom = a.y * __expf(a.x - new_max) + b.y * __expf(b.x - new_max);
    return make_float2(new_max, new_denom);
  }
};


template <uint32_t BLOCK_THREADS, BlockScanAlgorithm SCAN_ALGORITHM,
          BlockReduceAlgorithm REDUCE_ALGORITHM>
struct SamplingTempStorage {
  union {
    float deterministic_scan[BLOCK_THREADS / 64];
    typename BlockScan<float, BLOCK_THREADS, SCAN_ALGORITHM>::TempStorage scan;
    typename BlockReduce<float, BLOCK_THREADS, REDUCE_ALGORITHM>::TempStorage reduce;
    typename BlockReduce<int, BLOCK_THREADS, REDUCE_ALGORITHM>::TempStorage reduce_int;
    typename BlockReduce<ValueCount<float>, BLOCK_THREADS, REDUCE_ALGORITHM>::TempStorage
        reduce_value_count;
    typename BlockAdjacentDifference<bool, BLOCK_THREADS>::TempStorage adj_diff;
  } block_prim;
  struct {
    int32_t sampled_id;
    int32_t last_valid_id;
    float max_val;
    union {
      float value;
      ValueCount<float> pair;
    } block_aggregate;
  };
};


#define DISPATCH_COMPUTE_CAP_NUM_THREADS(compute_capacity, BLOCK_THREADS, ...) \
  if (compute_capacity.first >= 8) {                                           \
    constexpr uint32_t BLOCK_THREADS = 1024;                                   \
    __VA_ARGS__                                                                \
  } else {                                                                     \
    constexpr uint32_t BLOCK_THREADS = 512;                                    \
    __VA_ARGS__                                                                \
  }

  template <uint32_t VEC_SIZE, uint32_t BLOCK_THREADS, BlockScanAlgorithm SCAN_ALGORITHM,
          BlockReduceAlgorithm REDUCE_ALGORITHM>
__device__ __forceinline__ void DeterministicInclusiveSum(
    const float* in_data, float* out_data,
    SamplingTempStorage<BLOCK_THREADS, SCAN_ALGORITHM, REDUCE_ALGORITHM>* temp_storage) {
  float* smem_prefix_sum = temp_storage->block_prim.deterministic_scan;
  float thread_data[VEC_SIZE];
  float thread_sum = 0;
#pragma unroll
  for (uint32_t i = 0; i < VEC_SIZE; ++i) {
    thread_sum += in_data[i];
    thread_data[i] = thread_sum;
  }

  float thread_exclusive_prefix_sum = thread_sum;

#pragma unroll
  for (uint32_t offset = 1; offset < 64; offset *= 2) {
    float tmp = __shfl_up_sync(0xffffffffffffffff, thread_exclusive_prefix_sum, offset);
    if ((threadIdx.x + 1) % (offset * 2) == 0) {
      thread_exclusive_prefix_sum += tmp;
    }
  }

  float warp_sum = __shfl_sync(0xffffffffffffffff, thread_exclusive_prefix_sum, 0xffffffff);
  if (threadIdx.x % 64 == 63) {
    thread_exclusive_prefix_sum = 0;
  }

#pragma unroll
  for (uint32_t offset = 32; offset >= 1; offset /= 2) {
    float tmp = __shfl_xor_sync(0xffffffffffffffff, thread_exclusive_prefix_sum, offset);
    if ((threadIdx.x + 1) % (offset * 2) == 0) {
      thread_exclusive_prefix_sum = tmp + thread_exclusive_prefix_sum;
    }
    if ((threadIdx.x + 1) % (offset * 2) == offset) {
      thread_exclusive_prefix_sum = tmp;
    }
  }

  smem_prefix_sum[threadIdx.x / 64] = warp_sum; 
  __syncthreads();

  if (threadIdx.x < 64) {
    float warp_exclusive_prefix_sum =
        (threadIdx.x < BLOCK_THREADS / 64) ? smem_prefix_sum[threadIdx.x] : 0;

#pragma unroll
    for (uint32_t offset = 1; offset < 64; offset *= 2) {
      float tmp = __shfl_up_sync(0xffffffffffffffff, warp_exclusive_prefix_sum, offset);
      if ((threadIdx.x + 1) % (offset * 2) == 0) {
        warp_exclusive_prefix_sum += tmp;
      }
    }

    if (threadIdx.x % 64 == 63) {
      warp_exclusive_prefix_sum = 0;
    }

#pragma unroll
    for (uint32_t offset = 32; offset >= 1; offset /= 2) {
      float tmp = __shfl_xor_sync(0xffffffffffffffff, warp_exclusive_prefix_sum, offset);
      if ((threadIdx.x + 1) % (offset * 2) == 0) {
        warp_exclusive_prefix_sum = tmp + warp_exclusive_prefix_sum;
      }
      if ((threadIdx.x + 1) % (offset * 2) == offset) {
        warp_exclusive_prefix_sum = tmp;
      }
    }
    if (threadIdx.x < BLOCK_THREADS / 64) {
      smem_prefix_sum[threadIdx.x] = warp_exclusive_prefix_sum;
    }
  }
  __syncthreads();

#pragma unroll
  for (uint32_t i = 0; i < VEC_SIZE; ++i) {
    out_data[i] = smem_prefix_sum[threadIdx.x / 64] + thread_exclusive_prefix_sum + thread_data[i];
  }
}

template <uint32_t VEC_SIZE, uint32_t BLOCK_THREADS, BlockScanAlgorithm SCAN_ALGORITHM,
          BlockReduceAlgorithm REDUCE_ALGORITHM, bool DETERMINISTIC, typename Predicate>
__device__ __forceinline__ void DeviceSamplingFromProb(
    uint32_t i, uint32_t d, Predicate pred, float u, vec_t<float, VEC_SIZE> prob_vec,
    float& aggregate,
    SamplingTempStorage<BLOCK_THREADS, SCAN_ALGORITHM, REDUCE_ALGORITHM>* temp_storage) {
  const uint32_t tx = threadIdx.x;
  float prob_greater_than_threshold[VEC_SIZE];
  float inclusive_cdf[VEC_SIZE];
  bool greater_than_u[VEC_SIZE], valid[VEC_SIZE];
#pragma unroll
  for (uint32_t j = 0; j < VEC_SIZE; ++j) {
    prob_greater_than_threshold[j] = pred(prob_vec[j]) ? prob_vec[j] : 0;
    valid[j] = pred(prob_vec[j]) && (i * BLOCK_THREADS + tx) * VEC_SIZE + j < d;
  }
  float aggregate_local =
      BlockReduce<float, BLOCK_THREADS, REDUCE_ALGORITHM>(temp_storage->block_prim.reduce)
          .template Sum<VEC_SIZE>(prob_greater_than_threshold);
  if (tx == 0) {
    temp_storage->block_aggregate.value = aggregate_local;
  }
  __syncthreads();
  aggregate_local = temp_storage->block_aggregate.value;

  if (aggregate + aggregate_local > u) {
    if constexpr (DETERMINISTIC) {
      DeterministicInclusiveSum<VEC_SIZE, BLOCK_THREADS, SCAN_ALGORITHM, REDUCE_ALGORITHM>(
          prob_greater_than_threshold, inclusive_cdf, temp_storage);
    } else {
      BlockScan<float, BLOCK_THREADS, SCAN_ALGORITHM>(temp_storage->block_prim.scan)
          .template InclusiveSum<VEC_SIZE>(prob_greater_than_threshold, inclusive_cdf);

      __syncthreads();
    }

#pragma unroll
    for (uint32_t j = 0; j < VEC_SIZE; ++j) {
      greater_than_u[j] = (inclusive_cdf[j] + aggregate > u) && valid[j];
    }

    bool greater_than_u_diff[VEC_SIZE];
#ifdef FLASHINFER_CUB_SUBTRACTLEFT_DEFINED
    BlockAdjacentDifference<bool, BLOCK_THREADS>(temp_storage->block_prim.adj_diff)
        .SubtractLeft<VEC_SIZE>(greater_than_u, greater_than_u_diff, BoolDiffOp());
#else
    BlockAdjacentDifference<bool, BLOCK_THREADS>(temp_storage->block_prim.adj_diff)
        .template FlagHeads<VEC_SIZE>(greater_than_u_diff, greater_than_u, BoolDiffOp(), 0);
#endif
    __syncthreads();

#pragma unroll
    for (uint32_t j = 0; j < VEC_SIZE; ++j) {
      if (greater_than_u_diff[j]) {
        atomicMin(&(temp_storage->sampled_id), (i * BLOCK_THREADS + tx) * VEC_SIZE + j);
      }
    }
    __syncthreads();
  }

  // update the last valid index
  int valid_index[VEC_SIZE];
#pragma unroll
  for (uint32_t j = 0; j < VEC_SIZE; ++j) {
    if (valid[j]) {
      valid_index[j] = (i * BLOCK_THREADS + tx) * VEC_SIZE + j;
    } else {
      valid_index[j] = -1;
    }
  }
  int max_valid_index =
      BlockReduce<int, BLOCK_THREADS, REDUCE_ALGORITHM>(temp_storage->block_prim.reduce_int)
          .Reduce(valid_index, hipcub::Max());
  if (tx == 0 && max_valid_index != -1) {
    temp_storage->last_valid_id = max_valid_index;
  }
  __syncthreads();
  aggregate += aggregate_local;
}

template <uint32_t BLOCK_THREADS, BlockScanAlgorithm SCAN_ALGORITHM,
          BlockReduceAlgorithm REDUCE_ALGORITHM, uint32_t VEC_SIZE, bool DETERMINISTIC,
          typename DType, typename IdType>
__global__ void ChainSpeculativeSampling(DType* draft_probs, IdType* draft_token_ids,
                                         DType* uniform_samples, DType* target_probs,
                                         IdType* output_token_ids,
                                         IdType* output_accepted_token_num,
                                         IdType* output_emitted_draft_token_num,
                                         uint32_t num_speculative_tokens, uint32_t d) {
  const uint32_t bx = blockIdx.x, tx = threadIdx.x;
  const uint32_t row_idx = bx;

  extern __shared__ __align__(
      alignof(SamplingTempStorage<BLOCK_THREADS, SCAN_ALGORITHM, REDUCE_ALGORITHM>))
      uint8_t smem_sampling[];
  auto& temp_storage = 
      reinterpret_cast<SamplingTempStorage<BLOCK_THREADS, SCAN_ALGORITHM, REDUCE_ALGORITHM>&>(
          smem_sampling);

  uint32_t pos = num_speculative_tokens;
  for (uint32_t i = 0; i < num_speculative_tokens; ++i) {
    IdType draft_id = draft_token_ids[row_idx * num_speculative_tokens + i];
    float q = target_probs[(row_idx * (num_speculative_tokens + 1) + i) * d + draft_id],
          p = draft_probs[(row_idx * num_speculative_tokens + i) * d + draft_id];
    DType u = uniform_samples[row_idx * (num_speculative_tokens + 1) + i];
    if (u * p < q) {
      // accept the draft models output
      output_token_ids[row_idx * (num_speculative_tokens + 1) + i] = draft_id;
    } else {
      pos = i;
      break;
    }
  }

  if (tx == 0) {
    output_emitted_draft_token_num[row_idx] += pos + 1;
  }

  if (pos == num_speculative_tokens) {
    return;
  }

  // sample from relu(target_probs - draft_probs)
  float sum_relu_q_minus_p = 0;
  vec_t<float, VEC_SIZE> q_vec, p_vec;
  float relu_q_minus_p[VEC_SIZE];
#pragma unroll 2
  for (uint32_t i = 0; i < ceil_div(d, BLOCK_THREADS * VEC_SIZE); ++i) {
    q_vec.fill(0);
    p_vec.fill(0);
    if ((i * BLOCK_THREADS + tx) * VEC_SIZE < d) {
      q_vec.cast_load(target_probs + (row_idx * (num_speculative_tokens + 1) + pos) * d +
                      i * BLOCK_THREADS * VEC_SIZE + tx * VEC_SIZE);
      if (pos != num_speculative_tokens) {
        // there is no draft_probs for the bonus token
        p_vec.cast_load(draft_probs + (row_idx * num_speculative_tokens + pos) * d +
                        i * BLOCK_THREADS * VEC_SIZE + tx * VEC_SIZE);
      }
    }
#pragma unroll
    for (uint32_t j = 0; j < VEC_SIZE; ++j) {
      relu_q_minus_p[j] = max(q_vec[j] - p_vec[j], 0);;
    }
    sum_relu_q_minus_p +=
        BlockReduce<float, BLOCK_THREADS, REDUCE_ALGORITHM>(temp_storage.block_prim.reduce)
            .template Sum<VEC_SIZE>(relu_q_minus_p);
    __syncthreads();
  }
  if (tx == 0) {
    temp_storage.block_aggregate.value = sum_relu_q_minus_p;
  }
  // init the first rejected token to d
  temp_storage.sampled_id = d - 1;
  __syncthreads();
  sum_relu_q_minus_p = temp_storage.block_aggregate.value;
  DType u = uniform_samples[row_idx * (num_speculative_tokens + 1) +
                            min(pos + 1, num_speculative_tokens)] * sum_relu_q_minus_p;

  float aggregate_relu_q_minus_p(0);
#pragma unroll 2
  for (uint32_t i = 0; i < ceil_div(d, BLOCK_THREADS * VEC_SIZE); ++i) {
    q_vec.fill(0);
    p_vec.fill(0);
    if ((i * BLOCK_THREADS + tx) * VEC_SIZE < d) {
      q_vec.cast_load(target_probs + (row_idx * (num_speculative_tokens + 1) + pos) * d +
                      i * BLOCK_THREADS * VEC_SIZE + tx * VEC_SIZE);
      if (pos != num_speculative_tokens) {
        // there is no draft_probs for the bonus token
        p_vec.cast_load(draft_probs + (row_idx * num_speculative_tokens + pos) * d +
                        i * BLOCK_THREADS * VEC_SIZE + tx * VEC_SIZE);
      }
    }

    vec_t<float, VEC_SIZE> relu_q_minus_p_vec;
#pragma unroll
    for (uint32_t j = 0; j < VEC_SIZE; ++j) {
      relu_q_minus_p_vec[j] = max(q_vec[j] - p_vec[j], 0);
    }

    DeviceSamplingFromProb<VEC_SIZE, BLOCK_THREADS, SCAN_ALGORITHM, REDUCE_ALGORITHM,
                           DETERMINISTIC>(
        i, d, [&](float x) { return x > 0; }, u, relu_q_minus_p_vec, aggregate_relu_q_minus_p,
        &temp_storage);
    if (aggregate_relu_q_minus_p > u) {
      break;
    }
  }
  __syncthreads();
  int sampled_id = temp_storage.sampled_id;
  // if (sampled_id == d) {
  //   // NOTE(Zihao): this would happen when u is very close to 1
  //   // and the sum of probabilities is smaller than u
  //   // In this case, we use the last valid index as the sampled id
  //   sampled_id = temp_storage.last_valid_id;
  // }
  // set the first rejected token
  output_token_ids[row_idx * (num_speculative_tokens + 1) + pos] = sampled_id;
  // move to the next token
  pos++;

  // pad remaining tokens with -1
  for (; pos < num_speculative_tokens + 1; ++pos) {
    output_token_ids[row_idx * (num_speculative_tokens + 1) + pos] = -1;
  }
}

template <typename DType, typename IdType>
hipError_t ChainSpeculativeSampling(DType* draft_probs, IdType* draft_token_ids,
                                     DType* uniform_samples, DType* target_probs,
                                     IdType* output_token_ids,
                                     IdType* output_accepted_token_num,
                                     IdType* output_emitted_draft_token_num, uint32_t batch_size,
                                     uint32_t num_speculative_tokens, uint32_t d,
                                     bool deterministic, hipStream_t stream = 0) {
  constexpr uint32_t BLOCK_THREADS = 1024;
  const uint32_t vec_size = std::gcd(16 / sizeof(DType), d);

  const uint32_t smem_size = sizeof(SamplingTempStorage<BLOCK_THREADS, SCAN_ALGO, REDUCE_ALGO>);
  dim3 nblks(batch_size);
  dim3 nthrs(BLOCK_THREADS);
  DISPATCH_ALIGNED_VEC_SIZE(
      vec_size, VEC_SIZE, {DISPATCH_DETERMINISTIC(deterministic, DETERMINISTIC, {
        ChainSpeculativeSampling<BLOCK_THREADS, SCAN_ALGO, REDUCE_ALGO, VEC_SIZE,
            DETERMINISTIC, DType, IdType><<<nblks, nthrs, smem_size, stream>>>(
          draft_probs,
          draft_token_ids,
          uniform_samples,
          target_probs,
          output_token_ids,
          output_accepted_token_num,
          output_emitted_draft_token_num,
          num_speculative_tokens,
          d);
      })});
  return hipSuccess;
}

void chain_speculative_sampling(at::Tensor draft_probs, at::Tensor draft_token_ids,
                                at::Tensor uniform_samples, at::Tensor target_probs,
                                at::Tensor output_token_ids,
                                at::Tensor output_accepted_token_num,
                                at::Tensor output_emitted_draft_token_num, bool deterministic,
                                int64_t hip_stream) {
  auto device = draft_probs.device();
  unsigned int batch_size = draft_probs.size(0);
  unsigned int num_speculate_tokens = draft_probs.size(1);
  unsigned int vocab_size = draft_probs.size(2);

  hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);
  hipError_t status = ChainSpeculativeSampling<float, int>(
      static_cast<float*>(draft_probs.data_ptr()), static_cast<int*>(draft_token_ids.data_ptr()),
      static_cast<float*>(uniform_samples.data_ptr()), static_cast<float*>(target_probs.data_ptr()),
      static_cast<int*>(output_token_ids.data_ptr()),
      static_cast<int*>(output_accepted_token_num.data_ptr()),
      static_cast<int*>(output_emitted_draft_token_num.data_ptr()), batch_size, 
      num_speculate_tokens, vocab_size, deterministic, stream);

  TORCH_CHECK(status == hipSuccess, "ChainSpeculativeSampling failed with error code " +
                                         std::string(hipGetErrorString(status)));
}
