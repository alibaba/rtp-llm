#include "rtp_llm/cpp/cuda/cutlass/cutlass_kernels/fp8_group_gemm/fp8_group_gemm.h"

namespace rtp_llm {

int32_t get_sm_version_num() {
    int32_t major_capability, minor_capability;
    hipDeviceGetAttribute(&major_capability, hipDeviceAttributeComputeCapabilityMajor, 0);
    hipDeviceGetAttribute(&minor_capability, hipDeviceAttributeComputeCapabilityMinor, 0);
    int32_t version_num = major_capability * 10 + minor_capability;
    return version_num;
}

}  // namespace rtp_llm