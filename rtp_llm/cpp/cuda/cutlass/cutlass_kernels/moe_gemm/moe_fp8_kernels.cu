#include "hip/hip_runtime.h"
#include "rtp_llm/cpp/cuda/cutlass/cutlass_kernels/moe_gemm/moe_fp8_kernels.h"
// using namespace tensorrt_llm::common;
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wstrict-aliasing"

#include "cute/tensor.hpp"
#include "cutlass/conv/convolution.h"
// Order matters here, packed_stride.hpp is missing cute and convolution includes
#include "cutlass/util/packed_stride.hpp"

#include "cutlass/array.h"
#include "cutlass/epilogue/thread/activation.h"
#include "cutlass/numeric_conversion.h"
#include "cutlass/numeric_types.h"

#include "cutlass_extensions/epilogue/thread/fused_activations.h"

#pragma GCC diagnostic pop

#include "rtp_llm/cpp/cuda/trt_utils.h"
#include "rtp_llm/cpp/cuda/cuda_type_utils.cuh"
#include "rtp_llm/cpp/cuda/launch_utils.h"
#include "rtp_llm/cpp/cuda/reduce_kernel_utils.cuh"

#ifndef CUDART_VERSION
#error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 11050)
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#include <cub/util_type.cuh>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>
#else
#include "3rdparty/hipcub/hipcub.hpp"
#include "3rdparty/cub/device/device_radix_sort.cuh"
#include "3rdparty/cub/util_type.cuh"
#endif

namespace rtp_llm {

constexpr float FP8_E4M3_MAX             = 448.0f;
const int       EXPAND_THREADS_PER_BLOCK = 256;
template<class T, class U>
__host__ __device__ constexpr static U arrayConvert(T const& input) {
    using Type = typename U::Element;
    static_assert(T::kElements == U::kElements);
    U u;
#pragma unroll
    for (int i = 0; i < U::kElements; i++) {
        u[i] = static_cast<Type>(input[i]);
    }
    return u;
}

template<typename T>
__global__ void expandInputRowsContiguousKernel(T const*      unpermuted_input,
                                                float const*  fp8_scales,
                                                T*            permuted_output,
                                                float*        permuted_output_fp8_scales,
                                                float*        unpermuted_scales,
                                                float*        permuted_scales,
                                                int const*    source_rows,
                                                int const*    permuted_src_row_to_dst,
                                                int*          src_row_to_dst,
                                                int64_t const num_rows,
                                                int64_t const cols,
                                                int64_t       k) {
    int const permuted_source_row = blockIdx.x;
    int const expanded_source_row = source_rows[permuted_source_row];
    int const expanded_dest_row   = permuted_src_row_to_dst[permuted_source_row];
    // int64_t const expanded_source_row = expanded_dest_row_to_expanded_source_row[expanded_dest_row];
    // Load 128-bits per thread
    constexpr int64_t ELEM_PER_THREAD = 128 / cutlass::sizeof_bits<T>::value;
    using DataElem                    = cutlass::Array<T, ELEM_PER_THREAD>;

    // Duplicate and permute rows
    int const source_k_rank = expanded_source_row / num_rows;
    int const source_row    = expanded_source_row % num_rows;

    if (threadIdx.x == 0) {
        assert(expanded_dest_row <= INT32_MAX);
        src_row_to_dst[expanded_source_row] = static_cast<int>(expanded_dest_row);
    }

    auto const* source_row_ptr = reinterpret_cast<DataElem const*>(unpermuted_input + (int64_t)source_row * cols);
    auto*       dest_row_ptr   = reinterpret_cast<DataElem*>(permuted_output + (int64_t)expanded_dest_row * cols);

    int64_t const start_offset     = threadIdx.x;
    int64_t const stride           = EXPAND_THREADS_PER_BLOCK;
    int64_t const num_elems_in_col = cols / ELEM_PER_THREAD;

    for (int elem_index = start_offset; elem_index < num_elems_in_col; elem_index += stride) {
        dest_row_ptr[elem_index] = source_row_ptr[elem_index];
    }

    using ScaleDataElem    = cutlass::Array<float, 4>;
    int const   scale_cols = cols / 128;
    auto const* source_fp8_scales_row_ptr =
        reinterpret_cast<ScaleDataElem const*>(fp8_scales + (int64_t)source_row * scale_cols);
    auto* dest_fp8_scales_row_ptr =
        reinterpret_cast<ScaleDataElem*>(permuted_output_fp8_scales + (int64_t)expanded_dest_row * scale_cols);

    int const scale_num_elems_in_col = scale_cols / 4;

    for (int elem_index = start_offset; elem_index < scale_num_elems_in_col; elem_index += stride) {
        dest_fp8_scales_row_ptr[elem_index] = source_fp8_scales_row_ptr[elem_index];
    }
    if (permuted_scales && threadIdx.x == 0) {
        int64_t const source_k_idx         = source_row * k + source_k_rank;
        permuted_scales[expanded_dest_row] = unpermuted_scales[source_k_idx];
    }
}

template<typename T>
__global__ void expandInputRowsContiguousKernel_V2(T const*       unpermuted_input,
                                                   float const*   fp8_scales,
                                                   T*             permuted_output,
                                                   float*         permuted_output_fp8_scales,
                                                   float*         unpermuted_scales,
                                                   float*         permuted_scales,
                                                   int const*     source_rows,
                                                   int const*     permuted_src_row_to_dst,
                                                   int*           src_row_to_dst,
                                                   int64_t const* expert_first_token_offset,
                                                   size_t         num_experts_per_node,
                                                   int64_t const  num_rows,
                                                   int64_t const  cols,
                                                   int64_t        k) {
    int const permuted_source_row = blockIdx.x;
    if (permuted_source_row >= expert_first_token_offset[num_experts_per_node]) {
        return;
    }
    int const expanded_source_row = source_rows[permuted_source_row];
    int const expanded_dest_row   = permuted_src_row_to_dst[permuted_source_row];
    // int64_t const expanded_source_row = expanded_dest_row_to_expanded_source_row[expanded_dest_row];
    // Load 128-bits per thread
    constexpr int64_t ELEM_PER_THREAD = 128 / cutlass::sizeof_bits<T>::value;
    using DataElem                    = cutlass::Array<T, ELEM_PER_THREAD>;

    // Duplicate and permute rows
    int const source_k_rank = expanded_source_row / num_rows;
    int const source_row    = expanded_source_row % num_rows;

    if (threadIdx.x == 0) {
        assert(expanded_dest_row <= INT32_MAX);
        src_row_to_dst[expanded_source_row] = static_cast<int>(expanded_dest_row);
    }

    auto const* source_row_ptr = reinterpret_cast<DataElem const*>(unpermuted_input + (int64_t)source_row * cols);
    auto*       dest_row_ptr   = reinterpret_cast<DataElem*>(permuted_output + (int64_t)expanded_dest_row * cols);

    int64_t const start_offset     = threadIdx.x;
    int64_t const stride           = EXPAND_THREADS_PER_BLOCK;
    int64_t const num_elems_in_col = cols / ELEM_PER_THREAD;

    for (int elem_index = start_offset; elem_index < num_elems_in_col; elem_index += stride) {
        dest_row_ptr[elem_index] = source_row_ptr[elem_index];
    }

    using ScaleDataElem    = cutlass::Array<float, 4>;
    int const   scale_cols = cols / 128;
    auto const* source_fp8_scales_row_ptr =
        reinterpret_cast<ScaleDataElem const*>(fp8_scales + (int64_t)source_row * scale_cols);
    auto* dest_fp8_scales_row_ptr =
        reinterpret_cast<ScaleDataElem*>(permuted_output_fp8_scales + (int64_t)expanded_dest_row * scale_cols);

    int const scale_num_elems_in_col = scale_cols / 4;

    for (int elem_index = start_offset; elem_index < scale_num_elems_in_col; elem_index += stride) {
        dest_fp8_scales_row_ptr[elem_index] = source_fp8_scales_row_ptr[elem_index];
    }
    if (permuted_scales && threadIdx.x == 0) {
        int64_t const source_k_idx         = source_row * k + source_k_rank;
        permuted_scales[expanded_dest_row] = unpermuted_scales[source_k_idx];
    }
}

template<typename T>
void expandInputRowsKernelLauncherContiguous(T const*      unpermuted_input,
                                             float const*  fp8_scales,
                                             T*            permuted_output,
                                             float*        permuted_output_fp8_scales,
                                             float*        unpermuted_scales,
                                             float*        permuted_scales,
                                             int const*    source_rows,
                                             int const*    permuted_src_row_to_dst,
                                             int*          src_row_to_dst,
                                             int64_t const num_rows,
                                             int64_t const dest_num_rows,
                                             int64_t const cols,
                                             int const     k,
                                             hipStream_t  stream) {
    int64_t const blocks  = dest_num_rows;
    int64_t const threads = EXPAND_THREADS_PER_BLOCK;
    RTP_LLM_CHECK(cols % 512 == 0);
    expandInputRowsContiguousKernel<T><<<blocks, threads, 0, stream>>>(unpermuted_input,
                                                                       fp8_scales,
                                                                       permuted_output,
                                                                       permuted_output_fp8_scales,
                                                                       unpermuted_scales,
                                                                       permuted_scales,
                                                                       source_rows,
                                                                       permuted_src_row_to_dst,
                                                                       src_row_to_dst,
                                                                       num_rows,
                                                                       cols,
                                                                       k);
}

template void expandInputRowsKernelLauncherContiguous<__hip_bfloat16>(__hip_bfloat16 const* unpermuted_input,
                                                                     float const*         fp8_scales,
                                                                     __hip_bfloat16*       permuted_output,
                                                                     float*               permuted_output_fp8_scales,
                                                                     float*               unpermuted_scales,
                                                                     float*               permuted_scales,
                                                                     int const*           source_rows,
                                                                     int const*           permuted_src_row_to_dst,
                                                                     int*                 src_row_to_dst,
                                                                     int64_t const        num_rows,
                                                                     int64_t const        dest_num_rows,
                                                                     int64_t const        cols,
                                                                     int const            k,
                                                                     hipStream_t         stream);

#ifdef ENABLE_FP8
template void expandInputRowsKernelLauncherContiguous<__hip_fp8_e4m3_fnuz>(__hip_fp8_e4m3_fnuz const* unpermuted_input,
                                                                     float const*         fp8_scales,
                                                                     __hip_fp8_e4m3_fnuz*       permuted_output,
                                                                     float*               permuted_output_fp8_scales,
                                                                     float*               unpermuted_scales,
                                                                     float*               permuted_scales,
                                                                     int const*           source_rows,
                                                                     int const*           permuted_src_row_to_dst,
                                                                     int*                 src_row_to_dst,
                                                                     int64_t const        num_rows,
                                                                     int64_t const        dest_num_rows,
                                                                     int64_t const        cols,
                                                                     int const            k,
                                                                     hipStream_t         stream);
#endif

template<typename T>
void expandInputRowsKernelLauncherContiguous_V2(T const*       unpermuted_input,
                                                float const*   fp8_scales,
                                                T*             permuted_output,
                                                float*         permuted_output_fp8_scales,
                                                float*         unpermuted_scales,
                                                float*         permuted_scales,
                                                int const*     source_rows,
                                                int const*     permuted_src_row_to_dst,
                                                int*           src_row_to_dst,
                                                int64_t const* expert_first_token_offset,
                                                size_t         num_experts_per_node,
                                                int64_t const  num_rows,
                                                int64_t const  max_num_rows,
                                                int64_t const  cols,
                                                int const      k,
                                                hipStream_t   stream) {
    int64_t const blocks  = max_num_rows;
    int64_t const threads = EXPAND_THREADS_PER_BLOCK;
    RTP_LLM_CHECK(cols % 512 == 0);
    expandInputRowsContiguousKernel_V2<T><<<blocks, threads, 0, stream>>>(unpermuted_input,
                                                                          fp8_scales,
                                                                          permuted_output,
                                                                          permuted_output_fp8_scales,
                                                                          unpermuted_scales,
                                                                          permuted_scales,
                                                                          source_rows,
                                                                          permuted_src_row_to_dst,
                                                                          src_row_to_dst,
                                                                          expert_first_token_offset,
                                                                          num_experts_per_node,
                                                                          num_rows,
                                                                          cols,
                                                                          k);
}

template void expandInputRowsKernelLauncherContiguous_V2<__hip_bfloat16>(__hip_bfloat16 const* unpermuted_input,
                                                                        float const*         fp8_scales,
                                                                        __hip_bfloat16*       permuted_output,
                                                                        float*               permuted_output_fp8_scales,
                                                                        float*               unpermuted_scales,
                                                                        float*               permuted_scales,
                                                                        int const*           source_rows,
                                                                        int const*           permuted_src_row_to_dst,
                                                                        int*                 src_row_to_dst,
                                                                        int64_t const*       expert_first_token_offset,
                                                                        size_t               num_experts_per_node,
                                                                        int64_t const        num_rows,
                                                                        int64_t const        max_num_rows,
                                                                        int64_t const        cols,
                                                                        int const            k,
                                                                        hipStream_t         stream);

#ifdef ENABLE_FP8
template void expandInputRowsKernelLauncherContiguous_V2<__hip_fp8_e4m3_fnuz>(__hip_fp8_e4m3_fnuz const* unpermuted_input,
                                                                        float const*         fp8_scales,
                                                                        __hip_fp8_e4m3_fnuz*       permuted_output,
                                                                        float*               permuted_output_fp8_scales,
                                                                        float*               unpermuted_scales,
                                                                        float*               permuted_scales,
                                                                        int const*           source_rows,
                                                                        int const*           permuted_src_row_to_dst,
                                                                        int*                 src_row_to_dst,
                                                                        int64_t const*       expert_first_token_offset,
                                                                        size_t               num_experts_per_node,
                                                                        int64_t const        num_rows,
                                                                        int64_t const        max_num_rows,
                                                                        int64_t const        cols,
                                                                        int const            k,
                                                                        hipStream_t         stream);
#endif

__global__ void computeSrc2DstKernel(int64_t const* expert_first_token_offset,
                                     int*           permuted_src_row_to_dst,
                                     int*           masked_m,
                                     size_t         padding_size) {
    size_t const expert_id  = blockIdx.x;
    size_t const tid        = threadIdx.x;
    size_t       row_offset = static_cast<size_t>(expert_first_token_offset[expert_id]);
    size_t       curr_row   = static_cast<size_t>(expert_first_token_offset[expert_id + 1]) - row_offset;

    if (tid == 0) {
        masked_m[expert_id] = static_cast<int>(curr_row);
    }

    size_t padding_offset = expert_id * padding_size;
    for (size_t i = tid; i < curr_row; i += blockDim.x) {
        permuted_src_row_to_dst[row_offset + i] = padding_offset + i;
    }
}

void computeSrc2Dst(int64_t const* expert_first_token_offset,
                    int*           permuted_src_row_to_dst,
                    int*           masked_m,
                    size_t         num_experts_per_node,
                    size_t         padding_size,
                    hipStream_t   stream) {
    int64_t const blocks  = num_experts_per_node;
    int64_t const threads = 512;
    computeSrc2DstKernel<<<blocks, threads, 0, stream>>>(
        expert_first_token_offset, permuted_src_row_to_dst, masked_m, padding_size);
}

const int FINALIZE_THREADS_PER_BLOCK = 256;

// ============================== Gated Activation =================================
constexpr static int ACTIVATION_THREADS_PER_BLOCK = 256;

template<class GemmOutputType, class ScaleBiasType, template<class> class ActFn>
__global__ void doActivationContiguousKernel(__hip_fp8_e4m3_fnuz*        output_fp8,
                                             float*                output_fp8_scale,
                                             GemmOutputType const* gemm_result,
                                             ScaleBiasType const*  bias_ptr,
                                             bool                  bias_is_broadcast,
                                             int const*            src_row_to_dst,
                                             int64_t               inter_size,
                                             bool                  gated,
                                             int const*            permuted_experts) {
    int64_t const tid                 = threadIdx.x;
    int64_t const expanded_source_row = blockIdx.x;
    int const     expert              = permuted_experts[expanded_source_row];
    int64_t const token               = src_row_to_dst[expanded_source_row];
    if (token < 0) {
        return;
    }

    size_t gated_size_mul = gated ? 2 : 1;
    size_t gated_off      = gated ? inter_size : 0;

    gemm_result      = gemm_result + token * inter_size * gated_size_mul;
    output_fp8       = output_fp8 + token * inter_size;  // Aliases gemm_result for non-gated, non-fp8 cases
    output_fp8_scale = output_fp8_scale + token * inter_size / 128;
    // int64_t expert = source_k_rank;
    // if (bias_ptr)
    // {
    //     // TODO this is almost certainly faster as a linear scan
    //     expert = findTotalEltsLessThanTarget(expert_first_token_offset, num_experts, (int64_t) token + 1) - 1;
    // }

    if (bias_ptr) {
        size_t bias_offset =
            (bias_is_broadcast ? expert * inter_size * gated_size_mul : token * inter_size * gated_size_mul);
        bias_ptr = bias_ptr + bias_offset;
    }

    // Load 128-bits per thread, according to the smallest data type we read/write
    // constexpr int64_t ACTIVATION_ELEM_PER_THREAD
    //     = 128 / std::min(cutlass::sizeof_bits<T>::value, cutlass::sizeof_bits<GemmOutputType>::value);
    constexpr int64_t ACTIVATION_ELEM_PER_THREAD = 4;

    using BiasElem                = cutlass::Array<ScaleBiasType, ACTIVATION_ELEM_PER_THREAD>;
    using GemmResultElem          = cutlass::Array<GemmOutputType, ACTIVATION_ELEM_PER_THREAD>;
    using OutputElem              = cutlass::Array<__hip_fp8_e4m3_fnuz, ACTIVATION_ELEM_PER_THREAD>;
    using ComputeElem             = cutlass::Array<float, ACTIVATION_ELEM_PER_THREAD>;
    auto          gemm_result_vec = reinterpret_cast<GemmResultElem const*>(gemm_result);
    auto          output_vec      = reinterpret_cast<OutputElem*>(output_fp8);
    auto          bias_ptr_vec    = reinterpret_cast<BiasElem const*>(bias_ptr);
    int64_t const start_offset    = tid;
    int64_t const stride          = ACTIVATION_THREADS_PER_BLOCK;
    assert(inter_size % ACTIVATION_ELEM_PER_THREAD == 0);
    int64_t const num_elems_in_col = inter_size / ACTIVATION_ELEM_PER_THREAD;
    assert(gated_off % ACTIVATION_ELEM_PER_THREAD == 0);
    int64_t const gated_off_vec = gated_off / ACTIVATION_ELEM_PER_THREAD;

    ActFn<ComputeElem> fn{};
    for (int64_t elem_index = start_offset; elem_index < num_elems_in_col; elem_index += stride) {
        auto fc1_value = arrayConvert<GemmResultElem, ComputeElem>(gemm_result_vec[elem_index + gated_off_vec]);
        if (bias_ptr) {
            fc1_value = fc1_value + arrayConvert<BiasElem, ComputeElem>(bias_ptr_vec[elem_index + gated_off_vec]);
        }

        auto gate_act = fn(fc1_value);

        if (gated) {
            auto gate_mul = arrayConvert<GemmResultElem, ComputeElem>(gemm_result_vec[elem_index]);
            if (bias_ptr_vec) {
                gate_mul = gate_mul + arrayConvert<BiasElem, ComputeElem>(bias_ptr_vec[elem_index]);
            }
            gate_act = gate_act * gate_mul;
        }
        // plus<Array<T, N>> op;
        cutlass::maximum_absolute_value_reduction<ComputeElem, false> max_abs_op;
        float                                                         scale = max_abs_op((float)1e-4, gate_act);
        scale                                                               = warpReduceMax<float>(scale);
        scale                                                               = scale / FP8_E4M3_MAX;
        gate_act                                                            = gate_act * (1 / scale);
        output_vec[elem_index] = arrayConvert<ComputeElem, OutputElem>(gate_act);
        if (tid % 32 == 0) {
            output_fp8_scale[elem_index / 32] = scale;
        }
    }
}

template<class GemmOutputType, class ScaleBiasType, template<class> class ActFn>
__global__ void doActivationContiguousKernel_V2(__hip_fp8_e4m3_fnuz*        output_fp8,
                                                float*                output_fp8_scale,
                                                GemmOutputType const* gemm_result,
                                                ScaleBiasType const*  bias_ptr,
                                                bool                  bias_is_broadcast,
                                                int const*            src_row_to_dst,
                                                int64_t const*        expert_first_token_offset,
                                                size_t                num_experts_per_node,
                                                int64_t               inter_size,
                                                bool                  gated,
                                                int const*            permuted_experts) {
    int64_t const tid                 = threadIdx.x;
    int64_t const expanded_source_row = blockIdx.x;
    if (expanded_source_row >= expert_first_token_offset[num_experts_per_node]) {
        return;
    }
    int const     expert = permuted_experts[expanded_source_row];
    int64_t const token  = src_row_to_dst[expanded_source_row];
    if (token < 0) {
        return;
    }

    size_t gated_size_mul = gated ? 2 : 1;
    size_t gated_off      = gated ? inter_size : 0;

    gemm_result      = gemm_result + token * inter_size * gated_size_mul;
    output_fp8       = output_fp8 + token * inter_size;  // Aliases gemm_result for non-gated, non-fp8 cases
    output_fp8_scale = output_fp8_scale + token * inter_size / 128;
    // int64_t expert = source_k_rank;
    // if (bias_ptr)
    // {
    //     // TODO this is almost certainly faster as a linear scan
    //     expert = findTotalEltsLessThanTarget(expert_first_token_offset, num_experts, (int64_t) token + 1) - 1;
    // }

    if (bias_ptr) {
        size_t bias_offset =
            (bias_is_broadcast ? expert * inter_size * gated_size_mul : token * inter_size * gated_size_mul);
        bias_ptr = bias_ptr + bias_offset;
    }

    // Load 128-bits per thread, according to the smallest data type we read/write
    // constexpr int64_t ACTIVATION_ELEM_PER_THREAD
    //     = 128 / std::min(cutlass::sizeof_bits<T>::value, cutlass::sizeof_bits<GemmOutputType>::value);
    constexpr int64_t ACTIVATION_ELEM_PER_THREAD = 4;

    using BiasElem                = cutlass::Array<ScaleBiasType, ACTIVATION_ELEM_PER_THREAD>;
    using GemmResultElem          = cutlass::Array<GemmOutputType, ACTIVATION_ELEM_PER_THREAD>;
    using OutputElem              = cutlass::Array<__hip_fp8_e4m3_fnuz, ACTIVATION_ELEM_PER_THREAD>;
    using ComputeElem             = cutlass::Array<float, ACTIVATION_ELEM_PER_THREAD>;
    auto          gemm_result_vec = reinterpret_cast<GemmResultElem const*>(gemm_result);
    auto          output_vec      = reinterpret_cast<OutputElem*>(output_fp8);
    auto          bias_ptr_vec    = reinterpret_cast<BiasElem const*>(bias_ptr);
    int64_t const start_offset    = tid;
    int64_t const stride          = ACTIVATION_THREADS_PER_BLOCK;
    assert(inter_size % ACTIVATION_ELEM_PER_THREAD == 0);
    int64_t const num_elems_in_col = inter_size / ACTIVATION_ELEM_PER_THREAD;
    assert(gated_off % ACTIVATION_ELEM_PER_THREAD == 0);
    int64_t const gated_off_vec = gated_off / ACTIVATION_ELEM_PER_THREAD;

    ActFn<ComputeElem> fn{};
    for (int64_t elem_index = start_offset; elem_index < num_elems_in_col; elem_index += stride) {
        auto fc1_value = arrayConvert<GemmResultElem, ComputeElem>(gemm_result_vec[elem_index + gated_off_vec]);
        if (bias_ptr) {
            fc1_value = fc1_value + arrayConvert<BiasElem, ComputeElem>(bias_ptr_vec[elem_index + gated_off_vec]);
        }

        auto gate_act = fn(fc1_value);

        if (gated) {
            auto gate_mul = arrayConvert<GemmResultElem, ComputeElem>(gemm_result_vec[elem_index]);
            if (bias_ptr_vec) {
                gate_mul = gate_mul + arrayConvert<BiasElem, ComputeElem>(bias_ptr_vec[elem_index]);
            }
            gate_act = gate_act * gate_mul;
        }
        // plus<Array<T, N>> op;
        cutlass::maximum_absolute_value_reduction<ComputeElem, false> max_abs_op;
        float                                                         scale = max_abs_op((float)1e-4, gate_act);
        scale                                                               = warpReduceMax<float>(scale);
        scale                                                               = scale / FP8_E4M3_MAX;
        gate_act                                                            = gate_act * (1 / scale);
        output_vec[elem_index] = arrayConvert<ComputeElem, OutputElem>(gate_act);
        if (tid % 32 == 0) {
            output_fp8_scale[elem_index / 32] = scale;
        }
    }
}

template<class GemmOutputType, class ScaleBiasType, template<class> class ActFn>
__global__ void doActivationMaskedKernel(__hip_fp8_e4m3_fnuz*        output_fp8,
                                         float*                output_fp8_scale,
                                         GemmOutputType const* gemm_result,
                                         ScaleBiasType const*  bias_ptr,
                                         bool                  bias_is_broadcast,
                                         int64_t               token_num,
                                         int64_t               inter_size,
                                         bool                  gated,
                                         int const*            masked_m) {
    int64_t const tid          = threadIdx.x;
    const int     batch_idx    = blockIdx.x;
    const int64_t batch_stride = gridDim.y;
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.wait;");
#endif
    int const max_token = masked_m[batch_idx];
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.wait;");
#endif
    size_t gated_size_mul = gated ? 2 : 1;
    size_t gated_off      = gated ? inter_size : 0;
    gemm_result           = gemm_result + batch_idx * token_num * inter_size * gated_size_mul;
    output_fp8 = output_fp8 + batch_idx * token_num * inter_size;  // Aliases gemm_result for non-gated, non-fp8 cases
    output_fp8_scale                             = output_fp8_scale + batch_idx * token_num * inter_size / 128;
    constexpr int64_t ACTIVATION_ELEM_PER_THREAD = 8;

    using BiasElem                 = cutlass::Array<ScaleBiasType, ACTIVATION_ELEM_PER_THREAD>;
    using GemmResultElem           = cutlass::Array<GemmOutputType, ACTIVATION_ELEM_PER_THREAD>;
    using OutputElem               = cutlass::Array<__hip_fp8_e4m3_fnuz, ACTIVATION_ELEM_PER_THREAD>;
    using ComputeElem              = cutlass::Array<float, ACTIVATION_ELEM_PER_THREAD>;
    int64_t const start_offset     = tid;
    int64_t const stride           = ACTIVATION_THREADS_PER_BLOCK;
    int64_t const num_elems_in_col = inter_size / ACTIVATION_ELEM_PER_THREAD;
    assert(inter_size % ACTIVATION_ELEM_PER_THREAD == 0);
    assert(gated_off % ACTIVATION_ELEM_PER_THREAD == 0);
    int64_t const gated_off_vec = gated_off / ACTIVATION_ELEM_PER_THREAD;
    for (int token_idx = blockIdx.y; token_idx < max_token; token_idx += batch_stride) {
        auto gemm_result_vec =
            reinterpret_cast<GemmResultElem const*>(gemm_result + token_idx * inter_size * gated_size_mul);
        auto output_vec = reinterpret_cast<OutputElem*>(output_fp8 + token_idx * inter_size);

        ActFn<ComputeElem> fn{};
#pragma unroll 1
        for (int64_t elem_index = start_offset; elem_index < num_elems_in_col; elem_index += stride) {
            auto fc1_value = arrayConvert<GemmResultElem, ComputeElem>(gemm_result_vec[elem_index + gated_off_vec]);
            auto gate_act  = fn(fc1_value);
            if (gated) {
                auto gate_mul = arrayConvert<GemmResultElem, ComputeElem>(gemm_result_vec[elem_index]);
                gate_act      = gate_act * gate_mul;
            }
            // plus<Array<T, N>> op;
            cutlass::maximum_absolute_value_reduction<ComputeElem, false> max_abs_op;
            float                                                         scale = max_abs_op((float)1e-4, gate_act);
            static constexpr int THREADS_PER_ROW                                = 128 / ACTIVATION_ELEM_PER_THREAD;
#pragma unroll
            for (int mask = THREADS_PER_ROW / 2; mask > 0; mask /= 2) {
                scale = max(scale, __shfl_xor_sync(0xFFFFFFFF, scale, mask, THREADS_PER_ROW));
            }
            scale                  = scale / FP8_E4M3_MAX;
            gate_act               = gate_act * (1 / scale);
            output_vec[elem_index] = arrayConvert<ComputeElem, OutputElem>(gate_act);
            if (tid % THREADS_PER_ROW == 0) {
                const int64_t now_idx                             = elem_index / THREADS_PER_ROW;
                output_fp8_scale[now_idx * token_num + token_idx] = scale;
            }
        }
    }
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.launch_dependents;");
#endif
}

template<class GemmOutputType, class ScaleBiasType>
void doActivationContiguous(__hip_fp8_e4m3_fnuz*        output_fp8,
                            float*                fp8_scale,
                            GemmOutputType const* gemm_result,
                            ScaleBiasType const*  bias,
                            bool                  bias_is_broadcast,
                            int const*            src_row_to_dst,
                            int                   num_rows,
                            int64_t               inter_size,
                            ActivationType        activation_type,
                            int const*            permuted_experts,
                            hipStream_t          stream) {
    int64_t const blocks  = num_rows;
    int64_t const threads = ACTIVATION_THREADS_PER_BLOCK;
    RTP_LLM_CHECK(inter_size % 128 == 0);

    auto fn_list = std::array{
        &doActivationContiguousKernel<GemmOutputType, ScaleBiasType, cutlass::epilogue::thread::GELU>,     // Gelu
        &doActivationContiguousKernel<GemmOutputType, ScaleBiasType, cutlass::epilogue::thread::ReLu>,     // Relu
        &doActivationContiguousKernel<GemmOutputType, ScaleBiasType, cutlass::epilogue::thread::SiLu>,     // Silu
        &doActivationContiguousKernel<GemmOutputType, ScaleBiasType, cutlass::epilogue::thread::SiLu>,     // Swiglu
        &doActivationContiguousKernel<GemmOutputType, ScaleBiasType, cutlass::epilogue::thread::GELU>,     // Geglu
        &doActivationContiguousKernel<GemmOutputType, ScaleBiasType, cutlass::epilogue::thread::Identity>  // Identity
    };
    auto fn = fn_list[static_cast<int>(activation_type)];
    fn<<<blocks, threads, 0, stream>>>(output_fp8,
                                       fp8_scale,
                                       gemm_result,
                                       bias,
                                       bias_is_broadcast,
                                       src_row_to_dst,
                                       inter_size,
                                       isGatedActivation(activation_type),
                                       permuted_experts);
}

template void doActivationContiguous<__hip_bfloat16, __hip_bfloat16>(__hip_fp8_e4m3_fnuz*       output_fp8,
                                                                   float*               fp8_scale,
                                                                   __hip_bfloat16 const* gemm_result,
                                                                   __hip_bfloat16 const* bias,
                                                                   bool                 bias_is_broadcast,
                                                                   int const*           src_row_to_dst,
                                                                   int                  num_rows,
                                                                   int64_t              inter_size,
                                                                   ActivationType       activation_type,
                                                                   int const*           permuted_experts,
                                                                   hipStream_t         stream);

template<class GemmOutputType, class ScaleBiasType>
void doActivationContiguous_V2(__hip_fp8_e4m3_fnuz*        output_fp8,
                               float*                fp8_scale,
                               GemmOutputType const* gemm_result,
                               ScaleBiasType const*  bias,
                               bool                  bias_is_broadcast,
                               int const*            src_row_to_dst,
                               int64_t const*        expert_first_token_offset,
                               size_t                num_experts_per_node,
                               int                   max_num_rows,
                               int64_t               inter_size,
                               ActivationType        activation_type,
                               int const*            permuted_experts,
                               hipStream_t          stream) {
    int64_t const blocks  = max_num_rows;
    int64_t const threads = ACTIVATION_THREADS_PER_BLOCK;
    RTP_LLM_CHECK(inter_size % 128 == 0);

    auto fn_list = std::array{
        &doActivationContiguousKernel_V2<GemmOutputType, ScaleBiasType, cutlass::epilogue::thread::GELU>,  // Gelu
        &doActivationContiguousKernel_V2<GemmOutputType, ScaleBiasType, cutlass::epilogue::thread::ReLu>,  // Relu
        &doActivationContiguousKernel_V2<GemmOutputType, ScaleBiasType, cutlass::epilogue::thread::SiLu>,  // Silu
        &doActivationContiguousKernel_V2<GemmOutputType, ScaleBiasType, cutlass::epilogue::thread::SiLu>,  // Swiglu
        &doActivationContiguousKernel_V2<GemmOutputType, ScaleBiasType, cutlass::epilogue::thread::GELU>,  // Geglu
        &doActivationContiguousKernel_V2<GemmOutputType,
                                         ScaleBiasType,
                                         cutlass::epilogue::thread::Identity>  // Identity
    };
    auto fn = fn_list[static_cast<int>(activation_type)];
    fn<<<blocks, threads, 0, stream>>>(output_fp8,
                                       fp8_scale,
                                       gemm_result,
                                       bias,
                                       bias_is_broadcast,
                                       src_row_to_dst,
                                       expert_first_token_offset,
                                       num_experts_per_node,
                                       inter_size,
                                       isGatedActivation(activation_type),
                                       permuted_experts);
}

template void doActivationContiguous_V2<__hip_bfloat16, __hip_bfloat16>(__hip_fp8_e4m3_fnuz*       output_fp8,
                                                                      float*               fp8_scale,
                                                                      __hip_bfloat16 const* gemm_result,
                                                                      __hip_bfloat16 const* bias,
                                                                      bool                 bias_is_broadcast,
                                                                      int const*           src_row_to_dst,
                                                                      int64_t const*       expert_first_token_offset,
                                                                      size_t               num_experts_per_node,
                                                                      int                  max_num_rows,
                                                                      int64_t              inter_size,
                                                                      ActivationType       activation_type,
                                                                      int const*           permuted_experts,
                                                                      hipStream_t         stream);

template<class GemmOutputType, class ScaleBiasType>
void doActivationMasked(__hip_fp8_e4m3_fnuz*        output_fp8,
                        float*                fp8_scale,
                        GemmOutputType const* gemm_result,
                        ScaleBiasType const*  bias,
                        bool                  bias_is_broadcast,
                        int                   expert_num,
                        int                   token_num,
                        int64_t               inter_size,
                        ActivationType        activation_type,
                        int const*            masked_m,
                        hipStream_t          stream) {
    int64_t const threads = ACTIVATION_THREADS_PER_BLOCK;
    RTP_LLM_CHECK(inter_size % 128 == 0);
    RTP_LLM_CHECK(bias == nullptr);
    int  token_stride = 64;
    dim3 grid(expert_num, token_stride);
    dim3 thread(threads);

    auto fn_list = std::array{
        &doActivationMaskedKernel<GemmOutputType, ScaleBiasType, cutlass::epilogue::thread::GELU>,     // Gelu
        &doActivationMaskedKernel<GemmOutputType, ScaleBiasType, cutlass::epilogue::thread::ReLu>,     // Relu
        &doActivationMaskedKernel<GemmOutputType, ScaleBiasType, cutlass::epilogue::thread::SiLu>,     // Silu
        &doActivationMaskedKernel<GemmOutputType, ScaleBiasType, cutlass::epilogue::thread::SiLu>,     // Swiglu
        &doActivationMaskedKernel<GemmOutputType, ScaleBiasType, cutlass::epilogue::thread::GELU>,     // Geglu
        &doActivationMaskedKernel<GemmOutputType, ScaleBiasType, cutlass::epilogue::thread::Identity>  // Identity
    };
    auto fn = fn_list[static_cast<int>(activation_type)];
    LAUNCH_KERNEL_WITH_PDL(*fn,
                           grid,
                           thread,
                           0,
                           stream,
                           output_fp8,
                           fp8_scale,
                           gemm_result,
                           bias,
                           bias_is_broadcast,
                           token_num,
                           inter_size,
                           isGatedActivation(activation_type),
                           masked_m);
}

template void doActivationMasked<__hip_bfloat16, __hip_bfloat16>(__hip_fp8_e4m3_fnuz*       output_fp8,
                                                               float*               fp8_scale,
                                                               __hip_bfloat16 const* gemm_result,
                                                               __hip_bfloat16 const* bias,
                                                               bool                 bias_is_broadcast,
                                                               int                  expert_num,
                                                               int                  token_num,
                                                               int64_t              inter_size,
                                                               ActivationType       activation_type,
                                                               int const*           masked_m,
                                                               hipStream_t         stream);

}  // namespace rtp_llm
