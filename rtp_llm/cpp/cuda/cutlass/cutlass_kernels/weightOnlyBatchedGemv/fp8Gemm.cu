#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cutlass/numeric_conversion.h"
#include "rtp_llm/cpp/cuda/cutlass/cutlass_kernels/weightOnlyBatchedGemv/fp8Gemm.h"
#include <hipcub/hipcub.hpp>

namespace tensorrt_llm
{
namespace kernels
{
namespace fp8_gemm
{
template <typename InputType, typename OutputType, SizeType32 TILE_M, SizeType32 TILE_N, SizeType32 BLOCK_SIZE>
__global__ void fp8Gemm(InputType const* __restrict__ act, InputType const* __restrict__ weight, float alpha,
    OutputType* __restrict__ output, SizeType32 m, SizeType32 n, SizeType32 k)
{
    using VecType = int4;
    static constexpr SizeType32 kStepK = static_cast<SizeType32>(128 / (8 * sizeof(InputType)));
    static constexpr SizeType32 kTileK = kStepK * BLOCK_SIZE;
    auto tileIdM = static_cast<SizeType32>(blockIdx.x * TILE_M);
    auto tileIdN = static_cast<SizeType32>(blockIdx.y * TILE_N);
    auto tid = static_cast<SizeType32>(threadIdx.x);
    float tile_a[kStepK], tile_w[TILE_N * kStepK];
    float acc[TILE_M * TILE_N];

    static_assert(kStepK % 4 == 0);
    using CvtInputType
        = std::conditional_t<std::is_same_v<InputType, __hip_fp8_e4m3_fnuz>, cutlass::float_e4m3_t, cutlass::float_e5m2_t>;
    using Converter = cutlass::NumericArrayConverter<float, CvtInputType, 4>;
    using CvtSrcType = typename Converter::source_type;
    using CvtResType = typename Converter::result_type;
    static constexpr SizeType32 kCvtCount = static_cast<SizeType32>(sizeof(VecType) / sizeof(CvtSrcType));

#pragma unroll
    for (SizeType32 i = 0; i < TILE_M * TILE_N; ++i)
    {
        acc[i] = 0;
    }
    act += tileIdM * k;
    weight += tileIdN * k;
    output += tileIdM * n + tileIdN;
    for (SizeType32 idxK = tid * kStepK; idxK < k; idxK += kTileK)
    {
#pragma unroll
        for (SizeType32 i = 0; i < TILE_N; ++i)
        {
            auto tile_w_quantized = reinterpret_cast<VecType const*>(weight + i * k + idxK)[0];
#pragma unroll
            for (SizeType32 cvtIdx = 0; cvtIdx < kCvtCount; ++cvtIdx)
            {
                reinterpret_cast<CvtResType*>(tile_w)[i * kCvtCount + cvtIdx]
                    = Converter::convert(reinterpret_cast<CvtSrcType*>(&tile_w_quantized)[cvtIdx]);
            }
        }
#pragma unroll
        for (SizeType32 i = 0; i < TILE_M; ++i)
        {
            auto tile_a_quantized = reinterpret_cast<VecType const*>(act + i * k + idxK)[0];
#pragma unroll
            for (SizeType32 cvtIdx = 0; cvtIdx < kCvtCount; ++cvtIdx)
            {
                reinterpret_cast<CvtResType*>(tile_a)[cvtIdx]
                    = Converter::convert(reinterpret_cast<CvtSrcType*>(&tile_a_quantized)[cvtIdx]);
            }
#pragma unroll
            for (SizeType32 j = 0; j < TILE_N; ++j)
            {
#pragma unroll
                for (SizeType32 l = 0; l < kStepK; ++l)
                {
                    acc[i * TILE_N + j] = fma(tile_a[l], tile_w[j * kStepK + l], acc[i * TILE_N + j]);
                }
            }
        }
    }

    typedef hipcub::WarpReduce<float> WarpReduce;

    static constexpr SizeType32 kWarpSize = 32;
    static constexpr SizeType32 kWarpNum = BLOCK_SIZE / kWarpSize;
    SizeType32 warpId = tid / kWarpSize, laneId = tid % kWarpSize;
    __shared__ float shmem[TILE_M * TILE_N * kWarpNum];
    __shared__ typename WarpReduce::TempStorage tempStorage[kWarpNum];
#pragma unroll
    for (SizeType32 mi = 0; mi < TILE_M; ++mi)
    {
#pragma unroll
        for (SizeType32 ni = 0; ni < TILE_N; ++ni)
        {
            float val = WarpReduce(tempStorage[warpId]).Sum(acc[mi * TILE_N + ni]);
            if (laneId == 0)
            {
                shmem[mi * TILE_N + ni + warpId * TILE_M * TILE_N] = val;
            }
        }
    }
    __syncthreads();
#pragma unroll
    for (SizeType32 ii = tid; ii < TILE_M * TILE_N; ii += BLOCK_SIZE)
    {
        SizeType32 mid = ii / TILE_N, nid = ii % TILE_N;
        float val = 0;
#pragma unroll
        for (SizeType32 jj = 0; jj < kWarpNum; ++jj)
        {
            val += shmem[jj * TILE_M * TILE_N + ii];
        }
        output[mid * n + nid] = static_cast<OutputType>(val * alpha);
    }
}

template <typename InputType, typename OutputType, SizeType32 TILE_M, SizeType32 TILE_N, SizeType32 BLOCK_SIZE>
void fp8GemmKernel(Params& params, hipStream_t stream)
{
    dim3 block(BLOCK_SIZE);
    dim3 grid(params.m / TILE_M, params.n / TILE_N);
    fp8Gemm<InputType, OutputType, TILE_M, TILE_N, BLOCK_SIZE><<<grid, block, 0, stream>>>(
        reinterpret_cast<InputType const*>(params.act), reinterpret_cast<InputType const*>(params.weight), params.alpha,
        reinterpret_cast<OutputType*>(params.output), params.m, params.n, params.k);
}

template <typename InputType, typename OutputType>
void fp8GemmLauncher(Params& params, hipStream_t stream)
{
#define DISPATCH(TargetM, TILE_M, TILE_N, BLOCK_SIZE)                                                                  \
    if (params.m == TargetM)                                                                                           \
    {                                                                                                                  \
        fp8GemmKernel<InputType, OutputType, TILE_M, TILE_N, BLOCK_SIZE>(params, stream);                              \
        return;                                                                                                        \
    }
    DISPATCH(1, 1, 2, 128);
    DISPATCH(2, 2, 2, 128);
    DISPATCH(3, 3, 2, 128);
    DISPATCH(4, 4, 2, 128);
#undef DISPATCH
}

template void fp8GemmLauncher<__hip_fp8_e4m3_fnuz, float>(Params& params, hipStream_t stream);
template void fp8GemmLauncher<__hip_fp8_e4m3_fnuz, half>(Params& params, hipStream_t stream);
template void fp8GemmLauncher<__hip_fp8_e4m3_fnuz, __hip_bfloat16>(Params& params, hipStream_t stream);
} // namespace fp8_gemm
} // namespace kernels
} // namespace tensorrt_llm
