#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "rtp_llm/cpp/cuda/cuda_type_utils.cuh"
#include "rtp_llm/cpp/utils/Logger.h"
#include "rtp_llm/cpp/cuda/memory_utils.h"

namespace rtp_llm {

template<typename T>
void deviceMalloc(T** ptr, size_t size, bool is_random_initialize) {
    // RTP_LLM_CHECK_WITH_INFO(size >= ((size_t)0), "Ask deviceMalloc size " + std::to_string(size) + "< 0 is
    // invalid.");
    check_cuda_value(hipMalloc((void**)(ptr), sizeof(T) * size));
    if (is_random_initialize) {
        // Random initialization removed - function was unused
        check_cuda_value(hipMemset(static_cast<void*>(*ptr), 0, sizeof(T) * size));
    }
}

template void deviceMalloc(float** ptr, size_t size, bool is_random_initialize);
template void deviceMalloc(half** ptr, size_t size, bool is_random_initialize);
#ifdef ENABLE_BF16
template void deviceMalloc(__hip_bfloat16** ptr, size_t size, bool is_random_initialize);
#endif
template void deviceMalloc(uint16_t** ptr, size_t size, bool is_random_initialize);
template void deviceMalloc(int** ptr, size_t size, bool is_random_initialize);
template void deviceMalloc(bool** ptr, size_t size, bool is_random_initialize);
template void deviceMalloc(char** ptr, size_t size, bool is_random_initialize);
template void deviceMalloc(int8_t** ptr, size_t size, bool is_random_initialize);
template void deviceMalloc(uint8_t** ptr, size_t size, bool is_random_initialize);
#ifdef ENABLE_FP8
template void deviceMalloc(__hip_fp8_e4m3_fnuz** ptr, size_t size, bool is_random_initialize);
#endif

template<typename T>
void deviceFree(T*& ptr) {
    if (ptr != NULL) {
        check_cuda_value(hipFree(ptr));
        ptr = NULL;
    }
}

template void deviceFree(float*& ptr);
template void deviceFree(half*& ptr);
#ifdef ENABLE_BF16
template void deviceFree(__hip_bfloat16*& ptr);
#endif
template void deviceFree(unsigned short*& ptr);
template void deviceFree(int*& ptr);
template void deviceFree(bool*& ptr);
template void deviceFree(char*& ptr);
template void deviceFree(int8_t*& ptr);
template void deviceFree(uint8_t*& ptr);
#ifdef ENABLE_FP8
template void deviceFree(__hip_fp8_e4m3_fnuz*& ptr);
#endif

}  // namespace rtp_llm
