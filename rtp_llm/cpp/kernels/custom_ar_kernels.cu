#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "custom_ar_kernels.h"
#include "rtp_llm/cpp/cuda/cuda_type_utils.cuh"

#if USING_ROCM
#include "rtp_llm/cpp/rocm/cuda_shims.h"
#endif

#include <cassert>
#include <cstddef>

namespace rtp_llm {

typedef struct bf168 {
    __hip_bfloat162 x;
    __hip_bfloat162 y;
    __hip_bfloat162 z;
    __hip_bfloat162 w;
} bf168;

////////////////////////////////////////////////////////////////////////////////////////////////////

static inline __device__ uint32_t hadd2(const uint32_t& a, const uint32_t& b) {
#if USING_ROCM
    __half2 out = __hadd2(*reinterpret_cast<const __half2_raw*>(&a), *reinterpret_cast<const __half2_raw*>(&b));
    return *reinterpret_cast<uint32_t*>(&(out.data));
#else
    uint32_t c;
    asm volatile("add.f16x2 %0, %1, %2;\n" : "=r"(c) : "r"(a), "r"(b));
    return c;
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////

static inline __device__ uint32_t fadd(const uint32_t& a, const uint32_t& b) {
    uint32_t c;
#if USING_ROCM
    c = __float_as_uint(__uint_as_float(a) + __uint_as_float(b));
#else
    asm volatile("add.f32 %0, %1, %2;\n" : "=r"(c) : "r"(a), "r"(b));
#endif
    return c;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

static inline __device__ void st_flag_release(uint32_t const& flag, uint32_t* flag_addr) {
#if USING_ROCM
    __atomic_store((__attribute__((address_space(1))) uint32_t*)flag_addr,
                   (__attribute__((address_space(1))) uint32_t*)&flag,
                   __ATOMIC_RELEASE);
#else
#if __CUDA_ARCH__ >= 700
    asm volatile("st.global.release.sys.b32 [%1], %0;" ::"r"(flag), "l"(flag_addr));
#else
    __threadfence_system();
    asm volatile("st.global.volatile.b32 [%1], %0;" ::"r"(flag), "l"(flag_addr));
#endif
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////

static inline __device__ uint32_t ld_flag_acquire(uint32_t* flag_addr) {

    uint32_t flag;
#if USING_ROCM
    __atomic_load((__attribute__((address_space(1))) uint32_t*)flag_addr, &flag, __ATOMIC_ACQUIRE);
#else
#if __CUDA_ARCH__ >= 700
    asm volatile("ld.global.acquire.sys.b32 %0, [%1];" : "=r"(flag) : "l"(flag_addr));
#else
    asm volatile("ld.global.volatile.b32 %0, [%1];" : "=r"(flag) : "l"(flag_addr));
#endif
#endif
    return flag;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

// Type Converter that packs data format to 128 bits data type
template<typename T>
struct ARTypeConverter {
    using Type = uint4;
};

template<>
struct ARTypeConverter<__hip_bfloat16> {
    using Type = bf168;
};

// add two 128b data
template<typename T_IN, typename T_COMP>
inline __device__ T_IN add128b(T_IN a, T_IN b);

template<>
inline __device__ uint4 add128b<uint4, half>(uint4 a, uint4 b) {
    uint4 c;
    c.x = hadd2(a.x, b.x);
    c.y = hadd2(a.y, b.y);
    c.z = hadd2(a.z, b.z);
    c.w = hadd2(a.w, b.w);
    return c;
}

template<>
inline __device__ uint4 add128b<uint4, float>(uint4 a, uint4 b) {
    uint4 c;
    c.x = fadd(a.x, b.x);
    c.y = fadd(a.y, b.y);
    c.z = fadd(a.z, b.z);
    c.w = fadd(a.w, b.w);
    return c;
}

#ifdef ENABLE_BF16
template<>
inline __device__ bf168 add128b<bf168, __hip_bfloat16>(bf168 a, bf168 b) {
    bf168 c;
    c.x = bf16hadd2(a.x, b.x);
    c.y = bf16hadd2(a.y, b.y);
    c.z = bf16hadd2(a.z, b.z);
    c.w = bf16hadd2(a.w, b.w);
    return c;
}
#endif

// init 128bits data with 0
template<typename T>
inline __device__ T init_packed_type();

template<>
inline __device__ uint4 init_packed_type() {
    return make_uint4(0u, 0u, 0u, 0u);
}

template<>
inline __device__ bf168 init_packed_type() {
    bf168  val;
    uint4& val_u = reinterpret_cast<uint4&>(val);
    val_u        = make_uint4(0u, 0u, 0u, 0u);
    return val;
}

__inline__ __device__ void multi_gpu_barrier(uint32_t**     peer_barrier_ptrs,
                                             uint32_t const barrier_flag,
                                             size_t const   local_rank,
                                             size_t const   world_size,
                                             const size_t   tidx,
                                             const size_t   bidx) {
    // After this function, at least one block in each GPU has reached the barrier
    if (tidx < world_size) {
        // we can think of signals having the shape [world_size, world_size]
        // Dimension 0 is the "listening" dimension, dimension 1 is "emitting" dimension

        // Block 0 broadcasts its flag (local_rank on emitting dimension) to all receivers
        size_t offset = (barrier_flag % 2) ? world_size : 0;

        if (bidx == 0) {
            st_flag_release(barrier_flag, peer_barrier_ptrs[tidx] + offset + local_rank);
        }

        // All blocks check that corresponding block 0 on other GPUs have set the flag
        // No deadlock because block #0 is always the first block started
        uint32_t* peer_barrier_d = peer_barrier_ptrs[local_rank] + offset + tidx;
        while (ld_flag_acquire(peer_barrier_d) != barrier_flag) {}
    }

    __syncthreads();
}

__inline__ __device__ void block_barrier(uint32_t**     peer_barrier_ptrs,
                                         uint32_t const barrier_flag,
                                         size_t const   local_rank,
                                         size_t const   world_size,
                                         const size_t   tidx,
                                         const size_t   bidx,
                                         const size_t   grid_size) {
    // After this function, the block of id == bidx of each GPU has reached the barrier
    if (tidx < world_size) {
        // we can think of signals having the shape [world_size, 2, num_blocks, world_size]
        // (+ an offset on dim 2 to account for flags used in multi_gpu_barrier)
        // Dimension 0 is the "listening" dimension, dimension 3 is "emitting" dimension

        // Block broadcast its flag (local_rank on emitting dimension) to all receivers
        uint32_t flag_block_offset = world_size + bidx * world_size;

        if (barrier_flag % 2 == 1) {
            flag_block_offset += (grid_size + 1) * world_size;
        }

        st_flag_release(barrier_flag, peer_barrier_ptrs[tidx] + flag_block_offset + local_rank);

        // Blocks check that corresponding blocks on other GPUs have also set the flag
        uint32_t* peer_barrier_d = peer_barrier_ptrs[local_rank] + flag_block_offset + tidx;

        while (ld_flag_acquire(peer_barrier_d) != barrier_flag) {}
    }

    __syncthreads();
}

template<typename T, size_t RANKS_PER_NODE>
static __global__ void oneShotAllReduceKernel(CustomAllReduceParameters params, uint32_t barrier_flag) {
    // The block index.
    const size_t bidx = blockIdx.x;
    // The thread index with the block.
    const size_t tidx = threadIdx.x;

    // The number of elements packed into one for comms
    static constexpr size_t NUM_ELTS = std::is_same<T, float>::value ? 4 : 8;

    // Packed data type for comms
    using PackedType = typename ARTypeConverter<T>::Type;

    // The location in the destination array (load 8 fp16 or load 4 fp32 using LDG.128).
    size_t offset = bidx * params.elts_per_block + tidx * NUM_ELTS;
    // The end of the segment computed by that block.
    size_t max_offset = std::min((bidx + 1) * params.elts_per_block, params.elts_per_rank);

    multi_gpu_barrier(params.peer_barrier_ptrs, barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx);
    // The source pointers. Distributed round-robin for the different warps.
    const T* src_d[RANKS_PER_NODE];
#pragma unroll
    for (size_t ii = 0; ii < RANKS_PER_NODE; ++ii) {
        size_t rank = (params.local_rank + ii) % RANKS_PER_NODE;
        src_d[ii]   = (T*)(params.peer_comm_buffer_ptrs[rank]);
    }

    // Each block accumulates the values from the different GPUs on the same node.
    for (size_t iter_offset = offset; iter_offset < max_offset; iter_offset += blockDim.x * NUM_ELTS) {
        // Iterate over the different ranks/devices on the node to load the values.
        PackedType vals[RANKS_PER_NODE];
#pragma unroll
        for (size_t ii = 0; ii < RANKS_PER_NODE; ++ii) {
            vals[ii] = reinterpret_cast<const PackedType*>(&src_d[ii][iter_offset])[0];
        }

        // Sum the values from the different ranks.
        PackedType sums = init_packed_type<PackedType>();
#pragma unroll
        for (size_t ii = 0; ii < RANKS_PER_NODE; ++ii) {
            sums = add128b<PackedType, T>(sums, vals[ii]);
        }

        // Store to the destination buffer.
        reinterpret_cast<PackedType*>(&((T*)params.local_output_buffer_ptr)[iter_offset])[0] = sums;
    }
}

template<typename T, size_t RANKS_PER_NODE>
static __global__ void twoShotAllReduceKernel(CustomAllReduceParameters params, uint32_t barrier_flag) {

    // The block index.
    const size_t bidx = blockIdx.x;
    // The thread index with the block.
    const size_t tidx      = threadIdx.x;
    const size_t grid_size = gridDim.x;

    // The number of elements packed into one for comms
    static constexpr size_t NUM_ELTS = std::is_same<T, float>::value ? 4 : 8;

    // Packed data type for comms
    using PackedType = typename ARTypeConverter<T>::Type;

    // The location in the destination array (load 8 fp16 or load 4 fp32 using LDG.128).
    size_t offset = bidx * params.elts_per_block + tidx * NUM_ELTS + params.rank_offset;
    // The end of the segment computed by that block.
    size_t max_offset = offset + params.elts_per_block;

    multi_gpu_barrier(params.peer_barrier_ptrs, barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx);

    // The source pointers. Distributed round-robin for the different warps.
    T* src_d[RANKS_PER_NODE];
    // The destination ranks for round-robin gathering
    size_t dst_rank[RANKS_PER_NODE];
#pragma unroll
    for (size_t ii = 0; ii < RANKS_PER_NODE; ++ii) {
        size_t rank  = (params.local_rank + ii) % RANKS_PER_NODE;
        src_d[ii]    = (T*)(params.peer_comm_buffer_ptrs[rank]);
        dst_rank[ii] = rank;
    }

    // Each block accumulates the values from the different GPUs on the same node.
    for (size_t local_offset = offset; local_offset < max_offset; local_offset += blockDim.x * NUM_ELTS) {

        // Iterate over the different ranks/devices on the node to load the values.
        PackedType vals[RANKS_PER_NODE];
#pragma unroll
        for (size_t ii = 0; ii < RANKS_PER_NODE; ++ii) {
            vals[ii] = reinterpret_cast<const PackedType*>(&src_d[ii][local_offset])[0];
        }

        // Sum the values from the different ranks.
        PackedType sums = init_packed_type<PackedType>();
#pragma unroll
        for (size_t ii = 0; ii < RANKS_PER_NODE; ++ii) {
            sums = add128b<PackedType, T>(sums, vals[ii]);
        }

        // Store to the local buffer.
        reinterpret_cast<PackedType*>(&src_d[0][local_offset])[0] = sums;
    }

    block_barrier(params.peer_barrier_ptrs, barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx, grid_size);

    // Gather all needed elts_num from other intra-node ranks
    for (size_t local_offset = offset; local_offset < max_offset; local_offset += blockDim.x * NUM_ELTS) {
#pragma unroll
        for (size_t ii = 0; ii < RANKS_PER_NODE; ++ii) {
            // use round-robin gathering from other ranks
            size_t offset_rank = local_offset + ((int)dst_rank[ii] - (int)params.local_rank) * params.elts_per_rank;
            if (offset_rank < params.elts_total_num) {
                reinterpret_cast<PackedType*>(&((T*)params.local_output_buffer_ptr)[offset_rank])[0] =
                    reinterpret_cast<PackedType*>(&src_d[ii][offset_rank])[0];
            }
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void kernelLaunchConfig(CustomAllReduceParameters* param, size_t& blocks_per_grid, size_t& threads_per_block) {
    size_t data_type_bytes = param->data_type_size;
    assert(data_type_bytes == 2 || data_type_bytes == 4);

    size_t                ranks_per_node = param->ranks_per_node;
    AllReduceStrategyType kernel_algo    = param->kernel_algo;

    size_t elts_total_num  = param->elts_total_num;
    size_t elts_per_thread = 16 / data_type_bytes;
    switch (kernel_algo) {
        case AllReduceStrategyType::ONESHOT: {  // one stage all reduce algo
            assert(elts_total_num % elts_per_thread == 0);
            size_t const total_threads = roundUp(elts_total_num / elts_per_thread, WARP_SIZE);
            threads_per_block          = std::min((size_t)DEFAULT_BLOCK_SIZE, total_threads);
            blocks_per_grid =
                std::min(static_cast<size_t>(MAX_ALL_REDUCE_BLOCKS), divUp(total_threads, threads_per_block));
            param->elts_per_rank  = elts_total_num;
            param->rank_offset    = 0;
            param->elts_per_block = roundUp(divUp(elts_total_num, blocks_per_grid), elts_per_thread);

            // std::stringstream string_stream;
            // string_stream  << "[ONE] elts_total_num: " << elts_total_num << " blocks_per_grid: " << blocks_per_grid
            // << " total_threads: " << total_threads << " threads_per_block: " << threads_per_block << "
            // elts_per_thread: " << elts_per_thread << std::endl; RTP_LLM_LOG_INFO(string_stream.str());
            // RTP_LLM_LOG_INFO(string_stream.str());
            break;
        }
        case AllReduceStrategyType::TWOSHOT: {  // two stage all reduce algo
            size_t mod      = elts_per_thread * ranks_per_node * DEFAULT_BLOCK_SIZE * MAX_ALL_REDUCE_BLOCKS;
            size_t remain   = param->elts_total_num % mod;
            bool   half_mod = false;
            if (remain != 0) {
                size_t max_elts_num = param->max_elts_total_size / data_type_bytes;
                assert(max_elts_num % mod == 0);
                if (elts_total_num < mod * 2) {
                    mod      = elts_per_thread * ranks_per_node * DEFAULT_BLOCK_SIZE * MAX_ALL_REDUCE_BLOCKS / 2;
                    remain   = param->elts_total_num % mod;
                    half_mod = true;
                }
                elts_total_num += (mod - remain);
                elts_total_num = std::min(elts_total_num, max_elts_num);
            }

            assert(elts_total_num / (elts_per_thread * ranks_per_node) == 0);

            threads_per_block = DEFAULT_BLOCK_SIZE;
            blocks_per_grid   = MAX_ALL_REDUCE_BLOCKS;
            if (half_mod) {
                blocks_per_grid = MAX_ALL_REDUCE_BLOCKS / 2;
            }

            param->elts_per_rank  = elts_total_num / ranks_per_node;
            param->rank_offset    = param->rank * param->elts_per_rank;
            param->elts_per_block = roundUp(divUp(param->elts_per_rank, blocks_per_grid), elts_per_thread);

            // std::stringstream string_stream;
            // string_stream  << "[TWO] elts_total_num: " << elts_total_num << " blocks_per_grid: " << blocks_per_grid
            // << " total_threads: " << total_threads << " threads_per_block: " << threads_per_block << "
            // elts_per_thread: " << elts_per_thread << std::endl; RTP_LLM_LOG_INFO(string_stream.str());
            // RTP_LLM_LOG_INFO(string_stream.str());
            break;
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template<typename T, size_t RANKS_PER_NODE>
void invokeCustomAllReduceKernel(CustomAllReduceParameters* param, uint32_t barrier_flag, hipStream_t stream) {
    size_t blocks_per_grid = 1, threads_per_block = DEFAULT_BLOCK_SIZE;

    kernelLaunchConfig(param, blocks_per_grid, threads_per_block);

    if (param->kernel_algo == 0) {
        oneShotAllReduceKernel<T, RANKS_PER_NODE>
            <<<blocks_per_grid, threads_per_block, 0, stream>>>(*param, barrier_flag);
    } else {
        twoShotAllReduceKernel<T, RANKS_PER_NODE>
            <<<blocks_per_grid, threads_per_block, 0, stream>>>(*param, barrier_flag);
    }
}

template<typename T>
void invokeCustomAllReduceDispatch(CustomAllReduceParameters* param, uint32_t barrier_flag, hipStream_t stream) {
    switch (param->elts_per_rank) {
        case 2:
            invokeCustomAllReduceKernel<T, 2>(param, barrier_flag, stream);
            break;
        case 4:
            invokeCustomAllReduceKernel<T, 4>(param, barrier_flag, stream);
            break;
        case 8:
            invokeCustomAllReduceKernel<T, 8>(param, barrier_flag, stream);
            break;
        case 16:
            invokeCustomAllReduceKernel<T, 16>(param, barrier_flag, stream);
            break;
        default:
            throw std::runtime_error("not support custom ar world size: " + std::to_string(param->elts_per_rank));
            break;
    }
}

#define INSTANTIATE_GENERAL_CUSTOM_ALL_REDUCE_DISPATCH(T)                                                              \
    template void invokeCustomAllReduceDispatch<T>(                                                                    \
        CustomAllReduceParameters * param, uint32_t barrier_flag, hipStream_t stream);

#define INSTANTIATE_GENERAL_CUSTOM_ALL_REDUCE_KERNEL(T)                                                                \
    template void invokeCustomAllReduceKernel<T, 2>(                                                                   \
        CustomAllReduceParameters * param, uint32_t barrier_flag, hipStream_t stream);                                \
    template void invokeCustomAllReduceKernel<T, 4>(                                                                   \
        CustomAllReduceParameters * param, uint32_t barrier_flag, hipStream_t stream);                                \
    template void invokeCustomAllReduceKernel<T, 8>(                                                                   \
        CustomAllReduceParameters * param, uint32_t barrier_flag, hipStream_t stream);                                \
    template void invokeCustomAllReduceKernel<T, 16>(                                                                  \
        CustomAllReduceParameters * param, uint32_t barrier_flag, hipStream_t stream);

// Template instantiation

#ifdef ENABLE_BF16
INSTANTIATE_GENERAL_CUSTOM_ALL_REDUCE_DISPATCH(__hip_bfloat16)
#endif
INSTANTIATE_GENERAL_CUSTOM_ALL_REDUCE_DISPATCH(float)
INSTANTIATE_GENERAL_CUSTOM_ALL_REDUCE_DISPATCH(half)

#ifdef ENABLE_BF16
INSTANTIATE_GENERAL_CUSTOM_ALL_REDUCE_KERNEL(__hip_bfloat16)
#endif
INSTANTIATE_GENERAL_CUSTOM_ALL_REDUCE_KERNEL(float)
INSTANTIATE_GENERAL_CUSTOM_ALL_REDUCE_KERNEL(half)

}  // namespace rtp_llm
