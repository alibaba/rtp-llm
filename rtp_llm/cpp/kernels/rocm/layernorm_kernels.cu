#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "rtp_llm/cpp/cuda/cuda_type_utils.cuh"
#include "rtp_llm/cpp/kernels/rocm/layernorm_kernels.h"
#include "rtp_llm/cpp/cuda/reduce_kernel_utils.cuh"

#if ENABLE_TRITON
#include "rtp_llm/cpp/kernels/triton/layernorm_kernels.h"
#endif

#if USING_ROCM
#include "rtp_llm/cpp/rocm/cuda_shims.h"
#endif

// wont't support new features
namespace rtp_llm {

__device__ __forceinline__ int64_t loadOffset(int head_num, int size_per_head) {
    // [[q_head_1],[q_head_2]...[k_head_1],[k_head_2]...[v_head_1],[v_head_2]...]
    int head_id  = blockIdx.y;
    int batch_id = blockIdx.x;
    int offset   = batch_id * head_num * size_per_head + size_per_head * head_id;
    return offset;
}

__device__ __forceinline__ int64_t loadOffsetStrided(const int stride, const int n_elems) {
    return blockIdx.x * stride / n_elems;
}

template<typename T>
__global__ void
qkLayerNorm(T* __restrict qkv, const T* __restrict gamma, const float layernorm_eps, int head_num, int size_per_head) {
    constexpr auto   num_elems_T       = num_elems<T>::value;
    constexpr size_t warp_size         = 32;
    const int        vec_size_per_head = size_per_head / num_elems_T;
    const int        n_elems           = vec_size_per_head / warp_size;
    using float_packed_t               = typename packed_as<float, num_elems_T>::type;

    const int tid = threadIdx.x;

    __shared__ float s_mean;
    __shared__ float s_variance;
    float            mean     = 0.0f;
    float            variance = 0.0f;

    float local_sum = 0.0f;
    for (int i = 0; i < n_elems; i++) {
        auto index = loadOffset(head_num, vec_size_per_head) + tid * n_elems + i;
        auto val_f = cuda_cast<float_packed_t>(ldg(&qkv[index]));
        local_sum += cuda_sum<float>(val_f);
    }

    mean = warpReduceSum(local_sum);

    if (threadIdx.x == 0) {
        s_mean = mean / size_per_head;
    }
    __syncthreads();

    float local_var_sum = 0.0f;
    for (int i = 0; i < n_elems; i++) {
        auto index = loadOffset(head_num, vec_size_per_head) + tid * n_elems + i;
        auto val_f = cuda_cast<float_packed_t>(ldg(&qkv[index]));
        auto diff  = val_f - s_mean;
        local_var_sum += cuda_sum<float>(diff * diff);
    }
    variance = warpReduceSum(local_var_sum);

    if (threadIdx.x == 0) {
        s_variance = rsqrtf(variance / size_per_head + layernorm_eps);
    }
    __syncthreads();

    for (int i = 0; i < n_elems; i++) {
        auto index       = loadOffset(head_num, vec_size_per_head) + tid * n_elems + i;
        auto gamma_index = blockIdx.y * vec_size_per_head + tid * n_elems + i;
        auto val_f       = cuda_cast<float_packed_t>(ldg(&qkv[index]));
        auto val_gamma   = cuda_cast<float_packed_t>(gamma[gamma_index]);
        qkv[index]       = cuda_cast<T>((val_f - s_mean) * s_variance * val_gamma);
    }
}

template<typename T, bool IS_BIAS>
__global__ void layerNormWithStride(T* __restrict output,
                                    const int out_stride,
                                    const T* __restrict input,
                                    const int in_stride,
                                    const T* __restrict gamma,
                                    const T* __restrict beta,
                                    const float layernorm_eps,
                                    const int   n,  // 总特征维度
                                    const int   norm_size) {
    constexpr auto   num_elems_T = num_elems<T>::value;  // 向量化元素数
    constexpr size_t warp_size   = 32;
    const int        n_elems     = norm_size / num_elems_T / warp_size;
    using float_packed_t         = typename packed_as<float, num_elems_T>::type;

    const int tid        = threadIdx.x;
    const int sample_idx = blockIdx.x / (n / norm_size);  // 样本索引
    const int head_idx   = blockIdx.x % (n / norm_size);  // 头/窗口索引

    __shared__ float s_mean;
    __shared__ float s_variance;

    // 计算当前窗口的起始位置
    const T* sample_start   = input + sample_idx * (in_stride / num_elems_T);
    T*       output_start   = output + sample_idx * (out_stride / num_elems_T);
    const T* head_start     = sample_start + head_idx * (norm_size / num_elems_T);
    T*       out_head_start = output_start + head_idx * (norm_size / num_elems_T);

    // Stage 1: 计算均值
    float local_sum = 0.0f;
#pragma unroll
    for (int i = 0; i < n_elems; i++) {
        int  elem_idx = i * warp_size + tid;
        auto val_f    = cuda_cast<float_packed_t>(ldg(&head_start[elem_idx]));
        local_sum += cuda_sum<float>(val_f);
    }

    float mean = warpReduceSum(local_sum);
    if (tid == 0) {
        s_mean = mean / norm_size;
    }
    __syncthreads();

    float local_var_sum = 0.0f;
#pragma unroll
    for (int i = 0; i < n_elems; i++) {
        int  elem_idx = i * warp_size + tid;
        auto val_f    = cuda_cast<float_packed_t>(ldg(&head_start[elem_idx]));
        auto diff     = val_f - s_mean;
        local_var_sum += cuda_sum<float>(diff * diff);
    }

    float variance = warpReduceSum(local_var_sum);
    if (tid == 0) {
        s_variance = rsqrtf(variance / norm_size + layernorm_eps);
    }
    __syncthreads();

#pragma unroll
    for (int i = 0; i < n_elems; i++) {
        int  elem_idx = i * warp_size + tid;
        auto val_f    = cuda_cast<float_packed_t>(ldg(&head_start[elem_idx]));

        auto gamma_val = cuda_cast<float_packed_t>(gamma[elem_idx]);
        if (IS_BIAS) {
            auto beta_val = cuda_cast<float_packed_t>(beta[elem_idx]);

            val_f = (val_f - s_mean) * s_variance * gamma_val + beta_val;
        } else {
            val_f = (val_f - s_mean) * s_variance * gamma_val;
        }
        out_head_start[elem_idx] = cuda_cast<T>(val_f);
    }
}

template<typename T>
void invokeQkLayerNorm(T* __restrict qkv,
                       const T* __restrict gamma,
                       const float  layernorm_eps,
                       const int    tokens,
                       const int    head_num,
                       const int    head_num_kv,
                       const int    size_per_head,
                       hipStream_t stream) {
    constexpr size_t vec_size  = 2;
    constexpr size_t warp_size = 32;

    if (size_per_head % warp_size != 0) {
        throw std::invalid_argument("not supported size_per_head: " + std::to_string(size_per_head));
    }
    dim3 grid(tokens, head_num + head_num_kv);
    dim3 block(warp_size);

    int total_head_num = head_num + 2 * head_num_kv;
    using Tp           = typename packed_as<T, vec_size>::type;
    qkLayerNorm<Tp><<<grid, block, 0, stream>>>(
        reinterpret_cast<Tp*>(qkv), reinterpret_cast<const Tp*>(gamma), layernorm_eps, total_head_num, size_per_head);
}

template<typename T>
void invokeLayerNormWithStride(T* __restrict output,
                               const int out_stride,
                               const T* __restrict input,
                               const int in_stride,
                               const T* __restrict gamma,
                               const T* __restrict beta,
                               const float  layernorm_eps,
                               const int    m,
                               const int    n,
                               const int    norm_size,
                               hipStream_t stream) {
    constexpr size_t vec_size  = 2;
    constexpr size_t warp_size = 32;

    // 参数校验
    if (n % norm_size != 0) {
        throw std::invalid_argument("n:" + std::to_string(n)
                                    + " must be divisible by norm_size:" + std::to_string(norm_size));
    }
    if (norm_size % (warp_size * vec_size) != 0) {
        throw std::invalid_argument("norm_size must be multiple of " + std::to_string(warp_size * vec_size));
    }

    const int num_heads = n / norm_size;
    dim3      grid(m * num_heads);  // 每个block处理一个样本的一个头
    dim3      block(warp_size);

    using Tp     = typename packed_as<T, vec_size>::type;
    bool is_bias = beta != nullptr;
    if (is_bias) {
        layerNormWithStride<Tp, true><<<grid, block, 0, stream>>>(reinterpret_cast<Tp*>(output),
                                                                  out_stride,
                                                                  reinterpret_cast<const Tp*>(input),
                                                                  in_stride,
                                                                  reinterpret_cast<const Tp*>(gamma),
                                                                  reinterpret_cast<const Tp*>(beta),
                                                                  layernorm_eps,
                                                                  n,
                                                                  norm_size);
    } else {
        layerNormWithStride<Tp, false><<<grid, block, 0, stream>>>(reinterpret_cast<Tp*>(output),
                                                                   out_stride,
                                                                   reinterpret_cast<const Tp*>(input),
                                                                   in_stride,
                                                                   reinterpret_cast<const Tp*>(gamma),
                                                                   nullptr,
                                                                   layernorm_eps,
                                                                   n,
                                                                   norm_size);
    }
}

#define INSTANTIATE_QK_LAYERNORM(T)                                                                                    \
    template void invokeQkLayerNorm(T* __restrict qkv,                                                                 \
                                    const T* __restrict gamma,                                                         \
                                    const float  layernorm_eps,                                                        \
                                    const int    tokens,                                                               \
                                    const int    head_num,                                                             \
                                    const int    head_num_kv,                                                          \
                                    const int    size_per_head,                                                        \
                                    hipStream_t stream)
INSTANTIATE_QK_LAYERNORM(float);
INSTANTIATE_QK_LAYERNORM(half);
#ifdef ENABLE_BF16
INSTANTIATE_QK_LAYERNORM(__hip_bfloat16);
#endif
#undef INSTANTIATE_QK_LAYERNORM

#define INSTANTIATE_STRIDED_LAYERNORM(T)                                                                               \
    template void invokeLayerNormWithStride(T* __restrict output,                                                      \
                                            const int out_stride,                                                      \
                                            const T* __restrict input,                                                 \
                                            const int in_stride,                                                       \
                                            const T* __restrict gamma,                                                 \
                                            const T* __restrict beta,                                                  \
                                            const float  layernorm_eps,                                                \
                                            const int    m,                                                            \
                                            const int    n,                                                            \
                                            const int    norm_size,                                                    \
                                            hipStream_t stream);
INSTANTIATE_STRIDED_LAYERNORM(float);
INSTANTIATE_STRIDED_LAYERNORM(half);
#ifdef ENABLE_BF16
INSTANTIATE_STRIDED_LAYERNORM(__hip_bfloat16);
#endif
#undef INSTANTIATE_STRIDED_LAYERNORM

template<typename Tf, typename T, bool IS_BETA>
__inline__ __device__ Tf
compute_layernorm(Tf val, float s_mean, float s_variance, const T* gamma, const T* beta, int i) {
    Tf ret = (val - s_mean) * s_variance * cuda_cast<Tf>(gamma[i]);
    if (IS_BETA) {
        ret = ret + cuda_cast<Tf>(beta[i]);
    }
    return ret;
}

/* Computes the layernorm https://pytorch.org/docs/stable/generated/torch.nn.LayerNorm.html
 * normed_output <- ( (input - E[input]) / Sqrt(Var[input] + eps) ) * gamma + beta
 * input is [tokens, hidden_dim]. Mean and Variance are per-row (i.e. per-token)
 *
 * One CTA handles one row.
 *
 * with USE_DIFF_OF_SQUARES set to false:
 * First pass (loop) computes the mean.
 * Second computes the variance via Var[x] = E[(x - E[x])²].
 * Third pass computes and writes normed_output
 *
 * with USE_DIFF_OF_SQUARES set to true (may be faster but less accurate):
 * First pass (loop) computes the mean and variance via Var[x] = E[x²] - E[x]²
 * Second pass computes and writes normed_output
 *
 * use_shmem controls if we cache input values into shared memory
 *
 * Optional: with dynamic scaling, the last pass doesn't write immediately but finds the
 *           amax per row. A final pass scales to int8 accordingly, and writes output to
 *           normed_output_quant.
 */
template<typename T,
         typename QUANT_OUT_T,
         bool IS_OUTPUT,
         bool IS_BIAS,
         bool RESIDUAL,
         bool IS_BETA,
         bool RETURN_NORMED_OUTPUT,
         bool USE_DIFF_OF_SQUARES = false>
__global__ void generalLayerNorm(T*           output,
                                 T*           normed_output,
                                 const T*     input,
                                 const T*     bias,
                                 const T*     residual,
                                 const T*     gamma,
                                 const T*     beta,
                                 const float  eps,
                                 int          tokens,
                                 int          hidden_dim,
                                 const float* scale_orig_quant_per_tensor,
                                 float*       scale_orig_quant_per_token,
                                 QUANT_OUT_T* normed_output_quant) {
    constexpr auto num_elems_T = num_elems<T>::value;
    using quant_packed_t       = typename packed_as<QUANT_OUT_T, num_elems_T>::type;
    using Int32_Packed_T       = typename packed_as<int32_t, num_elems_T>::type;
    using float_packed_t       = typename packed_as<float, num_elems_T>::type;
    using T_scalar             = typename packed_as<T, 1>::type;

    extern __shared__ __align__(sizeof(float)) char _shmem[];
    T*                                              shmem = reinterpret_cast<T*>(_shmem);
    __shared__ float                                s_mean;
    __shared__ float                                s_variance;

    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;

    float mean          = 0.0f;
    float variance      = 0.0f;
    float local_sum     = 0.0f;
    float local_var_sum = 0.0f;

    const bool           with_per_token_scaling  = scale_orig_quant_per_token != nullptr;
    const bool           with_per_tensor_scaling = scale_orig_quant_per_tensor != nullptr;
    const float_packed_t scale_orig_quant =
        cuda_cast<float_packed_t>(with_per_tensor_scaling ? *scale_orig_quant_per_tensor : 0.0f);
    T_scalar  amax    = getAmax<QUANT_OUT_T>();
    const int n_elems = hidden_dim / num_elems_T;

    for (int i = tidx; i < n_elems; i += blockDim.x) {
        // const T val = input[bidx * n_elems + i];
        const int index = bidx * n_elems + i;
        T         val   = input[index];
        // const T val = input[index];
        if (IS_BIAS) {
            val = add(val, ldg(&bias[i]));
        }
        if (RESIDUAL) {
            val = add(val, ldg(&residual[index]));
        }
        if (IS_OUTPUT && !RETURN_NORMED_OUTPUT) {
            output[index] = val;
        }
        shmem[i] = val;

        const float_packed_t val_f = cuda_cast<float_packed_t>(val);
        local_sum += cuda_sum<float>(val_f);
        if (USE_DIFF_OF_SQUARES) {
            local_var_sum += cuda_sum<float>(val_f * val_f);
        }
    }

    if (USE_DIFF_OF_SQUARES) {
        float packed[2] = {local_sum, local_var_sum};
        blockReduceSumV2<float, 2>(packed);
        mean     = packed[0];
        variance = packed[1];
    } else {
        mean = blockReduceSum(local_sum);
    }

    if (threadIdx.x == 0) {
        mean   = mean / hidden_dim;
        s_mean = mean;
        if (USE_DIFF_OF_SQUARES) {
            variance   = (variance / hidden_dim) - (mean * mean);  // Var[x] = E[x²] - E[x]²
            s_variance = rsqrtf(variance + eps);
        }
    }
    __syncthreads();

    if (!USE_DIFF_OF_SQUARES) {
        for (int i = tidx; i < n_elems; i += blockDim.x) {
            const T        val  = shmem[i];
            float_packed_t diff = cuda_cast<float_packed_t>(val) - s_mean;
            local_var_sum += cuda_sum<float>(diff * diff);
        }
        variance = blockReduceSum(local_var_sum);

        if (threadIdx.x == 0) {
            s_variance = rsqrtf(variance / hidden_dim + eps);
        }
        __syncthreads();
    }

    for (int i = tidx; i < n_elems; i += blockDim.x) {
        const int            index = bidx * n_elems + i;
        const float_packed_t val_f = cuda_cast<float_packed_t>(shmem[i]);
        const T              val =
            cuda_cast<T>(compute_layernorm<float_packed_t, T, IS_BETA>(val_f, s_mean, s_variance, gamma, beta, i));
        if (RETURN_NORMED_OUTPUT && IS_OUTPUT) {
            output[index] = val;
        }

        if (with_per_token_scaling) {
            amax     = cuda_max(cuda_max<T_scalar, T>(cuda_abs(val)), amax);
            shmem[i] = val;
        } else if (with_per_tensor_scaling) {
            reinterpret_cast<quant_packed_t*>(normed_output_quant)[index] =
                cuda_cast<quant_packed_t>(cuda_cast<float_packed_t>(val) * scale_orig_quant);
        } else {
            normed_output[index] = val;
        }
    }

    if (with_per_token_scaling) {
        float       abs_max_f               = blockAllReduceMax(cuda_cast<float>(amax));
        const float scale_factor            = getScaleFactor<QUANT_OUT_T>();
        const float dynamic_per_token_scale = scale_factor / abs_max_f;
        for (int i = tidx; i < n_elems; i += blockDim.x) {
            const int      index = bidx * n_elems + i;
            float_packed_t val_f = cuda_cast<float_packed_t>(shmem[i]);
            reinterpret_cast<quant_packed_t*>(normed_output_quant)[index] =
                cuda_cast<quant_packed_t>(val_f * cuda_cast<float_packed_t>(dynamic_per_token_scale));
        }
        if (tidx == 0) {
            scale_orig_quant_per_token[bidx] = abs_max_f / scale_factor;
        }
    }
}

template<typename T,
         typename QUANT_OUT_T,
         bool IS_OUTPUT,
         bool IS_BIAS,
         bool RESIDUAL,
         bool IS_BETA,
         bool RETURN_NORMED_OUTPUT,
         bool USE_DIFF_OF_SQUARES>
void dispatch_layernorm_type_square_method(T*           output,
                                           T*           normed_output,
                                           const T*     input,
                                           const T*     bias,
                                           const T*     residual,
                                           const T*     gamma,
                                           const T*     beta,
                                           const float  eps,
                                           int          tokens,
                                           int          hidden_dim,
                                           const float* scale_orig_quant_per_tensor,
                                           float*       scale_orig_quant_per_token,
                                           QUANT_OUT_T* normed_output_quant,
                                           const dim3   grid,
                                           const dim3   block,
                                           const size_t shmem_size,
                                           hipStream_t stream) {
    if (shmem_size >= (48 << 10)) {
#if USING_CUDA
        hipError_t ret = hipFuncSetAttribute(reinterpret_cast<const void*>(generalLayerNorm<T),
                                                                QUANT_OUT_T,
                                                                IS_OUTPUT,
                                                                IS_BIAS,
                                                                RESIDUAL,
                                                                IS_BETA,
                                                                RETURN_NORMED_OUTPUT,
                                                                USE_DIFF_OF_SQUARES>,
                                               hipFuncAttributeMaxDynamicSharedMemorySize,
                                               shmem_size);
#endif
    }
    generalLayerNorm<T, QUANT_OUT_T, IS_OUTPUT, IS_BIAS, RESIDUAL, IS_BETA, RETURN_NORMED_OUTPUT, USE_DIFF_OF_SQUARES>
        <<<grid, block, shmem_size, stream>>>(output,
                                              normed_output,
                                              input,
                                              bias,
                                              residual,
                                              gamma,
                                              beta,
                                              eps,
                                              tokens,
                                              hidden_dim,
                                              scale_orig_quant_per_tensor,
                                              scale_orig_quant_per_token,
                                              normed_output_quant);
}

template<typename T,
         typename QUANT_OUT_T,
         bool IS_OUTPUT,
         bool IS_BIAS,
         bool RESIDUAL,
         bool IS_BETA,
         bool RETURN_NORMED_OUTPUT>
void dispatch_layernorm_return_normed(T*           output,
                                      T*           normed_output,
                                      const T*     input,
                                      const T*     bias,
                                      const T*     residual,
                                      const T*     gamma,
                                      const T*     beta,
                                      const float  eps,
                                      int          tokens,
                                      int          hidden_dim,
                                      const float* scale_orig_quant_per_tensor,
                                      float*       scale_orig_quant_per_token,
                                      QUANT_OUT_T* normed_output_quant,
                                      const dim3   grid,
                                      const dim3   block,
                                      const size_t shmem_size,
                                      hipStream_t stream,
                                      bool         use_diff_of_squares) {
    if (use_diff_of_squares) {
        dispatch_layernorm_type_square_method<T,
                                              QUANT_OUT_T,
                                              IS_OUTPUT,
                                              IS_BIAS,
                                              RESIDUAL,
                                              IS_BETA,
                                              RETURN_NORMED_OUTPUT,
                                              true>(output,
                                                    normed_output,
                                                    input,
                                                    bias,
                                                    residual,
                                                    gamma,
                                                    beta,
                                                    eps,
                                                    tokens,
                                                    hidden_dim,
                                                    scale_orig_quant_per_tensor,
                                                    scale_orig_quant_per_token,
                                                    normed_output_quant,
                                                    grid,
                                                    block,
                                                    shmem_size,
                                                    stream);
    } else {
        dispatch_layernorm_type_square_method<T,
                                              QUANT_OUT_T,
                                              IS_OUTPUT,
                                              IS_BIAS,
                                              RESIDUAL,
                                              IS_BETA,
                                              RETURN_NORMED_OUTPUT,
                                              false>(output,
                                                     normed_output,
                                                     input,
                                                     bias,
                                                     residual,
                                                     gamma,
                                                     beta,
                                                     eps,
                                                     tokens,
                                                     hidden_dim,
                                                     scale_orig_quant_per_tensor,
                                                     scale_orig_quant_per_token,
                                                     normed_output_quant,
                                                     grid,
                                                     block,
                                                     shmem_size,
                                                     stream);
    }
}

template<typename T, typename QUANT_OUT_T, bool IS_OUTPUT, bool IS_BIAS, bool RESIDUAL, bool IS_BETA>
void dispatch_layernorm_type(T*           output,
                             T*           normed_output,
                             const T*     input,
                             const T*     bias,
                             const T*     residual,
                             const T*     gamma,
                             const T*     beta,
                             const float  eps,
                             int          tokens,
                             int          hidden_dim,
                             const float* scale_orig_quant_per_tensor,
                             float*       scale_orig_quant_per_token,
                             QUANT_OUT_T* normed_output_quant,
                             const dim3   grid,
                             const dim3   block,
                             const size_t shmem_size,
                             hipStream_t stream,
                             bool         use_diff_of_squares,
                             bool         return_normed_output) {
    if (return_normed_output) {
        dispatch_layernorm_return_normed<T, QUANT_OUT_T, IS_OUTPUT, IS_BIAS, RESIDUAL, IS_BETA, true>(
            output,
            normed_output,
            input,
            bias,
            residual,
            gamma,
            beta,
            eps,
            tokens,
            hidden_dim,
            scale_orig_quant_per_tensor,
            scale_orig_quant_per_token,
            normed_output_quant,
            grid,
            block,
            shmem_size,
            stream,
            use_diff_of_squares);
    } else {
        dispatch_layernorm_return_normed<T, QUANT_OUT_T, IS_OUTPUT, IS_BIAS, RESIDUAL, IS_BETA, false>(
            output,
            normed_output,
            input,
            bias,
            residual,
            gamma,
            beta,
            eps,
            tokens,
            hidden_dim,
            scale_orig_quant_per_tensor,
            scale_orig_quant_per_token,
            normed_output_quant,
            grid,
            block,
            shmem_size,
            stream,
            use_diff_of_squares);
    }
}

template<typename T, typename QUANT_OUT_T, bool IS_OUTPUT, bool IS_BIAS, bool RESIUDAL>
void dispatch_layernorm_beta(T*           output,
                             T*           normed_output,
                             const T*     input,
                             const T*     bias,
                             const T*     residual,
                             const T*     gamma,
                             const T*     beta,
                             const float  eps,
                             int          tokens,
                             int          hidden_dim,
                             const float* scale_orig_quant_per_tensor,
                             float*       scale_orig_quant_per_token,
                             QUANT_OUT_T* normed_output_quant,
                             const dim3   grid,
                             const dim3   block,
                             const size_t shmem_size,
                             hipStream_t stream,
                             bool         use_diff_of_squares,
                             bool         return_normed_output) {
    if (beta != nullptr) {
        dispatch_layernorm_type<T, QUANT_OUT_T, IS_OUTPUT, IS_BIAS, RESIUDAL, true>(output,
                                                                                    normed_output,
                                                                                    input,
                                                                                    bias,
                                                                                    residual,
                                                                                    gamma,
                                                                                    beta,
                                                                                    eps,
                                                                                    tokens,
                                                                                    hidden_dim,
                                                                                    scale_orig_quant_per_tensor,
                                                                                    scale_orig_quant_per_token,
                                                                                    normed_output_quant,
                                                                                    grid,
                                                                                    block,
                                                                                    shmem_size,
                                                                                    stream,
                                                                                    use_diff_of_squares,
                                                                                    return_normed_output);
    } else {
        dispatch_layernorm_type<T, QUANT_OUT_T, IS_OUTPUT, IS_BIAS, RESIUDAL, false>(output,
                                                                                     normed_output,
                                                                                     input,
                                                                                     bias,
                                                                                     residual,
                                                                                     gamma,
                                                                                     beta,
                                                                                     eps,
                                                                                     tokens,
                                                                                     hidden_dim,
                                                                                     scale_orig_quant_per_tensor,
                                                                                     scale_orig_quant_per_token,
                                                                                     normed_output_quant,
                                                                                     grid,
                                                                                     block,
                                                                                     shmem_size,
                                                                                     stream,
                                                                                     use_diff_of_squares,
                                                                                     return_normed_output);
    }
}

template<typename T, typename QUANT_OUT_T, bool IS_OUTPUT, bool IS_BIAS>
void dispatch_layernorm_residual(T*           output,
                                 T*           normed_output,
                                 const T*     input,
                                 const T*     bias,
                                 const T*     residual,
                                 const T*     gamma,
                                 const T*     beta,
                                 const float  eps,
                                 int          tokens,
                                 int          hidden_dim,
                                 const float* scale_orig_quant_per_tensor,
                                 float*       scale_orig_quant_per_token,
                                 QUANT_OUT_T* normed_output_quant,
                                 const dim3   grid,
                                 const dim3   block,
                                 const size_t shmem_size,
                                 hipStream_t stream,
                                 bool         use_diff_of_squares,
                                 bool         return_normed_output) {
    if (residual != nullptr) {
        dispatch_layernorm_beta<T, QUANT_OUT_T, IS_OUTPUT, IS_BIAS, true>(output,
                                                                          normed_output,
                                                                          input,
                                                                          bias,
                                                                          residual,
                                                                          gamma,
                                                                          beta,
                                                                          eps,
                                                                          tokens,
                                                                          hidden_dim,
                                                                          scale_orig_quant_per_tensor,
                                                                          scale_orig_quant_per_token,
                                                                          normed_output_quant,
                                                                          grid,
                                                                          block,
                                                                          shmem_size,
                                                                          stream,
                                                                          use_diff_of_squares,
                                                                          return_normed_output);
    } else {
        dispatch_layernorm_beta<T, QUANT_OUT_T, IS_OUTPUT, IS_BIAS, false>(output,
                                                                           normed_output,
                                                                           input,
                                                                           bias,
                                                                           residual,
                                                                           gamma,
                                                                           beta,
                                                                           eps,
                                                                           tokens,
                                                                           hidden_dim,
                                                                           scale_orig_quant_per_tensor,
                                                                           scale_orig_quant_per_token,
                                                                           normed_output_quant,
                                                                           grid,
                                                                           block,
                                                                           shmem_size,
                                                                           stream,
                                                                           use_diff_of_squares,
                                                                           return_normed_output);
    }
}

template<typename T, typename QUANT_OUT_T, bool IS_OUTPUT>
void dispatch_layernorm_bias(T*           output,
                             T*           normed_output,
                             const T*     input,
                             const T*     bias,
                             const T*     residual,
                             const T*     gamma,
                             const T*     beta,
                             const float  eps,
                             int          tokens,
                             int          hidden_dim,
                             const float* scale_orig_quant_per_tensor,
                             float*       scale_orig_quant_per_token,
                             QUANT_OUT_T* normed_output_quant,
                             const dim3   grid,
                             const dim3   block,
                             const size_t shmem_size,
                             hipStream_t stream,
                             bool         use_diff_of_squares,
                             bool         return_normed_output) {
    if (bias != nullptr) {
        dispatch_layernorm_residual<T, QUANT_OUT_T, IS_OUTPUT, true>(output,
                                                                     normed_output,
                                                                     input,
                                                                     bias,
                                                                     residual,
                                                                     gamma,
                                                                     beta,
                                                                     eps,
                                                                     tokens,
                                                                     hidden_dim,
                                                                     scale_orig_quant_per_tensor,
                                                                     scale_orig_quant_per_token,
                                                                     normed_output_quant,
                                                                     grid,
                                                                     block,
                                                                     shmem_size,
                                                                     stream,
                                                                     use_diff_of_squares,
                                                                     return_normed_output);
    } else {
        dispatch_layernorm_residual<T, QUANT_OUT_T, IS_OUTPUT, false>(output,
                                                                      normed_output,
                                                                      input,
                                                                      bias,
                                                                      residual,
                                                                      gamma,
                                                                      beta,
                                                                      eps,
                                                                      tokens,
                                                                      hidden_dim,
                                                                      scale_orig_quant_per_tensor,
                                                                      scale_orig_quant_per_token,
                                                                      normed_output_quant,
                                                                      grid,
                                                                      block,
                                                                      shmem_size,
                                                                      stream,
                                                                      use_diff_of_squares,
                                                                      return_normed_output);
    }
}

template<typename T, typename QUANT_OUT_T>
void dispatch_layernorm_output(T*           output,
                               T*           normed_output,
                               const T*     input,
                               const T*     bias,
                               const T*     residual,
                               const T*     gamma,
                               const T*     beta,
                               const float  eps,
                               int          tokens,
                               int          hidden_dim,
                               const float* scale_orig_quant_per_tensor,
                               float*       scale_orig_quant_per_token,
                               QUANT_OUT_T* normed_output_quant,
                               const dim3   grid,
                               const dim3   block,
                               const size_t shmem_size,
                               hipStream_t stream,
                               bool         use_diff_of_squares,
                               bool         is_output,
                               bool         return_normed_output) {
    if (is_output) {
        dispatch_layernorm_bias<T, QUANT_OUT_T, true>(output,
                                                      normed_output,
                                                      input,
                                                      bias,
                                                      residual,
                                                      gamma,
                                                      beta,
                                                      eps,
                                                      tokens,
                                                      hidden_dim,
                                                      scale_orig_quant_per_tensor,
                                                      scale_orig_quant_per_token,
                                                      normed_output_quant,
                                                      grid,
                                                      block,
                                                      shmem_size,
                                                      stream,
                                                      use_diff_of_squares,
                                                      return_normed_output);
    } else {
        dispatch_layernorm_bias<T, QUANT_OUT_T, false>(output,
                                                       normed_output,
                                                       input,
                                                       bias,
                                                       residual,
                                                       gamma,
                                                       beta,
                                                       eps,
                                                       tokens,
                                                       hidden_dim,
                                                       scale_orig_quant_per_tensor,
                                                       scale_orig_quant_per_token,
                                                       normed_output_quant,
                                                       grid,
                                                       block,
                                                       shmem_size,
                                                       stream,
                                                       use_diff_of_squares,
                                                       return_normed_output);
    }
}

template<typename T, typename QUANT_OUT_T>
void invokeGeneralLayerNorm(T*           out,
                            T*           normed_output,
                            const T*     input,
                            const T*     gamma,
                            const T*     beta,
                            const float  eps,
                            const int    tokens,
                            const int    hidden_dim,
                            hipStream_t stream,
                            bool         use_diff_of_squares,
                            const float* scale,
                            float*       dynamic_scale,
                            QUANT_OUT_T* out_quant,
                            bool         return_normed_output) {
#if ENABLE_TRITON && !defined(ENABLE_FP8)
    if (hidden_dim <= 4096 && dynamic_scale == nullptr && scale == nullptr && beta != nullptr
        && (out == nullptr || return_normed_output == true)) {
        invokeTritonLayerNorm<T, QUANT_OUT_T, false>(out,
                                                     normed_output,
                                                     input,
                                                     (const T*)nullptr,
                                                     (const T*)nullptr,
                                                     gamma,
                                                     beta,
                                                     eps,
                                                     tokens,
                                                     hidden_dim,
                                                     stream,
                                                     use_diff_of_squares,
                                                     scale,
                                                     dynamic_scale,
                                                     out_quant,
                                                     return_normed_output);
        return;
    }
#endif

    dim3 grid(tokens);
    dim3 block(min(hidden_dim, 1024));
    // Make sure block.x is multiple of 32 for warp shuffle to work
    block.x = 32 * ((block.x + 31) / 32);

    constexpr size_t vec_size     = 2;
    const size_t     shmem_size   = hidden_dim * sizeof(T);
    const bool       use_vec_type = (hidden_dim % vec_size == 0)
                              && (std::is_same<T, half>::value
#ifdef ENABLE_BF16
                                  || std::is_same<T, __hip_bfloat16>::value
#endif
                              );

    if (use_vec_type) {
        using Tp = typename packed_as<T, vec_size>::type;
        dispatch_layernorm_output(reinterpret_cast<Tp*>(out),
                                  reinterpret_cast<Tp*>(normed_output),
                                  reinterpret_cast<const Tp*>(input),
                                  (const Tp*)nullptr,
                                  (const Tp*)nullptr,
                                  reinterpret_cast<const Tp*>(gamma),
                                  reinterpret_cast<const Tp*>(beta),
                                  eps,
                                  tokens,
                                  hidden_dim,
                                  scale,
                                  dynamic_scale,
                                  out_quant,
                                  grid,
                                  block,
                                  shmem_size,
                                  stream,
                                  use_diff_of_squares,
                                  out != nullptr,
                                  return_normed_output);
    } else {
        dispatch_layernorm_output(out,
                                  normed_output,
                                  (const T*)input,
                                  (const T*)nullptr,
                                  (const T*)nullptr,
                                  gamma,
                                  beta,
                                  eps,
                                  tokens,
                                  hidden_dim,
                                  scale,
                                  dynamic_scale,
                                  out_quant,
                                  grid,
                                  block,
                                  shmem_size,
                                  stream,
                                  use_diff_of_squares,
                                  out != nullptr,
                                  return_normed_output);
    }
}

template<typename T, typename QUANT_OUT_T>
void invokeGeneralAddBiasResidualLayerNorm(T*           out,
                                           T*           norm_output,
                                           const T*     input,
                                           const T*     bias,
                                           const T*     residual,
                                           const T*     gamma,
                                           const T*     beta,
                                           const float  eps,
                                           const int    tokens,
                                           const int    hidden_dim,
                                           hipStream_t stream,
                                           bool         use_diff_of_squares,
                                           const float* scale,
                                           float*       dynamic_scale,
                                           QUANT_OUT_T* out_quant,
                                           bool         return_normed_output) {
#if ENABLE_TRITON && !defined(ENABLE_FP8)
    if (hidden_dim <= 4096 && dynamic_scale == nullptr && scale == nullptr && beta != nullptr
        && (out == nullptr || return_normed_output == true)) {
        invokeTritonLayerNorm<T, QUANT_OUT_T, true>(out,
                                                    norm_output,
                                                    input,
                                                    bias,
                                                    residual,
                                                    gamma,
                                                    beta,
                                                    eps,
                                                    tokens,
                                                    hidden_dim,
                                                    stream,
                                                    use_diff_of_squares,
                                                    scale,
                                                    dynamic_scale,
                                                    out_quant,
                                                    return_normed_output);
        return;
    }
#endif

    dim3 grid(tokens);
    dim3 block(min(hidden_dim, 1024));
    // Make sure block.x is multiple of 32 for warp shuffle to work
    block.x = 32 * ((block.x + 31) / 32);

    constexpr size_t vec_size     = 2;
    const size_t     shmem_size   = hidden_dim * sizeof(T);
    const bool       use_vec_type = (hidden_dim % vec_size == 0)
                              && (std::is_same<T, half>::value
#ifdef ENABLE_BF16
                                  || std::is_same<T, __hip_bfloat16>::value
#endif
                              );

    if (use_vec_type) {
        using Tp = typename packed_as<T, vec_size>::type;
        dispatch_layernorm_output(reinterpret_cast<Tp*>(out),
                                  reinterpret_cast<Tp*>(norm_output),
                                  reinterpret_cast<const Tp*>(input),
                                  reinterpret_cast<const Tp*>(bias),
                                  reinterpret_cast<const Tp*>(residual),
                                  reinterpret_cast<const Tp*>(gamma),
                                  reinterpret_cast<const Tp*>(beta),
                                  eps,
                                  tokens,
                                  hidden_dim,
                                  scale,
                                  dynamic_scale,
                                  out_quant,
                                  grid,
                                  block,
                                  shmem_size,
                                  stream,
                                  use_diff_of_squares,
                                  true,
                                  return_normed_output);
    } else {
        dispatch_layernorm_output(out,
                                  norm_output,
                                  input,
                                  bias,
                                  residual,
                                  gamma,
                                  beta,
                                  eps,
                                  tokens,
                                  hidden_dim,
                                  scale,
                                  dynamic_scale,
                                  out_quant,
                                  grid,
                                  block,
                                  shmem_size,
                                  stream,
                                  use_diff_of_squares,
                                  true,
                                  return_normed_output);
    }
}

#define INSTANTIATE_GENERAL_LAYERNORM(T, QUANT_OUT_T)                                                                  \
    template void invokeGeneralLayerNorm(T*           out,                                                             \
                                         T*           normed_output,                                                   \
                                         const T*     input,                                                           \
                                         const T*     gamma,                                                           \
                                         const T*     beta,                                                            \
                                         const float  eps,                                                             \
                                         const int    tokens,                                                          \
                                         const int    hidden_dim,                                                      \
                                         hipStream_t stream,                                                          \
                                         bool         use_diff_of_squares,                                             \
                                         const float* scale,                                                           \
                                         float*       dynamic_scale,                                                   \
                                         QUANT_OUT_T* out_quant,                                                       \
                                         bool         return_normed_output);

INSTANTIATE_GENERAL_LAYERNORM(float, int8_t);
INSTANTIATE_GENERAL_LAYERNORM(half, int8_t);
#ifdef ENABLE_BF16
INSTANTIATE_GENERAL_LAYERNORM(__hip_bfloat16, int8_t);
#endif
#ifdef ENABLE_FP8
INSTANTIATE_GENERAL_LAYERNORM(float, __hip_fp8_e4m3_fnuz);
INSTANTIATE_GENERAL_LAYERNORM(half, __hip_fp8_e4m3_fnuz);
#ifdef ENABLE_BF16
INSTANTIATE_GENERAL_LAYERNORM(__hip_bfloat16, __hip_fp8_e4m3_fnuz);
#endif
#endif

#define INSTANTIATE_GENERAL_ADD_BIAS_RESIDUAL_LAYERNORM(T, QUANT_OUT_T)                                                \
    template void invokeGeneralAddBiasResidualLayerNorm(T*           out,                                              \
                                                        T*           norm_output,                                      \
                                                        const T*     input,                                            \
                                                        const T*     bias,                                             \
                                                        const T*     residual,                                         \
                                                        const T*     gamma,                                            \
                                                        const T*     beta,                                             \
                                                        const float  eps,                                              \
                                                        const int    tokens,                                           \
                                                        const int    hidden_dim,                                       \
                                                        hipStream_t stream,                                           \
                                                        bool         use_diff_of_squares,                              \
                                                        const float* scale,                                            \
                                                        float*       dynamic_scale,                                    \
                                                        QUANT_OUT_T* out_quant,                                        \
                                                        bool         return_normed_output);

INSTANTIATE_GENERAL_ADD_BIAS_RESIDUAL_LAYERNORM(float, int8_t);
INSTANTIATE_GENERAL_ADD_BIAS_RESIDUAL_LAYERNORM(half, int8_t);
#ifdef ENABLE_BF16
INSTANTIATE_GENERAL_ADD_BIAS_RESIDUAL_LAYERNORM(__hip_bfloat16, int8_t);
#endif
#ifdef ENABLE_FP8
INSTANTIATE_GENERAL_ADD_BIAS_RESIDUAL_LAYERNORM(float, __hip_fp8_e4m3_fnuz);
INSTANTIATE_GENERAL_ADD_BIAS_RESIDUAL_LAYERNORM(half, __hip_fp8_e4m3_fnuz);
#ifdef ENABLE_BF16
INSTANTIATE_GENERAL_ADD_BIAS_RESIDUAL_LAYERNORM(__hip_bfloat16, __hip_fp8_e4m3_fnuz);
#endif
#endif
}  // namespace rtp_llm
