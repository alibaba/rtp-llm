#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "rtp_llm/cpp/cuda/cuda_type_utils.cuh"
#include "rtp_llm/cpp/cuda/launch_utils.h"
#include "rtp_llm/cpp/cuda/reduce_kernel_utils.cuh"
#include "rtp_llm/cpp/kernels/rmsnormKernels.h"

#if USING_CUDA
#include "rtp_llm/cpp/cuda/cuda_host_utils.h"
#endif

#if USING_ROCM
#include "rtp_llm/cpp/rocm/cuda_shims.h"
#endif

namespace rtp_llm {

template<typename Tf, typename T, bool IS_BETA>
__inline__ __device__ Tf compute_rmsnorm(Tf val, float s_variance, const T* gamma, const T* beta, size_t i) {
    Tf ret = val * s_variance * cuda_cast<Tf>(gamma[i]);
    if (IS_BETA) {
        ret = ret + cuda_cast<Tf>(beta[i]);
    }
    return ret;
}

/* Computes the rmsnorm https://pytorch.org/docs/stable/generated/torch.nn.rmsnorm.html
 * normed_output <- ( input / Sqrt(E[input²] + eps) ) * gamma + beta
 * input is [tokens, hidden_dim]. Mean and Variance are per-row (i.e. per-token)
 *
 * One CTA handles one row.
 *
 *
 * use_shmem controls if we cache input values into shared memory
 *
 * Optional: with dynamic scaling, the last pass doesn't write immediately but finds the
 *           amax per row. A final pass scales to int8 accordingly, and writes output to
 *           normed_output_quant.
 */
template<typename T, bool IS_OUTPUT, bool IS_BIAS, bool RESIDUAL, bool IS_BETA, typename QUANT_OUT_T>
__global__ void generalRmsNorm(T*           output,
                               T*           normed_output,
                               const T*     input,
                               const T*     bias,
                               const T*     residual1,
                               const T*     residual2,
                               const T*     gamma,
                               const T*     beta,
                               const float  eps,
                               size_t       tokens,
                               size_t       hidden_dim,
                               const float* scale_orig_quant_per_tensor,
                               float*       scale_orig_quant_per_token,
                               QUANT_OUT_T* normed_output_quant) {
    constexpr auto num_elems_T = num_elems<T>::value;
    using quant_packed_t       = typename packed_as<QUANT_OUT_T, num_elems_T>::type;
    using float_packed_t       = typename packed_as<float, num_elems_T>::type;
    using T_scalar             = typename packed_as<T, 1>::type;

    extern __shared__ __align__(sizeof(float)) char _shmem[];
    T*                                              shmem = reinterpret_cast<T*>(_shmem);

    __shared__ float s_variance;

    const size_t tidx = threadIdx.x;
    const size_t bidx = blockIdx.x;

    float variance      = 0.0f;
    float local_var_sum = 0.0f;

    const size_t n_elems = hidden_dim / num_elems_T;

    const bool           with_per_token_scaling  = scale_orig_quant_per_token != nullptr;
    const bool           with_per_tensor_scaling = scale_orig_quant_per_tensor != nullptr;
    const bool           has_residual2           = residual2 != nullptr;
    const float_packed_t scale_orig_quant =
        cuda_cast<float_packed_t>(with_per_tensor_scaling ? *scale_orig_quant_per_tensor : 0.0f);
    T_scalar amax = getAmax<QUANT_OUT_T>();

    for (size_t i = tidx; i < n_elems; i += blockDim.x) {
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
        asm volatile("griddepcontrol.wait;");
#endif
        const size_t index = bidx * n_elems + i;
        T            val   = cuda_cast<T>(0.0f);
        // const T val = input[index];
        if (IS_BIAS) {
            val = add(val, ldg(&bias[i]));
        }
        if (RESIDUAL) {
            val = add(val, ldg(&residual1[index]));
            if (has_residual2) {
                val = add(val, ldg(&residual2[index]));
            }
        }
        if (IS_OUTPUT) {
            T in_val = input[index];
            val      = add(val, in_val);
        }

        shmem[i] = val;

        if (IS_OUTPUT) {
            output[index] = val;
        }
        const float_packed_t val_f = cuda_cast<float_packed_t>(val);

        local_var_sum += cuda_sum<float>(val_f * val_f);
    }

    float packed[1] = {local_var_sum};
    blockReduceSumV2<float, 1>(packed);
    variance = packed[0];

    if (threadIdx.x == 0) {
        variance   = (variance / hidden_dim);  // Var[x] = E[x²]
        s_variance = rsqrtf(variance + eps);
    }
    __syncthreads();
    const float scale_factor = getScaleFactor<QUANT_OUT_T>();
    for (size_t i = tidx; i < n_elems; i += blockDim.x) {
        const size_t         index = bidx * n_elems + i;
        const float_packed_t val_f = cuda_cast<float_packed_t>(shmem[i]);
        const T val = cuda_cast<T>(compute_rmsnorm<float_packed_t, T, IS_BETA>(val_f, s_variance, gamma, beta, i));

        if (with_per_token_scaling) {
            amax     = cuda_max(cuda_max<T_scalar, T>(cuda_abs(val)), amax);
            shmem[i] = val;
        } else if (with_per_tensor_scaling) {
            reinterpret_cast<quant_packed_t*>(normed_output_quant)[index] =
                cuda_cast<quant_packed_t>(cuda_cast<float_packed_t>(val) * scale_orig_quant);
        } else {
            normed_output[index] = val;
        }
    }

    if (with_per_token_scaling) {
        float       abs_max_f               = blockAllReduceMax(cuda_cast<float>(amax));
        const float dynamic_per_token_scale = scale_factor / abs_max_f;
        for (size_t i = tidx; i < n_elems; i += blockDim.x) {
            const size_t   index = bidx * n_elems + i;
            float_packed_t val_f = cuda_cast<float_packed_t>(shmem[i]);
            reinterpret_cast<quant_packed_t*>(normed_output_quant)[index] =
                cuda_cast<quant_packed_t>(val_f * cuda_cast<float_packed_t>(dynamic_per_token_scale));
        }
        if (tidx == 0) {
            scale_orig_quant_per_token[bidx] = abs_max_f / scale_factor;
        }
    }
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.launch_dependents;");
#endif
}

template<typename T, bool IS_BIAS>
__global__ void rmsNormWithStride(T* __restrict output,
                                  const size_t out_stride,
                                  const T* __restrict input,
                                  const size_t in_stride,
                                  const T* __restrict gamma,
                                  const T* __restrict bias,
                                  const float  eps,
                                  const size_t n,
                                  const size_t norm_size) {
    constexpr auto num_elems_T           = num_elems<T>::value;
    using float_packed_t                 = typename packed_as<float, num_elems_T>::type;
    constexpr size_t vec_size            = num_elems<T>::value;
    constexpr size_t warp_size           = 32;
    const size_t     elements_per_thread = norm_size / (warp_size * vec_size);

    const size_t sample_idx  = blockIdx.x / (n / norm_size);
    const size_t group_idx   = blockIdx.x % (n / norm_size);
    const T*     group_start = input + sample_idx * (in_stride / vec_size) + group_idx * (norm_size / vec_size);
    T*           dest_start  = output + sample_idx * (out_stride / vec_size) + group_idx * (norm_size / vec_size);

    __shared__ float smem_scale;

    float square_sum = 0.0f;
    for (size_t i = 0; i < elements_per_thread; ++i) {
        const size_t elem_idx   = i * warp_size + threadIdx.x;
        T            packed_val = group_start[elem_idx];
        auto         val        = cuda_cast<float_packed_t>(packed_val);

        square_sum += cuda_sum<float>(val * val);
    }

    float variance = warpReduceSum(square_sum) / norm_size;

    if (threadIdx.x == 0) {
        smem_scale = rsqrtf(variance + eps);
    }
    __syncthreads();

    for (size_t i = 0; i < elements_per_thread; ++i) {
        const size_t elem_idx   = i * warp_size + threadIdx.x;
        T            packed_val = group_start[elem_idx];

        const float_packed_t val_f = cuda_cast<float_packed_t>(packed_val);
        const T              val =
            cuda_cast<T>(compute_rmsnorm<float_packed_t, T, IS_BIAS>(val_f, smem_scale, gamma, bias, elem_idx));
        dest_start[elem_idx] = cuda_cast<T>(val);
    }
}

template<typename T, bool IS_OUTPUT, bool IS_BIAS, bool RESIDUAL, bool IS_BETA, typename QUANT_OUT_T = int8_t>
void dispatch_rmsnorm_type_square_method(T*           output,
                                         T*           normed_output,
                                         const T*     input,
                                         const T*     bias,
                                         const T*     residual1,
                                         const T*     residual2,
                                         const T*     gamma,
                                         const T*     beta,
                                         const float  eps,
                                         size_t       tokens,
                                         size_t       hidden_dim,
                                         const float* scale_orig_quant_per_tensor,
                                         float*       scale_orig_quant_per_token,
                                         QUANT_OUT_T* normed_output_quant,
                                         const dim3   grid,
                                         const dim3   block,
                                         const size_t shmem_size,
                                         hipStream_t stream) {
    if (shmem_size >= (48 << 10)) {
#if USING_CUDA
        check_cuda_value(hipFuncSetAttribute(reinterpret_cast<const void*>(generalRmsNorm<T), IS_OUTPUT, IS_BIAS, RESIDUAL, IS_BETA, QUANT_OUT_T>,
                                              hipFuncAttributeMaxDynamicSharedMemorySize,
                                              shmem_size));
#endif
    }
    LAUNCH_KERNEL_WITH_PDL((generalRmsNorm<T, IS_OUTPUT, IS_BIAS, RESIDUAL, IS_BETA, QUANT_OUT_T>),
                           grid,
                           block,
                           shmem_size,
                           stream,
                           output,
                           normed_output,
                           input,
                           bias,
                           residual1,
                           residual2,
                           gamma,
                           beta,
                           eps,
                           tokens,
                           hidden_dim,
                           scale_orig_quant_per_tensor,
                           scale_orig_quant_per_token,
                           normed_output_quant);
#if USING_CUDA
    check_cuda_value(hipPeekAtLastError());
    check_cuda_error();
#endif
}

template<typename T, bool IS_OUTPUT, bool IS_BIAS, bool RESIDUAL, typename QUANT_OUT_T = int8_t>
void dispatch_rmsnorm_beta(T*           output,
                           T*           normed_output,
                           const T*     input,
                           const T*     bias,
                           const T*     residual1,
                           const T*     residual2,
                           const T*     gamma,
                           const T*     beta,
                           const float  eps,
                           size_t       tokens,
                           size_t       hidden_dim,
                           const float* scale_orig_quant_per_tensor,
                           float*       scale_orig_quant_per_token,
                           QUANT_OUT_T* normed_output_quant,
                           const dim3   grid,
                           const dim3   block,
                           const size_t shmem_size,
                           hipStream_t stream) {
    if (beta != nullptr) {

        dispatch_rmsnorm_type_square_method<T, IS_OUTPUT, IS_BIAS, RESIDUAL, true, QUANT_OUT_T>(
            output,
            normed_output,
            input,
            bias,
            residual1,
            residual2,
            gamma,
            beta,
            eps,
            tokens,
            hidden_dim,
            scale_orig_quant_per_tensor,
            scale_orig_quant_per_token,
            normed_output_quant,
            grid,
            block,
            shmem_size,
            stream);
    } else {

        dispatch_rmsnorm_type_square_method<T, IS_OUTPUT, IS_BIAS, RESIDUAL, false, QUANT_OUT_T>(
            output,
            normed_output,
            input,
            bias,
            residual1,
            residual2,
            gamma,
            beta,
            eps,
            tokens,
            hidden_dim,
            scale_orig_quant_per_tensor,
            scale_orig_quant_per_token,
            normed_output_quant,
            grid,
            block,
            shmem_size,
            stream);
    }
}

template<typename T, bool IS_OUTPUT, bool IS_BIAS, typename QUANT_OUT_T>
void dispatch_rmsnorm_residual(T*           output,
                               T*           normed_output,
                               const T*     input,
                               const T*     bias,
                               const T*     residual1,
                               const T*     residual2,
                               const T*     gamma,
                               const T*     beta,
                               const float  eps,
                               size_t       tokens,
                               size_t       hidden_dim,
                               const float* scale_orig_quant_per_tensor,
                               float*       scale_orig_quant_per_token,
                               QUANT_OUT_T* normed_output_quant,
                               const dim3   grid,
                               const dim3   block,
                               const size_t shmem_size,
                               hipStream_t stream) {
    if (residual1 != nullptr) {

        dispatch_rmsnorm_beta<T, IS_OUTPUT, IS_BIAS, true, QUANT_OUT_T>(output,
                                                                        normed_output,
                                                                        input,
                                                                        bias,
                                                                        residual1,
                                                                        residual2,
                                                                        gamma,
                                                                        beta,
                                                                        eps,
                                                                        tokens,
                                                                        hidden_dim,
                                                                        scale_orig_quant_per_tensor,
                                                                        scale_orig_quant_per_token,
                                                                        normed_output_quant,
                                                                        grid,
                                                                        block,
                                                                        shmem_size,
                                                                        stream);
    } else {

        dispatch_rmsnorm_beta<T, IS_OUTPUT, IS_BIAS, false, QUANT_OUT_T>(output,
                                                                         normed_output,
                                                                         input,
                                                                         bias,
                                                                         residual1,
                                                                         residual2,
                                                                         gamma,
                                                                         beta,
                                                                         eps,
                                                                         tokens,
                                                                         hidden_dim,
                                                                         scale_orig_quant_per_tensor,
                                                                         scale_orig_quant_per_token,
                                                                         normed_output_quant,
                                                                         grid,
                                                                         block,
                                                                         shmem_size,
                                                                         stream);
    }
}

template<typename T, bool IS_OUTPUT, typename QUANT_OUT_T>
void dispatch_rmsnorm_bias(T*           output,
                           T*           normed_output,
                           const T*     input,
                           const T*     bias,
                           const T*     residual1,
                           const T*     residual2,
                           const T*     gamma,
                           const T*     beta,
                           const float  eps,
                           size_t       tokens,
                           size_t       hidden_dim,
                           const float* scale_orig_quant_per_tensor,
                           float*       scale_orig_quant_per_token,
                           QUANT_OUT_T* normed_output_quant,
                           const dim3   grid,
                           const dim3   block,
                           const size_t shmem_size,
                           hipStream_t stream) {
    if (bias != nullptr) {

        dispatch_rmsnorm_residual<T, IS_OUTPUT, true, QUANT_OUT_T>(output,
                                                                   normed_output,
                                                                   input,
                                                                   bias,
                                                                   residual1,
                                                                   residual2,
                                                                   gamma,
                                                                   beta,
                                                                   eps,
                                                                   tokens,
                                                                   hidden_dim,
                                                                   scale_orig_quant_per_tensor,
                                                                   scale_orig_quant_per_token,
                                                                   normed_output_quant,
                                                                   grid,
                                                                   block,
                                                                   shmem_size,
                                                                   stream);
    } else {

        dispatch_rmsnorm_residual<T, IS_OUTPUT, false, QUANT_OUT_T>(output,
                                                                    normed_output,
                                                                    input,
                                                                    bias,
                                                                    residual1,
                                                                    residual2,
                                                                    gamma,
                                                                    beta,
                                                                    eps,
                                                                    tokens,
                                                                    hidden_dim,
                                                                    scale_orig_quant_per_tensor,
                                                                    scale_orig_quant_per_token,
                                                                    normed_output_quant,
                                                                    grid,
                                                                    block,
                                                                    shmem_size,
                                                                    stream);
    }
}

template<typename T, typename QUANT_OUT_T>
void dispatch_rmsnorm_output(T*           output,
                             T*           normed_output,
                             const T*     input,
                             const T*     bias,
                             const T*     residual1,
                             const T*     residual2,
                             const T*     gamma,
                             const T*     beta,
                             const float  eps,
                             size_t       tokens,
                             size_t       hidden_dim,
                             const float* scale_orig_quant_per_tensor,
                             float*       scale_orig_quant_per_token,
                             QUANT_OUT_T* normed_output_quant,
                             const dim3   grid,
                             const dim3   block,
                             const size_t shmem_size,
                             hipStream_t stream,
                             bool         is_output) {
    if (is_output) {

        dispatch_rmsnorm_bias<T, true, QUANT_OUT_T>(output,
                                                    normed_output,
                                                    input,
                                                    bias,
                                                    residual1,
                                                    residual2,
                                                    gamma,
                                                    beta,
                                                    eps,
                                                    tokens,
                                                    hidden_dim,
                                                    scale_orig_quant_per_tensor,
                                                    scale_orig_quant_per_token,
                                                    normed_output_quant,
                                                    grid,
                                                    block,
                                                    shmem_size,
                                                    stream);
    } else {
        dispatch_rmsnorm_bias<T, false, QUANT_OUT_T>(output,
                                                     normed_output,
                                                     input,
                                                     bias,
                                                     residual1,
                                                     residual2,
                                                     gamma,
                                                     beta,
                                                     eps,
                                                     tokens,
                                                     hidden_dim,
                                                     scale_orig_quant_per_tensor,
                                                     scale_orig_quant_per_token,
                                                     normed_output_quant,
                                                     grid,
                                                     block,
                                                     shmem_size,
                                                     stream);
    }
}

template<typename T, typename QUANT_OUT_T>
void invokeGeneralRmsNorm(T*           out,
                          const T*     input,
                          const T*     gamma,
                          const T*     beta,
                          const float  eps,
                          const size_t tokens,
                          const size_t hidden_dim,
                          hipStream_t stream,
                          const float* scale,
                          float*       dynamic_scale,
                          QUANT_OUT_T* normed_output_quant) {
    constexpr size_t vec_size     = 2;
    const bool       use_vec_type = (hidden_dim % vec_size == 0)
                              && (std::is_same<T, half>::value
#ifdef ENABLE_BF16
                                  || std::is_same<T, __hip_bfloat16>::value
#endif
                              );

    dim3 grid(tokens);
    dim3 block(std::min(hidden_dim, 1024ul));
    if (use_vec_type) {
        block.x = std::min(hidden_dim / vec_size, 1024ul);
    }
    // Make sure block.x is multiple of 32 for warp shuffle to work
    block.x = 32 * ((block.x + 31) / 32);

    const size_t shmem_size = hidden_dim * sizeof(T);

    if (use_vec_type) {
        using Tp = typename packed_as<T, vec_size>::type;
        dispatch_rmsnorm_output(reinterpret_cast<Tp*>(out),
                                reinterpret_cast<Tp*>(out),
                                reinterpret_cast<Tp*>(out),
                                (const Tp*)nullptr,
                                reinterpret_cast<const Tp*>(input),
                                (const Tp*)nullptr,
                                reinterpret_cast<const Tp*>(gamma),
                                reinterpret_cast<const Tp*>(beta),
                                eps,
                                tokens,
                                hidden_dim,
                                scale,
                                dynamic_scale,
                                normed_output_quant,
                                grid,
                                block,
                                shmem_size,
                                stream,
                                false);
    } else {
        dispatch_rmsnorm_output(out,
                                out,
                                (const T*)out,
                                (const T*)nullptr,
                                input,
                                (const T*)nullptr,
                                gamma,
                                beta,
                                eps,
                                tokens,
                                hidden_dim,
                                scale,
                                dynamic_scale,
                                normed_output_quant,
                                grid,
                                block,
                                shmem_size,
                                stream,
                                false);
    }
}

template<typename T>
void invokeRmsNormWithStride(T* __restrict output,
                             const size_t out_stride,
                             const T* __restrict input,
                             const size_t in_stride,
                             const T* __restrict gamma,
                             const T* __restrict beta,
                             const float  layernorm_eps,
                             const size_t m,
                             const size_t n,
                             const size_t norm_size,
                             hipStream_t stream) {
    constexpr size_t vec_size  = 2;
    constexpr size_t warp_size = 32;

    // 参数校验
    if (n % norm_size != 0) {
        throw std::invalid_argument("n must be divisible by norm_size");
    }
    if (norm_size % (warp_size * vec_size) != 0) {
        throw std::invalid_argument("norm_size must be multiple of " + std::to_string(warp_size * vec_size));
    }

    const size_t num_heads = n / norm_size;
    dim3         grid(m * num_heads);  // 每个block处理一个样本的一个头
    dim3         block(warp_size);

    using Tp     = typename packed_as<T, vec_size>::type;
    bool is_bias = beta != nullptr;
    if (is_bias) {
        rmsNormWithStride<Tp, true><<<grid, block, 0, stream>>>(reinterpret_cast<Tp*>(output),
                                                                out_stride,
                                                                reinterpret_cast<const Tp*>(input),
                                                                in_stride,
                                                                reinterpret_cast<const Tp*>(gamma),
                                                                reinterpret_cast<const Tp*>(beta),
                                                                layernorm_eps,
                                                                n,
                                                                norm_size);
    } else {
        rmsNormWithStride<Tp, false><<<grid, block, 0, stream>>>(reinterpret_cast<Tp*>(output),
                                                                 out_stride,
                                                                 reinterpret_cast<const Tp*>(input),
                                                                 in_stride,
                                                                 reinterpret_cast<const Tp*>(gamma),
                                                                 reinterpret_cast<const Tp*>(beta),
                                                                 layernorm_eps,
                                                                 n,
                                                                 norm_size);
    }
#if USING_CUDA
    check_cuda_value(hipPeekAtLastError());
    check_cuda_error();
#endif
}

template<typename T, typename QUANT_OUT_T>
void invokeAddBiasResidualRmsNorm(T*           output,
                                  T*           normed_output,
                                  const T*     input,
                                  const T*     bias,
                                  const T*     residual,
                                  const T*     residual2,
                                  const T*     gamma,
                                  const T*     beta,
                                  const float  eps,
                                  const size_t tokens,
                                  const size_t hidden_dim,
                                  hipStream_t stream,
                                  const float* scale,
                                  float*       dynamic_scale,
                                  QUANT_OUT_T* normed_output_quant) {
    dim3 grid(tokens);
    dim3 block(std::min(hidden_dim, 1024ul));
    // Make sure block.x is multiple of 32 for warp shuffle to work
    block.x = 32 * ((block.x + 31) / 32);

    constexpr size_t vec_size     = 2;
    const size_t     shmem_size   = hidden_dim * sizeof(T);
    const bool       use_vec_type = (hidden_dim % vec_size == 0)
                              && (std::is_same<T, half>::value
#ifdef ENABLE_BF16
                                  || std::is_same<T, __hip_bfloat16>::value
#endif
                              );

    if (use_vec_type) {
        using Tp = typename packed_as<T, vec_size>::type;
        dispatch_rmsnorm_output(reinterpret_cast<Tp*>(output),
                                reinterpret_cast<Tp*>(normed_output),
                                reinterpret_cast<const Tp*>(input),
                                reinterpret_cast<const Tp*>(bias),
                                reinterpret_cast<const Tp*>(residual),
                                reinterpret_cast<const Tp*>(residual2),
                                reinterpret_cast<const Tp*>(gamma),
                                reinterpret_cast<const Tp*>(beta),
                                eps,
                                tokens,
                                hidden_dim,
                                scale,
                                dynamic_scale,
                                normed_output_quant,
                                grid,
                                block,
                                shmem_size,
                                stream,
                                true);
    } else {
        dispatch_rmsnorm_output(output,
                                normed_output,
                                input,
                                bias,
                                residual,
                                residual2,
                                gamma,
                                beta,
                                eps,
                                tokens,
                                hidden_dim,
                                scale,
                                dynamic_scale,
                                normed_output_quant,
                                grid,
                                block,
                                shmem_size,
                                stream,
                                true);
    }
}

#define INSTANTIATE_GENERAL_RMSNORM(T, QUANT_OUT_T)                                                                    \
    template void invokeGeneralRmsNorm(T*           out,                                                               \
                                       const T*     input,                                                             \
                                       const T*     gamma,                                                             \
                                       const T*     beta,                                                              \
                                       const float  eps,                                                               \
                                       const size_t tokens,                                                            \
                                       const size_t hidden_dim,                                                        \
                                       hipStream_t stream,                                                            \
                                       const float* scale,                                                             \
                                       float*       dynamic_scale,                                                     \
                                       QUANT_OUT_T* normed_output_quant);

INSTANTIATE_GENERAL_RMSNORM(float, int8_t);
INSTANTIATE_GENERAL_RMSNORM(half, int8_t);

#ifdef ENABLE_BF16
INSTANTIATE_GENERAL_RMSNORM(__hip_bfloat16, int8_t);
#endif

#ifdef ENABLE_FP8
INSTANTIATE_GENERAL_RMSNORM(float, __hip_fp8_e4m3_fnuz);
INSTANTIATE_GENERAL_RMSNORM(half, __hip_fp8_e4m3_fnuz);
#ifdef ENABLE_BF16
INSTANTIATE_GENERAL_RMSNORM(__hip_bfloat16, __hip_fp8_e4m3_fnuz);
#endif  // ENABLE_BF16
#endif  // ENABLE_FP8

#define INSTANTIATE_ADD_BIAS_RESL_RMSNORM(T, QUANT_OUT_T)                                                              \
    template void invokeAddBiasResidualRmsNorm(T*           output,                                                    \
                                               T*           normed_output,                                             \
                                               const T*     input,                                                     \
                                               const T*     bias,                                                      \
                                               const T*     resiudal,                                                  \
                                               const T*     resiudal2,                                                 \
                                               const T*     gamma,                                                     \
                                               const T*     beta,                                                      \
                                               const float  eps,                                                       \
                                               const size_t tokens,                                                    \
                                               const size_t hidden_dim,                                                \
                                               hipStream_t stream,                                                    \
                                               const float* scale,                                                     \
                                               float*       dynamic_scale,                                             \
                                               QUANT_OUT_T* normed_output_quant);

INSTANTIATE_ADD_BIAS_RESL_RMSNORM(float, int8_t);
INSTANTIATE_ADD_BIAS_RESL_RMSNORM(half, int8_t);
#ifdef ENABLE_BF16
INSTANTIATE_ADD_BIAS_RESL_RMSNORM(__hip_bfloat16, int8_t);
#endif

#ifdef ENABLE_FP8
INSTANTIATE_ADD_BIAS_RESL_RMSNORM(float, __hip_fp8_e4m3_fnuz);
INSTANTIATE_ADD_BIAS_RESL_RMSNORM(half, __hip_fp8_e4m3_fnuz);
#ifdef ENABLE_BF16
INSTANTIATE_ADD_BIAS_RESL_RMSNORM(__hip_bfloat16, __hip_fp8_e4m3_fnuz);
#endif  // ENABLE_BF16
#endif  // ENABLE_FP8

#define INSTANTIATE_STRIDED_RMSNORM(T)                                                                                 \
    template void invokeRmsNormWithStride(T* __restrict output,                                                        \
                                          const size_t out_stride,                                                     \
                                          const T* __restrict input,                                                   \
                                          const size_t in_stride,                                                      \
                                          const T* __restrict gamma,                                                   \
                                          const T* __restrict beta,                                                    \
                                          const float  layernorm_eps,                                                  \
                                          const size_t m,                                                              \
                                          const size_t n,                                                              \
                                          const size_t norm_size,                                                      \
                                          hipStream_t stream);
INSTANTIATE_STRIDED_RMSNORM(float);
INSTANTIATE_STRIDED_RMSNORM(half);
#ifdef ENABLE_BF16
INSTANTIATE_STRIDED_RMSNORM(__hip_bfloat16);
#endif
#undef INSTANTIATE_STRIDED_RMSNORM

}  // namespace rtp_llm
