#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <assert.h>
#include <type_traits>
#include "rtp_llm/cpp/cuda/cuda_type_utils.cuh"
#include "rtp_llm/cpp/cuda/cuda_fp8_utils.h"
#if USING_CUDA
#ifndef CUDART_VERSION
#error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 11050)
#include <hipcub/hipcub.hpp>
#else
#include "3rdparty/hipcub/hipcub.hpp"
#endif
#endif
#include "rtp_llm/cpp/kernels/gpt_kernels.h"

namespace rtp_llm {

template<typename T, bool USE_POS_EMB, bool USE_TYPE_ID_EMB, bool USE_MASK>
__global__ void embedding_lookup_kernel(T*            from_tensor,
                                        const T*      embedding_table,
                                        double        input_embedding_scalar,
                                        const T*      pos_table,
                                        const T*      type_table,
                                        const int*    input_ids,
                                        const int*    input_pos,
                                        const int*    input_type,
                                        const int*    input_mask,
                                        const int     token_num,
                                        const int64_t hidden_units) {
    for (int64_t index = blockIdx.x * blockDim.x + threadIdx.x; index < (int64_t)(token_num * hidden_units);
         index += blockDim.x * gridDim.x) {
        const int64_t token_index = index / hidden_units;
        const int64_t col_index   = index % hidden_units;
        const int     input_id    = input_ids[token_index];
        T             embedding   = (T)0.0f;
        T             pos_embed   = (T)0.0f;
        T             type_embed  = (T)0.0f;

        if constexpr (USE_POS_EMB) {
            assert(pos_table != nullptr);
            pos_embed = pos_table[input_pos[token_index] * hidden_units + col_index];
        }
        if constexpr (USE_TYPE_ID_EMB) {
            assert(type_table != nullptr);
            type_embed = type_table[input_type[token_index] * hidden_units + col_index];
        }
        if constexpr (USE_MASK) {
            assert(input_mask != nullptr);
            if (input_mask[token_index] == 0) {
                from_tensor[index] = pos_embed + type_embed;
                continue;
            }
        }

        embedding = embedding_table[input_id * hidden_units + col_index];

        // embedding *= input_embedding_scalar;
        if constexpr (std::is_same<T, __hip_bfloat16>::value) {
            embedding *= __double2bfloat16(input_embedding_scalar);
        } else if constexpr (std::is_same<T, __half>::value) {
            embedding *= static_cast<T>(input_embedding_scalar);
        } else {
            embedding *= input_embedding_scalar;
        }

        from_tensor[index] = embedding + pos_embed + type_embed;
    }
}

#define LDST128BITS(value) (reinterpret_cast<float4*>(&(value))[0])

template<typename VectorType, typename T, bool USE_POS_EMB, bool USE_TYPE_ID_EMB, bool USE_MASK>
__global__ void embedding_lookup_kernel_vec(T*            from_tensor,
                                            const T*      embedding_table,
                                            double        input_embedding_scalar,
                                            const T*      pos_table,
                                            const T*      type_table,
                                            const int*    input_ids,
                                            const int*    input_pos,
                                            const int*    input_type,
                                            const int*    input_mask,
                                            const int     token_num,
                                            const int64_t hidden_units) {
    const int64_t vector_size          = sizeof(VectorType) / sizeof(T);
    const int64_t aligned_hidden_units = hidden_units / vector_size;

    for (int64_t index = blockIdx.x * blockDim.x + threadIdx.x; index < (int64_t)(token_num * aligned_hidden_units);
         index += blockDim.x * gridDim.x) {
        const int64_t token_index = index / aligned_hidden_units;
        const int64_t col_index   = index % aligned_hidden_units;
        const int     input_id    = input_ids[token_index];

        VectorType embedding_vec = reinterpret_cast<const VectorType*>(
            &(embedding_table[input_id * hidden_units + col_index * vector_size]))[0];
        VectorType pos_embed_vec  = {.0f, .0f, .0f, .0f};
        VectorType type_embed_vec = {.0f, .0f, .0f, .0f};

        if constexpr (USE_POS_EMB) {
            assert(pos_table != nullptr);
            pos_embed_vec = LDST128BITS(pos_table[input_pos[token_index] * hidden_units + col_index * vector_size]);
        }
        if constexpr (USE_TYPE_ID_EMB) {
            assert(type_table != nullptr);
            type_embed_vec = LDST128BITS(type_table[input_pos[token_index] * hidden_units + col_index * vector_size]);
        }
        if constexpr (USE_MASK) {
            assert(input_mask != nullptr);
            if (input_mask[token_index] == 0) {
#pragma unroll
                for (int i = 0; i < vector_size; ++i) {
                    from_tensor[index * vector_size + i] =
                        reinterpret_cast<T*>(&pos_embed_vec)[i] + reinterpret_cast<T*>(&type_embed_vec)[i];
                }
                continue;
            }
        }

#pragma unroll
        for (int i = 0; i < vector_size; ++i) {
            if constexpr (std::is_same<T, __hip_bfloat16>::value) {
                reinterpret_cast<T*>(&embedding_vec)[i] *= __double2bfloat16(input_embedding_scalar);
            } else {
                reinterpret_cast<T*>(&embedding_vec)[i] *= static_cast<T>(input_embedding_scalar);
            }
            reinterpret_cast<T*>(&embedding_vec)[i] +=
                reinterpret_cast<T*>(&pos_embed_vec)[i] + reinterpret_cast<T*>(&type_embed_vec)[i];
        }

        LDST128BITS(from_tensor[index * vector_size]) = embedding_vec;
    }
}

#define INVOKE_WORD_EMBED_LOOKUP_VEC(USE_POS, USE_YPE, USE_MASK)                                                       \
    embedding_lookup_kernel_vec<float4, T, USE_POS, USE_YPE, USE_MASK>                                                 \
        <<<grid, block, 0, stream>>>(from_tensor,                                                                      \
                                     embedding_table,                                                                  \
                                     input_embedding_scalar,                                                           \
                                     pos_table,                                                                        \
                                     type_table,                                                                       \
                                     input_ids,                                                                        \
                                     input_pos,                                                                        \
                                     input_type,                                                                       \
                                     input_mask,                                                                       \
                                     token_num,                                                                        \
                                     hidden_units);

#define INVOKE_WORD_EMBED_LOOKUP(USE_POS, USE_YPE, USE_MASK)                                                           \
    embedding_lookup_kernel<T, USE_POS, USE_YPE, USE_MASK><<<grid, block, 0, stream>>>(from_tensor,                    \
                                                                                       embedding_table,                \
                                                                                       input_embedding_scalar,         \
                                                                                       pos_table,                      \
                                                                                       type_table,                     \
                                                                                       input_ids,                      \
                                                                                       input_pos,                      \
                                                                                       input_type,                     \
                                                                                       input_mask,                     \
                                                                                       token_num,                      \
                                                                                       hidden_units);

template<typename T>
void invokeEmebeddingLookupVec(T*           from_tensor,
                               const T*     embedding_table,
                               double       input_embedding_scalar,
                               const T*     pos_table,
                               const T*     type_table,
                               const int*   input_ids,
                               const int*   input_pos,
                               const int*   input_type,
                               const int*   input_mask,
                               const int    token_num,
                               const int    hidden_units,
                               hipStream_t stream) {
    using VectorType          = float4;
    const int64_t vector_size = sizeof(VectorType) / sizeof(T);
    assert(hidden_units % vector_size == 0);
    assert(!pos_table && !type_table && !input_mask);
    dim3 grid(std::min(token_num, 65536));
    dim3 block(std::min(int(hidden_units / vector_size), 1024));
    INVOKE_WORD_EMBED_LOOKUP_VEC(false, false, false);
}

template<typename T>
void invokeEmebeddingLookup(T*           from_tensor,
                            const T*     embedding_table,
                            double       input_embedding_scalar,
                            const T*     pos_table,
                            const T*     type_table,
                            const int*   input_ids,
                            const int*   input_pos,
                            const int*   input_type,
                            const int*   input_mask,
                            const int    token_num,
                            const int    hidden_units,
                            hipStream_t stream) {
    dim3 grid(std::min(token_num, 65536));
    dim3 block(std::min(hidden_units, 1024));
    if (!pos_table) {
        if (!type_table) {
            if (!input_mask) {
                INVOKE_WORD_EMBED_LOOKUP(false, false, false);
            } else {
                INVOKE_WORD_EMBED_LOOKUP(false, false, true);
            }
        } else {
            if (!input_mask) {
                INVOKE_WORD_EMBED_LOOKUP(false, true, false);
            } else {
                INVOKE_WORD_EMBED_LOOKUP(false, true, true);
            }
        }
    } else {
        if (!type_table) {
            if (!input_mask) {
                INVOKE_WORD_EMBED_LOOKUP(true, false, false);
            } else {
                INVOKE_WORD_EMBED_LOOKUP(true, false, true);
            }
        } else {
            if (!input_mask) {
                INVOKE_WORD_EMBED_LOOKUP(true, true, false);
            } else {
                INVOKE_WORD_EMBED_LOOKUP(true, true, true);
            }
        }
    }
}
#undef INVOKE_WORD_EMBED_LOOKUP

// PROMPT_SRC: 0 --> no prompts, 1 --> from loaded prompts, 2 --> from request prompts
template<typename T, bool OUTPUT_ID, int PROMPT_SRC>
__global__ void start_id_embedding_position_lookups_kernel(T*                    from_tensor,
                                                           int*                  output_ids,
                                                           const T*              embedding_table,
                                                           const T*              pos_table,
                                                           pPromptTuningParam<T> prompt_param,
                                                           const int*            input_ids,
                                                           const int             start_step,
                                                           const int             length,
                                                           const int             max_length,
                                                           const int             batch_size,
                                                           const int64_t         hidden_units) {
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < batch_size * length * hidden_units;
         index += blockDim.x * gridDim.x) {
        // transpose the input_ids [batch, length] (part of [batch, max_length]) to output_ids [length, batch]
        if (OUTPUT_ID && index < batch_size * max_length) {
            // for p/prompt_tuning (have prompt templates like [input1, prompt1, input2, prompt2])
            // we have to process it to like [input1, input2, prompt1, prompt2], and then remove the prompts during post
            // processing
            if (PROMPT_SRC > 0) {
                if (index < batch_size) {
                    int no_prompt_output_seq_id = 0;
#pragma unroll 1
                    for (int seq_id = 0; seq_id < max_length; seq_id++) {
                        int current_input_id = input_ids[index * max_length + seq_id];
                        if (current_input_id < prompt_param.p_prompt_tuning_id_start) {
                            output_ids[no_prompt_output_seq_id * batch_size + index] = current_input_id;
                            no_prompt_output_seq_id++;
                        }
                    }
                }
            } else {
                const int seq_id   = index % max_length;
                const int batch_id = index / max_length;
                if (seq_id < length) {
                    output_ids[seq_id * batch_size + batch_id] = input_ids[index];
                }
            }
        }

        // embedding lookup from word ids [batch, length] (part of [batch, max_length]) and [vocab, hidden] to generate
        // embedding [batch, length, hidden]
        const int word_index      = index / hidden_units;
        const int word_index_row  = word_index / length;  // batch_id
        const int word_index_col  = word_index % length;
        const int real_word_index = word_index_row * max_length + word_index_col;
        const int step            = start_step + word_index % length;
        const int col_index       = index % hidden_units;
        const int input_id        = input_ids == nullptr ? real_word_index : input_ids[real_word_index];
        const int prompt_id       = input_id - prompt_param.p_prompt_tuning_id_start;
        T         embedding       = (T)0.0f;
        if (PROMPT_SRC > 0 && prompt_id >= 0) {
            if (PROMPT_SRC == 1) {
                // from loaded prompt embedding tables
                embedding =
                    prompt_param.p_prompt_tuning_batch_weights[word_index_row][prompt_id * hidden_units + col_index];
            } else {
                // from request prompt embedding
                embedding =
                    prompt_param
                        .request_prompt_embedding[word_index_row * prompt_param.request_prompt_max_length * hidden_units
                                                  + prompt_id * hidden_units + col_index];
            }
        } else {
            embedding = embedding_table[input_id * hidden_units + col_index];
        }
        T pos_embed        = pos_table == nullptr ? (T)0.f : pos_table[(step - 1) * hidden_units + col_index];
        from_tensor[index] = embedding + pos_embed;
    }
}

#define WORD_POS_EMBEDDING_LOOPUP_KERNEL(OUTPUT_ID, PROMPT_SRC)                                                        \
    start_id_embedding_position_lookups_kernel<T, OUTPUT_ID, PROMPT_SRC><<<grid, block, 0, stream>>>(from_tensor,      \
                                                                                                     output_ids,       \
                                                                                                     embedding_table,  \
                                                                                                     pos_table,        \
                                                                                                     prompt_param,     \
                                                                                                     input_ids,        \
                                                                                                     start_step,       \
                                                                                                     length,           \
                                                                                                     max_length,       \
                                                                                                     batch_size,       \
                                                                                                     hidden_units);

template<typename T>
void invokeInputIdsEmbeddingLookupPosEncoding(T*                    from_tensor,
                                              int*                  output_ids,
                                              const T*              embedding_table,  // can also be inputs_embeds
                                              const T*              pos_table,
                                              pPromptTuningParam<T> prompt_param,
                                              const int*            input_ids,
                                              const int             start_step,
                                              const int             length,
                                              const int             max_length,
                                              const int             batch_size,
                                              const int             hidden_units,
                                              hipStream_t          stream) {
    dim3       grid(min(batch_size * length, 65536));
    dim3       block(min(hidden_units, 512));
    const bool has_output_ids = output_ids != nullptr;
    RTP_LLM_CHECK(!(has_output_ids && input_ids == nullptr));

    if (has_output_ids) {
        if (prompt_param.use_request_p_prompt_embedding) {
            WORD_POS_EMBEDDING_LOOPUP_KERNEL(true, 2);
        } else if (prompt_param.p_prompt_tuning_batch_weights != nullptr) {
            WORD_POS_EMBEDDING_LOOPUP_KERNEL(true, 1);
        } else {
            WORD_POS_EMBEDDING_LOOPUP_KERNEL(true, 0);
        }
    } else {
        if (prompt_param.use_request_p_prompt_embedding) {
            WORD_POS_EMBEDDING_LOOPUP_KERNEL(false, 2);
        } else if (prompt_param.p_prompt_tuning_batch_weights != nullptr) {
            WORD_POS_EMBEDDING_LOOPUP_KERNEL(false, 1);
        } else {
            WORD_POS_EMBEDDING_LOOPUP_KERNEL(false, 0);
        }
    }
}

template void invokeInputIdsEmbeddingLookupPosEncoding(float*                    from_tensor,
                                                       int*                      output_ids,
                                                       const float*              embedding_table,
                                                       const float*              pos_table,
                                                       pPromptTuningParam<float> prompt_param,
                                                       const int*                input_ids,
                                                       const int                 start_step,
                                                       const int                 length,
                                                       const int                 max_length,
                                                       const int                 batch_size,
                                                       const int                 hidden_units,
                                                       hipStream_t              stream);

template void invokeInputIdsEmbeddingLookupPosEncoding(half*                    from_tensor,
                                                       int*                     output_ids,
                                                       const half*              embedding_table,
                                                       const half*              pos_table,
                                                       pPromptTuningParam<half> prompt_param,
                                                       const int*               input_ids,
                                                       const int                start_step,
                                                       const int                length,
                                                       const int                max_length,
                                                       const int                batch_size,
                                                       const int                hidden_units,
                                                       hipStream_t             stream);

#ifdef ENABLE_BF16
template void invokeInputIdsEmbeddingLookupPosEncoding(__hip_bfloat16*                    from_tensor,
                                                       int*                              output_ids,
                                                       const __hip_bfloat16*              embedding_table,
                                                       const __hip_bfloat16*              pos_table,
                                                       pPromptTuningParam<__hip_bfloat16> prompt_param,
                                                       const int*                        input_ids,
                                                       const int                         start_step,
                                                       const int                         length,
                                                       const int                         max_length,
                                                       const int                         batch_size,
                                                       const int                         hidden_units,
                                                       hipStream_t                      stream);
#endif

#define INSTANTIATE_INVOKE_EMBEDDING_LOOKUP_VEC(T)                                                                     \
    template void invokeEmebeddingLookupVec(T*           from_tensor,                                                  \
                                            const T*     embedding_table,                                              \
                                            double       input_embedding_scalar,                                       \
                                            const T*     pos_table,                                                    \
                                            const T*     type_table,                                                   \
                                            const int*   input_ids,                                                    \
                                            const int*   input_pos,                                                    \
                                            const int*   input_type,                                                   \
                                            const int*   input_mask,                                                   \
                                            const int    token_num,                                                    \
                                            const int    hidden_units,                                                 \
                                            hipStream_t stream)

INSTANTIATE_INVOKE_EMBEDDING_LOOKUP_VEC(float);
INSTANTIATE_INVOKE_EMBEDDING_LOOKUP_VEC(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_EMBEDDING_LOOKUP_VEC(__hip_bfloat16);
#endif

#define INSTANTIATE_INVOKE_EMBEDDING_LOOKUP(T)                                                                         \
    template void invokeEmebeddingLookup(T*           from_tensor,                                                     \
                                         const T*     embedding_table,                                                 \
                                         double       input_embedding_scalar,                                          \
                                         const T*     pos_table,                                                       \
                                         const T*     type_table,                                                      \
                                         const int*   input_ids,                                                       \
                                         const int*   input_pos,                                                       \
                                         const int*   input_type,                                                      \
                                         const int*   input_mask,                                                      \
                                         const int    token_num,                                                       \
                                         const int    hidden_units,                                                    \
                                         hipStream_t stream)

INSTANTIATE_INVOKE_EMBEDDING_LOOKUP(float);
INSTANTIATE_INVOKE_EMBEDDING_LOOKUP(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_EMBEDDING_LOOKUP(__hip_bfloat16);
#endif

template<typename T>
__global__ void
inputIdsEmbeddingLookupPosEncodingSoftPrompt(inputIdsEmbeddingLookupPosEncodingSoftPromptParam<T> param) {
    // 1. Copy the input ids to output ids and transpose output ids to [seq_len, batch_size, beam_width].
    // 2. Embedding lookup by input ids and concat with soft prompt. The axis of concatenation is on axis of seq_len.

    // Assume batch size is 2 and prompts are [[t1, t2], [t3], [t4, t5]], input_ids are [[s1, s2], [s3], [s4]]
    // then the order of output_ids is
    // [ [?, ?, s1, s2]
    //   [?, s3, padding, padding]
    //   [?, ?, s4, padding] ]
    // and the order of embedding is
    // [ [t1, t2, s1, s2]
    //   [t3, s3, padding, padding]
    //   [t4, t5, s4, padding] ]
    // where "?" means undefined values and we should attach it.

    for (int index = blockIdx.x * blockDim.x + threadIdx.x;
         index < param.batch_size * param.beam_width * (param.max_prefix_soft_prompt_length + param.max_input_length)
                     * param.hidden_units;
         index += blockDim.x * gridDim.x) {
        // transpose the input_ids [batch, length] (part of [batch, beam, max_input_length]) to
        // output_ids [length, batch, beam].
        // ouptut_ids need to add padding in the beginning for soft prompting.

        if (index < param.batch_size * param.beam_width * param.max_input_length) {
            int       tmp_index = index;
            const int seq_id    = tmp_index % param.max_input_length;
            tmp_index           = (tmp_index - seq_id) / param.max_input_length;
            const int beam_id   = tmp_index % param.beam_width;
            tmp_index           = (tmp_index - beam_id) / param.beam_width;
            const int batch_id  = tmp_index % param.batch_size;
            if (seq_id < param.max_input_length) {
                param.output_ids[(param.prefix_soft_prompt_lengths[batch_id] + seq_id) * param.batch_size
                                     * param.beam_width
                                 + batch_id * param.beam_width + beam_id] = param.input_ids[index];
            }
        }

        // embedding lookup from word ids [batch, beam, length] (part of [batch, beam, max_input_length]), [vocab,
        // hidden] and [batch, max_prefix_soft_prompt_length, hidden] to generate embedding [batch, beam, length +
        // max_prefix_soft_prompt_length, hidden]
        int       tmp_index    = index;
        const int hidden_id    = tmp_index % param.hidden_units;
        tmp_index              = (tmp_index - hidden_id) / param.hidden_units;
        const int seq_id       = tmp_index % (param.max_prefix_soft_prompt_length + param.max_input_length);
        tmp_index              = (tmp_index - seq_id) / (param.max_prefix_soft_prompt_length + param.max_input_length);
        const int beam_id      = tmp_index % param.beam_width;
        tmp_index              = (tmp_index - beam_id) / param.beam_width;
        const int     batch_id = tmp_index % param.batch_size;
        const int64_t hidden_units = param.hidden_units;
        T             embedding =
            (seq_id < param.prefix_soft_prompt_lengths[batch_id]) ?
                            (T)param.prefix_soft_prompt_embedding[batch_id * param.max_prefix_soft_prompt_length * hidden_units
                                                      + seq_id * hidden_units + hidden_id] :
                            param.embedding_table[param.input_ids[batch_id * param.beam_width * param.max_input_length
                                                      + beam_id * param.max_input_length
                                                      + (seq_id - param.prefix_soft_prompt_lengths[batch_id])]
                                          * hidden_units
                                      + hidden_id];

        T pos_embed              = param.pos_table == nullptr ?
                                       (T)0.0f :
                                       param.pos_table[(param.start_step + seq_id - 1) * hidden_units + hidden_id];
        param.from_tensor[index] = embedding + pos_embed;

        if (seq_id == 0 && hidden_id == 0) {
            param.input_lengths[batch_id * param.beam_width + beam_id] += param.prefix_soft_prompt_lengths[batch_id];
        }
    }
}

template<typename T>
void invokeInputIdsEmbeddingLookupPosEncodingSoftPrompt(inputIdsEmbeddingLookupPosEncodingSoftPromptParam<T> param) {
    dim3 grid(min(param.batch_size * param.beam_width * (param.max_input_length + param.max_prefix_soft_prompt_length),
                  65536));
    dim3 block(min(param.hidden_units, 512));
    inputIdsEmbeddingLookupPosEncodingSoftPrompt<T><<<grid, block, 0, param.stream>>>(param);
}

template void
invokeInputIdsEmbeddingLookupPosEncodingSoftPrompt(inputIdsEmbeddingLookupPosEncodingSoftPromptParam<float> param);

template void
invokeInputIdsEmbeddingLookupPosEncodingSoftPrompt(inputIdsEmbeddingLookupPosEncodingSoftPromptParam<half> param);

#ifdef ENABLE_BF16
template void invokeInputIdsEmbeddingLookupPosEncodingSoftPrompt(
    inputIdsEmbeddingLookupPosEncodingSoftPromptParam<__hip_bfloat16> param);
#endif

// TODO Add half2 implementation
template<typename T>
__global__ void transposeAxis01(T* out, T* in, const int dim0, const int dim1, const int dim2) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < dim0 * dim1 * dim2) {
        const int input_dim2_index = index % dim2;
        index                      = (index - input_dim2_index) / dim2;
        const int input_dim1_index = index % dim1;
        index                      = (index - input_dim1_index) / dim1;
        const int input_dim0_index = index % dim0;

        out[input_dim1_index * dim0 * dim2 + input_dim0_index * dim2 + input_dim2_index] =
            in[input_dim0_index * dim1 * dim2 + input_dim1_index * dim2 + input_dim2_index];
    }
}

template<typename T>
void invokeTransposeAxis012(T* out, T* in, const int dim0, const int dim1, const int dim2, hipStream_t stream) {
    dim3 block(512);
    dim3 grid((int)(ceil(dim0 * dim1 * dim2 / 512.)));
    transposeAxis01<<<grid, block, 0, stream>>>(out, in, dim0, dim1, dim2);
}

template<typename T>
__global__ void transposeAxis01(T* out, T* in, const int* in_skipping_dim1, const int dim0, const int dim1) {
    // out: [dim1, dim0]
    // in: [dim0, dim1]
    // in_skipping_dim1: [dim1]

    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < dim0 * dim1) {
        const int input_dim1_index = index % dim1;
        index                      = (index - input_dim1_index) / dim1;
        const int input_dim0_index = index % dim0;
        const int in_offset        = in_skipping_dim1 == nullptr ? 0 : in_skipping_dim1[input_dim1_index] * dim1;

        out[input_dim1_index * dim0 + input_dim0_index] = in[in_offset + input_dim0_index * dim1 + input_dim1_index];
    }
}

template<typename T>
void invokeTransposeAxis01(T* out, T* in, const int dim0, const int dim1, hipStream_t stream) {
    dim3 block(512);
    dim3 grid((int)(ceil(dim0 * dim1 / 512.)));
    transposeAxis01<<<grid, block, 0, stream>>>(out, in, nullptr, dim0, dim1);
}

#define DEFINE_INVOKETRANSPOSE(T)                                                                                      \
    template void invokeTransposeAxis01(T* out, T* in, const int dim0, const int dim1, hipStream_t stream);           \
    template void invokeTransposeAxis012(                                                                              \
        T* out, T* in, const int dim0, const int dim1, const int dim2, hipStream_t stream)

DEFINE_INVOKETRANSPOSE(int32_t);
DEFINE_INVOKETRANSPOSE(int8_t);
DEFINE_INVOKETRANSPOSE(uint8_t);
DEFINE_INVOKETRANSPOSE(uint32_t);
DEFINE_INVOKETRANSPOSE(int64_t);
DEFINE_INVOKETRANSPOSE(uint64_t);
DEFINE_INVOKETRANSPOSE(float);
DEFINE_INVOKETRANSPOSE(half);
#ifdef ENABLE_BF16
DEFINE_INVOKETRANSPOSE(__hip_bfloat16);
#endif

#ifdef ENABLE_FP8
DEFINE_INVOKETRANSPOSE(__hip_fp8_e4m3_fnuz);
#endif

template<typename T>
__global__ void transposeAxis12(T* out, T* in, const int dim0, const int dim1, const int dim2, const int dim3) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < dim0 * dim1 * dim2 * dim3) {
        const int input_dim3_index = index % dim3;
        index                      = (index - input_dim3_index) / dim3;
        const int input_dim2_index = index % dim2;
        index                      = (index - input_dim2_index) / dim2;
        const int input_dim1_index = index % dim1;
        index                      = (index - input_dim1_index) / dim1;
        const int input_dim0_index = index % dim0;
        out[input_dim0_index * dim1 * dim2 * dim3 + input_dim2_index * dim1 * dim3 + input_dim1_index * dim3
            + input_dim3_index]    = in[input_dim0_index * dim1 * dim2 * dim3 + input_dim1_index * dim2 * dim3
                                     + input_dim2_index * dim3 + input_dim3_index];
    }
}

template<typename T>
void invokeTransposeAxis12(
    T* out, T* in, const int dim0, const int dim1, const int dim2, const int dim_3, hipStream_t stream) {
    dim3 block(512);
    dim3 grid((int)(ceil(dim0 * dim1 * dim2 * dim_3 / 512.)));
    transposeAxis12<<<grid, block, 0, stream>>>(out, in, dim0, dim1, dim2, dim_3);
}

template void invokeTransposeAxis12(
    float* out, float* in, const int dim0, const int dim1, const int dim2, const int dim_3, hipStream_t stream);

template void invokeTransposeAxis12(
    half* out, half* in, const int dim0, const int dim1, const int dim2, const int dim_3, hipStream_t stream);

template void invokeTransposeAxis12(
    int* out, int* in, const int dim0, const int dim1, const int dim2, const int dim_3, hipStream_t stream);

#ifdef ENABLE_BF16
template void invokeTransposeAxis12(__hip_bfloat16* out,
                                    __hip_bfloat16* in,
                                    const int      dim0,
                                    const int      dim1,
                                    const int      dim2,
                                    const int      dim_3,
                                    hipStream_t   stream);
#endif

template<typename T, bool PREFIX_PROMPT, bool IS_CAUSAL>
__global__ void buildDecoderAttentionMaskKernel(T*         attention_mask,
                                                const int* sequence_lengths,
                                                const int* prefix_prompt_lengths,
                                                const int  max_seq_len,
                                                const int  max_prompt_length) {
    // sequence_lengths: [batch_size]
    // attention_mask: [batch_size, 1, max_seq_len, max_seq_len + max_prompt_length]
    const int max_prompt_seq_length = max_seq_len + max_prompt_length;
    const int mask_size_per_seq     = max_seq_len * max_prompt_seq_length;
    attention_mask += blockIdx.x * mask_size_per_seq;
    const int seq_length    = sequence_lengths[blockIdx.x];
    const int prompt_length = PREFIX_PROMPT ? prefix_prompt_lengths[blockIdx.x] : 0;
    for (int i = threadIdx.x; i < mask_size_per_seq; i += blockDim.x) {
        int row_id       = i / max_prompt_seq_length;
        int col_id       = i % max_prompt_seq_length;
        int column_bound = IS_CAUSAL ? row_id + prompt_length : seq_length - 1;
        if (row_id < seq_length && col_id <= (column_bound)) {
            attention_mask[i] = (T)(1.0f);
        } else {
            attention_mask[i] = (T)(0.0f);
        }
    }
}

template<typename T>
void invokeBuildDecoderAttentionMask(T*           attention_mask,
                                     const int*   sequence_lengths,
                                     const int*   prefix_prompt_lengths,
                                     const int    batch_size,
                                     const int    max_seq_len,
                                     const int    max_prompt_length,
                                     const bool   is_causal,
                                     hipStream_t stream) {
#define RUN_KERNEL(has_prefix, is_causal)                                                                              \
    buildDecoderAttentionMaskKernel<T, has_prefix, is_causal><<<batch_size, 256, 0, stream>>>(                         \
        attention_mask, sequence_lengths, prefix_prompt_lengths, max_seq_len, max_prompt_length)

    if (max_prompt_length == 0) {
        if (is_causal) {
            RUN_KERNEL(false, true);
        } else {
            RUN_KERNEL(false, false);
        }
    } else {
        if (is_causal) {
            RUN_KERNEL(true, true);
        } else {
            RUN_KERNEL(true, false);
        }
    }
}

template void invokeBuildDecoderAttentionMask(float*       attention_mask,
                                              const int*   sequence_lengths,
                                              const int*   prefix_prompt_lengths,
                                              const int    batch_size,
                                              const int    max_seq_len,
                                              const int    max_prompt_length,
                                              const bool   is_causal,
                                              hipStream_t stream);
template void invokeBuildDecoderAttentionMask(half*        attention_mask,
                                              const int*   sequence_lengths,
                                              const int*   prefix_prompt_lengths,
                                              const int    batch_size,
                                              const int    max_seq_len,
                                              const int    max_prompt_length,
                                              const bool   is_causal,
                                              hipStream_t stream);
#ifdef ENABLE_BF16
template void invokeBuildDecoderAttentionMask(__hip_bfloat16* attention_mask,
                                              const int*     sequence_lengths,
                                              const int*     prefix_prompt_lengths,
                                              const int      batch_size,
                                              const int      max_seq_len,
                                              const int      max_prompt_length,
                                              const bool     is_causal,
                                              hipStream_t   stream);
#endif
#ifdef ENABLE_FP8
template void invokeBuildDecoderAttentionMask(__hip_fp8_e4m3_fnuz* attention_mask,
                                              const int*     sequence_lengths,
                                              const int*     prefix_prompt_lengths,
                                              const int      batch_size,
                                              const int      max_seq_len,
                                              const int      max_prompt_length,
                                              const bool     is_causal,
                                              hipStream_t   stream);
#endif

// The attention_mask only will be used in encode part, so just ignore the case when row_id >= length.
template<typename T>
__global__ void
buildGlmDecoderAttentionMaskKernel(T* attention_mask, const int* sequence_lengths, const int max_seq_len) {
    // sequence_lengths: [batch_size]
    // attention_mask: [batch_size, 1, max_seq_len, max_seq_len]
    attention_mask += blockIdx.x * max_seq_len * max_seq_len;
    const int seq_length = sequence_lengths[blockIdx.x];
    for (int i = threadIdx.x; i < max_seq_len * max_seq_len; i += blockDim.x) {
        int row_id = i / max_seq_len;
        int col_id = i % max_seq_len;
        if (row_id < seq_length && col_id <= row_id) {
            attention_mask[i] = (T)(1.0f);
        } else if (col_id < seq_length - 1) {
            attention_mask[i] = (T)(1.0f);
        } else {
            attention_mask[i] = (T)(0.0f);
        }
    }
}

template<typename T>
void invokeBuildGlmDecoderAttentionMask(
    T* attention_mask, const int* sequence_lengths, const int batch_size, const int max_seq_len, hipStream_t stream) {
    buildGlmDecoderAttentionMaskKernel<<<batch_size, 256, 0, stream>>>(attention_mask, sequence_lengths, max_seq_len);
}

template void invokeBuildGlmDecoderAttentionMask(float*       attention_mask,
                                                 const int*   sequence_lengths,
                                                 const int    batch_size,
                                                 const int    max_seq_len,
                                                 hipStream_t stream);
template void invokeBuildGlmDecoderAttentionMask(half*        attention_mask,
                                                 const int*   sequence_lengths,
                                                 const int    batch_size,
                                                 const int    max_seq_len,
                                                 hipStream_t stream);
#ifdef ENABLE_BF16
template void invokeBuildGlmDecoderAttentionMask(__hip_bfloat16* attention_mask,
                                                 const int*     sequence_lengths,
                                                 const int      batch_size,
                                                 const int      max_seq_len,
                                                 hipStream_t   stream);
#endif

template<typename T>
__launch_bounds__(1024, 1) __global__ void lookupHiddenStateOfLastToken(T*         from_tensor,
                                                                        const T*   hidden_state,
                                                                        const int* input_lengths,
                                                                        const int  batch_size,
                                                                        const int  hidden_units,
                                                                        const int  idx_offset) {
    for (int64_t index = (int64_t)blockIdx.x * blockDim.x + threadIdx.x; index < (int64_t)batch_size * hidden_units;
         index += (int64_t)blockDim.x * gridDim.x) {
        const int64_t col_index = index % hidden_units;
        const int64_t batch_id  = index / hidden_units;
        from_tensor[index] = hidden_state[((int64_t)input_lengths[batch_id] + idx_offset) * hidden_units + col_index];
    }
}

template<typename T>
__launch_bounds__(1024, 1) __global__ void lookupHiddenStateOfFirstToken(
    T* from_tensor, const T* hidden_state, const int* input_lengths, const int batch_size, const int hidden_units) {
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < batch_size * hidden_units;
         index += blockDim.x * gridDim.x) {
        const int col_index  = index % hidden_units;
        const int batch_id   = index / hidden_units;
        const int base_index = batch_id == 0 ? 0 : input_lengths[batch_id - 1] * hidden_units;
        from_tensor[index]   = hidden_state[base_index + col_index];
    }
}

template<typename T>
void invokeLookupHiddenStateOfLastToken(T*           from_tensor,
                                        const T*     hidden_state,
                                        const int*   input_lengths,
                                        const int    batch_size,
                                        const int    hidden_units,
                                        const int    idx_offset,
                                        hipStream_t stream) {
    const int grid_size = (int)(ceil(batch_size * hidden_units / 1024.));
    dim3      grid(min(grid_size, 65536));
    dim3      block(min(hidden_units, 1024));
    lookupHiddenStateOfLastToken<T>
        <<<grid, block, 0, stream>>>(from_tensor, hidden_state, input_lengths, batch_size, hidden_units, idx_offset);
}

template<typename T>
void invokeLookupHiddenStateOfFirstToken(T*           from_tensor,
                                         const T*     hidden_state,
                                         const int*   input_lengths,
                                         const int    batch_size,
                                         const int    hidden_units,
                                         hipStream_t stream) {
    const int grid_size = (int)(ceil(batch_size * hidden_units / 1024.));
    dim3      grid(min(grid_size, 65536));
    dim3      block(min(hidden_units, 1024));
    lookupHiddenStateOfFirstToken<T>
        <<<grid, block, 0, stream>>>(from_tensor, hidden_state, input_lengths, batch_size, hidden_units);
}

#define INSTANTIATE_INVOKE_LOOKUP_HIDDEN_OF_LAST(T)                                                                    \
    template void invokeLookupHiddenStateOfLastToken(T*           from_tensor,                                         \
                                                     const T*     hidden_state,                                        \
                                                     const int*   input_lengths,                                       \
                                                     const int    batch_size,                                          \
                                                     const int    hidden_units,                                        \
                                                     const int    idx_offset,                                          \
                                                     hipStream_t stream)

INSTANTIATE_INVOKE_LOOKUP_HIDDEN_OF_LAST(float);
INSTANTIATE_INVOKE_LOOKUP_HIDDEN_OF_LAST(half);
INSTANTIATE_INVOKE_LOOKUP_HIDDEN_OF_LAST(int32_t);
INSTANTIATE_INVOKE_LOOKUP_HIDDEN_OF_LAST(int8_t);
INSTANTIATE_INVOKE_LOOKUP_HIDDEN_OF_LAST(uint8_t);
INSTANTIATE_INVOKE_LOOKUP_HIDDEN_OF_LAST(uint32_t);
INSTANTIATE_INVOKE_LOOKUP_HIDDEN_OF_LAST(int64_t);
INSTANTIATE_INVOKE_LOOKUP_HIDDEN_OF_LAST(uint64_t);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_LOOKUP_HIDDEN_OF_LAST(__hip_bfloat16);
#endif

#ifdef ENABLE_FP8
INSTANTIATE_INVOKE_LOOKUP_HIDDEN_OF_LAST(__hip_fp8_e4m3_fnuz);
#endif

#define INSTANTIATE_INVOKE_LOOKUP_HIDDEN_OF_FIRST(T)                                                                   \
    template void invokeLookupHiddenStateOfFirstToken(T*           from_tensor,                                        \
                                                      const T*     hidden_state,                                       \
                                                      const int*   input_lengths,                                      \
                                                      const int    batch_size,                                         \
                                                      const int    hidden_units,                                       \
                                                      hipStream_t stream)

INSTANTIATE_INVOKE_LOOKUP_HIDDEN_OF_FIRST(float);
INSTANTIATE_INVOKE_LOOKUP_HIDDEN_OF_FIRST(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_LOOKUP_HIDDEN_OF_FIRST(__hip_bfloat16);
#endif

template<bool PREFIX_PROMPT>
__global__ void tileGptPromptInputs(int*       tiled_input_ids,
                                    int*       tiled_input_lengths,
                                    int*       tiled_prompt_lengths,
                                    const int* input_ids,
                                    const int* input_lengths,
                                    const int* prefix_prompt_lengths,
                                    const int  max_input_length) {
    if (threadIdx.x == 0) {
        tiled_input_lengths[blockIdx.x * gridDim.y + blockIdx.y] = input_lengths[blockIdx.x];
        if (PREFIX_PROMPT) {
            tiled_prompt_lengths[blockIdx.x * gridDim.y + blockIdx.y] = prefix_prompt_lengths[blockIdx.x];
        }
    }
    for (int index = threadIdx.x; index < max_input_length; index += blockDim.x) {
        tiled_input_ids[(blockIdx.x * gridDim.y + blockIdx.y) * max_input_length + index] =
            input_ids[blockIdx.x * max_input_length + index];
    }
}

void invokeTileGptPromptInputs(int*         tiled_input_ids,
                               int*         tiled_input_lengths,
                               int*         tiled_prompt_lengths,
                               const int*   input_ids,
                               const int*   input_lengths,
                               const int*   prefix_prompt_lengths,
                               const int    batch_size,
                               const int    beam_width,
                               const int    max_input_length,
                               hipStream_t stream) {
    dim3 grid(batch_size, beam_width);
    dim3 block(min(1024, max_input_length));
    if (prefix_prompt_lengths != nullptr) {
        tileGptPromptInputs<true><<<grid, block, 0, stream>>>(tiled_input_ids,
                                                              tiled_input_lengths,
                                                              tiled_prompt_lengths,
                                                              input_ids,
                                                              input_lengths,
                                                              prefix_prompt_lengths,
                                                              max_input_length);
    } else {
        tileGptPromptInputs<false><<<grid, block, 0, stream>>>(tiled_input_ids,
                                                               tiled_input_lengths,
                                                               tiled_prompt_lengths,
                                                               input_ids,
                                                               input_lengths,
                                                               prefix_prompt_lengths,
                                                               max_input_length);
    }
}

void invokeTileGptInputs(int*         tiled_input_ids,
                         int*         tiled_input_lengths,
                         const int*   input_ids,
                         const int*   input_lengths,
                         const int    batch_size,
                         const int    beam_width,
                         const int    max_input_length,
                         hipStream_t stream) {
    invokeTileGptPromptInputs(tiled_input_ids,
                              tiled_input_lengths,
                              nullptr,
                              input_ids,
                              input_lengths,
                              nullptr,
                              batch_size,
                              beam_width,
                              max_input_length,
                              stream);
}

#if USING_CUDA

template<int TB_SIZE>
__global__ void
find_context_dups(int* shared_contexts, const int* input_ids, const size_t batch_size, const size_t input_seq_len) {
    /* We compare all context pairs (i, j), with i (tgt) < j (src) , to detect duplicate
     * inputs. If there's a match between i and j, we store i at the
     * j-th position of shared_context. So that we know that j can be
     * represented by i. shared_contexts is initialized like shared_contexts[i] = i
     * and when there's a match, we actually use shared_contexts[j] = min(shared_contexts[j], i)
     * so that in the end, shared_contexts effectively contains an index
     * to the match with the lowest index context.
     * Note that shared_contexts[i] <= i, a property that will be used when uncompacting
     * inputs.
     */
    typedef hipcub::BlockReduce<int, TB_SIZE>       BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    __shared__ bool                              match;

    /* Each block is responsible for a (i, j) pair. To map the block space to
     * the i < j space, we need to convert a linear addressing to a triangle, of
     * size (batch_size * (batch_size - 1)) / 2
     * For more information, check https://en.wikipedia.org/wiki/Triangular_number
     */

    // blockIdx = [0, 1, 2, ... n(n-1)/2] -> base_index = [0, 1, 1, 2, 2, 2, 3, 3, 3, 3, ..., n - 2]
    const int base_index = floorf(0.5f * (sqrtf(1 + 8 * blockIdx.x) - 1));
    const int src_idx    = base_index + 1;  // base_index \in [1, batch_size)

    const int rev_base_index = base_index * (base_index + 1) / 2;
    const int tgt_idx        = blockIdx.x - rev_base_index;  // tgt_idx \in [0, src_idx)

    const int padded_length = TB_SIZE * ((input_seq_len + TB_SIZE - 1) / TB_SIZE);

    int sum = 0;
    for (int i = threadIdx.x; i < padded_length; i += TB_SIZE) {
        int compare =
            (i >= input_seq_len) ? 1 : input_ids[tgt_idx * input_seq_len + i] == input_ids[src_idx * input_seq_len + i];

        sum = BlockReduce(temp_storage).Sum(compare);

        if (threadIdx.x == 0) {
            match = (sum == TB_SIZE);
        }

        __syncthreads();

        if (!match) {
            break;
        }
    }

    if (threadIdx.x == 0 && match) {
        atomicMin(&shared_contexts[src_idx], tgt_idx);
    }
}

constexpr int DUPS_INDICES_BLOCK_SIZE = 128;

__global__ void generate_dups_indices(int*         batch_to_compact,
                                      int*         compact_to_batch,
                                      int*         compact_size,
                                      const int*   shared_contexts,
                                      const size_t batch_size,
                                      const size_t input_seq_len) {
    const int padded_batchsize = blockDim.x * ((batch_size + blockDim.x - 1) / blockDim.x);

    typedef hipcub::BlockScan<int, DUPS_INDICES_BLOCK_SIZE, hipcub::BLOCK_SCAN_WARP_SCANS> BlockScan;
    __shared__ typename BlockScan::TempStorage                                       temp_storage;
    __shared__ int                                                                   scan_offset;

    int scan = 0;
    for (int batch = threadIdx.x; batch < padded_batchsize; batch += blockDim.x) {
        bool masked     = (batch >= batch_size);
        bool first_iter = batch < blockDim.x;

        int is_first_occur = masked ? 0 : shared_contexts[batch] == batch;
        BlockScan(temp_storage).ExclusiveSum(is_first_occur, scan);

        if (!masked && is_first_occur) {
            int compact_idx = scan + (first_iter ? 0 : scan_offset);
            // Context rep. writes initial index
            batch_to_compact[batch]       = compact_idx;
            compact_to_batch[compact_idx] = batch;
        }

        if (threadIdx.x == blockDim.x - 1) {
            scan_offset = scan + is_first_occur + (first_iter ? 0 : scan_offset);
        }

        __syncthreads();

        if (!masked && !is_first_occur) {
            // Fill the rest of batch_to_compact based on what rep. wrote
            const int src_idx       = batch_to_compact[shared_contexts[batch]];
            batch_to_compact[batch] = src_idx;
        }
    }

    if (threadIdx.x == 0) {
        *compact_size = scan_offset;
    }
}

__global__ void init_shared_contexts(int* shared_contexts, const size_t batch_size) {
    const int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_idx >= batch_size) {
        return;
    }
    shared_contexts[global_idx] = global_idx;
}

void invokeFindContextDups(int*         shared_contexts,
                           int*         batch_to_compact,
                           int*         compact_to_batch,
                           int*         compact_size,
                           const int*   input_ids,
                           const size_t batch_size,
                           const size_t input_seq_len,
                           hipStream_t stream) {
    dim3 block{512};
    dim3 grid{((int)batch_size + block.x - 1) / block.x};
    init_shared_contexts<<<grid, block, 0, stream>>>(shared_contexts, batch_size);

    grid = dim3{(unsigned int)(batch_size * (batch_size - 1)) / 2};
    if (input_seq_len <= 128) {
        block = 128;
        find_context_dups<128><<<grid, block, 0, stream>>>(shared_contexts, input_ids, batch_size, input_seq_len);
    } else {
        block = 256;
        find_context_dups<256><<<grid, block, 0, stream>>>(shared_contexts, input_ids, batch_size, input_seq_len);
    }

    generate_dups_indices<<<1, DUPS_INDICES_BLOCK_SIZE, 0, stream>>>(
        batch_to_compact, compact_to_batch, compact_size, shared_contexts, batch_size, input_seq_len);
}
#endif

template<typename T>
__global__ void compact_inputs(T*         compact_input,
                               T*         compact_attention_mask,
                               int*       compact_input_lengths,
                               const T*   decoder_input,
                               const T*   decoder_mask,
                               const int* input_lengths,
                               const int* compact_idx,
                               size_t     compact_size,
                               size_t     seq_len,
                               size_t     hidden_dimension) {
    const int global_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (global_idx < compact_size * seq_len * hidden_dimension) {
        const int h_id     = global_idx % hidden_dimension;
        const int seq_id   = (global_idx / hidden_dimension) % seq_len;
        const int batch_id = global_idx / (hidden_dimension * seq_len);

        compact_input[global_idx] = decoder_input[(compact_idx[batch_id] * seq_len + seq_id) * hidden_dimension + h_id];
    }

    if (global_idx < compact_size * seq_len * seq_len) {
        const int seq1_id  = global_idx % seq_len;
        const int seq2_id  = (global_idx / seq_len) % seq_len;
        const int batch_id = global_idx / (seq_len * seq_len);

        compact_attention_mask[global_idx] =
            decoder_mask[(compact_idx[batch_id] * seq_len + seq2_id) * seq_len + seq1_id];
    }

    if (global_idx < compact_size) {
        compact_input_lengths[global_idx] = input_lengths[compact_idx[global_idx]];
    }
}

template<typename T>
void invokeCompactInputs(T*           compact_input,
                         T*           compact_attention_mask,
                         int*         compact_input_lengths,
                         const T*     decoder_input,
                         const T*     decoder_mask,
                         const int*   input_lengths,
                         const int*   compact_idx,
                         size_t       compact_size,
                         size_t       seq_len,
                         size_t       hidden_dimension,
                         hipStream_t stream) {
    /* Compact relevant decoder_layer inputs based on the identical contexts.
     * For example, decoder_input is [batch_size, seq_len, H]. It's compacted
     * into compact_input [compact_size, seq_len, H] such that
     * compact_input[i, ...] = decoder_input[compact_idx[i], ...] */
    const size_t elems_n = compact_size * seq_len * max(hidden_dimension, seq_len);
    const dim3   blockDim(512);
    const dim3   gridDim((elems_n + 512 - 1) / 512);

    compact_inputs<T><<<gridDim, blockDim, 0, stream>>>(compact_input,
                                                        compact_attention_mask,
                                                        compact_input_lengths,
                                                        decoder_input,
                                                        decoder_mask,
                                                        input_lengths,
                                                        compact_idx,
                                                        compact_size,
                                                        seq_len,
                                                        hidden_dimension);
}

#define INSTANTIATE_INVOKE_COMPACT_INPUTS(T)                                                                           \
    template void invokeCompactInputs<T>(T * compact_input,                                                            \
                                         T * compact_attention_mask,                                                   \
                                         int*         compact_input_lengths,                                           \
                                         const T*     decoder_input,                                                   \
                                         const T*     decoder_mask,                                                    \
                                         const int*   input_lengths,                                                   \
                                         const int*   compact_idx,                                                     \
                                         size_t       compact_size,                                                    \
                                         size_t       seq_len,                                                         \
                                         size_t       hidden_dimension,                                                \
                                         hipStream_t stream)
INSTANTIATE_INVOKE_COMPACT_INPUTS(half);
INSTANTIATE_INVOKE_COMPACT_INPUTS(float);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_COMPACT_INPUTS(__hip_bfloat16);
#endif
#undef INSTANTIATE_INVOKE_COMPACT_INPUTS

template<typename T>
__global__ void uncompact_outputs(T*         uncompact_buffer,
                                  const T*   compact_buffer,
                                  const int* batch_to_compact_idx,
                                  size_t     batch_size,
                                  size_t     buffer_stride) {
    /* Uncompact a buffer IN of size [Compact, Stride] into OUT of size [Batch, Stride]
     * so that \forall i, OUT[i, :] = IN[batch_to_compact_idx[i], :]
     */
    const int global_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (global_idx >= batch_size * buffer_stride) {
        return;
    }

    const int stride_idx = global_idx % buffer_stride;
    const int batch_idx  = global_idx / buffer_stride;

    const int src                = batch_to_compact_idx[batch_idx];
    uncompact_buffer[global_idx] = compact_buffer[src * buffer_stride + stride_idx];
}

template<typename T>
void invokeUnCompactOutputs(T*           uncompact_buffer,
                            const T*     compact_buffer,
                            const int*   batch_to_compact_idx,
                            size_t       batch_size,
                            size_t       buffer_stride,
                            hipStream_t stream) {
    const size_t num_elems = batch_size * buffer_stride;
    const dim3   blockDim(1024);
    const dim3   gridDim((num_elems + blockDim.x - 1) / blockDim.x);

    uncompact_outputs<T><<<gridDim, blockDim, 0, stream>>>(
        uncompact_buffer, compact_buffer, batch_to_compact_idx, batch_size, buffer_stride);
}

#define INSTANTIATE_INVOKE_UNCOMPACT_OUTPUTS(T)                                                                        \
    template void invokeUnCompactOutputs(T*           uncompact_buffer,                                                \
                                         const T*     compact_buffer,                                                  \
                                         const int*   batch_to_compact_idx,                                            \
                                         size_t       batch_size,                                                      \
                                         size_t       buffer_stride,                                                   \
                                         hipStream_t stream)
INSTANTIATE_INVOKE_UNCOMPACT_OUTPUTS(half);
INSTANTIATE_INVOKE_UNCOMPACT_OUTPUTS(float);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_UNCOMPACT_OUTPUTS(__hip_bfloat16);
#endif
#undef INSTANTIATE_INVOKE_UNCOMPACT_OUTPUTS

template<typename T>
__global__ void uncompact_caches(T*         uncompact_k_cache,
                                 T*         uncompact_v_cache,
                                 const T*   compact_k_cache,
                                 const T*   compact_v_cache,
                                 const int* batch_to_compact_idx,
                                 size_t     batch_size,
                                 size_t     num_heads,
                                 size_t     max_seq_len,
                                 size_t     seq_len,
                                 size_t     size_per_head,
                                 size_t     local_batch_size,
                                 size_t     ite) {
    const int hidden_dimension    = num_heads * size_per_head;
    const int num_elems_per_batch = seq_len * hidden_dimension;
    const int num_elems_cache     = batch_size * num_elems_per_batch;
    const int x_size              = 16 / sizeof(T);

    for (int global_idx = blockIdx.x * blockDim.x + threadIdx.x; global_idx < 2 * num_elems_cache;
         global_idx += blockDim.x * gridDim.x) {

        const bool     handle_k  = global_idx < num_elems_cache;
        const T* const cache_src = handle_k ? compact_k_cache : compact_v_cache;
        T* const       cache_dst = handle_k ? uncompact_k_cache : uncompact_v_cache;
        const int      idx       = handle_k ? global_idx : global_idx - num_elems_cache;

        const int src_offset = idx % num_elems_per_batch;
        const int batch_idx  = idx / num_elems_per_batch;
        const int batch_src  = batch_to_compact_idx[batch_idx] - ite * local_batch_size;

        if (batch_src < 0 || batch_src >= local_batch_size) {
            continue;
        }

        int dst_offset;
        if (handle_k) {
            const int i0 = idx % (x_size * seq_len);
            const int i1 = (idx / (x_size * seq_len)) % (num_heads * size_per_head / x_size);
            dst_offset   = i1 * max_seq_len * x_size + i0;
        } else {
            const int i0 = idx % (size_per_head * seq_len);
            const int i1 = (idx / (size_per_head * seq_len)) % (num_heads);
            dst_offset   = i1 * max_seq_len * size_per_head + i0;
        }

        cache_dst[batch_idx * max_seq_len * hidden_dimension + dst_offset] =
            cache_src[batch_src * num_elems_per_batch + src_offset];
    }
}

template<typename T>
void invokeUnCompactCaches(T*           uncompact_k_cache,
                           T*           uncompact_v_cache,
                           const T*     compact_k_cache,
                           const T*     compact_v_cache,
                           const int*   batch_to_compact_idx,
                           size_t       batch_size,
                           size_t       num_heads,
                           size_t       max_seq_len,
                           size_t       seq_len,
                           size_t       size_per_head,
                           size_t       local_batch_size,
                           size_t       ite,
                           hipStream_t stream) {
    const dim3 blockDim(512);
    const dim3 gridDim(1024);
    uncompact_caches<T><<<gridDim, blockDim, 0, stream>>>(uncompact_k_cache,
                                                          uncompact_v_cache,
                                                          compact_k_cache,
                                                          compact_v_cache,
                                                          batch_to_compact_idx,
                                                          batch_size,
                                                          num_heads,
                                                          max_seq_len,
                                                          seq_len,
                                                          size_per_head,
                                                          local_batch_size,
                                                          ite);
}

#define INSTANTIATE_INVOKE_UNCOMPACT_CACHES(T)                                                                         \
    template void invokeUnCompactCaches(T*           uncompact_k_cache,                                                \
                                        T*           uncompact_v_cache,                                                \
                                        const T*     compact_k_cache,                                                  \
                                        const T*     compact_v_cache,                                                  \
                                        const int*   batch_to_compact_idx,                                             \
                                        size_t       batch_size,                                                       \
                                        size_t       num_heads,                                                        \
                                        size_t       max_seq_len,                                                      \
                                        size_t       seq_len,                                                          \
                                        size_t       size_per_head,                                                    \
                                        size_t       local_batch_size,                                                 \
                                        size_t       ite,                                                              \
                                        hipStream_t stream)
INSTANTIATE_INVOKE_UNCOMPACT_CACHES(half);
INSTANTIATE_INVOKE_UNCOMPACT_CACHES(float);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_UNCOMPACT_CACHES(__hip_bfloat16);
#endif
#undef INSTANTIATE_INVOKE_UNCOMPACT_CACHES

template<bool PREFIX_PROMPT>
__global__ void update_padding_count(int*       total_padding_count,
                                     const int* input_lengths,
                                     const int* tiled_prompt_lengths,
                                     size_t     max_input_length,
                                     size_t     max_prompt_length,
                                     size_t     batch_size,
                                     size_t     beam_width) {
    const int gidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (gidx >= batch_size * beam_width) {
        return;
    }

    const int batch_idx = gidx / beam_width;

    total_padding_count[gidx] +=
        PREFIX_PROMPT ? (max_input_length + max_prompt_length - input_lengths[batch_idx] - tiled_prompt_lengths[gidx]) :
                        (max_input_length - input_lengths[batch_idx]);
}

void invokeUpdatePaddingCount(int*         total_padding_count,
                              const int*   input_lengths,
                              const int*   tiled_prompt_lengths,
                              size_t       max_input_length,
                              size_t       max_prompt_length,
                              size_t       batch_size,
                              size_t       beam_width,
                              hipStream_t stream) {
    dim3 blockSize(256);
    dim3 gridSize((batch_size * beam_width + blockSize.x - 1) / blockSize.x);

    if (tiled_prompt_lengths != nullptr) {
        update_padding_count<true><<<gridSize, blockSize, 0, stream>>>(total_padding_count,
                                                                       input_lengths,
                                                                       tiled_prompt_lengths,
                                                                       max_input_length,
                                                                       max_prompt_length,
                                                                       batch_size,
                                                                       beam_width);
    } else {
        update_padding_count<false><<<gridSize, blockSize, 0, stream>>>(total_padding_count,
                                                                        input_lengths,
                                                                        tiled_prompt_lengths,
                                                                        max_input_length,
                                                                        max_prompt_length,
                                                                        batch_size,
                                                                        beam_width);
    }
}

template<bool PREFIX_PROMPT>
__global__ void mask_padding_tokens(bool*        masked_tokens,
                                    const int*   input_lengths,
                                    const int*   tiled_prefix_prompt_lengths,
                                    const size_t memory_len,
                                    const size_t max_input_length,
                                    const size_t initial_step,
                                    size_t       beam_width) {
    const int seq_len = PREFIX_PROMPT ?
                            (input_lengths[blockIdx.x / beam_width] + tiled_prefix_prompt_lengths[blockIdx.x]) :
                            input_lengths[blockIdx.x / beam_width];
    for (int step = initial_step + seq_len + threadIdx.x; step < initial_step + max_input_length; step += blockDim.x) {
        masked_tokens[blockIdx.x * memory_len + step % memory_len] = true;
    }
}

void invokeMaskPaddingTokens(bool*        masked_tokens,
                             const int*   input_lengths,
                             const int*   tiled_prefix_prompt_lengths,
                             const size_t memory_len,
                             const size_t max_input_length,
                             const size_t initial_step,
                             size_t       batch_size,
                             size_t       beam_width,
                             hipStream_t stream) {
    dim3 blockSize(128);
    dim3 gridSize(batch_size * beam_width);
    if (tiled_prefix_prompt_lengths != nullptr) {
        mask_padding_tokens<true><<<gridSize, blockSize, 0, stream>>>(masked_tokens,
                                                                      input_lengths,
                                                                      tiled_prefix_prompt_lengths,
                                                                      memory_len,
                                                                      max_input_length,
                                                                      initial_step,
                                                                      beam_width);
    } else {
        mask_padding_tokens<false><<<gridSize, blockSize, 0, stream>>>(masked_tokens,
                                                                       input_lengths,
                                                                       tiled_prefix_prompt_lengths,
                                                                       memory_len,
                                                                       max_input_length,
                                                                       initial_step,
                                                                       beam_width);
    }
}

template<typename T>
__global__ void sum_length_dimension(
    float* out_buf, const T* in_buf, const size_t batch_size, const size_t input_length, const size_t hidden_dim) {
    const int bidx = blockIdx.x;

    for (int hidx = threadIdx.x; hidx < hidden_dim; hidx += blockDim.x) {
        float accum = 0.0f;
        for (int step = 0; step < input_length; step++) {
            accum += static_cast<float>(in_buf[(bidx * input_length + step) * hidden_dim + hidx]);
        }
        out_buf[bidx * hidden_dim + hidx] = accum;
    }
}

template<typename T>
void invokeSumLengthDimension(float*       out_buf,
                              const T*     in_buf,
                              const size_t batch_size,
                              const size_t input_length,
                              const size_t hidden_dim,
                              hipStream_t stream) {
    dim3 gridSize(batch_size);
    dim3 blockSize(256);

    sum_length_dimension<<<gridSize, blockSize, 0, stream>>>(out_buf, in_buf, batch_size, input_length, hidden_dim);
}

__global__ void ConvertOffsetToAddr(uint64_t*       block_addr,         // [l, b, 2, m]
                                    const uint64_t* k_cache_base_addr,  // [l]
                                    const uint64_t* v_cache_base_addr,
                                    const int*      offset,  // [b, m]
                                    int             layer_num,
                                    int             batch_size,
                                    int             max_block_num,
                                    int             block_size) {
    const int layer_stride = batch_size * 2 * max_block_num;
    const int batch_stride = 2 * max_block_num;
    for (size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < layer_num * batch_size * max_block_num;
         index += blockDim.x * gridDim.x) {
        const int    layer_index      = index / max_block_num / batch_size;
        const int    batch_index      = (index / max_block_num) % batch_size;
        const int    col_index        = index % max_block_num;
        const size_t block_offset     = (size_t)offset[batch_index * max_block_num + col_index] * block_size;
        const size_t block_addr_index = (size_t)layer_index * layer_stride + batch_index * batch_stride + col_index;
        block_addr[block_addr_index]  = k_cache_base_addr[layer_index] + block_offset;
        block_addr[block_addr_index + max_block_num] = v_cache_base_addr[layer_index] + block_offset;
    }
}

void invokeConvertOffsetToAddr(uint64_t*       block_addr,         // [l, b, 2, m]
                               const uint64_t* k_cache_base_addr,  // [l]
                               const uint64_t* v_cache_base_addr,
                               const int*      offset,  // [b, m]
                               int             layer_num,
                               int             batch_size,
                               int             max_block_num,
                               int             block_size,
                               hipStream_t    stream) {
    dim3 grid(min(batch_size * layer_num, 65536));
    dim3 block(min(max_block_num, 1024));
    ConvertOffsetToAddr<<<grid, block, 0, stream>>>(block_addr,         // [l, b, 2, m]
                                                    k_cache_base_addr,  // [l]
                                                    v_cache_base_addr,
                                                    offset,  // [b, m]
                                                    layer_num,
                                                    batch_size,
                                                    max_block_num,
                                                    block_size);
}

__global__ void ConvertOffsetToAddrOneLayer(uint64_t*      block_addr,  // [b, 2, m]
                                            const uint64_t k_cache_base_addr,
                                            const uint64_t v_cache_base_addr,
                                            const int*     offset,  // [b, m]
                                            int            batch_size,
                                            int            max_block_num,
                                            int            block_size) {
    const int batch_stride = 2 * max_block_num;
    for (size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < batch_size * max_block_num;
         index += blockDim.x * gridDim.x) {
        const int    batch_index      = index / max_block_num;
        const int    col_index        = index % max_block_num;
        const size_t block_offset     = (size_t)offset[batch_index * max_block_num + col_index] * block_size;
        const size_t block_addr_index = (size_t)batch_index * batch_stride + col_index;
        block_addr[block_addr_index]  = k_cache_base_addr + block_offset;
        block_addr[block_addr_index + max_block_num] = v_cache_base_addr + block_offset;
    }
}

__global__ void ConvertOffsetToBlockArrayData(int32_t*   offset_addr,
                                              const int* offset,  // [b, m]
                                              int        batch_size,
                                              int        max_block_num,
                                              int        kv_block_offset) {
    const int batch_stride = 2 * max_block_num;
    for (size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < batch_size * max_block_num;
         index += blockDim.x * gridDim.x) {
        const int     batch_index                     = index / max_block_num;
        const int     col_index                       = index % max_block_num;
        const int32_t block_offset                    = (int32_t)offset[batch_index * max_block_num + col_index];
        const int32_t block_addr_index                = (int32_t)batch_index * batch_stride + col_index;
        offset_addr[block_addr_index]                 = block_offset;
        offset_addr[block_addr_index + max_block_num] = block_offset + kv_block_offset;
    }
}

void invokeConvertOffsetToAddrOneLayer(uint64_t*      block_addr,  // [b, 2, m]
                                       const uint64_t k_cache_base_addr,
                                       const uint64_t v_cache_base_addr,
                                       const int*     offset,  // [b, m]
                                       int            batch_size,
                                       int            max_block_num,
                                       int            block_size,
                                       hipStream_t   stream) {
    dim3 grid(min(batch_size, 65536));
    dim3 block(min(max_block_num, 1024));
    ConvertOffsetToAddrOneLayer<<<grid, block, 0, stream>>>(block_addr,  // [b, 2, m]
                                                            k_cache_base_addr,
                                                            v_cache_base_addr,
                                                            offset,  // [b, m]
                                                            batch_size,
                                                            max_block_num,
                                                            block_size);
}

void invokeConvertOffsetToBlockArrayData(int32_t*     offset_addr,  // [b, 2, m]
                                         const int*   offset,       // [b, m]
                                         int          batch_size,
                                         int          max_block_num,
                                         int          kv_block_offset,
                                         hipStream_t stream) {
    dim3 grid(min(batch_size, 65536));
    dim3 block(min(max_block_num, 1024));
    ConvertOffsetToBlockArrayData<<<grid, block, 0, stream>>>(offset_addr,  // [b, 2, m]
                                                              offset,       // [b, m]
                                                              batch_size,
                                                              max_block_num,
                                                              kv_block_offset);
}

#define INSTANTIATE_INVOKE_SUM_LENGTH_DIMENSION(T)                                                                     \
    template void invokeSumLengthDimension(float*       out_buf,                                                       \
                                           const T*     in_buf,                                                        \
                                           const size_t batch_size,                                                    \
                                           const size_t input_length,                                                  \
                                           const size_t hidden_dim,                                                    \
                                           hipStream_t stream)
INSTANTIATE_INVOKE_SUM_LENGTH_DIMENSION(half);
INSTANTIATE_INVOKE_SUM_LENGTH_DIMENSION(float);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_SUM_LENGTH_DIMENSION(__hip_bfloat16);
#endif
#undef INSTANTIATE_INVOKE_SUM_LENGTH_DIMENSION

__global__ void getPaddingOffsetAndCuSeqLensKernel(
    int* padding_offset, int* cu_seqlens, const int* sequence_length, const int batch_size, const int max_seq_len) {
    // do cumulated sum
    int        total_seq_len        = 0;
    int        cum_offset           = 0;
    int        index                = 0;
    const bool calculate_cu_seqlens = cu_seqlens != nullptr;
    for (int i = 0; i < batch_size; i++) {
        const int seq_len = sequence_length[i];
        if (calculate_cu_seqlens) {
            cu_seqlens[i] = total_seq_len;
        }
        for (int j = 0; j < seq_len; j++) {
            padding_offset[index] = cum_offset;
            index++;
        }
        cum_offset += max_seq_len - seq_len;
        total_seq_len += seq_len;
    }
    if (calculate_cu_seqlens) {
        cu_seqlens[batch_size] = total_seq_len;
    }
}

__global__ void
getCuSeqLensKernel(int* cu_seqlens, const int* sequence_length, const int* prefix_length, const int batch_size) {
    // do cumulated sum
    int        total_seq_len     = 0;
    const bool has_prefix_length = prefix_length != nullptr;
    for (int i = 0; i < batch_size; i++) {
        int seq_len = sequence_length[i];
        if (has_prefix_length) {
            seq_len += prefix_length[i];
        }
        cu_seqlens[i] = total_seq_len;
        total_seq_len += seq_len;
    }
    cu_seqlens[batch_size] = total_seq_len;
}

void invokeGetPaddingOffsetAndCuSeqLens(int*         padding_offset,
                                        int*         cu_seqlens,
                                        const int*   sequence_lengths,
                                        const int    batch_size,
                                        const int    max_seq_len,
                                        hipStream_t stream) {
    getPaddingOffsetAndCuSeqLensKernel<<<1, 1, 0, stream>>>(
        padding_offset, cu_seqlens, sequence_lengths, batch_size, max_seq_len);
    check_cuda_error();
}

void invokeGetCuSeqLens(
    int* cu_seqlens, const int* sequence_length, const int* prefix_length, const int batch_size, hipStream_t stream) {
    getCuSeqLensKernel<<<1, 1, 0, stream>>>(cu_seqlens, sequence_length, prefix_length, batch_size);
    check_cuda_error();
}

template<typename T, int ELEM_PER_THREAD>
__global__ void scatter_add_stable_kernel(T const* src, int N, int K, int32_t const* index, T* out) {
    // 在输出位置上并行,每个线程负责一个输出位置的累加
    int64_t out_idx = blockIdx.x * blockDim.x + threadIdx.x;
    out_idx *= ELEM_PER_THREAD;

    // 计算当前输出元素对应的维度
    const int k     = out_idx % K;
    const int out_n = out_idx / K;

    if (out_n >= N)
        return;

// 对每个输入位置检查,如果它们映射到当前输出位置则累加
#pragma unroll
    for (int i = 0; i < ELEM_PER_THREAD; i++) {
        if (out_idx + i < (size_t)N * K) {
            T sum = out[out_idx + i];
            // 遍历所有输入,找到映射到当前输出位置的元素
            for (int in_n = 0; in_n < N; in_n++) {
                if (index[in_n] == out_n) {
                    sum = sum + src[in_n * K + k + i];
                }
            }
            out[out_idx + i] = sum;
        }
    }
}

template<typename T>
void invokeScatterAddStable(T const* src, int N, int K, int32_t const* index, T* out, hipStream_t stream) {
    const int  num_threads     = 256;
    const int  elem_per_thread = 4;
    const dim3 block(num_threads);
    RTP_LLM_CHECK(K % (elem_per_thread * 2) == 0);

    auto h_index = std::shared_ptr<int32_t[]>(new int32_t[N], std::default_delete<int32_t[]>());

    hipMemcpy(h_index.get(), index, N * sizeof(int32_t), hipMemcpyDeviceToHost);

    int32_t max_out_n = h_index[0];
    for (int i = 1; i < N; i++) {
        max_out_n = max(max_out_n, h_index[i]);
    }
    max_out_n++;

    if constexpr (std::is_same<T, float>::value) {
        const dim3 grid(((size_t)max_out_n * K + num_threads * elem_per_thread - 1) / (num_threads * elem_per_thread));
        scatter_add_stable_kernel<float, elem_per_thread><<<grid, block, 0, stream>>>(src, N, K, index, out);
    } else if (K % 2 == 0) {
#if USING_ROCM
        using Tp = typename rocm::packed_type_2<T>::type;
#else
        using Tp = typename packed_type_2<T>::type;
#endif
        const dim3 grid(((size_t)max_out_n * K / 2 + num_threads * elem_per_thread - 1)
                        / (num_threads * elem_per_thread));
        scatter_add_stable_kernel<Tp, elem_per_thread><<<grid, block, 0, stream>>>((Tp*)src, N, K / 2, index, (Tp*)out);
    } else {
        throw std::invalid_argument("scatter add unsupport type or K [%d]" + std::to_string(K));
    }
}

template<typename T, int ELEM_PER_THREAD>
__global__ void scatter_add_kernel(T const* src, int N, int K, int32_t const* index, T* out) {
    int64_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    thread_idx *= ELEM_PER_THREAD;
    // int offset = blockDim.x * gridDim.x;
    int     k       = thread_idx % K;
    int64_t new_idx = (int64_t)index[thread_idx / K] * K;
#pragma unroll
    for (int i = 0; i < ELEM_PER_THREAD; ++i) {
        if (thread_idx + i < (size_t)N * K) {
#if USING_ROCM
#ifdef ENABLE_BF16
            if constexpr (std::is_same<T, __hip_bfloat162>::value) {
                unsafeAtomicAdd(reinterpret_cast<__hip_bfloat162*>(out) + new_idx + k + i,
                                (__hip_bfloat162)src[thread_idx + i]);
            } else {
                unsafeAtomicAdd(out + new_idx + k + i, src[thread_idx + i]);
            }
#else
            unsafeAtomicAdd(out + new_idx + k + i, src[thread_idx + i]);
#endif
#else
            atomicAdd(out + new_idx + k + i, src[thread_idx + i]);
#endif
        }
    }
}

template<typename T>
void invokeScatterAdd(
    T const* src, int N, int K, int32_t const* index, T* out, bool use_stable_scatter_add, hipStream_t stream) {
    RTP_LLM_CHECK_WITH_INFO(N > 0 && K > 0, "N and K must be greater than 0");
    if (use_stable_scatter_add) {
        invokeScatterAddStable(src, N, K, index, out, stream);
        return;
    }
    const int  num_threads     = 256;
    const int  elem_per_thread = 4;
    const dim3 block(num_threads);
    RTP_LLM_CHECK(K % (elem_per_thread * 2) == 0);

    if constexpr (std::is_same<T, float>::value) {
        const dim3 grid(((size_t)N * K + num_threads * elem_per_thread - 1) / (num_threads * elem_per_thread));
        scatter_add_kernel<float, elem_per_thread><<<grid, block, 0, stream>>>(src, N, K, index, out);
    } else if (K % 2 == 0) {
#if USING_ROCM
        using Tp = typename rocm::packed_type_2<T>::type;
#else
        using Tp = typename packed_type_2<T>::type;
#endif
        const dim3 grid(((size_t)N * K / 2 + num_threads * elem_per_thread - 1) / (num_threads * elem_per_thread));
        scatter_add_kernel<Tp, elem_per_thread><<<grid, block, 0, stream>>>((Tp*)src, N, K / 2, index, (Tp*)out);

    } else {
        throw std::invalid_argument("scatter add unsupport type or K [%d]" + std::to_string(K));
    }
}

#define INSTANTIATE_INVOKE_SCATTER_ADD(T)                                                                              \
    template void invokeScatterAdd(                                                                                    \
        T const* src, int N, int K, int32_t const* index, T* out, bool use_stable_scatter_add, hipStream_t stream)

INSTANTIATE_INVOKE_SCATTER_ADD(half);
INSTANTIATE_INVOKE_SCATTER_ADD(float);

#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_SCATTER_ADD(__hip_bfloat16);
#endif
#undef INSTANTIATE_INVOKE_SCATTER_ADD

template<typename T>
__global__ void sliceDim1CopyKernel(T const* src, int dim0, int dim1, int dim1_start, int dim1_size, T* out) {
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < dim0 * dim1_size; index += blockDim.x * gridDim.x) {
        const int    col_index = index % dim1_size;
        const size_t batch_id  = index / dim1_size;
        out[index]             = src[batch_id * dim1 + dim1_start + col_index];
    }
}

template<typename T>
void invokeSliceDim1Copy(T const* src, int dim0, int dim1, int dim1_start, int dim1_size, T* out, hipStream_t stream) {
    if constexpr (std::is_same<uint8_t, T>::value) {
        if (dim1 % 16 == 0 && dim1_start % 16 == 0 && dim1_size % 16 == 0) {
            dim1 /= 16;
            dim1_start /= 16;
            dim1_size /= 16;
            const int grid_size = (int)(ceil((size_t)dim0 * dim1_size / 512.));
            dim3      grid(min(grid_size, 65536));
            dim3      block(512);
            sliceDim1CopyKernel<uint4>
                <<<grid, block, 0, stream>>>((uint4 const*)src, dim0, dim1, dim1_start, dim1_size, (uint4*)out);
        } else if (dim1 % 8 == 0 && dim1_start % 8 == 0 && dim1_size % 8 == 0) {
            dim1 /= 8;
            dim1_start /= 8;
            dim1_size /= 8;
            const int grid_size = (int)(ceil((size_t)dim0 * dim1_size / 512.));
            dim3      grid(min(grid_size, 65536));
            dim3      block(512);
            sliceDim1CopyKernel<uint2>
                <<<grid, block, 0, stream>>>((uint2 const*)src, dim0, dim1, dim1_start, dim1_size, (uint2*)out);
        } else if (dim1 % 4 == 0 && dim1_start % 4 == 0 && dim1_size % 4 == 0) {
            dim1 /= 4;
            dim1_start /= 4;
            dim1_size /= 4;
            const int grid_size = (int)(ceil((size_t)dim0 * dim1_size / 512.));
            dim3      grid(min(grid_size, 65536));
            dim3      block(512);
            sliceDim1CopyKernel<uint>
                <<<grid, block, 0, stream>>>((uint const*)src, dim0, dim1, dim1_start, dim1_size, (uint*)out);
        } else {
            const int grid_size = (int)(ceil((size_t)dim0 * dim1_size / 512.));
            dim3      grid(min(grid_size, 65536));
            dim3      block(512);
            sliceDim1CopyKernel<T><<<grid, block, 0, stream>>>(src, dim0, dim1, dim1_start, dim1_size, out);
        }
    } else {
        const int grid_size = (int)(ceil((size_t)dim0 * dim1_size / 512.));
        dim3      grid(min(grid_size, 65536));
        dim3      block(512);
        sliceDim1CopyKernel<T><<<grid, block, 0, stream>>>(src, dim0, dim1, dim1_start, dim1_size, out);
    }
}

#define INSTANTIATE_INVOKE_SlICE_DIM1_COPTY(T)                                                                         \
    template void invokeSliceDim1Copy(                                                                                 \
        T const* src, int dim0, int dim1, int dim1_start, int dim1_size, T* out, hipStream_t stream)

INSTANTIATE_INVOKE_SlICE_DIM1_COPTY(float);
INSTANTIATE_INVOKE_SlICE_DIM1_COPTY(half);
INSTANTIATE_INVOKE_SlICE_DIM1_COPTY(int32_t);
INSTANTIATE_INVOKE_SlICE_DIM1_COPTY(int8_t);
INSTANTIATE_INVOKE_SlICE_DIM1_COPTY(uint8_t);
INSTANTIATE_INVOKE_SlICE_DIM1_COPTY(uint32_t);
INSTANTIATE_INVOKE_SlICE_DIM1_COPTY(int64_t);
INSTANTIATE_INVOKE_SlICE_DIM1_COPTY(uint64_t);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_SlICE_DIM1_COPTY(__hip_bfloat16);
#endif

#ifdef ENABLE_FP8
INSTANTIATE_INVOKE_SlICE_DIM1_COPTY(__hip_fp8_e4m3_fnuz);
#endif

template<typename T>
__global__ void fakeBalanceExpertKernel(T* expert, float* expert_scales, int start, int expert_num, int size) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        expert[index]        = (start + index) % expert_num;
        expert_scales[index] = 1.0f;
    }
}

void fake_balance_expert(int* expert, float* expert_scales, int start, int expert_num, int size, hipStream_t stream) {
    fakeBalanceExpertKernel<int>
        <<<(size + 255) / 256, 256, 0, stream>>>(expert, expert_scales, start, expert_num, size);
}

void fake_balance_expert(
    int64_t* expert, float* expert_scales, int start, int expert_num, int size, hipStream_t stream) {
    fakeBalanceExpertKernel<int64_t>
        <<<(size + 255) / 256, 256, 0, stream>>>(expert, expert_scales, start, expert_num, size);
}

}  // namespace rtp_llm
