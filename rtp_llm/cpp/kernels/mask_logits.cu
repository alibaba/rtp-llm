#include "hip/hip_runtime.h"
#include "rtp_llm/cpp/kernels/mask_logits.h"
#if USING_CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>  // For half
#include <hip/hip_bf16.h>  // For __hip_bfloat16
#include "rtp_llm/cpp/cuda/cuda_utils.h"
#endif
#if USING_ROCM
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bfloat16.h>
#include "rtp_llm/cpp/rocm/hip_utils.h"
#endif

namespace rtp_llm {

#if USING_CUDA
#ifndef HIPRT_INF_FP16
#define HIPRT_INF_FP16 __ushort_as_half((unsigned short)0x7C00U)
#endif
#ifndef CUDART_INF_BF16
#define CUDART_INF_BF16 __ushort_as_bfloat16((unsigned short)0x7F80U)
#endif
#endif

#if USING_ROCM
#ifndef HIP_INF_FP16
#define HIP_INF_FP16 __ushort_as_half((unsigned short)0x7C00U)
#endif
#ifndef HIP_INF_BF16
#define HIP_INF_BF16 __ushort_as_bfloat16((unsigned short)0x7F80U)
#endif
#define HIPRT_INF_FP16 HIP_INF_FP16
#define CUDART_INF_BF16 HIP_INF_BF16
#endif

template<typename T>
__device__ T NegativeInfinity() {
    return -INFINITY;
}

template<>
__device__ __half NegativeInfinity<__half>() {
    return -HIPRT_INF_FP16;
}

template<>
__device__ __hip_bfloat16 NegativeInfinity<__hip_bfloat16>() {
    return -CUDART_INF_BF16;
}

// Batch version kernel for processing multiple beams
template<typename T>
__global__ void
mask_logits(const int batch_size, const int vocab_size, T* logits_batch, const uint8_t* __restrict__ mask_batch) {
    int batch_idx = blockIdx.y;
    int vocab_idx = threadIdx.x + (blockIdx.x * blockDim.x);

    if (batch_idx < batch_size && vocab_idx < vocab_size) {
        int global_idx = batch_idx * vocab_size + vocab_idx;
        if (mask_batch[global_idx]) {
            logits_batch[global_idx] = NegativeInfinity<T>();
        }
    }
}

template<typename T>
void invokeMaskLogits(T* logits_batch,
                      const uint8_t* __restrict__ mask_batch,
                      const int    batch_size,
                      const int    vocab_size,
                      hipStream_t stream) {
    dim3 block, grid;

    block.x = 64;
    block.y = 1;
    block.z = 1;
    grid.x  = (vocab_size + block.x - 1) / block.x;
    grid.y  = batch_size;
    grid.z  = 1;

    mask_logits<<<grid, block, 0, stream>>>(batch_size, vocab_size, logits_batch, mask_batch);

    check_cuda_error();
}

template void invokeMaskLogits<float>(float* logits_batch,
                                      const uint8_t* __restrict__ mask_batch,
                                      const int    batch_size,
                                      const int    vocab_size,
                                      hipStream_t stream);
template void invokeMaskLogits<half>(half* logits_batch,
                                     const uint8_t* __restrict__ mask_batch,
                                     const int    batch_size,
                                     const int    vocab_size,
                                     hipStream_t stream);
template void invokeMaskLogits<__hip_bfloat16>(__hip_bfloat16* logits_batch,
                                              const uint8_t* __restrict__ mask_batch,
                                              const int    batch_size,
                                              const int    vocab_size,
                                              hipStream_t stream);

}  // namespace rtp_llm
