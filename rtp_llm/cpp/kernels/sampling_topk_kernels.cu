#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdexcept>
#if USING_CUDA
#ifndef CUDART_VERSION
#error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 11050)
#include <hipcub/hipcub.hpp>
#else
#include "3rdparty/hipcub/hipcub.hpp"
#endif
#endif

#if USING_ROCM
#include <hipcub/hipcub.hpp>
#include "rtp_llm/cpp/rocm/cuda_shims.h"
#endif

#include "rtp_llm/cpp/cuda/reduce_kernel_utils.cuh"
#include "rtp_llm/cpp/kernels/sampling_topk_kernels.h"
#include "rtp_llm/cpp/utils/math_utils.h"

namespace rtp_llm {

__global__ void curandInitialize(hiprandState_t* state, const int size, const unsigned long long random_seed) {
    if (threadIdx.x + blockIdx.x * blockDim.x < size) {
        hiprand_init(random_seed, 0, 0, &state[blockIdx.x * blockDim.x + threadIdx.x]);
    }
}

void invokeCurandInitialize(hiprandState_t*           state,
                            const size_t             batch_size,
                            const unsigned long long random_seed,
                            hipStream_t             stream) {
    dim3 block(256);
    dim3 grid((int)(ceil(batch_size * 1.0 / 256)));
    curandInitialize<<<grid, block, 0, stream>>>(state, batch_size, random_seed);
}

__global__ void curandBatchInitialize(hiprandState_t* states, const int size, const unsigned long long* random_seeds) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        hiprand_init(random_seeds[idx], 0, 0, &states[idx]);
    }
}

void invokeCurandBatchInitialize(hiprandState_t*            states,
                                 const size_t              batch_size,
                                 const unsigned long long* random_seeds,
                                 hipStream_t              stream) {
    dim3 block(256);
    dim3 grid((int)(ceil(batch_size * 1.0 / 256)));
    curandBatchInitialize<<<grid, block, 0, stream>>>(states, batch_size, random_seeds);
}

template<typename T>
__global__ void addBiasEndMask(T*          logits,
                               const T*    bias,
                               const int*  end_ids,
                               const bool* finished,
                               const int   vocab_size,
                               const int   vocab_size_padded) {
    int  bid    = blockIdx.x;
    bool finish = finished != nullptr ? finished[bid] : false;
    int  offset = bid * vocab_size_padded;

    const bool IS_FP16   = std::is_same<T, half>::value;
    const T    MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;
    for (int tid = threadIdx.x; tid < vocab_size_padded; tid += blockDim.x) {
        if (tid >= vocab_size) {
            logits[offset + tid] = -MAX_T_VAL;
        } else if (finish) {
            logits[offset + tid] = (tid == end_ids[bid]) ? MAX_T_VAL : -MAX_T_VAL;
        } else {
            if (bias != nullptr) {
                logits[offset + tid] += bias[tid];
            }
        }
    }
}

template<typename T>
void invokeAddBiasEndMask(T*           logits,
                          const T*     bias,
                          const int*   end_ids,
                          const bool*  finished,
                          const int    batch_size,
                          const int    vocab_size,
                          const int    vocab_size_padded,
                          hipStream_t stream) {
    dim3 grid(batch_size);
    dim3 block(min(vocab_size_padded, 1024));
    /*n is the vocab_size, e.g., 30000, 7000.... vocab_size is usually very big. */
    addBiasEndMask<<<grid, block, 0, stream>>>(logits, bias, end_ids, finished, vocab_size, vocab_size_padded);
}

template void invokeAddBiasEndMask(float*       logits,
                                   const float* bias,
                                   const int*   end_ids,
                                   const bool*  finished,
                                   const int    batch_size,
                                   const int    vocab_size,
                                   const int    vocab_size_padded,
                                   hipStream_t stream);

template void invokeAddBiasEndMask(half*        logits,
                                   const half*  bias,
                                   const int*   end_ids,
                                   const bool*  finished,
                                   const int    batch_size,
                                   const int    vocab_size,
                                   const int    vocab_size_padded,
                                   hipStream_t stream);

template<typename T, int BLOCK_SIZE_, int BLOCKS_PER_BEAM_>
__global__ void topk_stage1(const T* __restrict log_probs,
                            T*          tmp_log_probs,
                            int*        topk_tmp_id_buf,
                            T*          topk_tmp_val_buf,
                            const bool* finished,
                            const int   max_top_k,
                            const int*  top_ks,
                            const int   vocab_size,
                            const int*  end_ids,
                            const bool* skip_decode) {
    typedef hipcub::BlockReduce<TopK_2<T>, BLOCK_SIZE_> BlockReduce;
    __shared__ typename BlockReduce::TempStorage     temp_storage;

    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    const int batch_id = bid / BLOCKS_PER_BEAM_;  // row id for log_probs
    if (skip_decode != nullptr && skip_decode[batch_id]) {
        return;
    }
    const int block_lane = bid % BLOCKS_PER_BEAM_;                              // block id for a beam
    const int k          = (top_ks != nullptr) ? top_ks[batch_id] : max_top_k;  // batch_id = batch index

    const int tmp_log_buf_index  = batch_id * vocab_size;
    const int tmp_topk_buf_index = batch_id * BLOCKS_PER_BEAM_ * max_top_k + block_lane * k;

    TopK_2<T>  partial;
    const bool IS_FP16   = std::is_same<T, half>::value;
    const T    MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;

    if (finished != nullptr && finished[batch_id] == true) {
        if (tid < k) {
            const int index = tmp_topk_buf_index + tid;
            if (block_lane == 0 && tid == 0) {
                const int end_id        = end_ids[batch_id];
                topk_tmp_id_buf[index]  = tmp_log_buf_index + end_id;
                topk_tmp_val_buf[index] = log_probs[tmp_log_buf_index + end_id];
            } else {
                topk_tmp_id_buf[index]  = -1;
                topk_tmp_val_buf[index] = -MAX_T_VAL;
            }
        }
        return;
    }

    for (int elem_id = tid + block_lane * BLOCK_SIZE_; elem_id < vocab_size;
         elem_id += BLOCK_SIZE_ * BLOCKS_PER_BEAM_) {
        int index            = elem_id + tmp_log_buf_index;
        tmp_log_probs[index] = log_probs[index];
    }

    for (int ite = 0; ite < k; ite++) {
        partial.init();
#pragma unroll
        for (int elem_id = tid + block_lane * BLOCK_SIZE_; elem_id < vocab_size;
             elem_id += BLOCK_SIZE_ * BLOCKS_PER_BEAM_) {
            int index = elem_id + tmp_log_buf_index;
            partial.insert(tmp_log_probs[index], index);
        }

        TopK_2<T> total = BlockReduce(temp_storage).Reduce(partial, reduce_topk_op_2<T>);

        if (tid == 0) {
            const int index         = tmp_topk_buf_index + ite;
            topk_tmp_id_buf[index]  = total.p;
            topk_tmp_val_buf[index] = total.u;
            if (total.p >= 0) {
                tmp_log_probs[total.p] = -MAX_T_VAL;
            }
        }
        __syncthreads();
    }
}

template<typename T, int BLOCK_SIZE_, int BLOCKS_PER_BEAM_, bool RECORD_PROB>
__global__ void topk_stage2_sampling(const int* __restrict topk_tmp_id_buf,
                                     T*             topk_tmp_val_buf,
                                     int*           ids,
                                     int*           sequence_length,
                                     bool*          finished,
                                     float*         cum_log_probs,
                                     float*         output_log_probs,
                                     float*         output_all_probs,
                                     const int      max_top_k,
                                     const int*     top_ks,
                                     const float    top_p,
                                     const float*   top_ps,
                                     hiprandState_t* curandstate,
                                     const int*     end_ids,
                                     const int      vocab_size,
                                     const bool*    skip_decode) {
    const bool IS_FP16   = std::is_same<T, half>::value;
    const T    MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;

    const int tid      = threadIdx.x;
    const int batch_id = blockIdx.x;
    if (skip_decode != nullptr && skip_decode[batch_id]) {
        return;
    }

    const int   k              = (top_ks != nullptr) ? top_ks[batch_id] : max_top_k;
    const float prob_threshold = (top_ps != nullptr) ? top_ps[batch_id] : top_p;
    const int   size           = k * BLOCKS_PER_BEAM_;
    const int   stride         = max_top_k * BLOCKS_PER_BEAM_;

    typedef hipcub::BlockReduce<TopK_2<float>, BLOCK_SIZE_> BlockReduce;
    __shared__ typename BlockReduce::TempStorage         temp_storage;
    extern __shared__ char                               array[];
    __shared__ float                                     rand_num;
    __shared__ float                                     s_sum;
    __shared__ float                                     s_max;
    T*                                                   s_val = topk_tmp_val_buf + batch_id * stride;
    int*                                                 s_id  = reinterpret_cast<int*>(array);
    if (tid == 0) {
        s_sum = 0.0f;
    }
    TopK_2<float> partial;

    if (finished != nullptr && finished[batch_id] == true) {
        ids[batch_id] = end_ids[batch_id];
        return;
    }

    float* s_val2 = reinterpret_cast<float*>(s_id + k);
    for (int ite = 0; ite < k; ite++) {
        partial.init();
#pragma unroll
        for (int i = tid; i < size; i += BLOCK_SIZE_) {
            partial.insert((float)s_val[i], i);
        }

        TopK_2<float> total = BlockReduce(temp_storage).Reduce(partial, reduce_topk_op_2<float>);

        if (tid == 0) {
            if (ite == 0) {
                s_max = total.u;
            }
            s_id[ite]      = total.p;
            s_val[total.p] = -MAX_T_VAL;

            // when cum_log_probs are computed, topk_tmp_val_buf (logits_buf_) are already pre-processed by
            // softmax_kernel
            total.u     = __expf(total.u - s_max);
            s_val2[ite] = total.u;
            s_sum += total.u;
        }
        __syncthreads();
    }

    //@miji TODO: use block sum to make it faster
    if constexpr (RECORD_PROB) {
        float prob_sum = 0;
        if (threadIdx.x == 0) {
            for (int i = 0; i < k; i++) {
                int   token_idx   = topk_tmp_id_buf[batch_id * stride + s_id[i]] % vocab_size;
                float origin_prob = __expf(logf(s_val2[i]) - logf(s_sum));
                prob_sum += origin_prob;
                output_all_probs[batch_id * vocab_size + token_idx] =
                    max(0.0, origin_prob - max(0.0, prob_sum - prob_threshold)) / prob_threshold;
                if (prob_sum >= prob_threshold) {
                    break;
                }
            }
        }
    }

    if (tid == 0) {
        rand_num = (float)hiprand_uniform(curandstate + blockIdx.x) * prob_threshold * s_sum;
        for (int i = 0; i < k; i++) {
            float exp_logit = s_val2[i];
            rand_num        = rand_num - exp_logit;
            if (rand_num <= 0.0f || i == k - 1) {
                ids[batch_id] = topk_tmp_id_buf[batch_id * stride + s_id[i]] % vocab_size;
                if (cum_log_probs != nullptr || output_log_probs != nullptr) {
                    float log_prob = logf(exp_logit) - logf(s_sum);
                    if (cum_log_probs != nullptr) {
                        cum_log_probs[batch_id] += log_prob;
                    }
                    if (output_log_probs != nullptr) {
                        // 'output_log_probs' is the probability induced by the top-k sampling.
                        // We normalize the probability 'exp_logit' of the selected token by
                        // the probability 's_sum' of a set of top-k tokens, meaning the log_prob
                        // is the probability of the selected token, conditioned on the event that
                        // it is selected, i.e.,
                        //   log_prob = log P(i | i is in top-k) = log(exp_logit / s_sum).
                        output_log_probs[batch_id] = log_prob;
                    }
                }
                break;
            }
        }
        if (sequence_length != nullptr && finished != nullptr) {
            sequence_length[batch_id] = finished[batch_id] ? sequence_length[batch_id] : sequence_length[batch_id] + 1;
            finished[batch_id]        = ids[batch_id] == end_ids[batch_id] ? true : false;
        }
    }
}

#define CASE_K(K_MIN, K_MAX, BLOCK_SIZE_1_, BLOCK_SIZE_2_, BLOCKS_PER_BEAM_, RECORD_PROB)                              \
    case K_MIN ... K_MAX:                                                                                              \
        topk_stage1<T, BLOCK_SIZE_1_, BLOCKS_PER_BEAM_>                                                                \
            <<<batch_size * BLOCKS_PER_BEAM_, BLOCK_SIZE_1_, 0, stream>>>(log_probs,                                   \
                                                                          temp_log_probs,                              \
                                                                          topk_tmp_id_buf,                             \
                                                                          topk_tmp_val_buf,                            \
                                                                          finished,                                    \
                                                                          max_top_k,                                   \
                                                                          top_ks,                                      \
                                                                          vocab_size,                                  \
                                                                          end_ids,                                     \
                                                                          skip_decode);                                \
        topk_stage2_sampling<T, BLOCK_SIZE_2_, BLOCKS_PER_BEAM_, RECORD_PROB>                                          \
            <<<batch_size, BLOCK_SIZE_2_, K_MAX * sizeof(int) + K_MAX * sizeof(float), stream>>>(topk_tmp_id_buf,      \
                                                                                                 topk_tmp_val_buf,     \
                                                                                                 ids,                  \
                                                                                                 sequence_length,      \
                                                                                                 finished,             \
                                                                                                 cum_log_probs,        \
                                                                                                 output_log_probs,     \
                                                                                                 output_all_probs,     \
                                                                                                 max_top_k,            \
                                                                                                 top_ks,               \
                                                                                                 top_p,                \
                                                                                                 top_ps,               \
                                                                                                 curandstate,          \
                                                                                                 end_ids,              \
                                                                                                 vocab_size,           \
                                                                                                 skip_decode);         \
        break;

template<typename T>
void invokeBatchTopKSampling(void*          workspace,
                             size_t&        workspace_size,
                             const T*       log_probs,
                             int*           ids,
                             int*           sequence_length,
                             bool*          finished,
                             float*         cum_log_probs,
                             float*         output_log_probs,
                             hiprandState_t* curandstate,
                             const int      max_top_k,
                             const int*     top_ks,
                             const float    top_p,
                             const float*   top_ps,
                             const int      vocab_size_padded,
                             const int*     end_ids,
                             float*         output_all_probs,
                             hipStream_t   stream,
                             const int      batch_size,
                             const bool*    skip_decode) {
    // Not allow an ambiguous inputs top_p and top_ps.
    assert(top_p == 1.0f || top_ps == nullptr);
    const int vocab_size              = vocab_size_padded;
    const int max_block_per_beam      = 8;
    int       temp_log_probs_buf_size = batch_size * vocab_size;                      // type float
    int       topk_tmp_ids_buf_size   = batch_size * max_top_k * max_block_per_beam;  // type int
    int       topk_tmp_val_buf_size   = batch_size * max_top_k * max_block_per_beam;  // type float

    // prevent memory misaligned address
    temp_log_probs_buf_size = (int)(ceil(temp_log_probs_buf_size / 4.)) * 4;
    topk_tmp_ids_buf_size   = (int)(ceil(topk_tmp_ids_buf_size / 4.)) * 4;
    topk_tmp_val_buf_size   = (int)(ceil(topk_tmp_val_buf_size / 4.)) * 4;

    if (workspace == nullptr) {
        workspace_size = sizeof(T) * temp_log_probs_buf_size + sizeof(int) * topk_tmp_ids_buf_size
                         + sizeof(T) * topk_tmp_val_buf_size;
        return;
    }

    T*   temp_log_probs   = (T*)workspace;
    int* topk_tmp_id_buf  = (int*)(temp_log_probs + temp_log_probs_buf_size);
    T*   topk_tmp_val_buf = (T*)(topk_tmp_id_buf + topk_tmp_ids_buf_size);
#define SWITCH_MAX_K(LOG_PROB)                                                                                         \
    switch (max_top_k) {                                                                                               \
        CASE_K(1, 16, 128, 128, 8, LOG_PROB);                                                                          \
        CASE_K(17, 32, 256, 128, 8, LOG_PROB);                                                                         \
        CASE_K(33, 64, 256, 256, 8, LOG_PROB);                                                                         \
        CASE_K(65, 1024, 256, 256, 8, LOG_PROB);                                                                       \
        default:                                                                                                       \
            throw std::domain_error(rtp_llm::fmtstr("top-k kernel supports 1<=k<=1024 but got k=%d", max_top_k));      \
    }
    if (output_all_probs) {
        SWITCH_MAX_K(true);
    } else {
        SWITCH_MAX_K(false);
    }
}

#undef CASE_K

template void invokeBatchTopKSampling(void*          workspace,
                                      size_t&        workspace_size,
                                      const float*   log_probs,
                                      int*           ids,
                                      int*           sequence_length,
                                      bool*          finished_buf,
                                      float*         cum_log_probs,
                                      float*         output_log_probs,
                                      hiprandState_t* curandstate,
                                      const int      max_top_k,
                                      const int*     top_ks,
                                      const float    top_p,
                                      const float*   top_ps,
                                      const int      vocab_size_padded,
                                      const int*     end_ids,
                                      float*         output_all_probs,
                                      hipStream_t   stream,
                                      const int      batch_size,
                                      const bool*    skip_decode);

template void invokeBatchTopKSampling(void*          workspace,
                                      size_t&        workspace_size,
                                      const half*    log_probs,
                                      int*           ids,
                                      int*           sequence_length,
                                      bool*          finished_buf,
                                      float*         cum_log_probs,
                                      float*         output_log_probs,
                                      hiprandState_t* curandstate,
                                      const int      max_top_k,
                                      const int*     top_ks,
                                      const float    top_p,
                                      const float*   top_ps,
                                      const int      vocab_size_padded,
                                      const int*     end_ids,
                                      float*         output_all_probs,
                                      hipStream_t   stream,
                                      const int      batch_size,
                                      const bool*    skip_decode);

template<typename T>
void invokeTopKSampling(void*          workspace,
                        size_t&        workspace_size,
                        const T*       log_probs,
                        int*           ids,
                        int*           sequence_length,
                        bool*          finished_buf,
                        float*         cum_log_probs,
                        float*         output_log_probs,
                        hiprandState_t* curandstate,
                        const int      top_k,
                        const float    top_p,
                        const int      vocab_size_padded,
                        const int*     end_ids,
                        float*         output_all_probs,
                        hipStream_t   stream,
                        const int      batch_size,
                        const bool*    skip_decode) {
    invokeBatchTopKSampling(workspace,
                            workspace_size,
                            log_probs,
                            ids,
                            sequence_length,
                            finished_buf,
                            cum_log_probs,
                            output_log_probs,
                            curandstate,
                            top_k,
                            nullptr,
                            top_p,
                            nullptr,
                            vocab_size_padded,
                            end_ids,
                            output_all_probs,
                            stream,
                            batch_size,
                            skip_decode);
}

template void invokeTopKSampling(void*          workspace,
                                 size_t&        workspace_size,
                                 const float*   log_probs,
                                 int*           ids,
                                 int*           sequence_length,
                                 bool*          finished_buf,
                                 float*         cum_log_probs,
                                 float*         output_log_probs,
                                 hiprandState_t* curandstate,
                                 const int      top_k,
                                 const float    top_p,
                                 const int      vocab_size_padded,
                                 const int*     end_ids,
                                 float*         output_all_probs,
                                 hipStream_t   stream,
                                 const int      batch_size,
                                 const bool*    skip_decode);

template void invokeTopKSampling(void*          workspace,
                                 size_t&        workspace_size,
                                 const half*    log_probs,
                                 int*           ids,
                                 int*           sequence_length,
                                 bool*          finished_buf,
                                 float*         cum_log_probs,
                                 float*         output_log_probs,
                                 hiprandState_t* curandstate,
                                 const int      top_k,
                                 const float    top_p,
                                 const int      vocab_size_padded,
                                 const int*     end_ids,
                                 float*         output_all_probs,
                                 hipStream_t   stream,
                                 const int      batch_size,
                                 const bool*    skip_decode);

template<uint TOP_K_MAX>
__global__ void setup_topk_runtime_args(int    batch_size,
                                        uint   top_k,
                                        uint*  top_ks,
                                        int    top_ks_size,
                                        float  top_p,
                                        float* top_ps,
                                        int    top_ps_size,
                                        bool*  skip_decode) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = index; i < batch_size; i += gridDim.x * blockDim.x) {
        uint  k = top_ks_size > 1 ? top_ks[i] : top_k;
        float p = top_ps_size > 1 ? top_ps[i] : top_p;
        if (k == 0 && p == 0.0f) {
            // FT's topp implementation does not support topp = 0.0f, but it equivalent to greedy search.
            // So, we set the topk = 1 as an alternative solution.
            k = 1;
        }
        if (k > 0 && p == 0.0f) {
            // for compatibility <= FT5.0.
            // This case corresponds to the old topk sampling, which is equivalent to
            // the old topk_topp sampling with topp=1.0f. TopKSamplingLayer and
            // TopKTopPSamplingLayer are now merged by TopKSamplingLayer. Thus, we
            // replace the case topk>0 and topp=0.0f by topk>0 and topp=1.0f for the
            // compatibility.
            p = 1.0f;
        }
        // Clip k value. A topk sampling kernel supports up to TOP_K_MAX=64.
        top_ks[i] = k > TOP_K_MAX ? TOP_K_MAX : k;
        if (k > TOP_K_MAX) {
            printf("[WARNING] topk (%d) is larger than max supported number (%d) for token %d"
                   " clip to max supported number %d. \n",
                   k,
                   TOP_K_MAX,
                   i,
                   top_ks[i]);
        }
        // Clip p value if it is out of range. range = [0.0, 1.0].
        top_ps[i] = p < 0.0f ? 0.0f : (p > 1.0f ? 1.0f : p);
        if (p < 0.0f || p > 1.0f) {
            printf("[WARNING] topp (%f) is out of range ([0.0, 1.0f]) for token %d"
                   " clip to closest number %f.\n",
                   p,
                   i,
                   top_ps[i]);
        }
        skip_decode[i] = k == 0;
    }
}

void invokeSetupTopKRuntimeArgs(int          batch_size,
                                uint         top_k,
                                uint*        top_ks,
                                int          top_ks_size,
                                float        top_p,
                                float*       top_ps,
                                int          top_ps_size,
                                bool*        skip_decode,
                                hipStream_t stream) {
    dim3 block(std::min((int)batch_size, 256));
    dim3 grid(div_up((int)batch_size, (int)block.x));
    // support top_k up to 1024.
    setup_topk_runtime_args<1024>
        <<<grid, block, 0, stream>>>(batch_size, top_k, top_ks, top_ks_size, top_p, top_ps, top_ps_size, skip_decode);
}

}  // namespace rtp_llm
