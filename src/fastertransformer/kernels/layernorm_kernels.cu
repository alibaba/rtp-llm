#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "src/fastertransformer/cuda/cuda_type_utils.cuh"
#include "src/fastertransformer/kernels/layernorm_kernels.h"
#include "src/fastertransformer/kernels/reduce_kernel_utils.cuh"

#if USING_ROCM
#include "src/fastertransformer/rocm/hip_utils.h"
#endif

// wont't support new features
namespace fastertransformer{
#if USING_ROCM
using namespace rocm;
#endif

__device__ __forceinline__ int64_t loadOffset(int head_num,
                                              int size_per_head)
{
    // [[q_head_1],[q_head_2]...[k_head_1],[k_head_2]...[v_head_1],[v_head_2]...]
    int head_id = blockIdx.y;
    int batch_id = blockIdx.x;
    int offset = batch_id * head_num * size_per_head + size_per_head * head_id;
    return offset;
}

template<typename T>
__global__ void qkLayerNorm(T* __restrict qkv,
                            const T* __restrict gamma,
                            const float layernorm_eps,
                            int head_num,
                            int size_per_head)
{
    constexpr auto num_elems_T = num_elems<T>::value;
    constexpr size_t warp_size = 32;
    const int vec_size_per_head = size_per_head / num_elems_T;
    const int n_elems = vec_size_per_head / warp_size;
    using float_packed_t = typename packed_as<float, num_elems_T>::type;

    const int tid = threadIdx.x;

    __shared__ float s_mean;
    __shared__ float s_variance;
    float            mean     = 0.0f;
    float            variance = 0.0f;

    float local_sum = 0.0f;
    for (int i = 0; i < n_elems; i++) {
        auto index = loadOffset(head_num, vec_size_per_head) + tid * n_elems + i;
        auto val_f = cuda_cast<float_packed_t>(ldg(&qkv[index]));
	local_sum += cuda_sum<float>(val_f);
    }

    mean = warpReduceSum(local_sum);

    if (threadIdx.x == 0) {
        s_mean = mean / size_per_head;
    }
    __syncthreads();

    float local_var_sum = 0.0f;
    for (int i = 0; i < n_elems; i++) {
        auto index = loadOffset(head_num, vec_size_per_head) + tid * n_elems + i;
        auto val_f = cuda_cast<float_packed_t>(ldg(&qkv[index]));
        auto diff = val_f - s_mean;
        local_var_sum += cuda_sum<float>(diff * diff);
    }
    variance = warpReduceSum(local_var_sum);

    if (threadIdx.x == 0) {
        s_variance = rsqrtf(variance / size_per_head + layernorm_eps);
    }
    __syncthreads();

    for (int i = 0; i < n_elems; i++) {
        auto index = loadOffset(head_num, vec_size_per_head) + tid * n_elems + i;
	auto gamma_index = blockIdx.y * vec_size_per_head + tid * n_elems + i;
        auto val_f = cuda_cast<float_packed_t>(ldg(&qkv[index]));
	auto val_gamma = cuda_cast<float_packed_t>(gamma[gamma_index]);
        qkv[index] = cuda_cast<T>((val_f - s_mean) * s_variance * val_gamma);
    }
}

template<typename T>
void invokeQkLayerNorm(T* __restrict qkv,
		       const T* __restrict gamma,
		       const float layernorm_eps,
		       const int tokens,
		       const int head_num,
		       const int head_num_kv,
		       const int size_per_head,
		       hipStream_t stream)
{
    constexpr size_t vec_size = 2;
    constexpr size_t warp_size = 32;

    if (size_per_head % warp_size != 0) {
        throw std::invalid_argument("not supported size_per_head: " + std::to_string(size_per_head));
    }
    dim3 grid(tokens, head_num + head_num_kv);
    dim3 block(warp_size);

    int total_head_num = head_num + 2 * head_num_kv;
    using Tp = typename packed_as<T, vec_size>::type;
    qkLayerNorm<Tp><<<grid, block, 0, stream>>>(reinterpret_cast<Tp*>(qkv), reinterpret_cast<const Tp*>(gamma),
						layernorm_eps, total_head_num, size_per_head);
}

#define INSTANTIATE_QK_LAYERNORM(T)				\
  template void invokeQkLayerNorm(T* __restrict qkv,		\
				  const T* __restrict gamma,	\
				  const float layernorm_eps,	\
				  const int tokens,		\
				  const int head_num,		\
				  const int head_num_kv,	\
				  const int size_per_head,	\
				  hipStream_t stream)
INSTANTIATE_QK_LAYERNORM(float);
INSTANTIATE_QK_LAYERNORM(half);
#ifdef ENABLE_BF16
INSTANTIATE_QK_LAYERNORM(__hip_bfloat16);
#endif


template <typename Tf, typename T, bool IS_BETA>
__inline__ __device__ Tf compute_layernorm(Tf val, float s_mean, float s_variance, const T* gamma, const T* beta, int i)
{
    Tf ret = (val - s_mean) * s_variance * cuda_cast<Tf>(gamma[i]);
    if (IS_BETA)
    {
        ret = ret + cuda_cast<Tf>(beta[i]);
    }
    return ret;
}

/* Computes the layernorm https://pytorch.org/docs/stable/generated/torch.nn.LayerNorm.html
 * normed_output <- ( (input - E[input]) / Sqrt(Var[input] + eps) ) * gamma + beta
 * input is [tokens, hidden_dim]. Mean and Variance are per-row (i.e. per-token)
 *
 * One CTA handles one row.
 *
 * with USE_DIFF_OF_SQUARES set to false:
 * First pass (loop) computes the mean.
 * Second computes the variance via Var[x] = E[(x - E[x])²].
 * Third pass computes and writes normed_output
 *
 * with USE_DIFF_OF_SQUARES set to true (may be faster but less accurate):
 * First pass (loop) computes the mean and variance via Var[x] = E[x²] - E[x]²
 * Second pass computes and writes normed_output
 *
 * use_shmem controls if we cache input values into shared memory
 *
 * Optional: with dynamic scaling, the last pass doesn't write immediately but finds the
 *           amax per row. A final pass scales to int8 accordingly, and writes output to
 *           normed_output_quant.
 */
template <typename T, bool IS_OUTPUT, bool IS_BIAS, bool RESIDUAL, bool IS_BETA, bool RETURN_NORMED_OUTPUT, bool USE_DIFF_OF_SQUARES = false>
__global__ void generalLayerNorm(T* output, T* normed_output, const T* input, const T* bias, const T* residual,
    const T* gamma, const T* beta, const float eps, int tokens, int hidden_dim,
    const float* scale_orig_quant_per_tensor, float* scale_orig_quant_per_token, int8_t* normed_output_quant)
{
    constexpr auto num_elems_T = num_elems<T>::value;
    using int8_packed_t = typename packed_as<int8_t, num_elems_T>::type;
    using Int32_Packed_T = typename packed_as<int32_t, num_elems<T>::value>::type;
    using float_packed_t = typename packed_as<float, num_elems_T>::type;
    using T_scalar = typename packed_as<T, 1>::type;

    extern __shared__ __align__(sizeof(float)) char _shmem[];
    T* shmem = reinterpret_cast<T*>(_shmem);
    __shared__ float s_mean;
    __shared__ float s_variance;

    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;

    float mean = 0.0f;
    float variance = 0.0f;
    float local_sum = 0.0f;
    float local_var_sum = 0.0f;

    const bool with_per_token_scaling = scale_orig_quant_per_token != nullptr;
    const bool with_per_tensor_scaling = scale_orig_quant_per_tensor != nullptr;
    const float_packed_t scale_orig_quant
        = cuda_cast<float_packed_t>(with_per_tensor_scaling ? *scale_orig_quant_per_tensor : 0.0f);
    T_scalar amax(1e-6f);

    const int n_elems = hidden_dim / num_elems_T;
    for (int i = tidx; i < n_elems; i += blockDim.x)
    {
        // const T val = input[bidx * n_elems + i];
        const int index = bidx * n_elems + i;
        T val = cuda_cast<T>(0.0f);
        // const T val = input[index];
        if (IS_BIAS)
        {
            val = add(val, ldg(&bias[i]));
        }
        if (RESIDUAL)
        {
            val = add(val, ldg(&residual[index]));
        }
        if (IS_OUTPUT)
        {
            T in_val;
            if (with_per_tensor_scaling)
            {
                in_val = cuda_cast<T>(cuda_cast<float_packed_t>(reinterpret_cast<const Int32_Packed_T*>(input)[index])
                    * scale_orig_quant);
            }
            else
            {
                in_val = input[index];
            }
            val = add(val, in_val);
            if (!RETURN_NORMED_OUTPUT) {
                output[index] = val;
            }
        }
        shmem[i] = val;

        const float_packed_t val_f = cuda_cast<float_packed_t>(val);
        local_sum += cuda_sum<float>(val_f);
        if (USE_DIFF_OF_SQUARES)
        {
            local_var_sum += cuda_sum<float>(val_f * val_f);
        }
    }

    if (USE_DIFF_OF_SQUARES)
    {
        float packed[2] = {local_sum, local_var_sum};
        blockReduceSumV2<float, 2>(packed);
        mean = packed[0];
        variance = packed[1];
    }
    else
    {
        mean = blockReduceSum(local_sum);
    }

    if (threadIdx.x == 0)
    {
        mean = mean / hidden_dim;
        s_mean = mean;
        if (USE_DIFF_OF_SQUARES)
        {
            variance = (variance / hidden_dim) - (mean * mean); // Var[x] = E[x²] - E[x]²
            s_variance = rsqrtf(variance + eps);
        }
    }
    __syncthreads();

    if (!USE_DIFF_OF_SQUARES)
    {
        for (int i = tidx; i < n_elems; i += blockDim.x)
        {
            const T val = shmem[i];
            float_packed_t diff = cuda_cast<float_packed_t>(val) - s_mean;
            local_var_sum += cuda_sum<float>(diff * diff);
        }
        variance = blockReduceSum(local_var_sum);

        if (threadIdx.x == 0)
        {
            s_variance = rsqrtf(variance / hidden_dim + eps);
        }
        __syncthreads();
    }

    for (int i = tidx; i < n_elems; i += blockDim.x)
    {
        const int index = bidx * n_elems + i;
        const float_packed_t val_f = cuda_cast<float_packed_t>(shmem[i]);
        const T val
            = cuda_cast<T>(compute_layernorm<float_packed_t, T, IS_BETA>(val_f, s_mean, s_variance, gamma, beta, i));
        if (RETURN_NORMED_OUTPUT && IS_OUTPUT) {
            output[index] = val;
        }
        if (with_per_token_scaling)
        {
            amax = cuda_max(cuda_max<T_scalar, T>(cuda_abs(val)), amax);
            shmem[i] = val;
        }
        else if (with_per_tensor_scaling)
        {
            reinterpret_cast<int8_packed_t*>(normed_output_quant)[index]
                = cuda_cast<int8_packed_t>(cuda_cast<float_packed_t>(val) * scale_orig_quant);
        }
        else
        {
            normed_output[index] = val;
        }
    }

    if (with_per_token_scaling)
    {
        float abs_max_f = blockAllReduceMax(cuda_cast<float>(amax));
        const float dynamic_per_token_scale = 127.f / abs_max_f;
        for (int i = tidx; i < n_elems; i += blockDim.x)
        {
            const int index = bidx * n_elems + i;
            float_packed_t val_f = cuda_cast<float_packed_t>(shmem[i]);
            reinterpret_cast<int8_packed_t*>(normed_output_quant)[index]
                = cuda_cast<int8_packed_t>(val_f * cuda_cast<float_packed_t>(dynamic_per_token_scale));
        }
        if (tidx == 0)
        {
            scale_orig_quant_per_token[bidx] = abs_max_f / 127.f;
        }
    }
}

template <typename T, bool IS_OUTPUT, bool IS_BIAS, bool RESIDUAL, bool IS_BETA, bool RETURN_NORMED_OUTPUT, bool USE_DIFF_OF_SQUARES>
void dispatch_layernorm_type_square_method(T* output, T* normed_output, const T* input, const T* bias,
    const T* residual, const T* gamma, const T* beta, const float eps, int tokens, int hidden_dim,
    const float* scale_orig_quant_per_tensor, float* scale_orig_quant_per_token, int8_t* normed_output_quant,
    const dim3 grid, const dim3 block, const size_t shmem_size, hipStream_t stream)
{
    if (shmem_size >= (48 << 10))
    {
#if USING_CUDA
        hipError_t ret
            = hipFuncSetAttribute(reinterpret_cast<const void*>(generalLayerNorm<T), IS_OUTPUT, IS_BIAS, RESIDUAL, IS_BETA, RETURN_NORMED_OUTPUT, USE_DIFF_OF_SQUARES>,
                hipFuncAttributeMaxDynamicSharedMemorySize, shmem_size);
#endif
    }
    generalLayerNorm<T, IS_OUTPUT, IS_BIAS, RESIDUAL, IS_BETA, RETURN_NORMED_OUTPUT, USE_DIFF_OF_SQUARES>
        <<<grid, block, shmem_size, stream>>>(output, normed_output, input, bias, residual, gamma, beta, eps, tokens,
            hidden_dim, scale_orig_quant_per_tensor, scale_orig_quant_per_token, normed_output_quant);
}

template <typename T, bool IS_OUTPUT, bool IS_BIAS, bool RESIDUAL, bool IS_BETA, bool RETURN_NORMED_OUTPUT>
void dispatch_layernorm_return_normed(T* output, T* normed_output, const T* input, const T* bias, const T* residual,
    const T* gamma, const T* beta, const float eps, int tokens, int hidden_dim,
    const float* scale_orig_quant_per_tensor, float* scale_orig_quant_per_token, int8_t* normed_output_quant,
    const dim3 grid, const dim3 block, const size_t shmem_size, hipStream_t stream, bool use_diff_of_squares)
{
    if (use_diff_of_squares)
    {
        dispatch_layernorm_type_square_method<T, IS_OUTPUT, IS_BIAS, RESIDUAL, IS_BETA, RETURN_NORMED_OUTPUT, true>(output, normed_output,
            input, bias, residual, gamma, beta, eps, tokens, hidden_dim, scale_orig_quant_per_tensor,
            scale_orig_quant_per_token, normed_output_quant, grid, block, shmem_size, stream);
    }
    else
    {
        dispatch_layernorm_type_square_method<T, IS_OUTPUT, IS_BIAS, RESIDUAL, IS_BETA, RETURN_NORMED_OUTPUT, false>(output, normed_output,
            input, bias, residual, gamma, beta, eps, tokens, hidden_dim, scale_orig_quant_per_tensor,
            scale_orig_quant_per_token, normed_output_quant, grid, block, shmem_size, stream);
    }
}

template <typename T, bool IS_OUTPUT, bool IS_BIAS, bool RESIDUAL, bool IS_BETA>
void dispatch_layernorm_type(T* output, T* normed_output, const T* input, const T* bias, const T* residual,
    const T* gamma, const T* beta, const float eps, int tokens, int hidden_dim,
    const float* scale_orig_quant_per_tensor, float* scale_orig_quant_per_token, int8_t* normed_output_quant,
    const dim3 grid, const dim3 block, const size_t shmem_size, hipStream_t stream, bool use_diff_of_squares, bool return_normed_output)
{
    if (return_normed_output)
    {
        dispatch_layernorm_return_normed<T, IS_OUTPUT, IS_BIAS, RESIDUAL, IS_BETA, true>(output, normed_output,
            input, bias, residual, gamma, beta, eps, tokens, hidden_dim, scale_orig_quant_per_tensor,
            scale_orig_quant_per_token, normed_output_quant, grid, block, shmem_size, stream, use_diff_of_squares);
    }
    else
    {
        dispatch_layernorm_return_normed<T, IS_OUTPUT, IS_BIAS, RESIDUAL, IS_BETA, false>(output, normed_output,
            input, bias, residual, gamma, beta, eps, tokens, hidden_dim, scale_orig_quant_per_tensor,
            scale_orig_quant_per_token, normed_output_quant, grid, block, shmem_size, stream, use_diff_of_squares);
    }
}

template <typename T, bool IS_OUTPUT, bool IS_BIAS, bool RESIUDAL>
void dispatch_layernorm_beta(T* output, T* normed_output, const T* input, const T* bias, const T* residual,
    const T* gamma, const T* beta, const float eps, int tokens, int hidden_dim,
    const float* scale_orig_quant_per_tensor, float* scale_orig_quant_per_token, int8_t* normed_output_quant,
    const dim3 grid, const dim3 block, const size_t shmem_size, hipStream_t stream, bool use_diff_of_squares, bool return_normed_output)
{
    if (beta != nullptr)
    {
        dispatch_layernorm_type<T, IS_OUTPUT, IS_BIAS, RESIUDAL, true>(output, normed_output, input, bias, residual,
            gamma, beta, eps, tokens, hidden_dim, scale_orig_quant_per_tensor, scale_orig_quant_per_token,
            normed_output_quant, grid, block, shmem_size, stream, use_diff_of_squares, return_normed_output);
    }
    else
    {
        dispatch_layernorm_type<T, IS_OUTPUT, IS_BIAS, RESIUDAL, false>(output, normed_output, input, bias, residual,
            gamma, beta, eps, tokens, hidden_dim, scale_orig_quant_per_tensor, scale_orig_quant_per_token,
            normed_output_quant, grid, block, shmem_size, stream, use_diff_of_squares, return_normed_output);
    }
}

template <typename T, bool IS_OUTPUT, bool IS_BIAS>
void dispatch_layernorm_residual(T* output, T* normed_output, const T* input, const T* bias, const T* residual,
    const T* gamma, const T* beta, const float eps, int tokens, int hidden_dim,
    const float* scale_orig_quant_per_tensor, float* scale_orig_quant_per_token, int8_t* normed_output_quant,
    const dim3 grid, const dim3 block, const size_t shmem_size, hipStream_t stream, bool use_diff_of_squares, bool return_normed_output)
{
    if (residual != nullptr)
    {
        dispatch_layernorm_beta<T, IS_OUTPUT, IS_BIAS, true>(output, normed_output, input, bias, residual, gamma, beta,
            eps, tokens, hidden_dim, scale_orig_quant_per_tensor, scale_orig_quant_per_token, normed_output_quant, grid,
            block, shmem_size, stream, use_diff_of_squares, return_normed_output);
    }
    else
    {
        dispatch_layernorm_beta<T, IS_OUTPUT, IS_BIAS, false>(output, normed_output, input, bias, residual, gamma, beta,
            eps, tokens, hidden_dim, scale_orig_quant_per_tensor, scale_orig_quant_per_token, normed_output_quant, grid,
            block, shmem_size, stream, use_diff_of_squares, return_normed_output);
    }
}

template <typename T, bool IS_OUTPUT>
void dispatch_layernorm_bias(T* output, T* normed_output, const T* input, const T* bias, const T* residual,
    const T* gamma, const T* beta, const float eps, int tokens, int hidden_dim,
    const float* scale_orig_quant_per_tensor, float* scale_orig_quant_per_token, int8_t* normed_output_quant,
    const dim3 grid, const dim3 block, const size_t shmem_size, hipStream_t stream, bool use_diff_of_squares, bool return_normed_output)
{
    if (bias != nullptr)
    {
        dispatch_layernorm_residual<T, IS_OUTPUT, true>(output, normed_output, input, bias, residual, gamma, beta, eps,
            tokens, hidden_dim, scale_orig_quant_per_tensor, scale_orig_quant_per_token, normed_output_quant, grid,
            block, shmem_size, stream, use_diff_of_squares, return_normed_output);
    }
    else
    {
        dispatch_layernorm_residual<T, IS_OUTPUT, false>(output, normed_output, input, bias, residual, gamma, beta, eps,
            tokens, hidden_dim, scale_orig_quant_per_tensor, scale_orig_quant_per_token, normed_output_quant, grid,
            block, shmem_size, stream, use_diff_of_squares, return_normed_output);
    }
}

template <typename T>
void dispatch_layernorm_output(T* output, T* normed_output, const T* input, const T* bias, const T* residual,
    const T* gamma, const T* beta, const float eps, int tokens, int hidden_dim,
    const float* scale_orig_quant_per_tensor, float* scale_orig_quant_per_token, int8_t* normed_output_quant,
    const dim3 grid, const dim3 block, const size_t shmem_size, hipStream_t stream, bool use_diff_of_squares,
    bool is_output, bool return_normed_output)
{
    if (is_output)
    {
        dispatch_layernorm_bias<T, true>(output, normed_output, input, bias, residual, gamma, beta, eps, tokens,
            hidden_dim, scale_orig_quant_per_tensor, scale_orig_quant_per_token, normed_output_quant, grid, block,
            shmem_size, stream, use_diff_of_squares, return_normed_output);
    }
    else
    {
        dispatch_layernorm_bias<T, false>(output, normed_output, input, bias, residual, gamma, beta, eps, tokens,
            hidden_dim, scale_orig_quant_per_tensor, scale_orig_quant_per_token, normed_output_quant, grid, block,
            shmem_size, stream, use_diff_of_squares, return_normed_output);
    }
}

template <typename T>
void invokeGeneralLayerNorm(T* out, const T* input, const T* gamma, const T* beta, const float eps, const int tokens,
    const int hidden_dim, hipStream_t stream, bool use_diff_of_squares, const float* scale, float* dynamic_scale,
    int8_t* out_quant, bool return_normed_output)
{
    dim3 grid(tokens);
    dim3 block(min(hidden_dim, 1024));
    // Make sure block.x is multiple of 32 for warp shuffle to work
    block.x = 32 * ((block.x + 31) / 32);

    constexpr size_t vec_size = 2;
    const size_t shmem_size = hidden_dim * sizeof(T);
    const bool use_vec_type = (hidden_dim % vec_size == 0)
        && (std::is_same<T, half>::value
#ifdef ENABLE_BF16
            || std::is_same<T, __hip_bfloat16>::value
#endif
        );

    if (use_vec_type)
    {
        using Tp = typename packed_as<T, vec_size>::type;
        dispatch_layernorm_output(reinterpret_cast<Tp*>(out), reinterpret_cast<Tp*>(out),
            reinterpret_cast<const Tp*>(out), (const Tp*) nullptr, reinterpret_cast<const Tp*>(input),
            reinterpret_cast<const Tp*>(gamma), reinterpret_cast<const Tp*>(beta), eps, tokens, hidden_dim, scale,
            dynamic_scale, out_quant, grid, block, shmem_size, stream, use_diff_of_squares, false, return_normed_output);
    }
    else
    {
        dispatch_layernorm_output(out, out, (const T*) out, (const T*) nullptr, input, gamma, beta, eps, tokens,
            hidden_dim, scale, dynamic_scale, out_quant, grid, block, shmem_size, stream, use_diff_of_squares, false, return_normed_output);
    }
}

template <typename T>
void invokeGeneralAddBiasResidualLayerNorm(T* out, T* norm_output, const T* input, const T* bias, const T* residual,
    const T* gamma, const T* beta, const float eps, const int tokens, const int hidden_dim, hipStream_t stream,
    bool use_diff_of_squares, const float* scale, float* dynamic_scale, int8_t* out_quant, bool return_normed_output)
{
    dim3 grid(tokens);
    dim3 block(min(hidden_dim, 1024));
    // Make sure block.x is multiple of 32 for warp shuffle to work
    block.x = 32 * ((block.x + 31) / 32);

    constexpr size_t vec_size = 2;
    const size_t shmem_size = hidden_dim * sizeof(T);
    const bool use_vec_type = (hidden_dim % vec_size == 0)
        && (std::is_same<T, half>::value
#ifdef ENABLE_BF16
            || std::is_same<T, __hip_bfloat16>::value
#endif
        );

    if (use_vec_type)
    {
        using Tp = typename packed_as<T, vec_size>::type;
        dispatch_layernorm_output(reinterpret_cast<Tp*>(out), reinterpret_cast<Tp*>(norm_output),
            reinterpret_cast<const Tp*>(input), reinterpret_cast<const Tp*>(bias),
            reinterpret_cast<const Tp*>(residual), reinterpret_cast<const Tp*>(gamma),
            reinterpret_cast<const Tp*>(beta), eps, tokens, hidden_dim, scale, dynamic_scale, out_quant, grid, block,
            shmem_size, stream, use_diff_of_squares, true, return_normed_output);
    }
    else
    {
        dispatch_layernorm_output(out, norm_output, input, bias, residual, gamma, beta, eps, tokens, hidden_dim, scale,
            dynamic_scale, out_quant, grid, block, shmem_size, stream, use_diff_of_squares, true, return_normed_output);
    }
}

#define INSTANTIATE_GENERAL_LAYERNORM(T)                                                                               \
    template void invokeGeneralLayerNorm(T* out, const T* input, const T* gamma, const T* beta, const float eps,       \
        const int tokens, const int hidden_dim, hipStream_t stream, bool use_diff_of_squares, const float* scale,     \
        float* dynamic_scale, int8_t* out_quant, bool return_normed_output);

INSTANTIATE_GENERAL_LAYERNORM(float);
INSTANTIATE_GENERAL_LAYERNORM(half);
#ifdef ENABLE_BF16
INSTANTIATE_GENERAL_LAYERNORM(__hip_bfloat16);
#endif

#define INSTANTIATE_GENERAL_ADD_BIAS_RESDIAUL_LAYERNORM(T)                                                             \
    template void invokeGeneralAddBiasResidualLayerNorm(T* out, T* norm_output, const T* input, const T* bias,         \
        const T* residual, const T* gamma, const T* beta, const float eps, const int tokens, const int hidden_dim,     \
        hipStream_t stream, bool use_diff_of_squares, const float* scale, float* dynamic_scale, int8_t* out_quant, bool return_normed_output);

INSTANTIATE_GENERAL_ADD_BIAS_RESDIAUL_LAYERNORM(float);
INSTANTIATE_GENERAL_ADD_BIAS_RESDIAUL_LAYERNORM(half);
#ifdef ENABLE_BF16
INSTANTIATE_GENERAL_ADD_BIAS_RESDIAUL_LAYERNORM(__hip_bfloat16);
#endif

} // namespace fastertransformer
