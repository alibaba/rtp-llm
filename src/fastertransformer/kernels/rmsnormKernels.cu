#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "src/fastertransformer/cuda/cuda_type_utils.cuh"
#include "src/fastertransformer/kernels/reduce_kernel_utils.cuh"
#include "src/fastertransformer/kernels/rmsnormKernels.h"

namespace fastertransformer
{

template <typename Tf, typename T, bool IS_BETA>
__inline__ __device__ Tf compute_rmsnorm(Tf val, float s_variance, const T* gamma, const T* beta, int i)
{
    Tf ret = val * s_variance * cuda_cast<Tf>(gamma[i]);
    if (IS_BETA)
    {
        ret = ret + cuda_cast<Tf>(beta[i]);
    }
    return ret;
}

/* Computes the rmsnorm https://pytorch.org/docs/stable/generated/torch.nn.rmsnorm.html
 * normed_output <- ( input / Sqrt(E[input²] + eps) ) * gamma + beta
 * input is [tokens, hidden_dim]. Mean and Variance are per-row (i.e. per-token)
 *
 * One CTA handles one row.
 *
 *
 * use_shmem controls if we cache input values into shared memory
 *
 * Optional: with dynamic scaling, the last pass doesn't write immediately but finds the
 *           amax per row. A final pass scales to int8 accordingly, and writes output to
 *           normed_output_quant.
 */
template <typename T, bool IS_OUTPUT, bool IS_BIAS, bool RESIDUAL, bool IS_BETA>
__global__ void generalRmsNorm(T* output, T* normed_output, const T* input, const T* bias, const T* residual1,
    const T* gamma, const T* beta, const float eps, int tokens, int hidden_dim,
    const float* scale_orig_quant_per_tensor, float* scale_orig_quant_per_token, int8_t* normed_output_quant)
{
    constexpr auto num_elems_T = num_elems<T>::value;
    using int8_packed_t = typename packed_as<int8_t, num_elems_T>::type;
    using Int32_Packed_T = typename packed_as<int32_t, num_elems<T>::value>::type;
    using float_packed_t = typename packed_as<float, num_elems_T>::type;
    using T_scalar = typename packed_as<T, 1>::type;

    extern __shared__ __align__(sizeof(float)) char _shmem[];
    T* shmem = reinterpret_cast<T*>(_shmem);

    __shared__ float s_variance;

    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;

    float variance = 0.0f;
    float local_var_sum = 0.0f;

    const int n_elems = hidden_dim / num_elems_T;

    const bool with_per_token_scaling = scale_orig_quant_per_token != nullptr;
    const bool with_per_tensor_scaling = scale_orig_quant_per_tensor != nullptr;
    const float_packed_t scale_orig_quant
        = cuda_cast<float_packed_t>(with_per_tensor_scaling ? *scale_orig_quant_per_tensor : 0.0f);
    T_scalar amax(1e-6f);

    for (int i = tidx; i < n_elems; i += blockDim.x)
    {
        const int index = bidx * n_elems + i;
        T val = cuda_cast<T>(0.0f);
        // const T val = input[index];
        if (IS_BIAS)
        {
            val = add(val, ldg(&bias[i]));
        }
        if (RESIDUAL)
        {
            val = add(val, ldg(&residual1[index]));
        }
        if (IS_OUTPUT)
        {
            T in_val;
            if (with_per_tensor_scaling)
            {
                in_val = cuda_cast<T>(
                    cuda_cast<float_packed_t>(reinterpret_cast<const Int32_Packed_T*>(input)[index]) * scale_orig_quant);
            }
            else
            {
                in_val = input[index];
            }
            val = add(val, in_val);
        }

        shmem[i] = val;

        if (IS_OUTPUT)
        {
            output[index] = val;
        }
        const float_packed_t val_f = cuda_cast<float_packed_t>(val);

        local_var_sum += cuda_sum<float>(val_f * val_f);
    }

    float packed[1] = {local_var_sum};
    blockReduceSumV2<float, 1>(packed);
    variance = packed[0];

    if (threadIdx.x == 0)
    {
        variance = (variance / hidden_dim); // Var[x] = E[x²]
        s_variance = rsqrtf(variance + eps);
    }
    __syncthreads();

    for (int i = tidx; i < n_elems; i += blockDim.x)
    {
        const int index = bidx * n_elems + i;
        const float_packed_t val_f = cuda_cast<float_packed_t>(shmem[i]);
        const T val = cuda_cast<T>(compute_rmsnorm<float_packed_t, T, IS_BETA>(val_f, s_variance, gamma, beta, i));

        if (with_per_token_scaling)
        {
            amax = cuda_max(cuda_max<T_scalar, T>(cuda_abs(val)), amax);
            shmem[i] = val;
        }
        else if (with_per_tensor_scaling)
        {
            reinterpret_cast<int8_packed_t*>(normed_output_quant)[index]
                = cuda_cast<int8_packed_t>(cuda_cast<float_packed_t>(val) * scale_orig_quant);
        }
        else
        {
            normed_output[index] = val;
        }
    }

    if (with_per_token_scaling)
    {
        float abs_max_f = blockAllReduceMax(cuda_cast<float>(amax));
        const float dynamic_per_token_scale = 127.f / abs_max_f;
        for (int i = tidx; i < n_elems; i += blockDim.x)
        {
            const int index = bidx * n_elems + i;
            float_packed_t val_f = cuda_cast<float_packed_t>(shmem[i]);
            reinterpret_cast<int8_packed_t*>(normed_output_quant)[index]
                = cuda_cast<int8_packed_t>(val_f * cuda_cast<float_packed_t>(dynamic_per_token_scale));
        }
        if (tidx == 0)
        {
            scale_orig_quant_per_token[bidx] = abs_max_f / 127.f;
        }
    }
}

template <typename T, bool IS_OUTPUT, bool IS_BIAS, bool RESIDUAL, bool IS_BETA>
void dispatch_rmsnorm_type_square_method(T* output, T* normed_output, const T* input, const T* bias, const T* residual1,
    const T* gamma, const T* beta, const float eps, int tokens, int hidden_dim,
    const float* scale_orig_quant_per_tensor, float* scale_orig_quant_per_token, int8_t* normed_output_quant,
    const dim3 grid, const dim3 block, const size_t shmem_size, hipStream_t stream)
{
    if (shmem_size >= (48 << 10))
    {
#if USING_CUDA
        hipError_t ret = hipFuncSetAttribute(reinterpret_cast<const void*>(generalRmsNorm<T), IS_OUTPUT, IS_BIAS, RESIDUAL, IS_BETA>,
            hipFuncAttributeMaxDynamicSharedMemorySize, shmem_size);
#endif
    }
    generalRmsNorm<T, IS_OUTPUT, IS_BIAS, RESIDUAL, IS_BETA><<<grid, block, shmem_size, stream>>>(output, normed_output,
        input, bias, residual1, gamma, beta, eps, tokens, hidden_dim, scale_orig_quant_per_tensor,
        scale_orig_quant_per_token, normed_output_quant);
}

template <typename T, bool IS_OUTPUT, bool IS_BIAS, bool RESIDUAL>
void dispatch_rmsnorm_beta(T* output, T* normed_output, const T* input, const T* bias, const T* residual1,
    const T* gamma, const T* beta, const float eps, int tokens, int hidden_dim,
    const float* scale_orig_quant_per_tensor, float* scale_orig_quant_per_token, int8_t* normed_output_quant,
    const dim3 grid, const dim3 block, const size_t shmem_size, hipStream_t stream)
{
    if (beta != nullptr)
    {

        dispatch_rmsnorm_type_square_method<T, IS_OUTPUT, IS_BIAS, RESIDUAL, true>(output, normed_output, input, bias,
            residual1, gamma, beta, eps, tokens, hidden_dim, scale_orig_quant_per_tensor, scale_orig_quant_per_token,
            normed_output_quant, grid, block, shmem_size, stream);
    }
    else
    {

        dispatch_rmsnorm_type_square_method<T, IS_OUTPUT, IS_BIAS, RESIDUAL, false>(output, normed_output, input, bias,
            residual1, gamma, beta, eps, tokens, hidden_dim, scale_orig_quant_per_tensor, scale_orig_quant_per_token,
            normed_output_quant, grid, block, shmem_size, stream);
    }
}

template <typename T, bool IS_OUTPUT, bool IS_BIAS>
void dispatch_rmsnorm_residual(T* output, T* normed_output, const T* input, const T* bias, const T* residual1,
    const T* gamma, const T* beta, const float eps, int tokens, int hidden_dim,
    const float* scale_orig_quant_per_tensor, float* scale_orig_quant_per_token, int8_t* normed_output_quant,
    const dim3 grid, const dim3 block, const size_t shmem_size, hipStream_t stream)
{
    if (residual1 != nullptr)
    {

        dispatch_rmsnorm_beta<T, IS_OUTPUT, IS_BIAS, true>(output, normed_output, input, bias, residual1, gamma, beta,
            eps, tokens, hidden_dim, scale_orig_quant_per_tensor, scale_orig_quant_per_token, normed_output_quant, grid,
            block, shmem_size, stream);
    }
    else
    {

        dispatch_rmsnorm_beta<T, IS_OUTPUT, IS_BIAS, false>(output, normed_output, input, bias, residual1, gamma, beta,
            eps, tokens, hidden_dim, scale_orig_quant_per_tensor, scale_orig_quant_per_token, normed_output_quant, grid,
            block, shmem_size, stream);
    }
}

template <typename T, bool IS_OUTPUT>
void dispatch_rmsnorm_bias(T* output, T* normed_output, const T* input, const T* bias, const T* residual1,
    const T* gamma, const T* beta, const float eps, int tokens, int hidden_dim,
    const float* scale_orig_quant_per_tensor, float* scale_orig_quant_per_token, int8_t* normed_output_quant,
    const dim3 grid, const dim3 block, const size_t shmem_size, hipStream_t stream)
{
    if (bias != nullptr)
    {

        dispatch_rmsnorm_residual<T, IS_OUTPUT, true>(output, normed_output, input, bias, residual1, gamma, beta, eps,
            tokens, hidden_dim, scale_orig_quant_per_tensor, scale_orig_quant_per_token, normed_output_quant, grid,
            block, shmem_size, stream);
    }
    else
    {

        dispatch_rmsnorm_residual<T, IS_OUTPUT, false>(output, normed_output, input, bias, residual1, gamma, beta, eps,
            tokens, hidden_dim, scale_orig_quant_per_tensor, scale_orig_quant_per_token, normed_output_quant, grid,
            block, shmem_size, stream);
    }
}

template <typename T>
void dispatch_rmsnorm_output(T* output, T* normed_output, const T* input, const T* bias, const T* residual1,
    const T* gamma, const T* beta, const float eps, int tokens, int hidden_dim,
    const float* scale_orig_quant_per_tensor, float* scale_orig_quant_per_token, int8_t* normed_output_quant,
    const dim3 grid, const dim3 block, const size_t shmem_size, hipStream_t stream, bool is_output)
{
    if (is_output)
    {

        dispatch_rmsnorm_bias<T, true>(output, normed_output, input, bias, residual1, gamma, beta, eps, tokens,
            hidden_dim, scale_orig_quant_per_tensor, scale_orig_quant_per_token, normed_output_quant, grid, block,
            shmem_size, stream);
    }
    else
    {
        dispatch_rmsnorm_bias<T, false>(output, normed_output, input, bias, residual1, gamma, beta, eps, tokens,
            hidden_dim, scale_orig_quant_per_tensor, scale_orig_quant_per_token, normed_output_quant, grid, block,
            shmem_size, stream);
    }
}

template <typename T>
void invokeGeneralRmsNorm(T* out, const T* input, const T* gamma, const T* beta, const float eps, const int tokens,
    const int hidden_dim, hipStream_t stream, const float* scale, float* dynamic_scale, int8_t* normed_output_quant)
{
    dim3 grid(tokens);
    dim3 block(min(hidden_dim, 1024));
    // Make sure block.x is multiple of 32 for warp shuffle to work
    block.x = 32 * ((block.x + 31) / 32);

    constexpr size_t vec_size = 2;
    const size_t shmem_size = hidden_dim * sizeof(T);
    const bool use_vec_type = (hidden_dim % vec_size == 0)
        && (std::is_same<T, half>::value
#ifdef ENABLE_BF16
            || std::is_same<T, __hip_bfloat16>::value
#endif
        );

    if (use_vec_type)
    {
        using Tp = typename packed_as<T, vec_size>::type;
        dispatch_rmsnorm_output(reinterpret_cast<Tp*>(out), reinterpret_cast<Tp*>(out), reinterpret_cast<Tp*>(out),
            (const Tp*) nullptr, reinterpret_cast<const Tp*>(input), reinterpret_cast<const Tp*>(gamma),
            reinterpret_cast<const Tp*>(beta), eps, tokens, hidden_dim, scale, dynamic_scale, normed_output_quant, grid,
            block, shmem_size, stream, false);
    }
    else
    {
        dispatch_rmsnorm_output(out, out, (const T*) out, (const T*) nullptr, input, gamma, beta, eps, tokens,
            hidden_dim, scale, dynamic_scale, normed_output_quant, grid, block, shmem_size, stream, false);
    }
}

template <typename T>
void invokeAddBiasResidualRmsNorm(T* output, T* normed_output, const T* input, const T* bias, const T* residual,
    const T* gamma, const T* beta, const float eps, const int tokens, const int hidden_dim, hipStream_t stream,
    const float* scale, float* dynamic_scale, int8_t* normed_output_quant)
{
    dim3 grid(tokens);
    dim3 block(min(hidden_dim, 1024));
    // Make sure block.x is multiple of 32 for warp shuffle to work
    block.x = 32 * ((block.x + 31) / 32);

    constexpr size_t vec_size = 2;
    const size_t shmem_size = hidden_dim * sizeof(T);
    const bool use_vec_type = (hidden_dim % vec_size == 0)
        && (std::is_same<T, half>::value
#ifdef ENABLE_BF16
            || std::is_same<T, __hip_bfloat16>::value
#endif
        );

    if (use_vec_type)
    {
        using Tp = typename packed_as<T, vec_size>::type;
        dispatch_rmsnorm_output(reinterpret_cast<Tp*>(output), reinterpret_cast<Tp*>(normed_output),
            reinterpret_cast<const Tp*>(input), reinterpret_cast<const Tp*>(bias),
            reinterpret_cast<const Tp*>(residual), reinterpret_cast<const Tp*>(gamma),
            reinterpret_cast<const Tp*>(beta), eps, tokens, hidden_dim, scale, dynamic_scale, normed_output_quant, grid,
            block, shmem_size, stream, true);
    }
    else
    {
        dispatch_rmsnorm_output(output, normed_output, input, bias, residual, gamma, beta, eps, tokens, hidden_dim,
            scale, dynamic_scale, normed_output_quant, grid, block, shmem_size, stream, true);
    }
}

#define INSTANTIATE_GENERAL_RMSNORM(T)                                                                                 \
    template void invokeGeneralRmsNorm(T* out, const T* input, const T* gamma, const T* beta, const float eps,         \
        const int tokens, const int hidden_dim, hipStream_t stream, const float* scale, float* dynamic_scale,         \
        int8_t* normed_output_quant);

INSTANTIATE_GENERAL_RMSNORM(float);
INSTANTIATE_GENERAL_RMSNORM(half);

#ifdef ENABLE_BF16
INSTANTIATE_GENERAL_RMSNORM(__hip_bfloat16);
#endif

#define INSTANTIATE_ADD_BIAS_RESL_RMSNORM(T)                                                                           \
    template void invokeAddBiasResidualRmsNorm(T* output, T* normed_output, const T* input, const T* bias,             \
        const T* resiudal, const T* gamma, const T* beta, const float eps, const int tokens, const int hidden_dim,     \
        hipStream_t stream, const float* scale, float* dynamic_scale, int8_t* normed_output_quant);

INSTANTIATE_ADD_BIAS_RESL_RMSNORM(float);
INSTANTIATE_ADD_BIAS_RESL_RMSNORM(half);
#ifdef ENABLE_BF16
INSTANTIATE_ADD_BIAS_RESL_RMSNORM(__hip_bfloat16);
#endif

} // namespace fastertransformer
