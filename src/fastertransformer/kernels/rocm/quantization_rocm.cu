#include "hip/hip_runtime.h"
#include "src/fastertransformer/utils/assert_utils.h"
#include "src/fastertransformer/kernels/rocm/quantization_rocm.h"
#include "src/fastertransformer/kernels/reduce_kernel_utils.cuh"
#include "src/fastertransformer/rocm/hip_utils.h"

namespace fastertransformer {
using namespace rocm;

/////////////////////////////////////////////////////////////////////////////////////////////////
// int4 col quant ///////////////////////////////////////////////////////////////////////////////
template<typename T>
__global__ void perColQuantization(const T*      src,
                                   const int64_t numRows,
                                   const int64_t numCols,
                                   const int64_t groupSize,
                                   uint8_t*      weightPtr,
                                   half*         scalePtr,
                                   half*         zerosPtr,
                                   float*        dbgfp  = nullptr,
                                   int*          dbgint = nullptr) {
    uint32_t colPckIdx = blockIdx.y;
    uint32_t rowGrpIdx = blockIdx.x;

    float       vall      = cuda_cast<float>(src[(rowGrpIdx * groupSize + threadIdx.x) * numCols + colPckIdx * 2 + 0]);
    float       valh      = cuda_cast<float>(src[(rowGrpIdx * groupSize + threadIdx.x) * numCols + colPckIdx * 2 + 1]);
    const float groupMaxl = blockAllReduceMax(vall);
    const float groupMaxh = blockAllReduceMax(valh);

    if (threadIdx.x == 0) {
        scalePtr[rowGrpIdx * numCols + colPckIdx * 2 + 0]  = groupMaxl / 7.0f;
        scalePtr[rowGrpIdx * numCols + colPckIdx * 2 + 1]  = groupMaxh / 7.0f;
        zerosPtr[rowGrpIdx * numCols + colPckIdx * 2 + 0] = 0;
        zerosPtr[rowGrpIdx * numCols + colPckIdx * 2 + 1] = 0;
    }

    const float scaleOrigQuantl = 7.f / groupMaxl;
    const float scaleOrigQuanth = 7.f / groupMaxh;

    int8_t tmpi8l = cuda_cast<int8_t>(cuda_cast<float>(vall) * scaleOrigQuantl);
    int8_t tmpi8h = cuda_cast<int8_t>(cuda_cast<float>(valh) * scaleOrigQuanth);

    uint8_t tmpu4l = tmpi8l & 0x0F;
    uint8_t tmpu4h = tmpi8h & 0x0F;

    uint8_t tmpu8 = tmpu4h;
    tmpu8         = tmpu8 << 4;
    tmpu8         = tmpu8 | tmpu4l;

    weightPtr[(rowGrpIdx * groupSize + threadIdx.x) * numCols / 2 + colPckIdx] = tmpu8;
}

template<typename T>
void invokePerColQuantizationInt4x2(const T*      src,
                                    const int64_t numRows,
                                    const int64_t numCols,
                                    const int64_t groupSize,
                                    uint8_t*      weightPtr,
                                    half*         scalePtr,
                                    half*         zerosPtr,
                                    hipStream_t  stream) {
    assert(numRows % groupSize == 0);
    const dim3 block(groupSize);
    const dim3 grid(numRows / groupSize, numCols / 2, 1);
    perColQuantization<T><<<grid, block, 0, stream>>>(src, numRows, numCols, groupSize, weightPtr, scalePtr, zerosPtr);
}

#define INSTANTIATE_INVOKE_PER_COL_QUANTIZATION_INT4X2(T)                                                              \
    template void invokePerColQuantizationInt4x2(const T*      src,                                                    \
                                                 const int64_t numRows,                                                \
                                                 const int64_t numCols,                                                \
                                                 const int64_t groupSize,                                              \
                                                 uint8_t*      weightPtr,                                              \
                                                 half*         scalePtr,                                               \
                                                 half*         zerosPtr,                                               \
                                                 hipStream_t  stream)
INSTANTIATE_INVOKE_PER_COL_QUANTIZATION_INT4X2(float);
INSTANTIATE_INVOKE_PER_COL_QUANTIZATION_INT4X2(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_PER_COL_QUANTIZATION_INT4X2(__hip_bfloat16);
#endif

/////////////////////////////////////////////////////////////////////////////////////////////////
// int4 col dequant /////////////////////////////////////////////////////////////////////////////
template<typename T>
__global__ void perColDequantization(T*            dst,
                                     const int64_t numRows,
                                     const int64_t numCols,
                                     const int64_t groupSize,
                                     const char4*  weightPtr,
                                     const half*   scalePtr,
                                     const half*   zerosPtr,
                                     float*        dbgfp  = nullptr,
                                     int*          dbgint = nullptr) {
    const uint8_t* pWeight   = (const uint8_t*)weightPtr;
    uint32_t       colPckIdx = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t       rowIdx    = blockIdx.y;
    uint32_t       rowGrpIdx = rowIdx / groupSize;
    if (colPckIdx >= numCols / 2)
        return;

    float scalel = cuda_cast<float>(scalePtr[rowGrpIdx * numCols + colPckIdx * 2 + 0]);
    float scaleh = cuda_cast<float>(scalePtr[rowGrpIdx * numCols + colPckIdx * 2 + 1]);
    float zerosl = cuda_cast<float>(zerosPtr[rowGrpIdx * numCols + colPckIdx * 2 + 0]);
    float zerosh = cuda_cast<float>(zerosPtr[rowGrpIdx * numCols + colPckIdx * 2 + 0]);

    uint8_t tmpu8 = pWeight[rowIdx * numCols / 2 + colPckIdx];

    uint8_t tmpu4l = tmpu8 & 0x0F;
    uint8_t tmpu4h = (tmpu8 >> 4) & 0x0F;

    if (tmpu4l & 0x08)
        tmpu4l |= 0xF0;
    if (tmpu4h & 0x08)
        tmpu4h |= 0xF0;
    int8_t tmpi4l = tmpu4l;
    int8_t tmpi4h = tmpu4h;

    float tmpfpl = cuda_cast<float>(tmpi4l);
    float tmpfph = cuda_cast<float>(tmpi4h);

    T vall = cuda_cast<T>(tmpfpl * scalel + zerosl);
    T valh = cuda_cast<T>(tmpfph * scaleh + zerosh);

    dst[rowIdx * numCols + colPckIdx * 2 + 0] = vall;
    dst[rowIdx * numCols + colPckIdx * 2 + 1] = valh;
}

template<typename T>
void invokePerColDequantizationInt4x2(T*            dst,
                                      const int64_t numRows,
                                      const int64_t numCols,
                                      const int64_t groupSize,
                                      const int8_t* weightPtr,
                                      half*         scalePtr,
                                      half*         zerosPtr,
                                      hipStream_t  stream) {
    assert(numRows % groupSize == 0);
    const dim3 block(numCols / 2 < 512 ? numCols / 2 : 512);
    const dim3 grid((numCols / 2 + block.x - 1) / block.x, numRows, 1);
    perColDequantization<T>
        <<<grid, block, 0, stream>>>(dst, numRows, numCols, groupSize, (char4*)weightPtr, scalePtr, zerosPtr);
}

#define INSTANTIATE_INVOKE_PER_COL_DEQUANTIZATION_INT4X2(T)                                                            \
    template void invokePerColDequantizationInt4x2(T*            dst,                                                  \
                                                   const int64_t numRows,                                              \
                                                   const int64_t numCols,                                              \
                                                   const int64_t groupSize,                                            \
                                                   const int8_t* weightPtr,                                            \
                                                   half*         scalePtr,                                             \
                                                   half*         zerosPtr,                                             \
                                                   hipStream_t  stream)
INSTANTIATE_INVOKE_PER_COL_DEQUANTIZATION_INT4X2(float);
INSTANTIATE_INVOKE_PER_COL_DEQUANTIZATION_INT4X2(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_PER_COL_DEQUANTIZATION_INT4X2(__hip_bfloat16);
#endif

/////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void quantizedKernel(char4* dst, const float4* src, const int64_t sizeDiv4, const float* scalePtr) {
    for (int64_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < sizeDiv4; idx += blockDim.x * gridDim.x) {
        const float  scale = __ldg(scalePtr);
        char4        tmp;
        const float4 floatTmp = __ldg(src + idx);
        tmp.x                 = cuda_cast<int8_t>(floatTmp.x * scale);
        tmp.y                 = cuda_cast<int8_t>(floatTmp.y * scale);
        tmp.z                 = cuda_cast<int8_t>(floatTmp.z * scale);
        tmp.w                 = cuda_cast<int8_t>(floatTmp.w * scale);
        dst[idx]              = tmp;
    }
}

__global__ void quantizedKernel(char4* dst, const half2* src, const int64_t sizeDiv4, const float* scalePtr) {
    for (int64_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < sizeDiv4; idx += blockDim.x * gridDim.x) {
        const float scale = __ldg(scalePtr);
        char4       tmp;
        int         srcId = idx << 1;

        const uint2 h2 = __ldg(reinterpret_cast<const uint2*>(src + srcId));

        const half2 half2Tmp  = reinterpret_cast<const half2&>(h2.x);
        const half2 half2Tmp2 = reinterpret_cast<const half2&>(h2.y);

        tmp.x    = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp.x) * scale);
        tmp.y    = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp.y) * scale);
        tmp.z    = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp2.x) * scale);
        tmp.w    = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp2.y) * scale);
        dst[idx] = tmp;
    }
}

template<typename T>
void invokeQuantization(
    int8_t* dst, const T* src, const int64_t size, const float* scalePtr, hipStream_t stream, int maxGridSize) {
    FT_CHECK_WITH_INFO(size % 4 == 0, "[ERROR][invokeQuantization] size should be a multiple of 4.\n");

    int numBlocks{static_cast<int>((size + 255) / 256)};
    if (maxGridSize == -1) {
        maxGridSize = numBlocks;
    }
    dim3 grid(std::min(numBlocks, maxGridSize));
    FT_CHECK_WITH_INFO(grid.x <= maxGridSize, "[ERROR][invokeQuantization] grid max size is exceeded\n");
    dim3 block(64);
    if (std::is_same_v<T, float>) {
        quantizedKernel<<<grid, block, 0, stream>>>((char4*)dst, (const float4*)src, size / 4, scalePtr);
    } else if (std::is_same_v<T, half>) {
        quantizedKernel<<<grid, block, 0, stream>>>((char4*)dst, (const half2*)src, size / 4, scalePtr);
    }
}

#define INSTANTIATE_INVOKE_QUANTIZATION(T)                                                                             \
    template void invokeQuantization(                                                                                  \
        int8_t* dst, const T* src, const int64_t size, const float* scalePtr, hipStream_t stream, int maxGridSize);

INSTANTIATE_INVOKE_QUANTIZATION(float);
INSTANTIATE_INVOKE_QUANTIZATION(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_QUANTIZATION(__hip_bfloat16);
#endif

template<typename T, bool IS_SMOOTHER, bool IS_SHIFT>
__global__ void perTokenQuantization(int8_t*       dst,
                                     const T*      src,
                                     const int64_t numRows,
                                     const int64_t numCols,
                                     float*        scalePtr,
                                     const float*  smoother,
                                     const float*  shift) {
    const T* srcRow = src + blockIdx.x * numCols;
    int8_t*  dstRow = dst + blockIdx.x * numCols;

    T localMax = 1e-6f;
    for (int i = threadIdx.x; i < numCols; i += blockDim.x) {
        T val = srcRow[i];
        if (IS_SMOOTHER) {
            val = cuda_cast<T>(val / cuda_cast<T>(smoother[i]));
        }
        if (IS_SHIFT) {
            val = cuda_cast<T>(val + cuda_cast<T>(shift[i]));
        }
        localMax = cuda_max(localMax, cuda_abs(val));
    }
    const float rowMax = blockAllReduceMax(cuda_cast<float>(localMax));

    if (threadIdx.x == 0) {
        scalePtr[blockIdx.x] = rowMax / 127.f;
    }

    const float scaleOrigQuant = 127.f / rowMax;
    for (int i = threadIdx.x; i < numCols; i += blockDim.x) {
        T val = srcRow[i];
        if (IS_SMOOTHER) {
            val = val / cuda_cast<T>(smoother[i]);
        }
        if (IS_SHIFT) {
            val = cuda_cast<T>(val + cuda_cast<T>(shift[i]));
        }
        dstRow[i] = cuda_cast<int8_t>(cuda_cast<float>(val) * scaleOrigQuant);
    }
}

template<typename T, bool IS_SMOOTHER>
void dispatch_per_token_quantization_shift(int8_t*       dst,
                                           const T*      src,
                                           const int64_t numRows,
                                           const int64_t numCols,
                                           float*        scalePtr,
                                           const float*  smoother,
                                           const float*  shift,
                                           hipStream_t  stream) {
    // each block is responsible for a single row
    const dim3 block(512);
    const dim3 grid(numRows);

    if (shift != nullptr) {
        perTokenQuantization<T, IS_SMOOTHER, true>
            <<<grid, block, 0, stream>>>(dst, src, numRows, numCols, scalePtr, smoother, shift);
    } else {
        perTokenQuantization<T, IS_SMOOTHER, false>
            <<<grid, block, 0, stream>>>(dst, src, numRows, numCols, scalePtr, smoother, nullptr);
    }
}

template<typename T>
void invokePerTokenQuantization(int8_t*       dst,
                                const T*      src,
                                const int64_t numRows,
                                const int64_t numCols,
                                float*        scalePtr,
                                const float*  smoother,
                                const float*  shift,
                                hipStream_t  stream) {
    if (smoother != nullptr) {
        dispatch_per_token_quantization_shift<T, true>(dst, src, numRows, numCols, scalePtr, smoother, shift, stream);
    } else {
        dispatch_per_token_quantization_shift<T, false>(dst, src, numRows, numCols, scalePtr, nullptr, shift, stream);
    }
}

#define INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(T)                                                                   \
    template void invokePerTokenQuantization(int8_t*       dst,                                                        \
                                             const T*      src,                                                        \
                                             const int64_t numRows,                                                    \
                                             const int64_t numCols,                                                    \
                                             float*        scalePtr,                                                   \
                                             const float*  smoother,                                                   \
                                             const float*  shift,                                                      \
                                             hipStream_t  stream)

INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(float);
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(__hip_bfloat16);
#endif

/////////////////////////////////////////////////////////////////////////////////////////////////
// int8 col quant ///////////////////////////////////////////////////////////////////////////////
template<typename T, bool IS_SMOOTHER, bool IS_SHIFT>
__global__ void perColQuantization(int8_t*       dst,
                                   const T*      src,
                                   const int64_t numRows,
                                   const int64_t numCols,
                                   half*         scalePtr,
                                   const float*  smoother,
                                   const float*  shift,
                                   float*        dbgfp  = nullptr,
                                   int*          dbgint = nullptr) {
    uint32_t colIdx = blockIdx.x;
    const T* srcCol = src + colIdx;
    int8_t*  dstCol = dst + colIdx;

    T localMax = 1e-6f;
    for (int rowIdx = threadIdx.x; rowIdx < numRows; rowIdx += blockDim.x) {
        T val = srcCol[rowIdx * numCols];
        if (IS_SMOOTHER) {
            val = cuda_cast<T>(val / cuda_cast<T>(smoother[rowIdx]));
        }
        if (IS_SHIFT) {
            val = cuda_cast<T>(val + cuda_cast<T>(shift[rowIdx]));
        }
        localMax = cuda_max(localMax, cuda_abs(val));
    }
    const float colMax = blockAllReduceMax(cuda_cast<float>(localMax));

    if (threadIdx.x == 0) {
        scalePtr[colIdx] = cuda_cast<half>(colMax / 128.f);
    }

    const float scaleOrigQuant = 128.f / colMax;
    for (int rowIdx = threadIdx.x; rowIdx < numRows; rowIdx += blockDim.x) {
        T val = srcCol[rowIdx * numCols];
        if (IS_SMOOTHER) {
            val = val / cuda_cast<T>(smoother[rowIdx]);
        }
        if (IS_SHIFT) {
            val = cuda_cast<T>(val + cuda_cast<T>(shift[rowIdx]));
        }
        dstCol[rowIdx * numCols] = cuda_cast<int8_t>(cuda_cast<float>(val) * scaleOrigQuant);
    }
}

template<typename T, bool IS_SMOOTHER>
void dispatch_per_col_quantization_shift(int8_t*       dst,
                                         const T*      src,
                                         const int64_t numRows,
                                         const int64_t numCols,
                                         half*         scalePtr,
                                         const float*  smoother,
                                         const float*  shift,
                                         hipStream_t  stream) {
    // each block is responsible for a single row
    const dim3 block(512);
    const dim3 grid(numCols);

    if (shift != nullptr) {
        perColQuantization<T, IS_SMOOTHER, true>
            <<<grid, block, 0, stream>>>(dst, src, numRows, numCols, scalePtr, smoother, shift);
    } else {
        perColQuantization<T, IS_SMOOTHER, false>
            <<<grid, block, 0, stream>>>(dst, src, numRows, numCols, scalePtr, smoother, nullptr);
    }
}

template<typename T>
void invokePerColQuantizationInt8(int8_t*       dst,
                                  const T*      src,
                                  const int64_t numRows,
                                  const int64_t numCols,
                                  half*         scalePtr,
                                  const float*  smoother,
                                  const float*  shift,
                                  hipStream_t  stream) {
    if (smoother != nullptr) {
        dispatch_per_col_quantization_shift<T, true>(dst, src, numRows, numCols, scalePtr, smoother, shift, stream);
    } else {
        dispatch_per_col_quantization_shift<T, false>(dst, src, numRows, numCols, scalePtr, nullptr, shift, stream);
    }
}

#define INSTANTIATE_INVOKE_PER_COL_QUANTIZATION_INT8(T)                                                                \
    template void invokePerColQuantizationInt8(int8_t*       dst,                                                      \
                                               const T*      src,                                                      \
                                               const int64_t numRows,                                                  \
                                               const int64_t numCols,                                                  \
                                               half*         scalePtr,                                                 \
                                               const float*  smoother,                                                 \
                                               const float*  shift,                                                    \
                                               hipStream_t  stream)

INSTANTIATE_INVOKE_PER_COL_QUANTIZATION_INT8(float);
INSTANTIATE_INVOKE_PER_COL_QUANTIZATION_INT8(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_PER_COL_QUANTIZATION_INT8(__hip_bfloat16);
#endif

/////////////////////////////////////////////////////////////////////////////////////////////////
// int8 col dequant /////////////////////////////////////////////////////////////////////////////
template<typename T, bool IS_SMOOTHER, bool IS_SHIFT>
__global__ void perColDequantization(T*            dst,
                                     const int8_t* src,
                                     const int64_t numRows,
                                     const int64_t numCols,
                                     const half*   scalePtr,
                                     const float*  smoother,
                                     const float*  shift,
                                     float*        dbgfp  = nullptr,
                                     int*          dbgint = nullptr) {
    uint32_t      colIdx = blockIdx.x;
    const int8_t* srcRow = src + colIdx;
    T*            dstRow = dst + colIdx;

    float scaleOrigQuant = cuda_cast<float>(scalePtr[colIdx]);
    if (IS_SMOOTHER) {
        scaleOrigQuant = scaleOrigQuant * smoother[colIdx];
    }
    if (IS_SHIFT) {
        scaleOrigQuant = scaleOrigQuant - shift[colIdx];
    }

    for (int rowIdx = threadIdx.x; rowIdx < numRows; rowIdx += blockDim.x) {
        uint8_t tmpi8 = srcRow[rowIdx * numCols];

        T val = cuda_cast<T>(cuda_cast<float>(tmpi8) * scaleOrigQuant);

        if (IS_SMOOTHER) {
            val = val * cuda_cast<T>(smoother[rowIdx]);
        }
        if (IS_SHIFT) {
            val = cuda_cast<T>(val - cuda_cast<T>(shift[rowIdx]));
        }

        dstRow[rowIdx * numCols] = val;
    }
}

template<typename T, bool IS_SMOOTHER>
void dispatch_per_col_dequantization_shift(T*            dst,
                                           const int8_t* src,
                                           const int64_t numRows,
                                           const int64_t numCols,
                                           half*         scalePtr,
                                           const float*  smoother,
                                           const float*  shift,
                                           hipStream_t  stream) {
    // each block is responsible for a single col
    const dim3 block(512);
    const dim3 grid(numCols);

    if (shift != nullptr) {
        perColDequantization<T, IS_SMOOTHER, true>
            <<<grid, block, 0, stream>>>(dst, src, numRows, numCols, scalePtr, smoother, shift);
    } else {
        perColDequantization<T, IS_SMOOTHER, false>
            <<<grid, block, 0, stream>>>(dst, src, numRows, numCols, scalePtr, smoother, nullptr);
    }
}

template<typename T>
void invokePerColDequantizationInt8(T*            dst,
                                    const int8_t* src,
                                    const int64_t numRows,
                                    const int64_t numCols,
                                    half*         scalePtr,
                                    const float*  smoother,
                                    const float*  shift,
                                    hipStream_t  stream) {
    if (smoother != nullptr) {
        dispatch_per_col_dequantization_shift<T, true>(dst, src, numRows, numCols, scalePtr, smoother, shift, stream);
    } else {
        dispatch_per_col_dequantization_shift<T, false>(dst, src, numRows, numCols, scalePtr, nullptr, shift, stream);
    }
}

#define INSTANTIATE_INVOKE_PER_COL_DEQUANTIZATION_INT8(T)                                                              \
    template void invokePerColDequantizationInt8(T*            dst,                                                    \
                                                 const int8_t* src,                                                    \
                                                 const int64_t numRows,                                                \
                                                 const int64_t numCols,                                                \
                                                 half*         scalePtr,                                               \
                                                 const float*  smoother,                                               \
                                                 const float*  shift,                                                  \
                                                 hipStream_t  stream)
INSTANTIATE_INVOKE_PER_COL_DEQUANTIZATION_INT8(float);
INSTANTIATE_INVOKE_PER_COL_DEQUANTIZATION_INT8(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_PER_COL_DEQUANTIZATION_INT8(__hip_bfloat16);
#endif

/////////////////////////////////////////////////////////////////////////////////////////////////
// int4 row dequant /////////////////////////////////////////////////////////////////////////////
template<typename T>
__global__ void perRowDequantization(T*            dst,
                                     const char4*  src,
                                     const int64_t numRows,
                                     const int64_t numCols,
                                     const half*   scalePtr,
                                     const half*   zerosPtr,
                                     const int64_t groupSize,
                                     float*        dbgfp  = nullptr,
                                     int*          dbgint = nullptr) {
    const uint8_t* pSrc      = (const uint8_t*)src;
    uint32_t       rowIdx    = blockIdx.y;
    uint32_t       colGrpIdx = blockIdx.x;
    uint32_t       colGrpNum = numCols / groupSize;

    float scale = cuda_cast<float>(scalePtr[rowIdx * colGrpNum + colGrpIdx]);
    float zeros = cuda_cast<float>(zerosPtr[rowIdx * colGrpNum + colGrpIdx]);
    // scale = 1.0f;
    // zeros = 0;

    uint8_t tmpu8 = pSrc[rowIdx * numCols / 2 + colGrpIdx * groupSize / 2 + threadIdx.x];

    uint8_t tmpu4l = tmpu8 & 0x0F;
    uint8_t tmpu4h = (tmpu8 >> 4) & 0x0F;

    if (tmpu4l & 0x08)
        tmpu4l |= 0xF0;
    if (tmpu4h & 0x08)
        tmpu4h |= 0xF0;
    int8_t tmpi4l = tmpu4l;
    int8_t tmpi4h = tmpu4h;

    float tmpfpl = cuda_cast<float>(tmpi4l);
    float tmpfph = cuda_cast<float>(tmpi4h);

    T vall = cuda_cast<T>(tmpfpl * scale);
    T valh = cuda_cast<T>(tmpfph * scale);

    dst[rowIdx * numCols + colGrpIdx * groupSize + threadIdx.x * 2 + 0] = vall;
    dst[rowIdx * numCols + colGrpIdx * groupSize + threadIdx.x * 2 + 1] = valh;
}

template<typename T>
void invokePerRowDequantizationInt4x2(T*            dst,
                                      const int8_t* src,
                                      const int64_t numRows,
                                      const int64_t numCols,
                                      half*         scalePtr,
                                      half*         zerosPtr,
                                      const int64_t groupSize,
                                      hipStream_t  stream) {
    const dim3 block(groupSize / 2);
    const dim3 grid(numCols / groupSize, numRows, 1);

    perRowDequantization<T>
        <<<grid, block, 0, stream>>>(dst, (char4*)src, numRows, numCols, scalePtr, zerosPtr, groupSize);
}

#define INSTANTIATE_INVOKE_PER_ROW_DEQUANTIZATION_INT4X2(T)                                                            \
    template void invokePerRowDequantizationInt4x2(T*            dst,                                                  \
                                                   const int8_t* src,                                                  \
                                                   const int64_t numRows,                                              \
                                                   const int64_t numCols,                                              \
                                                   half*         scalePtr,                                             \
                                                   half*         zerosPtr,                                             \
                                                   const int64_t groupSize,                                            \
                                                   hipStream_t  stream)
INSTANTIATE_INVOKE_PER_ROW_DEQUANTIZATION_INT4X2(float);
INSTANTIATE_INVOKE_PER_ROW_DEQUANTIZATION_INT4X2(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_PER_ROW_DEQUANTIZATION_INT4X2(__hip_bfloat16);
#endif
}  // namespace fastertransformer
