#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "src/fastertransformer/kernels/add_residual_kernels.h"
#include "src/fastertransformer/cuda/cuda_type_utils.cuh"
#include "src/fastertransformer/utils/assert_utils.h"

namespace fastertransformer {

template<typename T, int RESIDUAL_NUM, typename T2 = T>
__global__ void addBiasResidual(T*           output,
                                const T2*    input,
                                const T*     residual1,
                                const T*     residual2,
                                const T*     bias,
                                const float* scale_inter,
                                const float* scale_out,
                                const int    m,
                                const int    n)
{
    const int col_index = blockIdx.y * blockDim.x + threadIdx.x;
    if (col_index < n) {
        T bias_val = (bias == nullptr) ? (T)(0.0f) : bias[col_index];
        T in;
        if (std::is_same<T, T2>::value) {
            in = cuda_cast<T>(input[blockIdx.x * n + col_index]);  // cast required for compilation when T != T2
        }
        else {
            in = cuda_cast<float>(input[blockIdx.x * n + col_index]) * (*scale_inter) * (*scale_out);
        }

        if (RESIDUAL_NUM == 1) {
            output[blockIdx.x * n + col_index] = in + residual1[blockIdx.x * n + col_index] + bias_val;
        }
        else if (RESIDUAL_NUM == 2) {
            output[blockIdx.x * n + col_index] =
                in + residual1[blockIdx.x * n + col_index] + residual2[blockIdx.x * n + col_index] + bias_val;
        }
    }
}

template<typename T>
void invokeAddBiasResidual(T*           output,
                           const T*     input,
                           const T*     residual1,
                           const T*     residual2,
                           const T*     bias,
                           const float* scale_inter,
                           const float* scale_out,
                           const int    m,
                           const int    n,
                           hipStream_t stream)
{
    FT_CHECK_WITH_INFO(!((scale_inter == nullptr) ^ (scale_out == nullptr)),
                       "Cannot use `scale_inter` without `scale_out`");
    const bool should_scale_input = scale_inter != nullptr;
    int        blocks_per_row     = ceil(float(n) / 1024);
    dim3       grid(m, blocks_per_row);
    dim3       block(min(n, 1024));
    if (residual2 == nullptr) {
        if (should_scale_input) {
            addBiasResidual<T, 1><<<grid, block, 0, stream>>>(output,
                                                              reinterpret_cast<const int32_t*>(input),
                                                              residual1,
                                                              residual2,
                                                              bias,
                                                              scale_inter,
                                                              scale_out,
                                                              m,
                                                              n);
        }
        else {
            addBiasResidual<T, 1>
                <<<grid, block, 0, stream>>>(output, input, residual1, residual2, bias, nullptr, nullptr, m, n);
        }
    }
    else {
        if (should_scale_input) {
            addBiasResidual<T, 2><<<grid, block, 0, stream>>>(output,
                                                              reinterpret_cast<const int32_t*>(input),
                                                              residual1,
                                                              residual2,
                                                              bias,
                                                              scale_inter,
                                                              scale_out,
                                                              m,
                                                              n);
        }
        else {
            addBiasResidual<T, 2>
                <<<grid, block, 0, stream>>>(output, input, residual1, residual2, bias, nullptr, nullptr, m, n);
        }
    }
}

template<typename T>
__global__ void alphaAddBiasResidual(T* output, const T* input, const T* bias, const T alpha, const int m, const int n)
{
    const int col_index = blockIdx.y * blockDim.x + threadIdx.x;
    if (col_index < n) {
        T bias_val = (bias == nullptr) ? (T)(0.0f) : bias[col_index];
        output[blockIdx.x * n + col_index] =
            output[blockIdx.x * n + col_index] + input[blockIdx.x * n + col_index] * alpha + bias_val;
    }
}

template<typename T>
__global__ void alphaAddBiasResidual(T* output, const T* input, const T* residual, const T* bias, const T alpha, const int m, const int n)
{
    const int col_index = blockIdx.y * blockDim.x + threadIdx.x;
    if (col_index < n) {
        T bias_val = (bias == nullptr) ? (T)(0.0f) : bias[col_index];
        output[blockIdx.x * n + col_index] =
            residual[blockIdx.x * n + col_index] + input[blockIdx.x * n + col_index] * alpha + bias_val;
    }
}

template<typename T>
void invokeAlphaAddBiasResidual(
    T* output, const T* input, const T* residual, const T* bias, const T alpha, const int m, const int n, hipStream_t stream)
{
    int  blocks_per_row = ceil(float(n) / 1024);
    dim3 grid(m, blocks_per_row);
    dim3 block(min(n, 1024));
    if (residual) {
        alphaAddBiasResidual<<<grid, block, 0, stream>>>(output, input, residual, bias, alpha, m, n);
    } else {
        alphaAddBiasResidual<<<grid, block, 0, stream>>>(output, input, bias, alpha, m, n);
    }
}

template<typename T>
__global__ void addBiasAttentionFfnResidual(T*        block_output,
                                            const T*  ffn_output,
                                            const T*  attn_output,
                                            const T*  block_input,
                                            const T*  bias,
                                            const int m,
                                            const int n,
                                            const int block_input_tp_split)
{
    const int col_index = blockIdx.y * blockDim.x + threadIdx.x;
    if (col_index < n) {
        block_output[blockIdx.x * n + col_index] =
            ffn_output[blockIdx.x * n + col_index] + attn_output[blockIdx.x * n + col_index] + bias[col_index]
            + ((block_input != nullptr) ?
                   cuda_cast<T>((float)block_input[blockIdx.x * n + col_index] / (float)block_input_tp_split) :
                   static_cast<T>(0.0f));
    }
}

template<typename T>
__global__ void addBiasAttentionFfnResidual(T*        block_output,
                                            const T*  ffn_output,
                                            const T*  attn_output,
                                            const T*  bias,
                                            const int m,
                                            const int n,
                                            const int block_input_tp_split)
{
    const int col_index = blockIdx.y * blockDim.x + threadIdx.x;
    if (col_index < n) {
        const int global_index     = blockIdx.x * n + col_index;
        block_output[global_index] = add(cuda_cast<T>((float)block_output[global_index] / (float)block_input_tp_split),
                                         ffn_output[global_index],
                                         attn_output[global_index],
                                         bias[col_index]);
    }
}

template<typename T>
void invokeAddBiasAttentionFfnResidual(T*           block_output,
                                       const T*     ffn_output,
                                       const T*     attn_output,
                                       const T*     block_input,
                                       const T*     bias,
                                       const int    m,
                                       const int    n,
                                       const int    block_input_tp_split,
                                       hipStream_t stream)
{
    int  blocks_per_row = ceil(float(n) / 1024);
    dim3 grid(m, blocks_per_row);
    dim3 block(min(n, 1024));
    if (block_output == block_input) {
        addBiasAttentionFfnResidual<<<grid, block, 0, stream>>>(
            block_output, ffn_output, attn_output, bias, m, n, block_input_tp_split);
    }
    else {
        addBiasAttentionFfnResidual<<<grid, block, 0, stream>>>(
            block_output, ffn_output, attn_output, block_input, bias, m, n, block_input_tp_split);
    }
}

#define INSTANTIATE_INVOKE_ADD_BIAS_RESIDUAL(T)                                                                        \
    template void invokeAddBiasResidual(T*           output,                                                           \
                                        const T*     input,                                                            \
                                        const T*     residual1,                                                        \
                                        const T*     residual2,                                                        \
                                        const T*     bias,                                                             \
                                        const float* scale_inter,                                                      \
                                        const float* scale_out,                                                        \
                                        const int    m,                                                                \
                                        const int    n,                                                                \
                                        hipStream_t stream)
INSTANTIATE_INVOKE_ADD_BIAS_RESIDUAL(float);
INSTANTIATE_INVOKE_ADD_BIAS_RESIDUAL(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_ADD_BIAS_RESIDUAL(__hip_bfloat16);
#endif
#undef INSTANTIATE_INVOKE_ADD_BIAS_RESIDUAL

template void invokeAlphaAddBiasResidual(float*       output,
                                         const float* input,
                                         const float* residual,
                                         const float* bias,
                                         const float  alpha,
                                         const int    m,
                                         const int    n,
                                         hipStream_t stream);

template void invokeAlphaAddBiasResidual(half* output,
                                         const half* input,
                                         const half* residual,
                                         const half* bias,
                                         const half alpha,
                                         const int m,
                                         const int n,
                                         hipStream_t stream);

#ifdef ENABLE_BF16
template void invokeAlphaAddBiasResidual(__hip_bfloat16* output,
                                    const __hip_bfloat16* input,
                                    const __hip_bfloat16* residual,
                                    const __hip_bfloat16* bias,
                                    const __hip_bfloat16 alpha,
                                    const int m,
                                    const int n,
                                    hipStream_t stream);
#endif

template void invokeAddBiasAttentionFfnResidual(float*       block_output,
                                                const float* ffn_output,
                                                const float* attn_output,
                                                const float* input,
                                                const float* bias,
                                                const int    m,
                                                const int    n,
                                                const int    block_input_tp_split,
                                                hipStream_t stream);

template void invokeAddBiasAttentionFfnResidual(half*        block_output,
                                                const half*  ffn_output,
                                                const half*  attn_output,
                                                const half*  input,
                                                const half*  bias,
                                                const int    m,
                                                const int    n,
                                                const int    block_input_tp_split,
                                                hipStream_t stream);

#ifdef ENABLE_BF16
template void invokeAddBiasAttentionFfnResidual(__hip_bfloat16*       block_output,
                                                const __hip_bfloat16* ffn_output,
                                                const __hip_bfloat16* attn_output,
                                                const __hip_bfloat16* input,
                                                const __hip_bfloat16* bias,
                                                const int            m,
                                                const int            n,
                                                const int            block_input_tp_split,
                                                hipStream_t         stream);
#endif


/*******************  invokeAddBiasResidualCol32  ***********************/
// input1/input2/out matrix with layout of cublasLt CUBLASLT_ORDER_COL32 (m*n)
//(grid, block) must be (m, n/4)
// using char4
template<typename T>
__global__ void add_bias_input_COL32_int8I_DataTypeO(
    T* output, const int8_t* input1, const T* input2, const T* bias, int m, int n, const float* input1_deQFactor_ptr)
{
    const float input1_deQFactor = __ldg(input1_deQFactor_ptr);
    int         col_start        = threadIdx.x << 2;

    float  local_out[4];
    int    outIdx       = ((col_start & 0xffffffe0) * m + (blockIdx.x << 5) + (col_start & 31)) >> 2;
    char4* input1TmpPtr = (char4*)input1;
    char4  input1Tmp    = __ldg(input1TmpPtr + outIdx);

    int col_start_tmp = col_start;
    local_out[0] = static_cast<float>(input2[(outIdx << 2) + 0]) + static_cast<float>(input1Tmp.x) * input1_deQFactor
                   + static_cast<float>(__ldg(bias + col_start_tmp));
    col_start_tmp = col_start_tmp + 1;
    local_out[1]  = static_cast<float>(input2[(outIdx << 2) + 1]) + static_cast<float>(input1Tmp.y) * input1_deQFactor
                   + static_cast<float>(__ldg(bias + col_start_tmp));
    col_start_tmp = col_start_tmp + 1;
    local_out[2]  = static_cast<float>(input2[(outIdx << 2) + 2]) + static_cast<float>(input1Tmp.z) * input1_deQFactor
                   + static_cast<float>(__ldg(bias + col_start_tmp));
    col_start_tmp = col_start_tmp + 1;
    local_out[3]  = static_cast<float>(input2[(outIdx << 2) + 3]) + static_cast<float>(input1Tmp.w) * input1_deQFactor
                   + static_cast<float>(__ldg(bias + col_start_tmp));

    for (int i = 0; i < 4; i++) {
        output[(outIdx << 2) + i] = static_cast<T>(local_out[i]);
    }
}

template<>
__global__ void add_bias_input_COL32_int8I_DataTypeO(half4*        output,
                                                     const int8_t* input1,
                                                     const half4*  input2,
                                                     const half4*  bias,
                                                     int           m,
                                                     int           n,
                                                     const float*  input1_deQFactor_ptr)
{
    const float input1_deQFactor = __ldg(input1_deQFactor_ptr);
    int         col_start        = (blockIdx.x << 5) + (threadIdx.x << 2);
    int         row_start        = (blockIdx.y << 5) + (threadIdx.y);

    if (col_start < n && row_start < m) {
        half4  local_out;
        int    outIdx       = ((col_start & 0xffffffe0) * m + (row_start << 5) + (col_start & 31)) >> 2;
        char4* input1TmpPtr = (char4*)input1;
        char4  input1Tmp    = input1TmpPtr[outIdx];
        half4  input2Tmp    = input2[outIdx];
        half4  biasTmp      = bias[col_start >> 2];

        local_out.x = static_cast<half>((float)input1Tmp.x * input1_deQFactor + (float)biasTmp.x + (float)input2Tmp.x);
        local_out.y = static_cast<half>((float)input1Tmp.y * input1_deQFactor + (float)biasTmp.y + (float)input2Tmp.y);
        local_out.z = static_cast<half>((float)input1Tmp.z * input1_deQFactor + (float)biasTmp.z + (float)input2Tmp.z);
        local_out.w = static_cast<half>((float)input1Tmp.w * input1_deQFactor + (float)biasTmp.w + (float)input2Tmp.w);
        output[outIdx] = local_out;
    }
}

template<typename T>
void invokeAddBiasResidualCol32(T*            output,
                                const int8_t* input1,
                                const T*      input2,
                                const T*      bias,
                                int           m,
                                int           n,
                                hipStream_t  stream,
                                const float*  input1_deQFactor_ptr)
{
    dim3 grid((n + 31) / 32, (m + 31) / 32);
    dim3 block(8, 32);
    assert(block.x <= 1024);
    if (sizeof(T) == 2) {
        add_bias_input_COL32_int8I_DataTypeO<<<grid, block, 0, stream>>>(
            (half4*)output, input1, (const half4*)input2, (const half4*)bias, m, n, input1_deQFactor_ptr);
    }
    else {
        add_bias_input_COL32_int8I_DataTypeO<T>
            <<<grid, block, 0, stream>>>(output, input1, input2, bias, m, n, input1_deQFactor_ptr);
    }
}

template void invokeAddBiasResidualCol32(float*        output,
                                         const int8_t* input1,
                                         const float*  input2,
                                         const float*  bias,
                                         int           m,
                                         int           n,
                                         hipStream_t  stream,
                                         const float*  input1_deQFactor_ptr);

template void invokeAddBiasResidualCol32(half*         output,
                                         const int8_t* input1,
                                         const half*   input2,
                                         const half*   bias,
                                         int           m,
                                         int           n,
                                         hipStream_t  stream,
                                         const float*  input1_deQFactor_ptr);

/*******************  invokeAddBiasResidualCol32  ***********************/
// input1/input2/out matrix with layout of cublasLt CUBLASLT_ORDER_COL32 (m*n)
//(grid, block) must be (m, n/4)
// using char4
template<typename T>
__global__ void add_bias_input_COL32_int32I_DataTypeO(T*             output,
                                                      const int32_t* input1,
                                                      const T*       input2,
                                                      const T*       bias,
                                                      int            m,
                                                      int            n,
                                                      const float*   weight_amax,
                                                      const float*   input1_amax_ptr,
                                                      const int      scale_is_vector)
{
    int           col_start        = threadIdx.x << 2;
    const float4* weight_scale_ptr = (const float4*)weight_amax;
    const float4  weight_scale     = __ldg(weight_scale_ptr + threadIdx.x * scale_is_vector);
    const float   input1_deQ       = __ldg(input1_amax_ptr) / 127.0f;

    float local_out[4];
    int   outIdx       = ((col_start & 0xffffffe0) * m + (blockIdx.x << 5) + (col_start & 31)) >> 2;
    int4* input1TmpPtr = (int4*)input1;
    int4  input1Tmp    = input1TmpPtr[outIdx];

    int col_start_tmp = col_start;
    local_out[0]      = static_cast<float>(input2[(outIdx << 2) + 0])
                   + static_cast<float>(input1Tmp.x) * input1_deQ * weight_scale.x / 127.0f
                   + static_cast<float>(__ldg(bias + col_start_tmp));
    col_start_tmp = col_start_tmp + 1;
    local_out[1]  = static_cast<float>(input2[(outIdx << 2) + 1])
                   + static_cast<float>(input1Tmp.y) * input1_deQ * weight_scale.y / 127.0f
                   + static_cast<float>(__ldg(bias + col_start_tmp));
    col_start_tmp = col_start_tmp + 1;
    local_out[2]  = static_cast<float>(input2[(outIdx << 2) + 2])
                   + static_cast<float>(input1Tmp.z) * input1_deQ * weight_scale.z / 127.0f
                   + static_cast<float>(__ldg(bias + col_start_tmp));
    col_start_tmp = col_start_tmp + 1;
    local_out[3]  = static_cast<float>(input2[(outIdx << 2) + 3])
                   + static_cast<float>(input1Tmp.w) * input1_deQ * weight_scale.w / 127.0f
                   + static_cast<float>(__ldg(bias + col_start_tmp));

    for (int i = 0; i < 4; i++) {
        output[(outIdx << 2) + i] = static_cast<T>(local_out[i]);
    }
}

template<>
__global__ void add_bias_input_COL32_int32I_DataTypeO(half4*         output,
                                                      const int32_t* input1,
                                                      const half4*   input2,
                                                      const half4*   bias,
                                                      int            m,
                                                      int            n,
                                                      const float*   weight_amax,
                                                      const float*   input1_amax_ptr,
                                                      const int      scale_is_vector)
{
    int           col_start           = threadIdx.x << 2;
    const float4* weight_scale_ptr    = (const float4*)weight_amax;
    const float   weight_scale_single = __ldg(weight_amax);
    const float4  weight_scale =
        scale_is_vector == 1 ?
             __ldg(weight_scale_ptr + threadIdx.x * scale_is_vector) :
             make_float4(weight_scale_single, weight_scale_single, weight_scale_single, weight_scale_single);
    const float input1_deQ = __ldg(input1_amax_ptr) / 127.0f;

    float local_out[4];
    int   outIdx       = ((col_start & 0xffffffe0) * m + (blockIdx.x << 5) + (col_start & 31)) >> 2;
    int4* input1TmpPtr = (int4*)input1;
    int4  input1Tmp    = input1TmpPtr[outIdx];
    half4 input2Tmp    = input2[outIdx];
    half4 biasTmp      = bias[threadIdx.x];

    local_out[0] = static_cast<float>(input2Tmp.x)
                   + static_cast<float>(input1Tmp.x) * input1_deQ * weight_scale.x / 127.0f
                   + static_cast<float>(biasTmp.x);
    local_out[1] = static_cast<float>(input2Tmp.y)
                   + static_cast<float>(input1Tmp.y) * input1_deQ * weight_scale.y / 127.0f
                   + static_cast<float>(biasTmp.y);
    local_out[2] = static_cast<float>(input2Tmp.z)
                   + static_cast<float>(input1Tmp.z) * input1_deQ * weight_scale.z / 127.0f
                   + static_cast<float>(biasTmp.z);
    local_out[3] = static_cast<float>(input2Tmp.w)
                   + static_cast<float>(input1Tmp.w) * input1_deQ * weight_scale.w / 127.0f
                   + static_cast<float>(biasTmp.w);

    half4 outTmp;
    outTmp.x = static_cast<half>(local_out[0]);
    outTmp.y = static_cast<half>(local_out[1]);
    outTmp.z = static_cast<half>(local_out[2]);
    outTmp.w = static_cast<half>(local_out[3]);

    output[outIdx] = outTmp;
}

template<typename T>
void invokeAddBiasResidualCol32(T*             output,
                                const int32_t* input1,
                                const T*       input2,
                                const T*       bias,
                                int            m,
                                int            n,
                                hipStream_t   stream,
                                const float*   weight_amax,
                                const float*   input1_amax_ptr,
                                const int      scale_is_vector)
{
    dim3 grid(m);
    dim3 block(n / 4);
    assert(block.x <= 1024);
    if (sizeof(T) == 2) {
        add_bias_input_COL32_int32I_DataTypeO<<<grid, block, 0, stream>>>((half4*)output,
                                                                          input1,
                                                                          (const half4*)input2,
                                                                          (const half4*)bias,
                                                                          m,
                                                                          n,
                                                                          weight_amax,
                                                                          input1_amax_ptr,
                                                                          scale_is_vector);
    }
    else {
        add_bias_input_COL32_int32I_DataTypeO<T><<<grid, block, 0, stream>>>(
            output, input1, input2, bias, m, n, weight_amax, input1_amax_ptr, scale_is_vector);
    }
}

template void invokeAddBiasResidualCol32(float*       output,
                                         const int*   input1,
                                         const float* input2,
                                         const float* bias,
                                         int          m,
                                         int          n,
                                         hipStream_t stream,
                                         const float* weight_amax,
                                         const float* input1_amax_ptr,
                                         const int    scale_is_vector);

template void invokeAddBiasResidualCol32(half*        output,
                                         const int*   input1,
                                         const half*  input2,
                                         const half*  bias,
                                         int          m,
                                         int          n,
                                         hipStream_t stream,
                                         const float* weight_amax,
                                         const float* input1_amax_ptr,
                                         const int    scale_is_vector);

}  // namespace fastertransformer
