#include "hip/hip_runtime.h"
#include "src/fastertransformer/cuda/cuda_type_utils.cuh"
#include "src/fastertransformer/kernels/alpha_layernorm_kernels.h"
#include "src/fastertransformer/kernels/reduce_kernel_utils.cuh"

// wont't support new features
namespace fastertransformer{
template<typename T, int N>
__global__ void alphaAddBiasResidualPostLayerNorm(
    T* out, const T* input, const T* residual1, const T* bias, const T* gamma, const T* beta, T alpha, int m, int n) {
    __shared__ float s_mean;
    __shared__ float s_variance;
    float            mean     = 0.0f;
    float            variance = 0.0f;
    float            local_out_cache[N];

#pragma unroll N
    for (int idx = threadIdx.x, i = 0; idx < n && i < N; ++i) {
        float local_out =
            (float)(input[blockIdx.x * n + idx] + residual1[blockIdx.x * n + idx] * alpha + __ldg_func(&bias[idx]));
        mean += local_out;
        // save local_out to local_out_cache to save some recompute
        local_out_cache[i] = local_out;
        idx += blockDim.x;
    }

    mean = blockReduceSum<float>(mean);
    if (threadIdx.x == 0) {
        s_mean = mean / n;
    }
    __syncthreads();

#pragma unroll N
    for (int idx = threadIdx.x, i = 0; idx < n && i < N; ++i) {
        float local_out = local_out_cache[i];
        variance += (local_out - s_mean) * (local_out - s_mean);
        idx += blockDim.x;
    }
    variance = blockReduceSum<float>(variance);
    if (threadIdx.x == 0) {
        s_variance = variance / n + 1e-6f;
    }
    __syncthreads();

#pragma unroll N
    for (int idx = threadIdx.x, i = 0; idx < n && i < N; ++i) {
        float local_out           = local_out_cache[i];
        out[blockIdx.x * n + idx] = (T)(((local_out - s_mean) * rsqrtf(s_variance)) * (float)(__ldg_func(&gamma[idx]))
                                        + (float)(__ldg_func(&beta[idx])));
        idx += blockDim.x;
    }
}

template<typename T>
__global__ void generalAlphaAddBiasResidualPostLayerNorm(
    T* out, const T* input, const T* residual1, const T* bias, const T* gamma, const T* beta, T alpha, int m, int n) {
    __shared__ float s_mean;
    __shared__ float s_variance;
    float            mean     = 0.0f;
    float            variance = 0.0f;

    for (int idx = threadIdx.x; idx < n; idx += blockDim.x) {
        float local_out =
            (float)(input[blockIdx.x * n + idx] + residual1[blockIdx.x * n + idx] * alpha + __ldg_func(&bias[idx]));
        mean += local_out;
        // save local_out to out to save some recompute
        out[blockIdx.x * n + idx] = local_out;
    }

    mean = blockReduceSum<float>(mean);
    if (threadIdx.x == 0) {
        s_mean = mean / n;
    }
    __syncthreads();

    for (int idx = threadIdx.x; idx < n; idx += blockDim.x) {
        float local_out = out[blockIdx.x * n + idx];
        variance += (local_out - s_mean) * (local_out - s_mean);
    }
    variance = blockReduceSum<float>(variance);
    if (threadIdx.x == 0) {
        s_variance = variance / n + 1e-6f;
    }
    __syncthreads();

    for (int idx = threadIdx.x; idx < n; idx += blockDim.x) {
        float local_out           = out[blockIdx.x * n + idx];
        out[blockIdx.x * n + idx] = (T)(((local_out - s_mean) * rsqrtf(s_variance)) * (float)(__ldg_func(&gamma[idx]))
                                        + (float)(__ldg_func(&beta[idx])));
    }
}

template<>
__global__ void generalAlphaAddBiasResidualPostLayerNorm(half*       out,
                                                         const half* input,
                                                         const half* residual1,
                                                         const half* bias,
                                                         const half* gamma,
                                                         const half* beta,
                                                         half        alpha,
                                                         int         m,
                                                         int         n) {
    __shared__ float s_mean;
    __shared__ float s_variance;
    float            mean     = 0.0f;
    float            variance = 0.0f;

    half2        alpha2       = make_half2(alpha, alpha);
    half2*       out_ptr      = (half2*)out;
    const half2* input_ptr    = (const half2*)input;
    const half2* residual_ptr = (const half2*)residual1;
    const half2* bias_ptr     = (const half2*)bias;
    const half2* gamma_ptr    = (const half2*)gamma;
    const half2* beta_ptr     = (const half2*)beta;

    float local_out = 0.0f;
    for (int idx = threadIdx.x; idx < n / 2; idx += blockDim.x) {
        int    id  = blockIdx.x * n / 2 + idx;
        half2  tmp = __hadd2(__hadd2(input_ptr[id], __hmul2(residual_ptr[id], alpha2)), __ldg_func(&bias_ptr[idx]));
        float2 local_out_fp2 = __half22float2(tmp);
        local_out += local_out_fp2.x;
        local_out += local_out_fp2.y;
        // save tmp to out_ptr to save some recomputation
        out_ptr[id] = tmp;
    }

    mean = blockReduceSum<float>(local_out);
    if (threadIdx.x == 0) {
        s_mean = mean / n;
    }
    __syncthreads();

    for (int idx = threadIdx.x; idx < n / 2; idx += blockDim.x) {
        int    id            = blockIdx.x * n / 2 + idx;
        float2 local_out_fp2 = __half22float2(out_ptr[id]);
        variance += (local_out_fp2.x - s_mean) * (local_out_fp2.x - s_mean);
        variance += (local_out_fp2.y - s_mean) * (local_out_fp2.y - s_mean);
    }

    variance = blockReduceSum<float>(variance);
    if (threadIdx.x == 0) {
        s_variance = rsqrtf(variance / n + 1e-6f);
    }
    __syncthreads();

    for (int idx = threadIdx.x; idx < n / 2; idx += blockDim.x) {
        int    id            = blockIdx.x * n / 2 + idx;
        float2 local_out_fp2 = __half22float2(out_ptr[id]);
        float2 gamma_val     = __half22float2(__ldg_func(&gamma_ptr[idx]));
        float2 beta_val      = __half22float2(__ldg_func(&beta_ptr[idx]));
        local_out_fp2.x      = (local_out_fp2.x - s_mean) * s_variance * gamma_val.x + beta_val.x;
        local_out_fp2.y      = (local_out_fp2.y - s_mean) * s_variance * gamma_val.y + beta_val.y;
        out_ptr[id]          = __float22half2_rn(local_out_fp2);
    }
}

template<typename T>
__global__ void alphaAddBiasResidualPostLayerNormV2(T* out,
                                                    const T* __restrict input,
                                                    const T* __restrict residual1,
                                                    const T* __restrict bias,
                                                    const T* __restrict gamma,
                                                    const T* __restrict beta,
                                                    T   alpha,
                                                    int n) {
    const int ite = 4;
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    __shared__ float s_mean;
    __shared__ float s_variance;
    float            mean     = 0.0f;
    float            variance = 0.0f;
    float            local_out[ite];

    float sum = 0.0f;
#pragma unroll
    for (int i = 0; i < ite; i++) {
        int col_id   = i * blockDim.x + tid;
        int id       = bid * n + col_id;
        local_out[i] = (float)(input[id] + __ldg_func(&residual1[id]) * alpha + __ldg_func(&bias[col_id]));
        sum += local_out[i];
    }

    mean = blockReduceSum<float>(sum);
    if (tid == 0) {
        s_mean = mean / n;
    }
    __syncthreads();

    float var = 0.0f;
#pragma unroll
    for (int i = 0; i < ite; i++) {
        float diff = local_out[i] - s_mean;
        var += diff * diff;
    }

    variance = blockReduceSum<float>(var);
    if (tid == 0) {
        s_variance = rsqrtf(variance / n + 1e-6f);
    }
    __syncthreads();

#pragma unroll
    for (int i = 0; i < ite; i++) {
        int col_id = i * blockDim.x + tid;
        int id     = bid * n + col_id;
        out[id]    = (T)((local_out[i] - s_mean) * s_variance * (float)__ldg_func(&gamma[col_id])
                      + (float)__ldg_func(&beta[col_id]));
    }
}

template<>
__global__ void alphaAddBiasResidualPostLayerNormV2(half* out,
                                                    const half* __restrict input,
                                                    const half* __restrict residual1,
                                                    const half* __restrict bias,
                                                    const half* __restrict gamma,
                                                    const half* __restrict beta,
                                                    half alpha,
                                                    int  n) {
    const int        ite = 4;
    const int        tid = threadIdx.x;
    const int        bid = blockIdx.x;
    __shared__ float s_mean;
    __shared__ float s_variance;
    float            mean     = 0.0f;
    float            variance = 0.0f;
    half2            local_out_half2[ite];

    half2        alpha2       = make_half2(alpha, alpha);
    half2*       out_ptr      = (half2*)out;
    const half2* input_ptr    = (const half2*)input;
    const half2* residual_ptr = (const half2*)residual1;
    const half2* bias_ptr     = (const half2*)bias;
    const half2* gamma_ptr    = (const half2*)gamma;
    const half2* beta_ptr     = (const half2*)beta;

    // float sum = 0.0f;
    half2 sum = __float2half2_rn(0.0f);
#pragma unroll
    for (int i = 0; i < ite; i++) {
        int col_id         = i * blockDim.x + tid;
        int id             = bid * n / 2 + col_id;
        local_out_half2[i] = input_ptr[id] + __ldg_func(&residual_ptr[id]) * alpha2 + __ldg_func(&bias_ptr[col_id]);
        sum += local_out_half2[i];
    }

    mean = blockReduceSum<float>((float)(sum.x + sum.y));
    if (threadIdx.x == 0) {
        s_mean = mean / n;
    }
    __syncthreads();

    float var      = 0.0f;
    half2 s_mean_2 = __float2half2_rn(s_mean);
#pragma unroll
    for (int i = 0; i < ite; i++) {
        local_out_half2[i] = local_out_half2[i] - s_mean_2;
        float v1           = (float)local_out_half2[i].x;
        float v2           = (float)local_out_half2[i].y;
        var += v1 * v1 + v2 * v2;
    }

    variance = blockReduceSum<float>(var);
    if (threadIdx.x == 0) {
        s_variance = rsqrtf(variance / n + 1e-6f);
    }
    __syncthreads();

    half2 s_var_2 = __float2half2_rn(s_variance);
#pragma unroll
    for (int i = 0; i < ite; i++) {
        int col_id  = i * blockDim.x + tid;
        int id      = bid * n / 2 + col_id;
        out_ptr[id] = local_out_half2[i] * s_var_2 * __ldg_func(&gamma_ptr[col_id]) + __ldg_func(&beta_ptr[col_id]);
    }
}

template<typename T>
void invokeAlphaAddBiasResidualLayerNorm(T*           out,
                                         const T*     input,
                                         const T*     residual1,
                                         const T*     bias,
                                         const T*     gamma,
                                         const T*     beta,
                                         T            alpha,
                                         int          m,
                                         int          n,
                                         hipStream_t stream) {
    dim3 grid(m);
    dim3 block(std::min(n, 1024));
    if (n == 768 || n == 1024) {
        alphaAddBiasResidualPostLayerNormV2<T>
            <<<grid, n / 4, 0, stream>>>(out, input, residual1, bias, gamma, beta, alpha, n);
    } else {
        block.x       = std::min(n, 1024);
        int num_trips = (n + block.x - 1) / block.x;
        if (num_trips == 1) {
            alphaAddBiasResidualPostLayerNorm<T, 1>
                <<<grid, block, 0, stream>>>(out, input, residual1, bias, gamma, beta, alpha, m, n);
        } else if (num_trips == 2) {
            alphaAddBiasResidualPostLayerNorm<T, 2>
                <<<grid, block, 0, stream>>>(out, input, residual1, bias, gamma, beta, alpha, m, n);
        } else {
            generalAlphaAddBiasResidualPostLayerNorm<T>
                <<<grid, block, 0, stream>>>(out, input, residual1, bias, gamma, beta, alpha, m, n);
        }
    }
}

template<>
void invokeAlphaAddBiasResidualLayerNorm(half*        out,
                                         const half*  input,
                                         const half*  residual1,
                                         const half*  bias,
                                         const half*  gamma,
                                         const half*  beta,
                                         half         alpha,
                                         int          m,
                                         int          n,
                                         hipStream_t stream) {
    dim3 grid(m);
    dim3 block(std::min(n, 1024));

    if (m >= 512 && (n == 768 || n == 1024)) {
        alphaAddBiasResidualPostLayerNormV2<half>
            <<<grid, n / 8, 0, stream>>>(out, input, residual1, bias, gamma, beta, alpha, n);
    } else {
        block.x       = std::min(n, 1024);
        int num_trips = (n + block.x - 1) / block.x;
        if (num_trips == 1) {
            alphaAddBiasResidualPostLayerNorm<half, 1>
                <<<grid, block, 0, stream>>>(out, input, residual1, bias, gamma, beta, alpha, m, n);
        } else if (num_trips == 2) {
            alphaAddBiasResidualPostLayerNorm<half, 2>
                <<<grid, block, 0, stream>>>(out, input, residual1, bias, gamma, beta, alpha, m, n);
        } else {
            generalAlphaAddBiasResidualPostLayerNorm<half>
                <<<grid, block, 0, stream>>>(out, input, residual1, bias, gamma, beta, alpha, m, n);
        }
    }
}

#define INVOKE_ALPHA_ADD_BIAS_RES_LN(T)                                                                                \
    template void invokeAlphaAddBiasResidualLayerNorm(T*           out,                                                \
                                                      const T*     input,                                              \
                                                      const T*     residual1,                                          \
                                                      const T*     bias,                                               \
                                                      const T*     gamma,                                              \
                                                      const T*     beta,                                               \
                                                      T            alpha,                                              \
                                                      int          m,                                                  \
                                                      int          n,                                                  \
                                                      hipStream_t stream);
INVOKE_ALPHA_ADD_BIAS_RES_LN(float)
INVOKE_ALPHA_ADD_BIAS_RES_LN(half)
#ifdef ENABLE_BF16
INVOKE_ALPHA_ADD_BIAS_RES_LN(__hip_bfloat16)
#endif

template<typename T, int N>
__global__ void addBiasResidualPostLayerNorm(
    T* out, const T* input, const T* bias, const T* gamma, const T* beta, const float layernorm_eps, int m, int n) {
    __shared__ float s_mean;
    __shared__ float s_variance;
    float            mean     = 0.0f;
    float            variance = 0.0f;
    float            local_out_cache[N];

#pragma unroll N
    for (int idx = threadIdx.x, i = 0; idx < n && i < N; ++i) {
        float local_out = (float)(add(out[blockIdx.x * n + idx], input[blockIdx.x * n + idx], ldg(&bias[idx])));
        mean += local_out;
        // save local_out to local_out_cache to save some recompute
        local_out_cache[i] = local_out;
        idx += blockDim.x;
    }

    mean = blockReduceSum<float>(mean);
    if (threadIdx.x == 0) {
        s_mean = mean / n;
    }
    __syncthreads();

#pragma unroll N
    for (int idx = threadIdx.x, i = 0; idx < n && i < N; ++i) {
        float local_out = local_out_cache[i];
        variance += (local_out - s_mean) * (local_out - s_mean);
        idx += blockDim.x;
    }
    variance = blockReduceSum<float>(variance);
    if (threadIdx.x == 0) {
        s_variance = variance / n + layernorm_eps;
    }
    __syncthreads();

#pragma unroll N
    for (int idx = threadIdx.x, i = 0; idx < n && i < N; ++i) {
        float local_out = local_out_cache[i];
        out[blockIdx.x * n + idx] =
            (T)(((local_out - s_mean) * rsqrtf(s_variance)) * (float)(ldg(&gamma[idx])) + (float)(ldg(&beta[idx])));
        idx += blockDim.x;
    }
}

template<typename T>
__global__ void generalAddBiasResidualPostLayerNorm(
    T* out, const T* input, const T* bias, const T* gamma, const T* beta, const float layernorm_eps, int m, int n) {
    using T2 = typename TypeConverter<T>::Type;
    __shared__ float s_mean;
    __shared__ float s_variance;
    float            mean     = 0.0f;
    float            variance = 0.0f;

    T2*       out_ptr   = (T2*)out;
    const T2* input_ptr = (const T2*)input;
    const T2* bias_ptr  = (const T2*)bias;
    const T2* gamma_ptr = (const T2*)gamma;
    const T2* beta_ptr  = (const T2*)beta;

    float local_out = 0.0f;
    for (int idx = threadIdx.x; idx < n / 2; idx += blockDim.x) {
        int    id            = blockIdx.x * n / 2 + idx;
        T2     tmp           = hadd2(hadd2(out_ptr[id], input_ptr[id]), ldg(&bias_ptr[idx]));
        float2 local_out_fp2 = cuda_cast<float2>(tmp);
        local_out += local_out_fp2.x;
        local_out += local_out_fp2.y;
        // save tmp to out_ptr to save some recomputation
        out_ptr[id] = tmp;
    }

    mean = blockReduceSum<float>(local_out);
    if (threadIdx.x == 0) {
        s_mean = mean / n;
    }
    __syncthreads();

    for (int idx = threadIdx.x; idx < n / 2; idx += blockDim.x) {
        int    id            = blockIdx.x * n / 2 + idx;
        float2 local_out_fp2 = cuda_cast<float2>(out_ptr[id]);
        variance += (local_out_fp2.x - s_mean) * (local_out_fp2.x - s_mean);
        variance += (local_out_fp2.y - s_mean) * (local_out_fp2.y - s_mean);
    }

    variance = blockReduceSum<float>(variance);
    if (threadIdx.x == 0) {
        s_variance = rsqrtf(variance / n + layernorm_eps);
    }
    __syncthreads();

    for (int idx = threadIdx.x; idx < n / 2; idx += blockDim.x) {
        int    id            = blockIdx.x * n / 2 + idx;
        float2 local_out_fp2 = cuda_cast<float2>(out_ptr[id]);
        float2 gamma_val     = cuda_cast<float2>(ldg(&gamma_ptr[idx]));
        float2 beta_val      = cuda_cast<float2>(ldg(&beta_ptr[idx]));
        local_out_fp2.x      = (local_out_fp2.x - s_mean) * s_variance * gamma_val.x + beta_val.x;
        local_out_fp2.y      = (local_out_fp2.y - s_mean) * s_variance * gamma_val.y + beta_val.y;
        out_ptr[id]          = cuda_cast<T2>(local_out_fp2);
    }
}

template<>
__global__ void generalAddBiasResidualPostLayerNorm(float*       out,
                                                    const float* input,
                                                    const float* bias,
                                                    const float* gamma,
                                                    const float* beta,
                                                    const float  layernorm_eps,
                                                    int          m,
                                                    int          n) {
    __shared__ float s_mean;
    __shared__ float s_variance;
    float            mean     = 0.0f;
    float            variance = 0.0f;

    for (int idx = threadIdx.x; idx < n; idx += blockDim.x) {
        float local_out = (float)(out[blockIdx.x * n + idx] + input[blockIdx.x * n + idx] + __ldg_func(&bias[idx]));
        mean += local_out;
        // save local_out to out to save some recompute
        out[blockIdx.x * n + idx] = local_out;
    }

    mean = blockReduceSum<float>(mean);
    if (threadIdx.x == 0) {
        s_mean = mean / n;
    }
    __syncthreads();

    for (int idx = threadIdx.x; idx < n; idx += blockDim.x) {
        float local_out = out[blockIdx.x * n + idx];
        variance += (local_out - s_mean) * (local_out - s_mean);
    }
    variance = blockReduceSum<float>(variance);
    if (threadIdx.x == 0) {
        s_variance = rsqrtf(variance / n + layernorm_eps);
    }
    __syncthreads();

    for (int idx = threadIdx.x; idx < n; idx += blockDim.x) {
        float local_out           = out[blockIdx.x * n + idx];
        out[blockIdx.x * n + idx] = (float)(((local_out - s_mean) * s_variance) * (float)(__ldg_func(&gamma[idx]))
                                            + (float)(__ldg_func(&beta[idx])));
    }
}

template<typename T>
__global__ void addBiasResidualPostLayerNormV2(T* out,
                                               const T* __restrict input,
                                               const T* __restrict bias,
                                               const T* __restrict gamma,
                                               const T* __restrict beta,
                                               const float layernorm_eps,
                                               int         n) {
    using T2             = typename TypeConverter<T>::Type;
    const int        ite = 4;
    const int        tid = threadIdx.x;
    const int        bid = blockIdx.x;
    __shared__ float s_mean;
    __shared__ float s_variance;
    float            mean     = 0.0f;
    float            variance = 0.0f;
    T2               local_out_half2[ite];

    T2*       out_ptr   = (T2*)out;
    const T2* input_ptr = (const T2*)input;
    const T2* bias_ptr  = (const T2*)bias;
    const T2* gamma_ptr = (const T2*)gamma;
    const T2* beta_ptr  = (const T2*)beta;

    // float sum = 0.0f;
    T2 sum = cuda_cast<T2>(0.0f);
#pragma unroll
    for (int i = 0; i < ite; i++) {
        int col_id         = i * blockDim.x + tid;
        int id             = bid * n / 2 + col_id;
        local_out_half2[i] = add(out_ptr[id], ldg(&input_ptr[id]), ldg(&bias_ptr[col_id]));
        sum                = add(sum, local_out_half2[i]);
    }

    mean = blockReduceSum<float>((float)(sum.x + sum.y));
    if (threadIdx.x == 0) {
        s_mean = mean / n;
    }
    __syncthreads();

    float var      = 0.0f;
    T2    s_mean_2 = cuda_cast<T2>(s_mean);
#pragma unroll
    for (int i = 0; i < ite; i++) {
        local_out_half2[i] = hsub2(local_out_half2[i], s_mean_2);
        float v1           = (float)local_out_half2[i].x;
        float v2           = (float)local_out_half2[i].y;
        var += v1 * v1 + v2 * v2;
    }

    variance = blockReduceSum<float>(var);
    if (tid == 0) {
        s_variance = rsqrtf(variance / n + layernorm_eps);
    }
    __syncthreads();

    T2 s_var_2 = cuda_cast<T2>(s_variance);
#pragma unroll
    for (int i = 0; i < ite; i++) {
        int col_id  = i * blockDim.x + tid;
        int id      = bid * n / 2 + col_id;
        out_ptr[id] = fma(local_out_half2[i], s_var_2, ldg(&gamma_ptr[col_id]), ldg(&beta_ptr[col_id]));
    }
}

template<>
__global__ void addBiasResidualPostLayerNormV2(float* out,
                                               const float* __restrict input,
                                               const float* __restrict bias,
                                               const float* __restrict gamma,
                                               const float* __restrict beta,
                                               const float layernorm_eps,
                                               int         n) {
    const int ite = 4;
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    __shared__ float s_mean;
    __shared__ float s_variance;
    float            mean     = 0.0f;
    float            variance = 0.0f;
    float            local_out[ite];

    float sum = 0.0f;
#pragma unroll
    for (int i = 0; i < ite; i++) {
        int col_id   = i * blockDim.x + tid;
        int id       = bid * n + col_id;
        local_out[i] = (float)(out[id] + __ldg_func(&input[id]) + __ldg_func(&bias[col_id]));
        sum += local_out[i];
    }

    mean = blockReduceSum<float>(sum);
    if (tid == 0) {
        s_mean = mean / n;
    }
    __syncthreads();

    float var = 0.0f;
#pragma unroll
    for (int i = 0; i < ite; i++) {
        float diff = local_out[i] - s_mean;
        var += diff * diff;
    }

    variance = blockReduceSum<float>(var);
    if (tid == 0) {
        s_variance = rsqrtf(variance / n + layernorm_eps);
    }
    __syncthreads();

#pragma unroll
    for (int i = 0; i < ite; i++) {
        int col_id = i * blockDim.x + tid;
        int id     = bid * n + col_id;
        out[id]    = (float)((local_out[i] - s_mean) * s_variance * (float)__ldg_func(&gamma[col_id])
                          + (float)__ldg_func(&beta[col_id]));
    }
}

template<typename T>
void invokeAddBiasResidualLayerNorm(T*           out,
                                    const T*     input,
                                    const T*     bias,
                                    const T*     gamma,
                                    const T*     beta,
                                    const float  layernorm_eps,
                                    int          m,
                                    int          n,
                                    hipStream_t stream) {
    dim3 grid(m);
    dim3 block(std::min(n, 1024));

    if (m >= 512 && (n == 768 || n == 1024)) {
        addBiasResidualPostLayerNormV2<T><<<grid, n / 8, 0, stream>>>(out, input, bias, gamma, beta, layernorm_eps, n);
    } else {
        block.x       = std::min(n, 1024);
        int num_trips = (n + block.x - 1) / block.x;
        if (num_trips == 1) {
            addBiasResidualPostLayerNorm<T, 1>
                <<<grid, block, 0, stream>>>(out, input, bias, gamma, beta, layernorm_eps, m, n);
        } else if (num_trips == 2) {
            addBiasResidualPostLayerNorm<T, 2>
                <<<grid, block, 0, stream>>>(out, input, bias, gamma, beta, layernorm_eps, m, n);
        } else {
            generalAddBiasResidualPostLayerNorm<T>
                <<<grid, block, 0, stream>>>(out, input, bias, gamma, beta, layernorm_eps, m, n);
        }
    }
}

template<>
void invokeAddBiasResidualLayerNorm(float*       out,
                                    const float* input,
                                    const float* bias,
                                    const float* gamma,
                                    const float* beta,
                                    const float  layernorm_eps,
                                    int          m,
                                    int          n,
                                    hipStream_t stream) {
    dim3 grid(m);
    dim3 block(std::min(n, 1024));
    if (n == 768 || n == 1024) {
        addBiasResidualPostLayerNormV2<float>
            <<<grid, n / 4, 0, stream>>>(out, input, bias, gamma, beta, layernorm_eps, n);
    } else {
        block.x       = std::min(n, 1024);
        int num_trips = (n + block.x - 1) / block.x;
        if (num_trips == 1) {
            addBiasResidualPostLayerNorm<float, 1>
                <<<grid, block, 0, stream>>>(out, input, bias, gamma, beta, layernorm_eps, m, n);
        } else if (num_trips == 2) {
            addBiasResidualPostLayerNorm<float, 2>
                <<<grid, block, 0, stream>>>(out, input, bias, gamma, beta, layernorm_eps, m, n);
        } else {
            generalAddBiasResidualPostLayerNorm<float>
                <<<grid, block, 0, stream>>>(out, input, bias, gamma, beta, layernorm_eps, m, n);
        }
    }
}

#define INVOKE_ADD_BIAS_RES_LN(T)                                                                                      \
    template void invokeAddBiasResidualLayerNorm(T*           out,                                                     \
                                                 const T*     input,                                                   \
                                                 const T*     bias,                                                    \
                                                 const T*     gamma,                                                   \
                                                 const T*     beta,                                                    \
                                                 const float  layernorm_eps,                                           \
                                                 int          m,                                                       \
                                                 int          n,                                                       \
                                                 hipStream_t stream);
INVOKE_ADD_BIAS_RES_LN(float)
INVOKE_ADD_BIAS_RES_LN(half)
#ifdef ENABLE_BF16
INVOKE_ADD_BIAS_RES_LN(__hip_bfloat16)
#endif

template<typename T, bool DYNAMIC_SCALING = false>
__global__ void generalLayerNormWithPadding(const T* __restrict input,
                                            const T* __restrict gamma,
                                            const T* __restrict beta,
                                            T*          normed_output,
                                            const float layernorm_eps,
                                            int         m,
                                            int         real_n,
                                            int         padding_n,
                                            float*      scale,
                                            float*      dynamic_scale,
                                            const int   int8_mode) {
    const int tid = threadIdx.x;

    extern __shared__ __align__(sizeof(float)) char _shmem[];
    T*                                              shmem = reinterpret_cast<T*>(_shmem);

    __shared__ float s_mean;
    __shared__ float s_variance;
    float            mean     = 0.0f;
    float            variance = 0.0f;

    using Int8_Packed_T  = typename packed_as<int8_t, num_elems<T>::value>::type;
    using Int32_Packed_T = typename packed_as<int32_t, num_elems<T>::value>::type;
    using Float_Packed_T = typename packed_as<float, num_elems<T>::value>::type;
    using Scalar_T       = typename packed_as<T, 1>::type;

    const Float_Packed_T scale_to_int = cuda_cast<Float_Packed_T>(int8_mode == 2 ? *scale : 0.0f);

    float local_sum = 0.0f;
    for (int i = tid; i < real_n; i += blockDim.x) {
        local_sum += (float)(ldg(&input[blockIdx.x * padding_n + i]));
    }

    mean = blockReduceSum(local_sum);

    if (threadIdx.x == 0) {
        s_mean = mean / real_n;
    }
    __syncthreads();

    float local_var_sum = 0.0f;
    for (int i = tid; i < real_n; i += blockDim.x) {
        float diff = (float)(ldg(&input[blockIdx.x * padding_n + i])) - s_mean;
        local_var_sum += diff * diff;
    }
    variance = blockReduceSum(local_var_sum);

    if (threadIdx.x == 0) {
        s_variance = rsqrtf(variance / real_n + layernorm_eps);
    }
    __syncthreads();

    Scalar_T abs_max(1e-6f);

    for (int i = tid; i < real_n; i += blockDim.x) {
        const int index    = blockIdx.x * padding_n + i;
        float     beta_val = (beta == nullptr) ? 0.0f : (float)ldg(&beta[i]);
        T         val      = (T)((((float)input[index] - s_mean) * s_variance) * (float)(ldg(&gamma[i])) + beta_val);

        if (DYNAMIC_SCALING) {
            abs_max  = cuda_max(cuda_max<Scalar_T, T>(cuda_abs(val)), abs_max);
            shmem[i] = val;
        } else if (int8_mode == 2) {
            reinterpret_cast<Int8_Packed_T*>(normed_output)[index] =
                cuda_cast<Int8_Packed_T>(cuda_cast<Float_Packed_T>(val) * scale_to_int);
        } else {
            normed_output[index] = val;
        }
    }

    if (DYNAMIC_SCALING) {
        float          abs_max_f               = blockAllReduceMax(cuda_cast<float>(abs_max));
        const Scalar_T dynamic_per_token_scale(127. / abs_max_f);
        for (int i = tid; i < real_n; i += blockDim.x) {
            const int index                                        = blockIdx.x * padding_n + i;
            reinterpret_cast<Int8_Packed_T*>(normed_output)[index] = cuda_cast<Int8_Packed_T>(
                cuda_cast<Float_Packed_T>(shmem[i]) * cuda_cast<Float_Packed_T>(dynamic_per_token_scale));
        }
        if (threadIdx.x == 0) {
            dynamic_scale[blockIdx.x] = (*scale * abs_max_f) / 127.f;
        }
    }
}

template<typename T>
void invokeGeneralLayerNormWithPadding(T*           out,
                                       const T*     input,
                                       const T*     gamma,
                                       const T*     beta,
                                       const float  layernorm_eps,
                                       const int    m,
                                       const int    real_n,
                                       const int    padding_n,
                                       float*       scale,
                                       float*       dynamic_scale,
                                       const int    int8_mode,
                                       hipStream_t stream,
                                       int          opt_version) {
    dim3       grid(m);
    const bool dynamic_quant = dynamic_scale != nullptr;

    dim3 block(min(real_n, 1024));

    /* For general cases, n is equal to hidden_units, e.g., 512/1024.
        Since we have warp shuffle inside the code, block.x % 32 should be 0.
    */
    block.x = 32 * ((block.x + 31) / 32);

    /* should pay attention to the rsqrt precision*/
    if (dynamic_quant) {
        size_t maxbytes = real_n * sizeof(T);
        if (maxbytes >= (48 << 10)) {
#if USING_CUDA
            check_cuda_error(hipFuncSetAttribute(reinterpret_cast<const void*>(
                generalLayerNormWithPadding<T), true>, hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes));
#endif
        }
        generalLayerNormWithPadding<T, true><<<grid, block, maxbytes, stream>>>(
            input, gamma, beta, out, layernorm_eps, m, real_n, padding_n, scale, dynamic_scale, int8_mode);
    } else {
        generalLayerNormWithPadding<T, false><<<grid, block, 0, stream>>>(
            input, gamma, beta, out, layernorm_eps, m, real_n, padding_n, scale, dynamic_scale, int8_mode);
    }
}

#define INVOKE_GENERAL_LN_WITH_PADDING(T)                                                                              \
    template void invokeGeneralLayerNormWithPadding(T*           out,                                                  \
                                                    const T*     input,                                                \
                                                    const T*     gamma,                                                \
                                                    const T*     beta,                                                 \
                                                    const float  layernorm_eps,                                        \
                                                    const int    m,                                                    \
                                                    const int    real_n,                                               \
                                                    const int    padding_n,                                            \
                                                    float*       scale,                                                \
                                                    float*       dynamic_scale,                                        \
                                                    const int    int8_mode,                                            \
                                                    hipStream_t stream,                                               \
                                                    int          opt_version);
INVOKE_GENERAL_LN_WITH_PADDING(float)
INVOKE_GENERAL_LN_WITH_PADDING(half)
#ifdef ENABLE_BF16
INVOKE_GENERAL_LN_WITH_PADDING(__hip_bfloat16)
#endif

}