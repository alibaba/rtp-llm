#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "src/fastertransformer/utils/assert_utils.h"
#include "src/fastertransformer/kernels/quantization_tensor.h"
#include "src/fastertransformer/kernels/reduce_kernel_utils.cuh"
#if USING_CUDA
#include "src/fastertransformer/cuda/cuda_type_utils.cuh"
#include "src/fastertransformer/cuda/cuda_utils.h"
#endif
#if USING_ROCM
#include "src/fastertransformer/rocm/hip_utils.h"
#endif

namespace fastertransformer
{
#if USING_ROCM
using namespace rocm;
#endif

__global__ void quantizedKernel(char4* dst, const float4* src, const int64_t sizeDiv4, const float* scalePtr)
{
    for (int64_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < sizeDiv4; idx += blockDim.x * gridDim.x)
    {
        const float scale = __ldg(scalePtr);
        char4 tmp;
        const float4 floatTmp = __ldg(src + idx);
        tmp.x = cuda_cast<int8_t>(floatTmp.x * scale);
        tmp.y = cuda_cast<int8_t>(floatTmp.y * scale);
        tmp.z = cuda_cast<int8_t>(floatTmp.z * scale);
        tmp.w = cuda_cast<int8_t>(floatTmp.w * scale);
        dst[idx] = tmp;
    }
}

__global__ void quantizedKernel(char4* dst, const half2* src, const int64_t sizeDiv4, const float* scalePtr)
{
    for (int64_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < sizeDiv4; idx += blockDim.x * gridDim.x)
    {
        const float scale = __ldg(scalePtr);
        char4 tmp;
        int srcId = idx << 1;

        const uint2 h2 = __ldg(reinterpret_cast<const uint2*>(src + srcId));

        const half2 half2Tmp = reinterpret_cast<const half2&>(h2.x);
        const half2 half2Tmp2 = reinterpret_cast<const half2&>(h2.y);

        tmp.x = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp.x) * scale);
        tmp.y = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp.y) * scale);
        tmp.z = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp2.x) * scale);
        tmp.w = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp2.y) * scale);
        dst[idx] = tmp;
    }
}

template <typename T>
void invokeQuantization(
    int8_t* dst, const T* src, const int64_t size, const float* scalePtr, hipStream_t stream, int maxGridSize)
{
    FT_CHECK_WITH_INFO(size % 4 == 0, "[ERROR][invokeQuantization] size should be a multiple of 4.\n");

    int numBlocks{static_cast<int>((size + 255) / 256)};
    if (maxGridSize == -1) {
        maxGridSize = numBlocks;
    }
    dim3 grid(std::min(numBlocks, maxGridSize));
    FT_CHECK_WITH_INFO(grid.x <= maxGridSize, "[ERROR][invokeQuantization] grid max size is exceeded\n");
    dim3 block(64);
    if (std::is_same_v<T, float>)
    {
        quantizedKernel<<<grid, block, 0, stream>>>((char4*) dst, (const float4*) src, size / 4, scalePtr);
    }
    else if (std::is_same_v<T, half>)
    {
        quantizedKernel<<<grid, block, 0, stream>>>((char4*) dst, (const half2*) src, size / 4, scalePtr);
    }
}

#define INSTANTIATE_INVOKE_QUANTIZATION(T)                                                                        \
template void invokeQuantization(                                                                                 \
    int8_t* dst, const T* src, const int64_t size, const float* scalePtr, hipStream_t stream, int maxGridSize);

INSTANTIATE_INVOKE_QUANTIZATION(float);
INSTANTIATE_INVOKE_QUANTIZATION(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_QUANTIZATION(__hip_bfloat16);
#endif

template <typename T, bool IS_SMOOTHER, bool IS_SHIFT>
__global__ void perTokenQuantization(
    int8_t* dst, const T* src, const int64_t numRows, const int64_t numCols, float* scalePtr, const float* smoother, const float* shift)
{
    const T* srcRow = src + blockIdx.x * numCols;
    int8_t* dstRow = dst + blockIdx.x * numCols;

    T localMax = 1e-6f;
    for (int i = threadIdx.x; i < numCols; i += blockDim.x)
    {
        T val = srcRow[i];
        if(IS_SMOOTHER){
            val = cuda_cast<T>(val / cuda_cast<T>(smoother[i]));
        }
        if(IS_SHIFT){
            val = cuda_cast<T>(val + cuda_cast<T>(shift[i]));
        }
        localMax = cuda_max(localMax, cuda_abs(val));
    }
    const float rowMax = blockAllReduceMax(cuda_cast<float>(localMax));

    if (threadIdx.x == 0)
    {
        scalePtr[blockIdx.x] = rowMax / 127.f;
    }

    const float scaleOrigQuant = 127.f / rowMax;
    for (int i = threadIdx.x; i < numCols; i += blockDim.x)
    {
        T val = srcRow[i];
        if(IS_SMOOTHER){
            val = val / cuda_cast<T>(smoother[i]);
        }
        if(IS_SHIFT){
            val = cuda_cast<T>(val + cuda_cast<T>(shift[i]));
        }
        dstRow[i] = cuda_cast<int8_t>(cuda_cast<float>(val) * scaleOrigQuant);
    }
}

template <typename T, bool IS_SMOOTHER>
void dispatch_per_token_quantization_shift(
    int8_t* dst, const T* src, const int64_t numRows, const int64_t numCols, float* scalePtr, const float* smoother, const float* shift, hipStream_t stream)
{
    // each block is responsible for a single row
    const dim3 block(512);
    const dim3 grid(numRows);

    if(shift != nullptr){
        perTokenQuantization<T, IS_SMOOTHER, true><<<grid, block, 0, stream>>>(dst, src, numRows, numCols, scalePtr, smoother, shift);
    }
    else{
        perTokenQuantization<T, IS_SMOOTHER, false><<<grid, block, 0, stream>>>(dst, src, numRows, numCols, scalePtr, smoother, nullptr);
    }
}

template<typename T>
void invokePerTokenQuantization(
    int8_t* dst, const T* src, const int64_t numRows, const int64_t numCols, float* scalePtr, const float* smoother, const float* shift, hipStream_t stream)
{
    if(smoother != nullptr){
        dispatch_per_token_quantization_shift<T, true>(dst, src, numRows, numCols, scalePtr, smoother, shift, stream);
    }
    else{
        dispatch_per_token_quantization_shift<T, false>(dst, src, numRows, numCols, scalePtr, nullptr, shift, stream);
    }

}

#define INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(T)                                                                   \
    template void invokePerTokenQuantization(                                                                          \
        int8_t* dst, const T* src, const int64_t numRows, const int64_t numCols, float* scalePtr, const float* smoother, const float* shift, hipStream_t stream)

INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(float);
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(__hip_bfloat16);
#endif

/////////////////////////////////////////////////////////////////////////////////////////////////
// int8 col quant ///////////////////////////////////////////////////////////////////////////////
template<typename T, bool IS_SMOOTHER, bool IS_SHIFT>
__global__ void perColQuantization(int8_t*       dst,
                                   const T*      src,
                                   const int64_t numRows,
                                   const int64_t numCols,
                                   half*         scalePtr,
                                   const float*  smoother,
                                   const float*  shift,
                                   float*        dbgfp  = nullptr,
                                   int*          dbgint = nullptr) {
    uint32_t colIdx = blockIdx.x;
    const T* srcCol = src + colIdx;
    int8_t*  dstCol = dst + colIdx;

    T localMax = 1e-6f;
    for (int rowIdx = threadIdx.x; rowIdx < numRows; rowIdx += blockDim.x) {
        T val = srcCol[rowIdx * numCols];
        if (IS_SMOOTHER) {
            val = cuda_cast<T>(val / cuda_cast<T>(smoother[rowIdx]));
        }
        if (IS_SHIFT) {
            val = cuda_cast<T>(val + cuda_cast<T>(shift[rowIdx]));
        }
        localMax = cuda_max(localMax, cuda_abs(val));
    }
    const float colMax = blockAllReduceMax(cuda_cast<float>(localMax));

    if (threadIdx.x == 0) {
        scalePtr[colIdx] = cuda_cast<half>(colMax / 128.f);
    }

    const float scaleOrigQuant = 128.f / colMax;
    for (int rowIdx = threadIdx.x; rowIdx < numRows; rowIdx += blockDim.x) {
        T val = srcCol[rowIdx * numCols];
        if (IS_SMOOTHER) {
            val = val / cuda_cast<T>(smoother[rowIdx]);
        }
        if (IS_SHIFT) {
            val = cuda_cast<T>(val + cuda_cast<T>(shift[rowIdx]));
        }
        dstCol[rowIdx * numCols] = cuda_cast<int8_t>(cuda_cast<float>(val) * scaleOrigQuant);
    }
}

template<typename T, bool IS_SMOOTHER>
void dispatch_per_col_quantization_shift(int8_t*       dst,
                                         const T*      src,
                                         const int64_t numRows,
                                         const int64_t numCols,
                                         half*         scalePtr,
                                         const float*  smoother,
                                         const float*  shift,
                                         hipStream_t  stream) {
    // each block is responsible for a single row
    const dim3 block(512);
    const dim3 grid(numCols);

    if (shift != nullptr) {
        perColQuantization<T, IS_SMOOTHER, true>
            <<<grid, block, 0, stream>>>(dst, src, numRows, numCols, scalePtr, smoother, shift);
    } else {
        perColQuantization<T, IS_SMOOTHER, false>
            <<<grid, block, 0, stream>>>(dst, src, numRows, numCols, scalePtr, smoother, nullptr);
    }
}

template<typename T>
void invokePerColQuantizationInt8(int8_t*       dst,
                                  const T*      src,
                                  const int64_t numRows,
                                  const int64_t numCols,
                                  half*         scalePtr,
                                  const float*  smoother,
                                  const float*  shift,
                                  hipStream_t  stream) {
    if (smoother != nullptr) {
        dispatch_per_col_quantization_shift<T, true>(dst, src, numRows, numCols, scalePtr, smoother, shift, stream);
    } else {
        dispatch_per_col_quantization_shift<T, false>(dst, src, numRows, numCols, scalePtr, nullptr, shift, stream);
    }
}

#define INSTANTIATE_INVOKE_PER_COL_QUANTIZATION_INT8(T)                                                                \
    template void invokePerColQuantizationInt8(int8_t*       dst,                                                      \
                                               const T*      src,                                                      \
                                               const int64_t numRows,                                                  \
                                               const int64_t numCols,                                                  \
                                               half*         scalePtr,                                                 \
                                               const float*  smoother,                                                 \
                                               const float*  shift,                                                    \
                                               hipStream_t  stream)

INSTANTIATE_INVOKE_PER_COL_QUANTIZATION_INT8(float);
INSTANTIATE_INVOKE_PER_COL_QUANTIZATION_INT8(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_PER_COL_QUANTIZATION_INT8(__hip_bfloat16);
#endif

/////////////////////////////////////////////////////////////////////////////////////////////////
// int8 col dequant /////////////////////////////////////////////////////////////////////////////
template<typename T, bool IS_SMOOTHER, bool IS_SHIFT>
__global__ void perColDequantization(T*            dst,
                                     const int8_t* src,
                                     const int64_t numRows,
                                     const int64_t numCols,
                                     const half*   scalePtr,
                                     const float*  smoother,
                                     const float*  shift,
                                     float*        dbgfp  = nullptr,
                                     int*          dbgint = nullptr) {
    uint32_t      colIdx = blockIdx.x;
    const int8_t* srcRow = src + colIdx;
    T*            dstRow = dst + colIdx;

    float scaleOrigQuant = cuda_cast<float>(scalePtr[colIdx]);
    if (IS_SMOOTHER) {
        scaleOrigQuant = scaleOrigQuant * smoother[colIdx];
    }
    if (IS_SHIFT) {
        scaleOrigQuant = scaleOrigQuant - shift[colIdx];
    }

    for (int rowIdx = threadIdx.x; rowIdx < numRows; rowIdx += blockDim.x) {
        uint8_t tmpi8 = srcRow[rowIdx * numCols];

        T val = cuda_cast<T>(cuda_cast<float>(tmpi8) * scaleOrigQuant);

        if (IS_SMOOTHER) {
            val = val * cuda_cast<T>(smoother[rowIdx]);
        }
        if (IS_SHIFT) {
            val = cuda_cast<T>(val - cuda_cast<T>(shift[rowIdx]));
        }

        dstRow[rowIdx * numCols] = val;
    }
}

template<typename T, bool IS_SMOOTHER>
void dispatch_per_col_dequantization_shift(T*            dst,
                                           const int8_t* src,
                                           const int64_t numRows,
                                           const int64_t numCols,
                                           half*         scalePtr,
                                           const float*  smoother,
                                           const float*  shift,
                                           hipStream_t  stream) {
    // each block is responsible for a single col
    const dim3 block(512);
    const dim3 grid(numCols);

    if (shift != nullptr) {
        perColDequantization<T, IS_SMOOTHER, true>
            <<<grid, block, 0, stream>>>(dst, src, numRows, numCols, scalePtr, smoother, shift);
    } else {
        perColDequantization<T, IS_SMOOTHER, false>
            <<<grid, block, 0, stream>>>(dst, src, numRows, numCols, scalePtr, smoother, nullptr);
    }
}

template<typename T>
void invokePerColDequantizationInt8(T*            dst,
                                    const int8_t* src,
                                    const int64_t numRows,
                                    const int64_t numCols,
                                    half*         scalePtr,
                                    const float*  smoother,
                                    const float*  shift,
                                    hipStream_t  stream) {
    if (smoother != nullptr) {
        dispatch_per_col_dequantization_shift<T, true>(dst, src, numRows, numCols, scalePtr, smoother, shift, stream);
    } else {
        dispatch_per_col_dequantization_shift<T, false>(dst, src, numRows, numCols, scalePtr, nullptr, shift, stream);
    }
}

#define INSTANTIATE_INVOKE_PER_COL_DEQUANTIZATION_INT8(T)                                                              \
    template void invokePerColDequantizationInt8(T*            dst,                                                    \
                                                 const int8_t* src,                                                    \
                                                 const int64_t numRows,                                                \
                                                 const int64_t numCols,                                                \
                                                 half*         scalePtr,                                               \
                                                 const float*  smoother,                                               \
                                                 const float*  shift,                                                  \
                                                 hipStream_t  stream)
INSTANTIATE_INVOKE_PER_COL_DEQUANTIZATION_INT8(float);
INSTANTIATE_INVOKE_PER_COL_DEQUANTIZATION_INT8(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_PER_COL_DEQUANTIZATION_INT8(__hip_bfloat16);
#endif

/////////////////////////////////////////////////////////////////////////////////////////////////
// int4 col quant ///////////////////////////////////////////////////////////////////////////////
template<typename T, bool IS_SMOOTHER, bool IS_SHIFT>
__global__ void perColQuantization(char4*        dst,
                                   const T*      src,
                                   const int64_t numRows,
                                   const int64_t numCols,
                                   const int64_t numColsBlk,
                                   half*         scalePtr,
                                   const float*  smoother,
                                   const float*  shift,
                                   float*        dbgfp  = nullptr,
                                   int*          dbgint = nullptr) {
    uint8_t* pDst      = (uint8_t*)dst;
    uint32_t colBlkIdx = blockIdx.x;
    const T* srcCol    = src + colBlkIdx * numColsBlk;
    uint8_t* dstCol    = pDst + colBlkIdx * numColsBlk / 2;

    T localMax = 1e-6f;
    for (int rowIdx = threadIdx.x; rowIdx < numRows; rowIdx += blockDim.x) {
        for (int colInBlkIdx = 0; colInBlkIdx < numColsBlk; colInBlkIdx++) {
            T val = srcCol[rowIdx * numCols + colInBlkIdx];
            if (IS_SMOOTHER) {
                val = cuda_cast<T>(val / cuda_cast<T>(smoother[colBlkIdx]));
            }
            if (IS_SHIFT) {
                val = cuda_cast<T>(val + cuda_cast<T>(shift[colBlkIdx]));
            }
            localMax = cuda_max(localMax, cuda_abs(val));
        }
    }
    const float colBlkMax = blockAllReduceMax(cuda_cast<float>(localMax));

    if (threadIdx.x == 0) {
        scalePtr[colBlkIdx] = colBlkMax / 8.0f;
    }

    const float scaleOrigQuant = 8.f / colBlkMax;
    for (int rowIdx = threadIdx.x; rowIdx < numRows; rowIdx += blockDim.x) {
        // one loop process 2 cols of intput, and 1 col of uint8_t output
        for (int colInBlkIdx = 0; colInBlkIdx < numColsBlk / 2; colInBlkIdx++) {
            T vall = srcCol[rowIdx * numCols + colInBlkIdx * 2];
            T valh = srcCol[rowIdx * numCols + colInBlkIdx * 2 + 1];
            if (IS_SMOOTHER) {
                vall = vall / cuda_cast<T>(smoother[colBlkIdx]);
                valh = valh / cuda_cast<T>(smoother[colBlkIdx]);
            }
            if (IS_SHIFT) {
                vall = cuda_cast<T>(vall + cuda_cast<T>(shift[colBlkIdx]));
                valh = cuda_cast<T>(valh + cuda_cast<T>(shift[colBlkIdx]));
            }

            int8_t tmpi8l = cuda_cast<int8_t>(cuda_cast<float>(vall) * scaleOrigQuant);
            int8_t tmpi8h = cuda_cast<int8_t>(cuda_cast<float>(valh) * scaleOrigQuant);
            int8_t tmpi4l = tmpi8l & 0x0F;
            int8_t tmpi4h = tmpi8h & 0x0F;

            uint8_t tmpuint = tmpi4l;
            tmpuint         = tmpuint << 4;
            tmpuint         = tmpuint | tmpi4h;

            dstCol[rowIdx * numCols / 2 + colInBlkIdx] = tmpuint;
        }
    }
}

template<typename T, bool IS_SMOOTHER>
void dispatch_per_col_quantization_shift(char4*        dst,
                                         const T*      src,
                                         const int64_t numRows,
                                         const int64_t numCols,
                                         half*         scalePtr,
                                         const float*  smoother,
                                         const float*  shift,
                                         hipStream_t  stream) {
    // each block is responsible for a block cols, share the same scale
    const int colBlk = 2;
    assert(colBlk % 2 == 0);
    assert(numCols % colBlk == 0);

    const dim3 block(512);
    const dim3 grid(numCols / colBlk);

    if (shift != nullptr) {
        perColQuantization<T, IS_SMOOTHER, true>
            <<<grid, block, 0, stream>>>(dst, src, numRows, numCols, colBlk, scalePtr, smoother, shift);
    } else {
        perColQuantization<T, IS_SMOOTHER, false>
            <<<grid, block, 0, stream>>>(dst, src, numRows, numCols, colBlk, scalePtr, smoother, nullptr);
    }
}

template<typename T>
void invokePerColQuantizationInt4x2(int8_t*       dst,
                                    const T*      src,
                                    const int64_t numRows,
                                    const int64_t numCols,
                                    half*         scalePtr,
                                    const float*  smoother,
                                    const float*  shift,
                                    hipStream_t  stream) {
    if (smoother != nullptr) {
        dispatch_per_col_quantization_shift<T, true>(
            (char4*)dst, src, numRows, numCols, scalePtr, smoother, shift, stream);
    } else {
        dispatch_per_col_quantization_shift<T, false>(
            (char4*)dst, src, numRows, numCols, scalePtr, nullptr, shift, stream);
    }
}

#define INSTANTIATE_INVOKE_PER_COL_QUANTIZATION_INT4X2(T)                                                              \
    template void invokePerColQuantizationInt4x2(int8_t*       dst,                                                    \
                                                 const T*      src,                                                    \
                                                 const int64_t numRows,                                                \
                                                 const int64_t numCols,                                                \
                                                 half*         scalePtr,                                               \
                                                 const float*  smoother,                                               \
                                                 const float*  shift,                                                  \
                                                 hipStream_t  stream)
INSTANTIATE_INVOKE_PER_COL_QUANTIZATION_INT4X2(float);
INSTANTIATE_INVOKE_PER_COL_QUANTIZATION_INT4X2(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_PER_COL_QUANTIZATION_INT4X2(__hip_bfloat16);
#endif

/////////////////////////////////////////////////////////////////////////////////////////////////
// int4 col dequant /////////////////////////////////////////////////////////////////////////////
template<typename T, bool IS_SMOOTHER, bool IS_SHIFT>
__global__ void perColDequantization(T*            dst,
                                     const char4*  src,
                                     const int64_t numRows,
                                     const int64_t numCols,
                                     const int64_t numColsBlk,
                                     const half*   scalePtr,
                                     const float*  smoother,
                                     const float*  shift,
                                     float*        dbgfp  = nullptr,
                                     int*          dbgint = nullptr) {
    const uint8_t* pSrc      = (const uint8_t*)src;
    uint32_t       colBlkIdx = blockIdx.x;

    float scaleOrigQuant = scalePtr[colBlkIdx];
    if (IS_SMOOTHER) {
        scaleOrigQuant = scaleOrigQuant * smoother[colBlkIdx];
    }
    if (IS_SHIFT) {
        scaleOrigQuant = scaleOrigQuant - shift[colBlkIdx];
    }

    for (int rowIdx = threadIdx.x; rowIdx < numRows; rowIdx += blockDim.x) {
        // one loop process 1 col uint8 input, and 2 cols of output
        for (int colInBlkIdx = 0; colInBlkIdx < numColsBlk / 2; colInBlkIdx++) {
            uint8_t tmpu8 = pSrc[rowIdx * numCols / 2 + colBlkIdx * numColsBlk / 2 + colInBlkIdx];

            uint8_t tmpi4l = tmpu8 & 0x0F;
            uint8_t tmpi4h = (tmpu8 >> 4) & 0x0F;

            T vall = cuda_cast<T>(cuda_cast<float>(tmpi4l) * scaleOrigQuant);
            T valh = cuda_cast<T>(cuda_cast<float>(tmpi4h) * scaleOrigQuant);

            if (IS_SMOOTHER) {
                vall = vall * cuda_cast<T>(smoother[colBlkIdx]);
                valh = valh * cuda_cast<T>(smoother[colBlkIdx]);
            }
            if (IS_SHIFT) {
                vall = cuda_cast<T>(vall - cuda_cast<T>(shift[colBlkIdx]));
                valh = cuda_cast<T>(valh - cuda_cast<T>(shift[colBlkIdx]));
            }

            dst[rowIdx * numCols + colBlkIdx * numColsBlk + colInBlkIdx * 2 + 0] = valh;
            dst[rowIdx * numCols + colBlkIdx * numColsBlk + colInBlkIdx * 2 + 1] = vall;
        }
    }
}

template<typename T, bool IS_SMOOTHER>
void dispatch_per_col_dequantization_shift(T*            dst,
                                           const char4*  src,
                                           const int64_t numRows,
                                           const int64_t numCols,
                                           half*         scalePtr,
                                           const float*  smoother,
                                           const float*  shift,
                                           hipStream_t  stream) {
    // each block is responsible for a block cols, share the same scale
    const int colBlk = 2;
    assert(colBlk % 2 == 0);
    assert(numCols % colBlk == 0);

    const dim3 block(512);
    const dim3 grid(numCols / colBlk);

    if (shift != nullptr) {
        perColDequantization<T, IS_SMOOTHER, true>
            <<<grid, block, 0, stream>>>(dst, src, numRows, numCols, colBlk, scalePtr, smoother, shift);
    } else {
        perColDequantization<T, IS_SMOOTHER, false>
            <<<grid, block, 0, stream>>>(dst, src, numRows, numCols, colBlk, scalePtr, smoother, nullptr);
    }
}

template<typename T>
void invokePerColDequantizationInt4x2(T*            dst,
                                      const int8_t* src,
                                      const int64_t numRows,
                                      const int64_t numCols,
                                      half*         scalePtr,
                                      const float*  smoother,
                                      const float*  shift,
                                      hipStream_t  stream) {
    if (smoother != nullptr) {
        dispatch_per_col_dequantization_shift<T, true>(
            dst, (char4*)src, numRows, numCols, scalePtr, smoother, shift, stream);
    } else {
        dispatch_per_col_dequantization_shift<T, false>(
            dst, (char4*)src, numRows, numCols, scalePtr, nullptr, shift, stream);
    }
}

#define INSTANTIATE_INVOKE_PER_COL_DEQUANTIZATION_INT4X2(T)                                                            \
    template void invokePerColDequantizationInt4x2(T*            dst,                                                  \
                                                   const int8_t* src,                                                  \
                                                   const int64_t numRows,                                              \
                                                   const int64_t numCols,                                              \
                                                   half*         scalePtr,                                             \
                                                   const float*  smoother,                                             \
                                                   const float*  shift,                                                \
                                                   hipStream_t  stream)
INSTANTIATE_INVOKE_PER_COL_DEQUANTIZATION_INT4X2(float);
INSTANTIATE_INVOKE_PER_COL_DEQUANTIZATION_INT4X2(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_PER_COL_DEQUANTIZATION_INT4X2(__hip_bfloat16);
#endif

}  // namespace fastertransformer
