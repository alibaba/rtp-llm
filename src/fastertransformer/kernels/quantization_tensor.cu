#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "src/fastertransformer/utils/assert_utils.h"
#include "src/fastertransformer/kernels/quantization_tensor.h"
#include "src/fastertransformer/kernels/reduce_kernel_utils.cuh"
#if USING_CUDA
#include "src/fastertransformer/cuda/cuda_type_utils.cuh"
#include "src/fastertransformer/cuda/cuda_utils.h"
#endif
#if USING_ROCM
#include "src/fastertransformer/rocm/hip_utils.h"
#endif

namespace fastertransformer {
#if USING_ROCM
using namespace rocm;
#endif

__global__ void quantizedKernel(char4* dst, const float4* src, const int64_t sizeDiv4, const float* scalePtr) {
    for (int64_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < sizeDiv4; idx += blockDim.x * gridDim.x) {
        const float  scale = __ldg(scalePtr);
        char4        tmp;
        const float4 floatTmp = __ldg(src + idx);
        tmp.x                 = cuda_cast<int8_t>(floatTmp.x * scale);
        tmp.y                 = cuda_cast<int8_t>(floatTmp.y * scale);
        tmp.z                 = cuda_cast<int8_t>(floatTmp.z * scale);
        tmp.w                 = cuda_cast<int8_t>(floatTmp.w * scale);
        dst[idx]              = tmp;
    }
}

__global__ void quantizedKernel(char4* dst, const half2* src, const int64_t sizeDiv4, const float* scalePtr) {
    for (int64_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < sizeDiv4; idx += blockDim.x * gridDim.x) {
        const float scale = __ldg(scalePtr);
        char4       tmp;
        int         srcId = idx << 1;

        const uint2 h2 = __ldg(reinterpret_cast<const uint2*>(src + srcId));

        const half2 half2Tmp  = reinterpret_cast<const half2&>(h2.x);
        const half2 half2Tmp2 = reinterpret_cast<const half2&>(h2.y);

        tmp.x    = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp.x) * scale);
        tmp.y    = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp.y) * scale);
        tmp.z    = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp2.x) * scale);
        tmp.w    = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp2.y) * scale);
        dst[idx] = tmp;
    }
}

template<typename T>
void invokeQuantization(
    int8_t* dst, const T* src, const int64_t size, const float* scalePtr, hipStream_t stream, int maxGridSize) {
    FT_CHECK_WITH_INFO(size % 4 == 0, "[ERROR][invokeQuantization] size should be a multiple of 4.\n");

    int numBlocks{static_cast<int>((size + 255) / 256)};
    if (maxGridSize == -1) {
        maxGridSize = numBlocks;
    }
    dim3 grid(std::min(numBlocks, maxGridSize));
    FT_CHECK_WITH_INFO(grid.x <= maxGridSize, "[ERROR][invokeQuantization] grid max size is exceeded\n");
    dim3 block(64);
    if (std::is_same_v<T, float>) {
        quantizedKernel<<<grid, block, 0, stream>>>((char4*)dst, (const float4*)src, size / 4, scalePtr);
    } else if (std::is_same_v<T, half>) {
        quantizedKernel<<<grid, block, 0, stream>>>((char4*)dst, (const half2*)src, size / 4, scalePtr);
    }
}

#define INSTANTIATE_INVOKE_QUANTIZATION(T)                                                                             \
    template void invokeQuantization(                                                                                  \
        int8_t* dst, const T* src, const int64_t size, const float* scalePtr, hipStream_t stream, int maxGridSize);

INSTANTIATE_INVOKE_QUANTIZATION(float);
INSTANTIATE_INVOKE_QUANTIZATION(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_QUANTIZATION(__hip_bfloat16);
#endif

template<typename T, bool IS_SMOOTHER, bool IS_SHIFT>
__global__ void perTokenQuantization(int8_t*       dst,
                                     const T*      src,
                                     const int64_t numRows,
                                     const int64_t numCols,
                                     float*        scalePtr,
                                     const float*  smoother,
                                     const float*  shift) {
    const T* srcRow = src + blockIdx.x * numCols;
    int8_t*  dstRow = dst + blockIdx.x * numCols;

    T localMax = 1e-6f;
    for (int i = threadIdx.x; i < numCols; i += blockDim.x) {
        T val = srcRow[i];
        if (IS_SMOOTHER) {
            val = cuda_cast<T>(val / cuda_cast<T>(smoother[i]));
        }
        if (IS_SHIFT) {
            val = cuda_cast<T>(val + cuda_cast<T>(shift[i]));
        }
        localMax = cuda_max(localMax, cuda_abs(val));
    }
    const float rowMax = blockAllReduceMax(cuda_cast<float>(localMax));

    if (threadIdx.x == 0) {
        scalePtr[blockIdx.x] = rowMax / 127.f;
    }

    const float scaleOrigQuant = 127.f / rowMax;
    for (int i = threadIdx.x; i < numCols; i += blockDim.x) {
        T val = srcRow[i];
        if (IS_SMOOTHER) {
            val = val / cuda_cast<T>(smoother[i]);
        }
        if (IS_SHIFT) {
            val = cuda_cast<T>(val + cuda_cast<T>(shift[i]));
        }
        dstRow[i] = cuda_cast<int8_t>(cuda_cast<float>(val) * scaleOrigQuant);
    }
}

template<typename T, bool IS_SMOOTHER>
void dispatch_per_token_quantization_shift(int8_t*       dst,
                                           const T*      src,
                                           const int64_t numRows,
                                           const int64_t numCols,
                                           float*        scalePtr,
                                           const float*  smoother,
                                           const float*  shift,
                                           hipStream_t  stream) {
    // each block is responsible for a single row
    const dim3 block(512);
    const dim3 grid(numRows);

    if (shift != nullptr) {
        perTokenQuantization<T, IS_SMOOTHER, true>
            <<<grid, block, 0, stream>>>(dst, src, numRows, numCols, scalePtr, smoother, shift);
    } else {
        perTokenQuantization<T, IS_SMOOTHER, false>
            <<<grid, block, 0, stream>>>(dst, src, numRows, numCols, scalePtr, smoother, nullptr);
    }
}

template<typename T>
void invokePerTokenQuantization(int8_t*       dst,
                                const T*      src,
                                const int64_t numRows,
                                const int64_t numCols,
                                float*        scalePtr,
                                const float*  smoother,
                                const float*  shift,
                                hipStream_t  stream) {
    if (smoother != nullptr) {
        dispatch_per_token_quantization_shift<T, true>(dst, src, numRows, numCols, scalePtr, smoother, shift, stream);
    } else {
        dispatch_per_token_quantization_shift<T, false>(dst, src, numRows, numCols, scalePtr, nullptr, shift, stream);
    }
}

#define INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(T)                                                                   \
    template void invokePerTokenQuantization(int8_t*       dst,                                                        \
                                             const T*      src,                                                        \
                                             const int64_t numRows,                                                    \
                                             const int64_t numCols,                                                    \
                                             float*        scalePtr,                                                   \
                                             const float*  smoother,                                                   \
                                             const float*  shift,                                                      \
                                             hipStream_t  stream)

INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(float);
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(__hip_bfloat16);
#endif

/////////////////////////////////////////////////////////////////////////////////////////////////
// int8 col quant ///////////////////////////////////////////////////////////////////////////////
template<typename T, bool IS_SMOOTHER, bool IS_SHIFT>
__global__ void perColQuantization(int8_t*       dst,
                                   const T*      src,
                                   const int64_t numRows,
                                   const int64_t numCols,
                                   half*         scalePtr,
                                   const float*  smoother,
                                   const float*  shift,
                                   float*        dbgfp  = nullptr,
                                   int*          dbgint = nullptr) {
    uint32_t colIdx = blockIdx.x;
    const T* srcCol = src + colIdx;
    int8_t*  dstCol = dst + colIdx;

    T localMax = 1e-6f;
    for (int rowIdx = threadIdx.x; rowIdx < numRows; rowIdx += blockDim.x) {
        T val = srcCol[rowIdx * numCols];
        if (IS_SMOOTHER) {
            val = cuda_cast<T>(val / cuda_cast<T>(smoother[rowIdx]));
        }
        if (IS_SHIFT) {
            val = cuda_cast<T>(val + cuda_cast<T>(shift[rowIdx]));
        }
        localMax = cuda_max(localMax, cuda_abs(val));
    }
    const float colMax = blockAllReduceMax(cuda_cast<float>(localMax));

    if (threadIdx.x == 0) {
        scalePtr[colIdx] = cuda_cast<half>(colMax / 128.f);
    }

    const float scaleOrigQuant = 128.f / colMax;
    for (int rowIdx = threadIdx.x; rowIdx < numRows; rowIdx += blockDim.x) {
        T val = srcCol[rowIdx * numCols];
        if (IS_SMOOTHER) {
            val = val / cuda_cast<T>(smoother[rowIdx]);
        }
        if (IS_SHIFT) {
            val = cuda_cast<T>(val + cuda_cast<T>(shift[rowIdx]));
        }
        dstCol[rowIdx * numCols] = cuda_cast<int8_t>(cuda_cast<float>(val) * scaleOrigQuant);
    }
}

template<typename T, bool IS_SMOOTHER>
void dispatch_per_col_quantization_shift(int8_t*       dst,
                                         const T*      src,
                                         const int64_t numRows,
                                         const int64_t numCols,
                                         half*         scalePtr,
                                         const float*  smoother,
                                         const float*  shift,
                                         hipStream_t  stream) {
    // each block is responsible for a single row
    const dim3 block(512);
    const dim3 grid(numCols);

    if (shift != nullptr) {
        perColQuantization<T, IS_SMOOTHER, true>
            <<<grid, block, 0, stream>>>(dst, src, numRows, numCols, scalePtr, smoother, shift);
    } else {
        perColQuantization<T, IS_SMOOTHER, false>
            <<<grid, block, 0, stream>>>(dst, src, numRows, numCols, scalePtr, smoother, nullptr);
    }
}

template<typename T>
void invokePerColQuantizationInt8(int8_t*       dst,
                                  const T*      src,
                                  const int64_t numRows,
                                  const int64_t numCols,
                                  half*         scalePtr,
                                  const float*  smoother,
                                  const float*  shift,
                                  hipStream_t  stream) {
    if (smoother != nullptr) {
        dispatch_per_col_quantization_shift<T, true>(dst, src, numRows, numCols, scalePtr, smoother, shift, stream);
    } else {
        dispatch_per_col_quantization_shift<T, false>(dst, src, numRows, numCols, scalePtr, nullptr, shift, stream);
    }
}

#define INSTANTIATE_INVOKE_PER_COL_QUANTIZATION_INT8(T)                                                                \
    template void invokePerColQuantizationInt8(int8_t*       dst,                                                      \
                                               const T*      src,                                                      \
                                               const int64_t numRows,                                                  \
                                               const int64_t numCols,                                                  \
                                               half*         scalePtr,                                                 \
                                               const float*  smoother,                                                 \
                                               const float*  shift,                                                    \
                                               hipStream_t  stream)

INSTANTIATE_INVOKE_PER_COL_QUANTIZATION_INT8(float);
INSTANTIATE_INVOKE_PER_COL_QUANTIZATION_INT8(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_PER_COL_QUANTIZATION_INT8(__hip_bfloat16);
#endif

/////////////////////////////////////////////////////////////////////////////////////////////////
// int8 col dequant /////////////////////////////////////////////////////////////////////////////
template<typename T, bool IS_SMOOTHER, bool IS_SHIFT>
__global__ void perColDequantization(T*            dst,
                                     const int8_t* src,
                                     const int64_t numRows,
                                     const int64_t numCols,
                                     const half*   scalePtr,
                                     const float*  smoother,
                                     const float*  shift,
                                     float*        dbgfp  = nullptr,
                                     int*          dbgint = nullptr) {
    uint32_t      colIdx = blockIdx.x;
    const int8_t* srcRow = src + colIdx;
    T*            dstRow = dst + colIdx;

    float scaleOrigQuant = cuda_cast<float>(scalePtr[colIdx]);
    if (IS_SMOOTHER) {
        scaleOrigQuant = scaleOrigQuant * smoother[colIdx];
    }
    if (IS_SHIFT) {
        scaleOrigQuant = scaleOrigQuant - shift[colIdx];
    }

    for (int rowIdx = threadIdx.x; rowIdx < numRows; rowIdx += blockDim.x) {
        uint8_t tmpi8 = srcRow[rowIdx * numCols];

        T val = cuda_cast<T>(cuda_cast<float>(tmpi8) * scaleOrigQuant);

        if (IS_SMOOTHER) {
            val = val * cuda_cast<T>(smoother[rowIdx]);
        }
        if (IS_SHIFT) {
            val = cuda_cast<T>(val - cuda_cast<T>(shift[rowIdx]));
        }

        dstRow[rowIdx * numCols] = val;
    }
}

template<typename T, bool IS_SMOOTHER>
void dispatch_per_col_dequantization_shift(T*            dst,
                                           const int8_t* src,
                                           const int64_t numRows,
                                           const int64_t numCols,
                                           half*         scalePtr,
                                           const float*  smoother,
                                           const float*  shift,
                                           hipStream_t  stream) {
    // each block is responsible for a single col
    const dim3 block(512);
    const dim3 grid(numCols);

    if (shift != nullptr) {
        perColDequantization<T, IS_SMOOTHER, true>
            <<<grid, block, 0, stream>>>(dst, src, numRows, numCols, scalePtr, smoother, shift);
    } else {
        perColDequantization<T, IS_SMOOTHER, false>
            <<<grid, block, 0, stream>>>(dst, src, numRows, numCols, scalePtr, smoother, nullptr);
    }
}

template<typename T>
void invokePerColDequantizationInt8(T*            dst,
                                    const int8_t* src,
                                    const int64_t numRows,
                                    const int64_t numCols,
                                    half*         scalePtr,
                                    const float*  smoother,
                                    const float*  shift,
                                    hipStream_t  stream) {
    if (smoother != nullptr) {
        dispatch_per_col_dequantization_shift<T, true>(dst, src, numRows, numCols, scalePtr, smoother, shift, stream);
    } else {
        dispatch_per_col_dequantization_shift<T, false>(dst, src, numRows, numCols, scalePtr, nullptr, shift, stream);
    }
}

#define INSTANTIATE_INVOKE_PER_COL_DEQUANTIZATION_INT8(T)                                                              \
    template void invokePerColDequantizationInt8(T*            dst,                                                    \
                                                 const int8_t* src,                                                    \
                                                 const int64_t numRows,                                                \
                                                 const int64_t numCols,                                                \
                                                 half*         scalePtr,                                               \
                                                 const float*  smoother,                                               \
                                                 const float*  shift,                                                  \
                                                 hipStream_t  stream)
INSTANTIATE_INVOKE_PER_COL_DEQUANTIZATION_INT8(float);
INSTANTIATE_INVOKE_PER_COL_DEQUANTIZATION_INT8(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_PER_COL_DEQUANTIZATION_INT8(__hip_bfloat16);
#endif

/////////////////////////////////////////////////////////////////////////////////////////////////
// int4 col quant ///////////////////////////////////////////////////////////////////////////////
template<typename T>
__global__ void perColQuantization(char4*        dst,
                                   const T*      src,
                                   const int64_t numRows,
                                   const int64_t numCols,
                                   const int64_t numColsBlk,
                                   half*         scalePtr,
                                   const float*  smoother,
                                   const float*  shift,
                                   float*        dbgfp  = nullptr,
                                   int*          dbgint = nullptr) {
    uint8_t* pDst      = (uint8_t*)dst;
    uint32_t colBlkIdx = blockIdx.x;
    const T* srcCol    = src + colBlkIdx * numColsBlk;
    uint8_t* dstCol    = pDst + colBlkIdx * numColsBlk / 2;

    T localMax = 1e-6f;
    for (int rowIdx = threadIdx.x; rowIdx < numRows; rowIdx += blockDim.x) {
        for (int colInBlkIdx = 0; colInBlkIdx < numColsBlk; colInBlkIdx++) {
            T val = srcCol[rowIdx * numCols + colInBlkIdx];

            localMax = cuda_max(localMax, cuda_abs(val));
        }
    }
    const float colBlkMax = blockAllReduceMax(cuda_cast<float>(localMax));

    if (threadIdx.x == 0) {
        scalePtr[colBlkIdx] = colBlkMax / 8.0f;
    }

    const float scaleOrigQuant = 8.f / colBlkMax;
    for (int rowIdx = threadIdx.x; rowIdx < numRows; rowIdx += blockDim.x) {
        // one loop process 2 cols of intput, and 1 col of uint8_t output
        for (int colInBlkIdx = 0; colInBlkIdx < numColsBlk / 2; colInBlkIdx++) {
            T vall = srcCol[rowIdx * numCols + colInBlkIdx * 2];
            T valh = srcCol[rowIdx * numCols + colInBlkIdx * 2 + 1];

            int8_t tmpi8l = cuda_cast<int8_t>(cuda_cast<float>(vall) * scaleOrigQuant);
            int8_t tmpi8h = cuda_cast<int8_t>(cuda_cast<float>(valh) * scaleOrigQuant);
            int8_t tmpi4l = tmpi8l & 0x0F;
            int8_t tmpi4h = tmpi8h & 0x0F;

            uint8_t tmpuint = tmpi4l;
            tmpuint         = tmpuint << 4;
            tmpuint         = tmpuint | tmpi4h;

            dstCol[rowIdx * numCols / 2 + colInBlkIdx] = tmpuint;
        }
    }
}

template<typename T>
void invokePerColQuantizationInt4x2(int8_t*       dst,
                                    const T*      src,
                                    const int64_t numRows,
                                    const int64_t numCols,
                                    half*         scalePtr,
                                    const float*  smoother,
                                    const float*  shift,
                                    hipStream_t  stream) {

    const int colBlk = 2;
    assert(colBlk % 2 == 0);
    assert(numCols % colBlk == 0);

    const dim3 block(512);
    const dim3 grid(numCols / colBlk);

    // perColQuantization<T>
    //     <<<grid, block, 0, stream>>>(dst, src, numRows, numCols, colBlk, scalePtr, smoother, nullptr);
}

#define INSTANTIATE_INVOKE_PER_COL_QUANTIZATION_INT4X2(T)                                                              \
    template void invokePerColQuantizationInt4x2(int8_t*       dst,                                                    \
                                                 const T*      src,                                                    \
                                                 const int64_t numRows,                                                \
                                                 const int64_t numCols,                                                \
                                                 half*         scalePtr,                                               \
                                                 const float*  smoother,                                               \
                                                 const float*  shift,                                                  \
                                                 hipStream_t  stream)
INSTANTIATE_INVOKE_PER_COL_QUANTIZATION_INT4X2(float);
INSTANTIATE_INVOKE_PER_COL_QUANTIZATION_INT4X2(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_PER_COL_QUANTIZATION_INT4X2(__hip_bfloat16);
#endif

/////////////////////////////////////////////////////////////////////////////////////////////////
// int4 col dequant /////////////////////////////////////////////////////////////////////////////
template<typename T>
__global__ void perColDequantization(T*            dst,
                                     const char4*  src,
                                     const int64_t numRows,
                                     const int64_t numCols,
                                     const half*   scalePtr,
                                     const half*   zerosPtr,
                                     const int64_t groupSize,
                                     float*        dbgfp  = nullptr,
                                     int*          dbgint = nullptr) {
    const uint8_t* pSrc      = (const uint8_t*)src;
    uint32_t       colPckIdx = blockIdx.y;
    uint32_t       rowBlkIdx = blockIdx.x;

    float scalel = cuda_cast<float>(scalePtr[rowBlkIdx * numCols + colPckIdx * 2 + 0]);
    float scaleh = cuda_cast<float>(scalePtr[rowBlkIdx * numCols + colPckIdx * 2 + 1]);
    float zerosl = cuda_cast<float>(zerosPtr[rowBlkIdx * numCols + colPckIdx * 2 + 0]);
    float zerosh = cuda_cast<float>(zerosPtr[rowBlkIdx * numCols + colPckIdx * 2 + 1]);

    uint8_t tmpu8 = pSrc[(groupSize * rowBlkIdx + threadIdx.x) * numCols / 2 + colPckIdx];

    uint8_t tmpu4l = tmpu8 & 0x0F;
    uint8_t tmpu4h = (tmpu8 >> 4) & 0x0F;

    if (tmpu4l & 0x08)
        tmpu4l |= 0xF0;
    if (tmpu4h & 0x08)
        tmpu4h |= 0xF0;
    int8_t tmpi4l = tmpu4l;
    int8_t tmpi4h = tmpu4h;

    float tmpfpl = cuda_cast<float>(tmpi4l);
    float tmpfph = cuda_cast<float>(tmpi4h);

    T vall = cuda_cast<T>(tmpfpl * scalel + zerosl);
    T valh = cuda_cast<T>(tmpfph * scaleh + zerosh);

    dst[(groupSize * rowBlkIdx + threadIdx.x) * numCols + colPckIdx * 2 + 0] = vall;
    dst[(groupSize * rowBlkIdx + threadIdx.x) * numCols + colPckIdx * 2 + 1] = valh;
}

template<typename T>
void invokePerColDequantizationInt4x2(T*            dst,
                                      const int8_t* src,
                                      const int64_t numRows,
                                      const int64_t numCols,
                                      half*         scalePtr,
                                      half*         zerosPtr,
                                      const int64_t groupSize,
                                      hipStream_t  stream) {
    const dim3 block(groupSize);
    const dim3 grid(numRows / groupSize, numCols / 2, 1);

    perColDequantization<T>
        <<<grid, block, 0, stream>>>(dst, (char4*)src, numRows, numCols, scalePtr, zerosPtr, groupSize);
}

#define INSTANTIATE_INVOKE_PER_COL_DEQUANTIZATION_INT4X2(T)                                                            \
    template void invokePerColDequantizationInt4x2(T*            dst,                                                  \
                                                   const int8_t* src,                                                  \
                                                   const int64_t numRows,                                              \
                                                   const int64_t numCols,                                              \
                                                   half*         scalePtr,                                             \
                                                   half*         zerosPtr,                                             \
                                                   const int64_t groupSize,                                            \
                                                   hipStream_t  stream)
INSTANTIATE_INVOKE_PER_COL_DEQUANTIZATION_INT4X2(float);
INSTANTIATE_INVOKE_PER_COL_DEQUANTIZATION_INT4X2(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_PER_COL_DEQUANTIZATION_INT4X2(__hip_bfloat16);
#endif

/////////////////////////////////////////////////////////////////////////////////////////////////
// int4 row dequant /////////////////////////////////////////////////////////////////////////////
template<typename T>
__global__ void perRowDequantization(T*            dst,
                                     const char4*  src,
                                     const int64_t numRows,
                                     const int64_t numCols,
                                     const half*   scalePtr,
                                     const half*   zerosPtr,
                                     const int64_t groupSize,
                                     float*        dbgfp  = nullptr,
                                     int*          dbgint = nullptr) {
    const uint8_t* pSrc      = (const uint8_t*)src;
    uint32_t       rowIdx    = blockIdx.y;
    uint32_t       colGrpIdx = blockIdx.x;
    uint32_t       colGrpNum = numCols / groupSize;

    float scale = cuda_cast<float>(scalePtr[rowIdx * colGrpNum + colGrpIdx]);
    float zeros = cuda_cast<float>(zerosPtr[rowIdx * colGrpNum + colGrpIdx]);
    // scale = 1.0f;
    // zeros = 0;

    uint8_t tmpu8 = pSrc[rowIdx * numCols / 2 + colGrpIdx * groupSize / 2 + threadIdx.x];

    uint8_t tmpu4l = tmpu8 & 0x0F;
    uint8_t tmpu4h = (tmpu8 >> 4) & 0x0F;

    if (tmpu4l & 0x08)
        tmpu4l |= 0xF0;
    if (tmpu4h & 0x08)
        tmpu4h |= 0xF0;
    int8_t tmpi4l = tmpu4l;
    int8_t tmpi4h = tmpu4h;

    float tmpfpl = cuda_cast<float>(tmpi4l);
    float tmpfph = cuda_cast<float>(tmpi4h);

    T vall = cuda_cast<T>(tmpfpl * scale);
    T valh = cuda_cast<T>(tmpfph * scale);

    dst[rowIdx * numCols + colGrpIdx * groupSize + threadIdx.x * 2 + 0] = vall;
    dst[rowIdx * numCols + colGrpIdx * groupSize + threadIdx.x * 2 + 1] = valh;
}

template<typename T>
void invokePerRowDequantizationInt4x2(T*            dst,
                                      const int8_t* src,
                                      const int64_t numRows,
                                      const int64_t numCols,
                                      half*         scalePtr,
                                      half*         zerosPtr,
                                      const int64_t groupSize,
                                      hipStream_t  stream) {
    const dim3 block(groupSize / 2);
    const dim3 grid(numCols / groupSize, numRows, 1);

    perRowDequantization<T>
        <<<grid, block, 0, stream>>>(dst, (char4*)src, numRows, numCols, scalePtr, zerosPtr, groupSize);
}

#define INSTANTIATE_INVOKE_PER_ROW_DEQUANTIZATION_INT4X2(T)                                                            \
    template void invokePerRowDequantizationInt4x2(T*            dst,                                                  \
                                                   const int8_t* src,                                                  \
                                                   const int64_t numRows,                                              \
                                                   const int64_t numCols,                                              \
                                                   half*         scalePtr,                                             \
                                                   half*         zerosPtr,                                             \
                                                   const int64_t groupSize,                                            \
                                                   hipStream_t  stream)
INSTANTIATE_INVOKE_PER_ROW_DEQUANTIZATION_INT4X2(float);
INSTANTIATE_INVOKE_PER_ROW_DEQUANTIZATION_INT4X2(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_PER_ROW_DEQUANTIZATION_INT4X2(__hip_bfloat16);
#endif
}  // namespace fastertransformer
